
#include <hip/hip_runtime.h>
#include <cstdio>
int main()
{
  int count = 0;
  if (hipSuccess != hipGetDeviceCount(&count)) return -1;
  if (count == 0) return -1;
  for (int device = 0; device < count; ++device)
  {
    hipDeviceProp_t prop;
    if (hipSuccess == hipGetDeviceProperties(&prop, device))
      std::printf("%d.%d ", prop.major, prop.minor);
  }
  return 0;
}
