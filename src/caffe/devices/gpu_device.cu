#include "hip/hip_runtime.h"
// Copyright 2014 BVLC and contributors.

#include <math_functions.h>  // CUDA's, not caffe's, for fabs, signbit
#include <thrust/device_vector.h>
#include <thrust/functional.h>  // thrust::plus
#include <thrust/reduce.h>
#include <cmath>
#include <cstdlib>
#include <cstring>

#include "caffe/common.hpp"
#include "caffe/device.hpp"

namespace caffe {

template <typename Dtype>
__global__ void set_kernel(const int n, const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = alpha;
  }
}

template <typename Dtype>
void GPUDevice<Dtype>::set(const int N, const Dtype alpha, Dtype* Y) {
  if (alpha == 0) {
    CUDA_CHECK(hipMemset(Y, 0, sizeof(Dtype) * N));
    return;
  }
  // NOLINT_NEXT_LINE(whitespace/operators)
  set_kernel<Dtype><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template <typename Dtype>
__global__ void add_scalar_kernel(const int n, const Dtype alpha, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] += alpha;
  }
}

template <typename Dtype>
void GPUDevice<Dtype>::add_scalar(const int N, const Dtype alpha, Dtype* Y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_scalar_kernel<Dtype><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, alpha, Y);
}

template <typename Dtype>
__global__ void add_kernel(const int n, const Dtype* a, const Dtype* b,
                           Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] + b[index];
  }
}

template <typename Dtype>
void GPUDevice<Dtype>::add(const int N, const Dtype* a, const Dtype* b,
                           Dtype* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  add_kernel<Dtype><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void sub_kernel(const int n, const Dtype* a, const Dtype* b,
                           Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] - b[index];
  }
}

template <typename Dtype>
void GPUDevice<Dtype>::sub(const int N, const Dtype* a, const Dtype* b,
                           Dtype* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  sub_kernel<Dtype><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void mul_kernel(const int n, const Dtype* a, const Dtype* b,
                           Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] * b[index];
  }
}

template <typename Dtype>
void GPUDevice<Dtype>::mul(const int N, const Dtype* a, const Dtype* b,
                           Dtype* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  mul_kernel<Dtype><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void div_kernel(const int n, const Dtype* a, const Dtype* b,
                           Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] / b[index];
  }
}

template <typename Dtype>
void GPUDevice<Dtype>::div(const int N, const Dtype* a, const Dtype* b,
                           Dtype* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  div_kernel<Dtype><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, b, y);
}

template <typename Dtype>
__global__ void powx_kernel(const int n, const Dtype* a, const Dtype alpha,
                            Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = pow(a[index], alpha);
  }
}

template <typename Dtype>
void GPUDevice<Dtype>::powx(const int N, const Dtype* a, const Dtype alpha,
                            Dtype* y) {
  // NOLINT_NEXT_LINE(whitespace/operators)
  powx_kernel<Dtype><<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, a, alpha, y);
}

template <typename Dtype>
__global__ void sign_kernel(const int n, const Dtype* x, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = (Dtype(0) < x[index]) - (x[index] < Dtype(0));
  }
}

template <typename Dtype>
void GPUDevice<Dtype>::sign(const int n, const Dtype* x, Dtype* y) {
  /* NOLINT_NEXT_LINE(whitespace/operators) */
  sign_kernel<Dtype><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(
      n, x, y);
}

template <typename Dtype>
__global__ void sgnbit_kernel(const int n, const Dtype* x, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = signbit(x[index]);
  }
}

template <typename Dtype>
void GPUDevice<Dtype>::sgnbit(const int n, const Dtype* x, Dtype* y) {
  /* NOLINT_NEXT_LINE(whitespace/operators) */
  sgnbit_kernel<Dtype><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(
      n, x, y);
}

template <typename Dtype>
__global__ void fabs_kernel(const int n, const Dtype* x, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = fabs(x[index]);
  }
}

template <typename Dtype>
void GPUDevice<Dtype>::fabs(const int n, const Dtype* x, Dtype* y) {
  /* NOLINT_NEXT_LINE(whitespace/operators) */
  fabs_kernel<Dtype><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(
      n, x, y);
}

template <typename Dtype>
__global__ void sqr_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = a[index] * a[index];
  }
}

template <typename Dtype>
void GPUDevice<Dtype>::sqr(const int n, const Dtype* a, Dtype* y) {
  /* NOLINT_NEXT_LINE(whitespace/operators) */
  sqr_kernel<Dtype><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(
      n, a, y);
}

template <typename Dtype>
__global__ void exp_kernel(const int n, const Dtype* a, Dtype* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = exp(a[index]);
  }
}

template <typename Dtype>
void GPUDevice<Dtype>::exp(const int n, const Dtype* a, Dtype* y) {
  /* NOLINT_NEXT_LINE(whitespace/operators) */
  exp_kernel<Dtype><<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(
      n, a, y);
}

__global__ void popc_kernel(const int n, const float* a,
    const float* b, uint8_t* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = __popc(static_cast<uint32_t>(a[index]) ^
                      static_cast<uint32_t>(b[index]));
  }
}

__global__ void popcll_kernel(const int n, const double* a,
    const double* b, uint8_t* y) {
  CUDA_KERNEL_LOOP(index, n) {
    y[index] = __popcll(static_cast<uint64_t>(a[index]) ^
                        static_cast<uint64_t>(b[index]));
  }
}

template <>
void GPUDevice<float>::hamming_distance(const int n, const float* x,
                                        const float* y, int* out) {
  // TODO: Fix caffe_gpu_hamming_distance (see failing unit test
  // TestHammingDistanceGPU in test_math_functions.cpp).
  NOT_IMPLEMENTED;
  thrust::device_vector<uint8_t> popcounts(n);
  // NOLINT_NEXT_LINE(whitespace/operators)
  popc_kernel<<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(
      n, x, y, thrust::raw_pointer_cast(popcounts.data()));
  *out = thrust::reduce(popcounts.begin(), popcounts.end(),
                        (uint32_t) 0, thrust::plus<uint32_t>());
}

template <>
void GPUDevice<double>::hamming_distance(const int n, const double* x,
                                         const double* y, int* out) {
  // TODO: Fix caffe_gpu_hamming_distance (see failing unit test
  // TestHammingDistanceGPU in test_math_functions.cpp).
  NOT_IMPLEMENTED;
  thrust::device_vector<uint8_t> popcounts(n);
  // NOLINT_NEXT_LINE(whitespace/operators)
  popcll_kernel<<<CAFFE_GET_BLOCKS(n), CAFFE_CUDA_NUM_THREADS>>>(
      n, x, y, thrust::raw_pointer_cast(popcounts.data()));
  *out = thrust::reduce(popcounts.begin(), popcounts.end(),
                        /* NOLINT_NEXT_LINE(build/include_what_you_use) */
                        (uint32_t) 0, thrust::plus<uint32_t>());
}

template <>
void GPUDevice<float>::rng_uniform(const int n, const float a, const float b,
                                   float* r) {
  CURAND_CHECK(hiprandGenerateUniform(Caffe::curand_generator(), r, n));
  const float range = b - a;
  if (range != static_cast<float>(1)) {
    GetDevice<float>(Caffe::GPU)->scal(n, range, r);
  }
  if (a != static_cast<float>(0)) {
    GetDevice<float>(Caffe::GPU)->add_scalar(n, a, r);
  }
}

template <>
void GPUDevice<double>::rng_uniform(const int n, const double a, const double b,
                                    double* r) {
  CURAND_CHECK(hiprandGenerateUniformDouble(Caffe::curand_generator(), r, n));
  const double range = b - a;
  if (range != static_cast<double>(1)) {
    GetDevice<double>(Caffe::GPU)->scal(n, range, r);
  }
  if (a != static_cast<double>(0)) {
    GetDevice<double>(Caffe::GPU)->add_scalar(n, a, r);
  }
}

template <>
void GPUDevice<float>::rng_gaussian(const int n, const float mu,
                                    const float sigma, float* r) {
  CURAND_CHECK(
      hiprandGenerateNormal(Caffe::curand_generator(), r, n, mu, sigma));
}

template <>
void GPUDevice<double>::rng_gaussian(const int n, const double mu,
                                     const double sigma, double* r) {
  CURAND_CHECK(
      hiprandGenerateNormalDouble(Caffe::curand_generator(), r, n, mu, sigma));
}


template <typename Dtype>
__global__ void im2col_gpu_kernel(const int n, const Dtype* data_im,
                                  const int height, const int width,
                                  const int ksize, const int pad,
                                  const int stride, const int height_col,
                                  const int width_col, Dtype* data_col) {
  CUDA_KERNEL_LOOP(index, n) {
    int w_out = index % width_col;
    int h_index = index / width_col;
    int h_out = h_index % height_col;
    int channel_in = h_index / height_col;
    int channel_out = channel_in * ksize * ksize;
    int h_in = h_out * stride - pad;
    int w_in = w_out * stride - pad;
    Dtype* data_col_ptr = data_col;
    data_col_ptr += (channel_out * height_col + h_out) * width_col + w_out;
    const Dtype* data_im_ptr = data_im;
    data_im_ptr += (channel_in * height + h_in) * width + w_in;
    for (int i = 0; i < ksize; ++i) {
      for (int j = 0; j < ksize; ++j) {
        int h = h_in + i;
        int w = w_in + j;
        *data_col_ptr = (h >= 0 && w >= 0 && h < height && w < width) ?
            data_im_ptr[i * width + j] : 0;
        data_col_ptr += height_col * width_col;
      }
    }
  }
}

template <typename Dtype>
void GPUDevice<Dtype>::im2col(const Dtype* data_im, const int channels,
                              const int height, const int width,
                              const int ksize, const int pad, const int stride,
                              Dtype* data_col) {
  // We are going to launch channels * height_col * width_col kernels, each
  // kernel responsible for copying a single-channel grid.
  int height_col = (height + 2 * pad - ksize) / stride + 1;
  int width_col = (width + 2 * pad - ksize) / stride + 1;
  int num_kernels = channels * height_col * width_col;
  // NOLINT_NEXT_LINE(whitespace/operators)
  im2col_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
                             CAFFE_CUDA_NUM_THREADS>>>(
      num_kernels, data_im, height, width, ksize, pad, stride, height_col,
      width_col, data_col);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void col2im_gpu_kernel(const int n, const Dtype* data_col,
                                  const int height, const int width,
                                  const int channels, const int ksize,
                                  const int pad, const int stride,
                                  const int height_col, const int width_col,
                                  Dtype* data_im) {
  CUDA_KERNEL_LOOP(index, n) {
    Dtype val = 0;
    int w = index % width + pad;
    int h = (index / width) % height + pad;
    int c = index / (width * height);
    // compute the start and end of the output
    int w_col_start = (w < ksize) ? 0 : (w - ksize) / stride + 1;
    int w_col_end = min(w / stride + 1, width_col);
    int h_col_start = (h < ksize) ? 0 : (h - ksize) / stride + 1;
    int h_col_end = min(h / stride + 1, height_col);
    /*
    for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
      for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
        // the col location: [c * width * height + h_out, w_out]
        int c_col = c * ksize * ksize + (h - h_col * stride) * ksize + (w - w_col * stride);
        val += data_col[(c_col * height_col + h_col) * width_col + w_col];
      }
    }
    */
    // equivalent implementation
    int offset = (c * ksize * ksize + h * ksize + w) * height_col * width_col;
    int coeff_h_col = (1 - stride * ksize * height_col) * width_col;
    int coeff_w_col = (1 - stride * height_col * width_col);
    for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
      for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
        val += data_col[offset + h_col * coeff_h_col + w_col * coeff_w_col];
      }
    }
    data_im[index] = val;
  }
}

template <typename Dtype>
void GPUDevice<Dtype>::col2im(const Dtype* data_col, const int channels,
                              const int height, const int width,
                              const int ksize, const int pad, const int stride,
                              Dtype* data_im) {
  // CUDA_CHECK(hipMemset(data_im, 0,
  //            sizeof(Dtype) * height * width * channels));
  int height_col = (height + 2 * pad - ksize) / stride + 1;
  int width_col = (width + 2 * pad - ksize) / stride + 1;
  int num_kernels = channels * height * width;
  // To avoid involving atomic operations, we will launch one kernel per
  // bottom dimension, and then in the kernel add up the top dimensions.
  // NOLINT_NEXT_LINE(whitespace/operators)
  col2im_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
                             CAFFE_CUDA_NUM_THREADS>>>(
      num_kernels, data_col, height, width, channels, ksize, pad, stride,
      height_col, width_col, data_im);
  CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_CLASS(GPUDevice);

}  // namespace caffe
