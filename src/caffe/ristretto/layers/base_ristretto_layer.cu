#include "hip/hip_runtime.h"
#include "ristretto/base_ristretto_layer.hpp"
#include "ristretto/base_ristretto_layer.cuh"

namespace caffe {

template <typename Dtype>
void BaseRistrettoLayer<Dtype>::QuantizeWeights_gpu(
      vector<shared_ptr<Blob<Dtype> > > weights_quantized, const int rounding,
      const bool bias_term) {
  Dtype* weight = weights_quantized[0]->mutable_gpu_data();
  const int cnt_weight = weights_quantized[0]->count();
  switch (precision_) {
  case QuantizationParameter_Precision_MINIFLOAT:
    Trim2MiniFloat_gpu(weight, cnt_weight, fp_mant_, fp_exp_, rounding);
    if (bias_term) {
      Trim2MiniFloat_gpu(weights_quantized[1]->mutable_gpu_data(),
          weights_quantized[1]->count(), fp_mant_, fp_exp_, rounding);
    }
    break;
  case QuantizationParameter_Precision_DYNAMIC_FIXED_POINT:
    Trim2FixedPoint_gpu(weight, cnt_weight, bw_params_, rounding, fl_params_);
    if (bias_term) {
      Trim2FixedPoint_gpu(weights_quantized[1]->mutable_gpu_data(),
          weights_quantized[1]->count(), bw_params_, rounding, fl_params_);
    }
    break;
  case QuantizationParameter_Precision_INTEGER_POWER_OF_2_WEIGHTS:
    Trim2IntegerPowerOf2_gpu(weight, cnt_weight, pow_2_min_exp_, pow_2_max_exp_,
        rounding);
    // Don't trim bias
    break;
  default:
    LOG(FATAL) << "Unknown trimming mode: " << precision_;
    break;
  }
}

template <typename Dtype>
void BaseRistrettoLayer<Dtype>::QuantizeLayerInputs_gpu(
    Dtype* data, const int count) {
  switch (precision_) {
    case QuantizationParameter_Precision_INTEGER_POWER_OF_2_WEIGHTS:
    case QuantizationParameter_Precision_DYNAMIC_FIXED_POINT:
      Trim2FixedPoint_gpu(data, count, bw_layer_in_, rounding_, fl_layer_in_);
      break;
    case QuantizationParameter_Precision_MINIFLOAT:
      Trim2MiniFloat_gpu(data, count, fp_mant_, fp_exp_, rounding_);
      break;
    default:
      LOG(FATAL) << "Unknown trimming mode: " << precision_;
      break;
  }
}

template <typename Dtype>
void BaseRistrettoLayer<Dtype>::QuantizeLayerOutputs_gpu(Dtype* data,
      const int count) {
  switch (precision_) {
    case QuantizationParameter_Precision_INTEGER_POWER_OF_2_WEIGHTS:
    case QuantizationParameter_Precision_DYNAMIC_FIXED_POINT:
      Trim2FixedPoint_gpu(data, count, bw_layer_out_, rounding_, fl_layer_out_);
      break;
    case QuantizationParameter_Precision_MINIFLOAT:
      Trim2MiniFloat_gpu(data, count, fp_mant_, fp_exp_, rounding_);
      break;
    default:
      LOG(FATAL) << "Unknown trimming mode: " << precision_;
      break;
  }
}

template <typename Dtype>
__global__ void Trim2FixedPoint_kernel(Dtype* data, const int cnt,
      const int bit_width, const int rounding, const int fl) {
	CUDA_KERNEL_LOOP(index, cnt) {
    // Saturate data
    Dtype max_data = (powf(2, bit_width - 1) - 1) * powf(2, -fl);
    Dtype min_data = -powf(2, bit_width - 1) * powf(2, -fl);
    data[index] = fmax(fmin(data[index], max_data), min_data);
    // Round data
    data[index] /= powf(2, -fl);
    switch (rounding) {
    case QuantizationParameter_Rounding_NEAREST:
      data[index] = rint(data[index]);
      break;
    case QuantizationParameter_Rounding_STOCHASTIC:
      data[index] = __float2int_rd(data[index] + RandUniform_device(index));
      break;
    default:
      break;
    }
    data[index] *= powf(2, -fl);
	}
}

template <typename Dtype>
void BaseRistrettoLayer<Dtype>::Trim2FixedPoint_gpu(Dtype* data, const int cnt,
      const int bit_width, const int rounding, int fl) {
  Trim2FixedPoint_kernel<<<CAFFE_GET_BLOCKS(cnt), CAFFE_CUDA_NUM_THREADS>>>(
      data, cnt, bit_width, rounding, fl);
}

template <typename Dtype>
__global__ void Trim2MiniFloat_kernel(Dtype* data, const int cnt,
      const int bw_mant, const int bw_exp, const int rounding){
	CUDA_KERNEL_LOOP(index, cnt) {
    Trim2MiniFloat_device(&data[index], bw_mant, bw_exp, rounding, index);
	}
}

template <typename Dtype>
void BaseRistrettoLayer<Dtype>::Trim2MiniFloat_gpu(Dtype* data,
      const int cnt, const int bw_mant, const int bw_exp, const int rounding) {
  Trim2MiniFloat_kernel<<<CAFFE_GET_BLOCKS(cnt), CAFFE_CUDA_NUM_THREADS>>>(
      data, cnt, bw_mant, bw_exp, rounding);
}

template <typename Dtype>
__global__ void Trim2IntegerPowerOf2_kernel(Dtype* data, const int cnt,
      const int min_exp, const int max_exp, const int rounding) {
	CUDA_KERNEL_LOOP(index, cnt) {
    float exponent = log2f(fabs((float)data[index]));
    int sign = data[index] >= 0 ? 1 : -1;
    switch (rounding) {
    case QuantizationParameter_Rounding_NEAREST:
      exponent = rint(exponent);
      break;
    case QuantizationParameter_Rounding_STOCHASTIC:
      exponent = __float2int_rd(exponent + RandUniform_device(index));
      break;
    default:
      break;
    }
    exponent = fmaxf(fminf(exponent, max_exp), min_exp);
    data[index] = sign * powf(2, exponent);
	}
}

template <typename Dtype>
void BaseRistrettoLayer<Dtype>::Trim2IntegerPowerOf2_gpu(Dtype* data,
      const int cnt, const int min_exp, const int max_exp, const int rounding) {
  Trim2IntegerPowerOf2_kernel<<<CAFFE_GET_BLOCKS(cnt), CAFFE_CUDA_NUM_THREADS>>>(
      data, cnt, min_exp, max_exp, rounding);
}

// Explicit instantiations
template void BaseRistrettoLayer<double>::QuantizeWeights_gpu(
    vector<shared_ptr<Blob<double> > > weights_quantized, const int rounding,
    const bool bias_term);
template void BaseRistrettoLayer<float>::QuantizeWeights_gpu(
    vector<shared_ptr<Blob<float> > > weights_quantized, const int rounding,
    const bool bias_term);
template void BaseRistrettoLayer<double>::QuantizeLayerInputs_gpu(double* data,
    const int count);
template void BaseRistrettoLayer<float>::QuantizeLayerInputs_gpu(float* data,
    const int count);
template void BaseRistrettoLayer<double>::QuantizeLayerOutputs_gpu(
    double* top_data, const int top_count);
template void BaseRistrettoLayer<float>::QuantizeLayerOutputs_gpu(
    float* top_data, const int top_count);
template void BaseRistrettoLayer<double>::Trim2FixedPoint_gpu(double* data,
    const int cnt, const int bit_width, const int rounding, int fl);
template void BaseRistrettoLayer<float>::Trim2FixedPoint_gpu(float* data,
    const int cnt, const int bit_width, const int rounding, int fl);
template void BaseRistrettoLayer<double>::Trim2MiniFloat_gpu(double* data,
    const int cnt, const int bw_mant, const int bw_exp, const int rounding);
template void BaseRistrettoLayer<float>::Trim2MiniFloat_gpu(float* data,
    const int cnt, const int bw_mant, const int bw_exp, const int rounding);
template void BaseRistrettoLayer<double>::Trim2IntegerPowerOf2_gpu(double* data,
    const int cnt, const int min_exp, const int max_exp, const int rounding);
template void BaseRistrettoLayer<float>::Trim2IntegerPowerOf2_gpu(float* data,
    const int cnt, const int min_exp, const int max_exp, const int rounding);

}  // namespace caffe


