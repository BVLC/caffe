// Copyright 2014 BVLC and contributors.

#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
Dtype FlattenLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      vector<Blob<Dtype>*>* top) {
  (*top)[0]->ShareData(*bottom[0]);
  return Dtype(0.);
}

template <typename Dtype>
void FlattenLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, vector<Blob<Dtype>*>* bottom) {
  (*bottom)[0]->ShareDiff(*top[0]);
}

INSTANTIATE_CLASS(FlattenLayer);

}  // namespace caffe
