#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/layers/triplet_loss_layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void TripletLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  Dtype margin = this->layer_param_.triplet_loss_param().margin();
  Dtype losstype = this->layer_param_.triplet_loss_param().losstype();
  int num_negatives = this->layer_param_.triplet_loss_param().num_negatives();
  int use_pair = this->layer_param_.triplet_loss_param().use_pair();
  CHECK_EQ(bottom[0]->num()%(2 + num_negatives), 0);
  Dtype loss(0.0);
  int dim = bottom[0]->count()/bottom[0]->num();
  int num_set = bottom[0]->num()/(2 + num_negatives);
  if (losstype == 0) {
  for (int i = 0; i < num_set; ++i) {
    caffe_gpu_sub(
        dim,
        bottom[0]->gpu_data() + (2 + num_negatives)*i*dim,  // reference
        bottom[0]->gpu_data() + ((2 + num_negatives)*i + 1)*dim,  // positive
        diff_pos.mutable_gpu_data() + i*dim);  // reference-pose_close
    caffe_gpu_dot(
        dim,
        diff_pos.gpu_data() + i*dim,
        diff_pos.gpu_data() + i*dim,
        dist_sq_pos.mutable_cpu_data() + i);
    // a b is a similar pair for pair wise
    // loss accumulated by the pair wise part
    if (use_pair == 1) {
        loss += dist_sq_pos.cpu_data()[i];
    }
    for (int triplet = 0; triplet < num_negatives; ++triplet) {
      // Triplet loss accumulation
      // a and negative[triplet] is a similar pair for triplet
      dist_sq_.mutable_cpu_data()[i] = dist_sq_pos.cpu_data()[i];
      // Loss component calculated from negative part
      caffe_gpu_sub(
          dim,
          bottom[0]->gpu_data() + (2 + num_negatives)*i*dim,  // reference
          bottom[0]->gpu_data() + ((2 + num_negatives)*i + 2 + triplet)*dim,
          diff_neg.mutable_gpu_data() + i*dim);  // reference-negative
      caffe_gpu_dot(
          dim,
          diff_neg.gpu_data() + i*dim,
          diff_neg.gpu_data() + i*dim,
          dist_sq_neg.mutable_cpu_data() + i);
      // a and negative[triplet] is a dissimilar pair for triplet
      dist_sq_.mutable_cpu_data()[i] -= dist_sq_neg.cpu_data()[i];
      // loss accumulated accumulated by the triplet part
      loss += std::max(margin + dist_sq_.cpu_data()[i], Dtype(0.0));
    }
  }
  loss = loss / static_cast<Dtype>(num_set) / Dtype(2);
  top[0]->mutable_cpu_data()[0] = loss;
  } else if (losstype == 1) {
  for (int i = 0; i < num_set; ++i) {
    caffe_gpu_sub(
        dim,
        bottom[0]->gpu_data() + (2 + num_negatives)*i*dim,  // reference
        bottom[0]->gpu_data() + ((2 + num_negatives)*i + 1)*dim,  // positive
        diff_pos.mutable_gpu_data() + i*dim);  // reference-pose_close
    // Loss component calculated from reference and close one
    caffe_gpu_dot(
        dim,
        diff_pos.gpu_data() + i*dim,
        diff_pos.gpu_data() + i*dim,
        dist_sq_pos.mutable_cpu_data() + i);
    // a b is a similar pair for pair wise
    // loss accumulated by the pair wise part
    if (use_pair == 1) {
        loss += dist_sq_pos.cpu_data()[i];
    }
    for (int triplet = 0; triplet < num_negatives; ++triplet) {
      dist_sq_.mutable_cpu_data()[i] = dist_sq_pos.mutable_cpu_data()[i];
      dist_sq_.mutable_cpu_data()[i] += margin;
      // Loss component calculated from negative part
      caffe_gpu_sub(
          dim,
          bottom[0]->gpu_data() + (2 + num_negatives)*i*dim,  // reference
          bottom[0]->gpu_data() + ((2 + num_negatives)*i + 2 + triplet)*dim,
          diff_neg.mutable_gpu_data() + i*dim);  // reference-negative
      caffe_gpu_dot(
          dim,
          diff_neg.gpu_data() + i*dim,
          diff_neg.gpu_data() + i*dim,
          dist_sq_neg.mutable_cpu_data() + i);
      // a and negative[triplet] is a dissimilar pair for triplet
      dist_sq_.mutable_cpu_data()[i] = 1 - \
        dist_sq_neg.cpu_data()[i] / dist_sq_.mutable_cpu_data()[i];
      // loss accumulated accumulated by the triplet part
      loss += std::max(dist_sq_.cpu_data()[i], Dtype(0.0));
    }
  }
  loss = loss / static_cast<Dtype>(num_set) / Dtype(2);
  top[0]->mutable_cpu_data()[0] = loss;
  } else if (losstype == 2) {
  for (int i = 0; i < num_set; ++i) {
    caffe_gpu_sub(
      dim,
      bottom[0]->gpu_data() +
      (2 + num_negatives)*i*dim,  // reference
      bottom[0]->gpu_data() +
      ((2 + num_negatives)*i + 1)*dim,  // positive
      diff_pos.mutable_gpu_data() + i*dim);  // reference-pose_close
    // Loss component calculated from reference and close one
    caffe_gpu_dot(
      dim,
      diff_pos.gpu_data() + i*dim,
      diff_pos.gpu_data() + i*dim,
      dist_sq_pos.mutable_cpu_data() + i);
    // a b is a similar pair for pair wise
    // loss accumulated by the pair wise part
    if (use_pair == 1) {
    loss += dist_sq_pos.cpu_data()[i];
    }
    for (int triplet = 0; triplet < num_negatives; ++triplet) {
    dist_sq_.mutable_cpu_data()[i] = exp(dist_sq_pos.mutable_cpu_data()[i]);
    dist_sq_.mutable_cpu_data()[i] += margin;
    // Loss component calculated from negative part
    caffe_gpu_sub(
      dim,
      bottom[0]->gpu_data() +
      (2 + num_negatives)*i*dim,  // reference
      bottom[0]->gpu_data() +
      ((2 + num_negatives)*i + 2 + triplet)*dim,
      diff_neg.mutable_gpu_data() + i*dim);  // reference-negative
    caffe_gpu_dot(
      dim,
      diff_neg.gpu_data() + i*dim,
      diff_neg.gpu_data() + i*dim,
      dist_sq_neg.mutable_cpu_data() + i);
    // a and negative[triplet] is a dissimilar pair for triplet
    dist_sq_.mutable_cpu_data()[i] = 1 - \
    exp(dist_sq_neg.cpu_data()[i]) / dist_sq_.mutable_cpu_data()[i];
    // loss accumulated accumulated by the triplet part
    loss += std::max(dist_sq_.cpu_data()[i], Dtype(0.0));
    }
  }
  loss = loss / static_cast<Dtype>(num_set) / Dtype(2);
  top[0]->mutable_cpu_data()[0] = loss;
  }
}

template <typename Dtype>
void TripletLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  Dtype margin = this->layer_param_.triplet_loss_param().margin();
  Dtype losstype = this->layer_param_.triplet_loss_param().losstype();
  int num_negatives = this->layer_param_.triplet_loss_param().num_negatives();
  int use_pair = this->layer_param_.triplet_loss_param().use_pair();
  int dim = bottom[0]->count()/bottom[0]->num();
  int num_set = bottom[0]->num()/(2 + num_negatives);
  if (losstype == 0) {
  // BP for feat1(extracted from reference)
  for (int i = 0; i < 1; ++i) {
    if (propagate_down[0]) {
      const Dtype sign = 1;
      const Dtype alpha = sign * top[0]->cpu_diff()[0] /
          static_cast<Dtype>(num_set);
      for (int j = 0; j < num_set; ++j) {
        Dtype* bout = bottom[0]->mutable_gpu_diff();
      // the pair part
      if (use_pair == 1) {
        caffe_gpu_axpby(
            dim,
            alpha,
            diff_pos.gpu_data() + (j*dim),
            Dtype(0.0),
            bout + ((2 + num_negatives)*j + i)*dim);
      } else {
        caffe_gpu_axpby(
            dim,
            Dtype(0.0),
            diff_pos.gpu_data() + (j*dim),
            Dtype(0.0),
            bout + ((2 + num_negatives)*j + i)*dim);
      }
        // the num_negatives triplet part
        for (int triplet = 0; triplet < num_negatives; ++triplet) {
          caffe_gpu_sub(
              dim,
              bottom[0]->gpu_data() + (2 + num_negatives)*j*dim,  // reference
              bottom[0]->gpu_data() + ((2 + num_negatives)*j + 2 + triplet)*dim,
              diff_neg.mutable_gpu_data() + j*dim);  // reference-negative
          caffe_gpu_dot(
              dim,
              diff_neg.gpu_data() + j*dim,
              diff_neg.gpu_data() + j*dim,
              dist_sq_neg.mutable_cpu_data() + j);
          // Triplet loss accumulation
          // a and negative[triplet] is a similar pair for triplet
          dist_sq_.mutable_cpu_data()[j] = dist_sq_pos.cpu_data()[j];
          // a and negative[triplet] is a dissimilar pair for triplet
          dist_sq_.mutable_cpu_data()[j] -= dist_sq_neg.cpu_data()[j];
          // Loss component calculated from negative part
          if ((margin + dist_sq_.cpu_data()[j]) > Dtype(0.0)) {
            // similar pair in triplet
            caffe_gpu_axpby(
                dim,
                alpha,
                diff_pos.gpu_data() + (j*dim),
                Dtype(1.0),
                bout + (2 + num_negatives)*j*dim);
            // dissimilar pair in triplet
            caffe_gpu_axpby(
                dim,
                -alpha,
                diff_neg.gpu_data() + (j*dim),
                Dtype(1.0),
                bout + ((2 + num_negatives)*j + i)*dim);
          }
        }
      }
    }
  }
  // BP for feat2(extracted from the closest sample)
  for (int i = 1; i < 2; ++i) {
    if (propagate_down[0]) {
      const Dtype sign = -1;
      const Dtype alpha = sign * top[0]->cpu_diff()[0] /
          static_cast<Dtype>(num_set);
      for (int j = 0; j < num_set; ++j) {
        Dtype* bout = bottom[0]->mutable_gpu_diff();
        // the pair part
        if (use_pair == 1) {
        caffe_gpu_axpby(
            dim,
            alpha,
            diff_pos.gpu_data() + (j*dim),
            Dtype(0.0),
            bout + ((2 + num_negatives)*j + i)*dim);
        } else {
            caffe_gpu_axpby(
            dim,
            Dtype(0.0),
            diff_pos.gpu_data() + (j*dim),
            Dtype(0.0),
            bout + ((2 + num_negatives)*j + i)*dim);
        }
        // the num_negatives triplet part
        for (int triplet = 0; triplet < num_negatives; ++triplet) {
          caffe_gpu_sub(
              dim,
              bottom[0]->gpu_data() + (2 + num_negatives)*j*dim,  // reference
              bottom[0]->gpu_data() + ((2 + num_negatives)*j + 2 + triplet)*dim,
              diff_neg.mutable_gpu_data() + j*dim);  // reference-negative
          // Triplet loss accumulation
          // a and negative[triplet] is a similar pair for triplet
          dist_sq_.mutable_cpu_data()[j] = dist_sq_pos.cpu_data()[j];
          caffe_gpu_dot(
              dim,
              diff_neg.gpu_data() + j*dim,
              diff_neg.gpu_data() + j*dim,
              dist_sq_neg.mutable_cpu_data() + j);
          // a and negative[triplet] is a dissimilar pair for triplet
          dist_sq_.mutable_cpu_data()[j] -= dist_sq_neg.cpu_data()[j];
          if ((margin + dist_sq_.cpu_data()[j]) > Dtype(0.0)) {
            // similar pair in triplet
            caffe_gpu_axpby(
                dim,
                alpha,
                diff_pos.gpu_data() + (j*dim),
                Dtype(1.0),
                bout + ((2 + num_negatives)*j + i)*dim);
          }
        }
      }
    }
  }
  // BP for negative feature used in the num_negatives triplet part
  for (int i = 2; i < 2 + num_negatives; ++i) {
    if (propagate_down[0]) {
      const Dtype sign = 1;
      const Dtype alpha = sign * top[0]->cpu_diff()[0] /
          static_cast<Dtype>(num_set);
      for (int j = 0; j < num_set; ++j) {
        Dtype* bout = bottom[0]->mutable_gpu_diff();
        caffe_gpu_sub(
            dim,
            bottom[0]->gpu_data() + (2 + num_negatives)*j*dim,  // reference
            bottom[0]->gpu_data() + ((2 + num_negatives)*j + i)*dim,
            diff_neg.mutable_gpu_data() + j*dim);  // reference-negative
        // Triplet loss accumulation
        // a and negative[triplet] is a similar pair for triplet
        dist_sq_.mutable_cpu_data()[j] = dist_sq_pos.cpu_data()[j];
        caffe_gpu_dot(
            dim,
            diff_neg.gpu_data() + j*dim,
            diff_neg.gpu_data() + j*dim,
            dist_sq_neg.mutable_cpu_data() + j);
        // a and negative[triplet] is a dissimilar pair for triplet
        dist_sq_.mutable_cpu_data()[j] -= dist_sq_neg.cpu_data()[j];
        if ((margin + dist_sq_.cpu_data()[j]) > Dtype(0.0)) {
          // dissimilar pairs
          caffe_gpu_axpby(
              dim,
              alpha,
              diff_neg.gpu_data() + (j*dim),
              Dtype(0.0),
              bout + ((2 + num_negatives)*j + i)*dim);
        } else {
            caffe_gpu_set(dim, Dtype(0),
                bout + ((2 + num_negatives)*j + i)*dim);
        }
      }
    }
  }
  } else if (losstype == 1) {
  for (int i = 0; i < 1; ++i) {
    // BP for data1(feat1)
    if (propagate_down[0]) {
      const Dtype sign = 1;
      const Dtype alpha = sign * top[0]->cpu_diff()[0] /
          static_cast<Dtype>(num_set);
      for (int j = 0; j < num_set; ++j) {
        Dtype* bout = bottom[0]->mutable_gpu_diff();
      // the pair part
      if (use_pair == 1) {
        caffe_gpu_axpby(
            dim,
            alpha,
            diff_pos.gpu_data() + (j*dim),
            Dtype(0.0),
            bout + ((2 + num_negatives)*j + i)*dim);
        } else {
        caffe_gpu_axpby(
            dim,
            Dtype(0.0),
            diff_pos.gpu_data() + (j*dim),
            Dtype(0.0),
            bout + ((2 + num_negatives)*j + i)*dim);
        }
        // the num_negatives triplet part
        for (int triplet = 0; triplet < num_negatives; ++triplet) {
          dist_sq_.mutable_cpu_data()[j] = dist_sq_pos.cpu_data()[j];
          dist_sq_.mutable_cpu_data()[j] += margin;
          // Loss component calculated from negative part
          caffe_gpu_sub(
              dim,
              bottom[0]->gpu_data() + (2 + num_negatives)*j*dim,  // reference
              bottom[0]->gpu_data() + ((2 + num_negatives)*j + 2 + triplet)*dim,
              diff_neg.mutable_gpu_data() + j*dim);  // reference-negative
          caffe_gpu_dot(
              dim,
              diff_neg.gpu_data() + j*dim,
              diff_neg.gpu_data() + j*dim,
              dist_sq_neg.mutable_cpu_data() + j);
          // a and negative[triplet] is a dissimilar pair for triplet
          dist_sq_.mutable_cpu_data()[j] = 1 - \
            dist_sq_neg.cpu_data()[j] / dist_sq_.cpu_data()[j];
          // loss accumulated accumulated by the triplet part
          if ((dist_sq_.cpu_data()[j]) > Dtype(0.0)) {
            caffe_gpu_axpby(
                dim,
                alpha*dist_sq_neg.mutable_cpu_data()[j]/
          ((dist_sq_pos.cpu_data()[j]+margin)*
           (dist_sq_pos.cpu_data()[j]+margin)),
                diff_pos.gpu_data() + (j*dim),
                Dtype(1.0),
                bout + ((2 + num_negatives)*j + i)*dim);
            caffe_gpu_axpby(
                dim,
                -alpha/(dist_sq_pos.cpu_data()[j] + margin),
                diff_neg.gpu_data() + (j*dim),
                Dtype(1.0),
                bout + ((2 + num_negatives)*j + i)*dim);
          }
        }
      }
    }
  }
  for (int i = 1; i < 2; ++i) {
    // BP for positive data(feat2)
    if (propagate_down[0]) {
      const Dtype sign = -1;
      const Dtype alpha = sign * top[0]->cpu_diff()[0] /
          static_cast<Dtype>(num_set);
      for (int j = 0; j < num_set; ++j) {
        Dtype* bout = bottom[0]->mutable_gpu_diff();
      // the pair part
      if (use_pair == 1) {
        caffe_gpu_axpby(
            dim,
            alpha,
            diff_pos.gpu_data() + (j*dim),
            Dtype(0.0),
            bout + ((2 + num_negatives)*j + i)*dim);
      } else {
      caffe_gpu_axpby(
          dim,
          Dtype(0.0),
          diff_pos.gpu_data() + (j*dim),
          Dtype(0.0),
          bout + ((2 + num_negatives)*j + i)*dim);
      }
        // the num_negatives triplet part
        for (int triplet = 0; triplet < num_negatives; ++triplet) {
          dist_sq_.mutable_cpu_data()[j] = dist_sq_pos.cpu_data()[j];
          dist_sq_.mutable_cpu_data()[j] += margin;
          // Loss component calculated from negative part
          caffe_gpu_sub(
              dim,
              bottom[0]->gpu_data() + (2 + num_negatives)*j*dim,  // reference
              bottom[0]->gpu_data() + ((2 + num_negatives)*j + 2 + triplet)*dim,
              diff_neg.mutable_gpu_data() + j*dim);  // reference-negative
          caffe_gpu_dot(
              dim,
              diff_neg.gpu_data() + j*dim,
              diff_neg.gpu_data() + j*dim,
              dist_sq_neg.mutable_cpu_data() + j);
          // a and negative[triplet] is a dissimilar pair for triplet
          dist_sq_.mutable_cpu_data()[j] = 1 - \
            dist_sq_neg.cpu_data()[j] / dist_sq_.mutable_cpu_data()[j];
          // loss accumulated accumulated by the triplet part
          if ((dist_sq_.cpu_data()[j]) > Dtype(0.0)) {
            caffe_gpu_axpby(
                dim,
                alpha*dist_sq_neg.cpu_data()[j]/
          ((dist_sq_pos.cpu_data()[j]+margin)*
           (dist_sq_pos.cpu_data()[j]+margin)),
                diff_pos.gpu_data() + (j*dim),
                Dtype(1.0),
                bout + ((2 + num_negatives)*j + i)*dim);
          }
        }
      }
    }
  }
  for (int i = 2; i < 2 + num_negatives; ++i) {
    // BP for negative data(feat3)
    if (propagate_down[0]) {
      const Dtype sign = 1;
      const Dtype alpha = sign * top[0]->cpu_diff()[0] /
          static_cast<Dtype>(num_set);
      for (int j = 0; j < num_set; ++j) {
        Dtype* bout = bottom[0]->mutable_gpu_diff();
        dist_sq_.mutable_cpu_data()[j] = dist_sq_pos.cpu_data()[j];
        dist_sq_.mutable_cpu_data()[j] += margin;
        // Loss component calculated from negative part
        caffe_gpu_sub(
            dim,
            bottom[0]->gpu_data() + (2 + num_negatives)*j*dim,  // reference
            bottom[0]->gpu_data() + ((2 + num_negatives)*j + i)*dim,
            diff_neg.mutable_gpu_data() + j*dim);  // reference-negative
        caffe_gpu_dot(
            dim,
            diff_neg.gpu_data() + j*dim,
            diff_neg.gpu_data() + j*dim,
            dist_sq_neg.mutable_cpu_data() + j);
        // a and negative[triplet] is a dissimilar pair for triplet
        dist_sq_.mutable_cpu_data()[j] = 1 - \
          dist_sq_neg.cpu_data()[j] / dist_sq_.cpu_data()[j];
        // loss accumulated accumulated by the triplet part
        if ((dist_sq_.cpu_data()[j]) > Dtype(0.0)) {
          caffe_gpu_axpby(
              dim,
              alpha/(dist_sq_pos.cpu_data()[j] + margin),
              diff_neg.gpu_data() + (j*dim),
              Dtype(0.0),
              bout + ((2 + num_negatives)*j + i)*dim);
        } else {
            caffe_gpu_set(dim, Dtype(0),
                bout + ((2 + num_negatives)*j + i)*dim);
        }
      }
    }
  }
  } else if (losstype == 2) {
  for (int i = 0; i < 1; ++i) {
    // BP for data1(feat1)
    if (propagate_down[0]) {
    const Dtype sign = 1;
    const Dtype alpha = sign * top[0]->cpu_diff()[0] /
    static_cast<Dtype>(num_set);
    for (int j = 0; j < num_set; ++j) {
      Dtype* bout = bottom[0]->mutable_cpu_diff();
      // the pair part
      if (use_pair == 1) {
      caffe_gpu_axpby(
        dim,
        alpha,
        diff_pos.gpu_data() + (j*dim),
        Dtype(0.0),
        bout + ((2 + num_negatives)*j + i)*dim);
      } else {
      caffe_gpu_axpby(
        dim,
          Dtype(0.0),
        diff_pos.gpu_data() + (j*dim),
        Dtype(0.0),
        bout + ((2 + num_negatives)*j + i)*dim);
      }
      // the num_negatives triplet part
      for (int triplet = 0; triplet < num_negatives; ++triplet) {
      dist_sq_.mutable_cpu_data()[j] =
      exp(dist_sq_pos.mutable_cpu_data()[j]);
      dist_sq_.mutable_cpu_data()[j] += margin;
      // Loss component calculated from negative part
      caffe_gpu_sub(
        dim,
        bottom[0]->gpu_data()+(2 + num_negatives)*j*dim,  // reference
        bottom[0]->gpu_data()+((2 + num_negatives)*j + 2 + triplet)*dim,
        diff_neg.mutable_gpu_data() + j*dim);  // reference-negative
      caffe_gpu_dot(
        dim,
        diff_neg.gpu_data()+j*dim,
        diff_neg.gpu_data()+j*dim,
        dist_sq_neg.mutable_cpu_data() + j);
      // a and negative[triplet] is a dissimilar pair for triplet
      dist_sq_.mutable_cpu_data()[j] = 1 - \
      exp(dist_sq_neg.cpu_data()[j]) / dist_sq_.cpu_data()[j];
      // loss accumulated accumulated by the triplet part
      if ((dist_sq_.cpu_data()[j]) > Dtype(0.0)) {
        caffe_gpu_axpby(
          dim,
          alpha*
            Dtype(exp(dist_sq_neg.cpu_data()[j]))*
            Dtype(exp(dist_sq_pos.cpu_data()[j]))/
            (Dtype((exp(dist_sq_pos.cpu_data()[j]))+margin)*
            (Dtype(exp(dist_sq_pos.cpu_data()[j]))+margin)),
          diff_pos.gpu_data() + (j*dim),
          Dtype(1.0),
          bout + ((2 + num_negatives)*j + i)*dim);
        caffe_cpu_axpby(
          dim,
          -alpha*
          Dtype(exp(dist_sq_neg.cpu_data()[j]))/
            (Dtype(exp(dist_sq_pos.cpu_data()[j]))+margin),
          diff_neg.cpu_data() + (j*dim),
          Dtype(1.0),
          bout + ((2 + num_negatives)*j + i)*dim);
      }
      }
    }
    }
  }
  for (int i = 1; i < 2; ++i) {
    // BP for positive data(feat2)
    if (propagate_down[0]) {
    const Dtype sign = -1;
    const Dtype alpha = sign * top[0]->cpu_diff()[0] /
    static_cast<Dtype>(num_set);
    for (int j = 0; j < num_set; ++j) {
      Dtype* bout = bottom[0]->mutable_cpu_diff();
      // the pair part
      if (use_pair == 1) {
      caffe_gpu_axpby(
        dim,
        alpha,
        diff_pos.gpu_data() + (j*dim),
        Dtype(0.0),
        bout + ((2 + num_negatives)*j + i)*dim);
      } else {
      caffe_gpu_axpby(
        dim,
        Dtype(0.0),
        diff_pos.gpu_data() + (j*dim),
        Dtype(0.0),
        bout + ((2 + num_negatives)*j + i)*dim);
      }
      // the num_negatives triplet part
      for (int triplet = 0; triplet < num_negatives; ++triplet) {
      dist_sq_.mutable_cpu_data()[j] =
      exp(dist_sq_pos.cpu_data()[j]);
      dist_sq_.mutable_cpu_data()[j] += margin;
      // Loss component calculated from negative part
      caffe_gpu_sub(
        dim,
        bottom[0]->gpu_data()+(2 + num_negatives)*j*dim,  // reference
        bottom[0]->gpu_data()+((2 + num_negatives)*j + 2 + triplet)*dim,
        diff_neg.mutable_gpu_data() + j*dim);  // reference-negative
      caffe_gpu_dot(
        dim,
        diff_neg.gpu_data()+j*dim,
        diff_neg.gpu_data()+j*dim,
        dist_sq_neg.mutable_cpu_data() + j);
      // a and negative[triplet] is a dissimilar pair for triplet
      dist_sq_.mutable_cpu_data()[j] = 1 - \
      exp(dist_sq_neg.cpu_data()[j]) / dist_sq_.cpu_data()[j];
      // loss accumulated accumulated by the triplet part
      if ((dist_sq_.cpu_data()[j]) > Dtype(0.0)) {
        caffe_gpu_axpby(
          dim,
          alpha*
          Dtype(exp(dist_sq_neg.cpu_data()[j]))*
          Dtype(exp(dist_sq_pos.cpu_data()[j]))/
            ((Dtype(exp(dist_sq_pos.cpu_data()[j]))+margin)*
            (Dtype(exp(dist_sq_pos.cpu_data()[j]))+margin)),
          diff_pos.gpu_data() + (j*dim),
          Dtype(1.0),
          bout + ((2 + num_negatives)*j + i)*dim);
      }
      }
    }
    }
  }
  for (int i = 2; i < 2 + num_negatives; ++i) {
    // BP for negative data(feat3)
    if (propagate_down[0]) {
    const Dtype sign = 1;
    const Dtype alpha = sign * top[0]->cpu_diff()[0] /
    static_cast<Dtype>(num_set);
    for (int j = 0; j < num_set; ++j) {
      Dtype* bout = bottom[0]->mutable_cpu_diff();
      dist_sq_.mutable_cpu_data()[j] =
      exp(dist_sq_pos.cpu_data()[j]);
      dist_sq_.mutable_cpu_data()[j] += margin;
      // Loss component calculated from negative part
      caffe_gpu_sub(
        dim,
        bottom[0]->gpu_data()+(2 + num_negatives)*j*dim,  // reference
        bottom[0]->gpu_data()+((2 + num_negatives)*j + i)*dim,
        diff_neg.mutable_gpu_data() + j*dim);  // reference-negative
      caffe_gpu_dot(
        dim,
        diff_neg.gpu_data()+j*dim,
        diff_neg.gpu_data()+j*dim,
        dist_sq_neg.mutable_cpu_data() + j);
      // a and negative[triplet] is a dissimilar pair for triplet
      dist_sq_.mutable_cpu_data()[j] = 1 - \
      exp(dist_sq_neg.cpu_data()[j]) / dist_sq_.cpu_data()[j];
      // loss accumulated accumulated by the triplet part
      if ((dist_sq_.cpu_data()[j]) > Dtype(0.0)) {
      caffe_gpu_axpby(
        dim,
        alpha*Dtype(exp(dist_sq_neg.cpu_data()[j]))/
        (Dtype(exp(dist_sq_pos.cpu_data()[j]))+margin),
        diff_neg.gpu_data() + (j*dim),
        Dtype(0.0),
        bout + ((2 + num_negatives)*j + i)*dim);
      } else {
      caffe_set(dim, Dtype(0), bout + ((2 + num_negatives)*j + i)*dim);
      }
    }
    }
  }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(TripletLossLayer);

}  // namespace caffe
