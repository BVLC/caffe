#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/vision_layers.hpp"

namespace caffe {

// Copy (one line per thread) from one array to another, with arbitrary
// strides in the last two dimensions.
template <typename Dtype>
__global__ void copy_kernel(const int n, const int height, const int width,
    const int src_outer_stride, const int src_inner_stride,
    const int dest_outer_stride, const int dest_inner_stride,
    const Dtype* src, Dtype* dest) {
  CUDA_KERNEL_LOOP(index, n) {
    int src_start = index / height * src_outer_stride
                  + index % height * src_inner_stride;
    int dest_start = index / height * dest_outer_stride
                   + index % height * dest_inner_stride;
    for (int i = 0; i < width; ++i) {
      dest[dest_start + i] = src[src_start + i];
    }
  }
}

template <typename Dtype>
void CropLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int lines = top[0]->count() / top[0]->width();

  // NOLINT_NEXT_LINE(whitespace/operators)
  copy_kernel<<<CAFFE_GET_BLOCKS(lines), CAFFE_CUDA_NUM_THREADS>>>(
      lines, top[0]->height(), top[0]->width(),
      bottom[0]->height() * bottom[0]->width(), bottom[0]->width(),
      top[0]->height() * top[0]->width(), top[0]->width(),
      bottom_data + bottom[0]->offset(0, 0, crop_h_, crop_w_), top_data);
}

template <typename Dtype>
void CropLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const int lines = top[0]->count() / top[0]->width();

  if (propagate_down[0]) {
    caffe_gpu_set(bottom[0]->count(), static_cast<Dtype>(0), bottom_diff);
    // NOLINT_NEXT_LINE(whitespace/operators)
    copy_kernel<<<CAFFE_GET_BLOCKS(lines), CAFFE_CUDA_NUM_THREADS>>>(
        lines, top[0]->height(), top[0]->width(),
        top[0]->height() * top[0]->width(), top[0]->width(),
        bottom[0]->height() * bottom[0]->width(), bottom[0]->width(),
        top_diff, bottom_diff + bottom[0]->offset(0, 0, crop_h_, crop_w_));
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(CropLayer);

}  // namespace caffe
