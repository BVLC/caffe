#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layers/embed_layer.hpp"
#include "caffe/util/gpu_util.cuh"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void EmbedForward(const int nthreads, const Dtype* bottom_data,
    const Dtype* weight, const int M, const int N, const int K,
    Dtype* top_data) {
  CUDA_KERNEL_LOOP(top_index, nthreads) {
    const int n = top_index / N;
    const int d = top_index % N;
    const int index = static_cast<int>(bottom_data[n]);
    const int weight_index = index * N + d;
    top_data[top_index] = weight[weight_index];
  }
}

template <typename Dtype>
void EmbedLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const Dtype* weight = this->blobs_[0]->gpu_data();
  const int count = top[0]->count();
  EmbedForward<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, weight, M_, N_, K_, top_data);
  if (bias_term_) {
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, N_, 1, Dtype(1),
        bias_multiplier_.gpu_data(),
        this->blobs_[1]->gpu_data(), Dtype(1), top_data);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(EmbedLayer);

}  // namespace caffe
