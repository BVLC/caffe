#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/blob.hpp"
#include "caffe/common.hpp"
#include "caffe/common_layers.hpp"
#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void EmbedForward(const int nthreads, const Dtype* bottom_data,
    const Dtype* weight, const int M, const int N, const int K,
    Dtype* top_data) {
  CUDA_KERNEL_LOOP(top_index, nthreads) {
    const int n = top_index / N;
    const int d = top_index % N;
    const int index = static_cast<int>(bottom_data[n]);
    const int weight_index = index * N + d;
    top_data[top_index] = weight[weight_index];
  }
}

template <typename Dtype>
__global__ void EmbedBackward(const int nthreads, const Dtype* bottom_data,
    const Dtype* top_diff, const int M, const int N, const int K,
    Dtype* weight_diff) {
  CUDA_KERNEL_LOOP(weight_index, nthreads) {
    const int index = weight_index / N;
    const int output_index = weight_index % N;
    for (int n = 0; n < M; ++n) {
      if (static_cast<int>(bottom_data[n]) == index) {
        weight_diff[weight_index] += top_diff[n * N + output_index];
      }
    }
  }
}

template <typename Dtype>
void EmbedLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const Dtype* weight = this->blobs_[0]->gpu_data();
  const int count = top[0]->count();
  EmbedForward<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, weight, M_, N_, K_, top_data);
  if (bias_term_) {
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, N_, 1, Dtype(1),
        bias_multiplier_.gpu_data(),
        this->blobs_[1]->gpu_data(), Dtype(1), top_data);
  }
}

template <typename Dtype>
void EmbedLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  CHECK(!propagate_down[0]) << "Can't backpropagate to EmbedLayer input.";
  if (this->param_propagate_down_[0]) {
    const int count = this->blobs_[0]->count();
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
    EmbedBackward<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, top_diff, M_, N_, K_, weight_diff);
  }
  if (bias_term_ && this->param_propagate_down_[1]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();
    caffe_gpu_gemv<Dtype>(CblasTrans, M_, N_, Dtype(1), top_diff,
        bias_multiplier_.gpu_data(), Dtype(1), bias_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(EmbedLayer);

}  // namespace caffe
