#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layers/embed_layer.hpp"
#ifdef USE_CUDA
#include "caffe/util/gpu_util.cuh"
#endif  // USE_CUDA
#include "caffe/util/math_functions.hpp"

#ifdef USE_GREENTEA
#include "caffe/greentea/greentea.hpp"
#include "caffe/greentea/greentea_math_functions.hpp"
#endif


namespace caffe {

#ifdef USE_CUDA
template <typename Dtype>
__global__ void EmbedForward(const int_tp nthreads, const Dtype* bottom_data,
    const Dtype* weight, const int_tp M, const int_tp N, const int_tp K,
    Dtype* top_data) {
  CUDA_KERNEL_LOOP(top_index, nthreads) {
    const int_tp n = top_index / N;
    const int_tp d = top_index % N;
    const int_tp index = static_cast<int_tp>(bottom_data[n]);
    const int_tp weight_index = index * N + d;
    top_data[top_index] = weight[weight_index];
  }
}

template <typename Dtype>
__global__ void EmbedBackward(const int_tp nthreads, const Dtype* bottom_data,
    const Dtype* top_diff, const int_tp M, const int_tp N, const int_tp K,
    Dtype* weight_diff) {
  CUDA_KERNEL_LOOP(top_index, nthreads) {
    const int_tp n = top_index / N;
    const int_tp d = top_index % N;
    const int_tp index = static_cast<int_tp>(bottom_data[n]);
    const int_tp weight_index = index * N + d;
    caffe_gpu_atomic_add(top_diff[top_index], weight_diff + weight_index);
  }
}
#endif  // USE_CUDA

template <typename Dtype>
void EmbedLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const Dtype* weight = this->blobs_[0]->gpu_data();
  const int_tp count = top[0]->count();
  if (this->get_device()->backend() == BACKEND_CUDA) {
#ifdef USE_CUDA

    EmbedForward<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
    CUDA_KERNEL(CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS)(
        count, bottom_data, weight, M_, N_, K_, top_data);
    if (bias_term_) {
      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, M_, N_, 1, Dtype(1),
                            bias_multiplier_.gpu_data(),
                            this->blobs_[1]->gpu_data(), Dtype(1), top_data);
    }
#endif  // USE_CUDA
    } else {
#ifdef USE_GREENTEA
      viennacl::ocl::context &ctx = viennacl::ocl::get_context(
          this->device_->id());
      viennacl::ocl::program &program = this->device_->program();

      viennacl::ocl::kernel &oclk_embed = program.get_kernel(
          CL_KERNEL_SELECT("embed_forward"));
      viennacl::ocl::enqueue(
          oclk_embed(count, WrapHandle((cl_mem) bottom_data, &ctx),
                    WrapHandle((cl_mem) weight, &ctx), M_, N_, K_,
                    WrapHandle((cl_mem) top_data, &ctx)),
          ctx.get_queue());

    if (bias_term_) {
      greentea_gpu_gemm<Dtype>(this->get_device()->id(), CblasNoTrans,
                               CblasNoTrans, M_, N_, 1, Dtype(1),
                               (cl_mem) (bias_multiplier_.gpu_data()), 0,
                               (cl_mem) (this->blobs_[1]->gpu_data()), 0,
                               Dtype(1), (cl_mem) top_data, 0);
    }

#endif  // USE_GREENTEA
    }
}

template <typename Dtype>
void EmbedLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  CHECK(!propagate_down[0]) << "Can't backpropagate to EmbedLayer input.";
  if (this->param_propagate_down_[0]) {
    const int_tp top_count = top[0]->count();
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
    if (this->get_device()->backend() == BACKEND_CUDA) {
#ifdef USE_CUDA
    EmbedBackward<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
        CUDA_KERNEL(CAFFE_GET_BLOCKS(top_count), CAFFE_CUDA_NUM_THREADS)(
        top_count, bottom_data, top_diff, M_, N_, K_, weight_diff);
#endif  // USE_CUDA
    } else {
#ifdef USE_GREENTEA
      viennacl::ocl::context &ctx = viennacl::ocl::get_context(
          this->device_->id());
      viennacl::ocl::program &program = this->device_->program();

      viennacl::ocl::kernel &oclk_embed = program.get_kernel(
          CL_KERNEL_SELECT("embed_backward"));
      viennacl::ocl::enqueue(
          oclk_embed(top_count, WrapHandle((cl_mem) bottom_data, &ctx),
                     WrapHandle((cl_mem) top_diff, &ctx), M_, N_, K_,
                     WrapHandle((cl_mem) weight_diff, &ctx)),
          ctx.get_queue());
#endif  // USE_GREENTEA
    }
  }
  if (bias_term_ && this->param_propagate_down_[1]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();
    if (this->get_device()->backend() == BACKEND_CUDA) {
#ifdef USE_CUDA
    caffe_gpu_gemv<Dtype>(CblasTrans, M_, N_, Dtype(1), top_diff,
        bias_multiplier_.gpu_data(), Dtype(1), bias_diff);
#endif  // USE_CUDA
    } else {
#ifdef USE_GREENTEA
      greentea_gpu_gemv<Dtype>(this->get_device()->id(), CblasTrans, M_, N_,
                               Dtype(1), (cl_mem) top_diff, 0,
                               (cl_mem) (bias_multiplier_.gpu_data()), 0,
                               Dtype(1), (cl_mem) bias_diff, 0);
#endif  // USE_GREENTEA
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(EmbedLayer);

}  // namespace caffe

