#include "hip/hip_runtime.h"
#include <algorithm>
#include <cmath>
#include <string>
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/util/hdf5.hpp"
#include "caffe/util/math_functions.hpp"

#include "hdf5.h"
#include "hdf5_hl.h"
#include "stdint.h"

#include "caffe/layers/wasserstein_loss_layer.hpp"

#define DISTANCE_DATASET_NAME "data"

namespace caffe {

template <typename Dtype>
void WassersteinLossLayer<Dtype>::Forward_gpu(
  const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const Dtype* ypred = bottom[0]->gpu_data();
  Dtype* input_ylabel = bottom[1]->mutable_gpu_data();
  Dtype* ylabel = input_ylabel;

  const Dtype* K = K_.gpu_data();
  int num = bottom[0]->num();
  int count = bottom[0]->count();
  int dim = count / num;
  float lambda = this->layer_param_.wasserstein_param().lambda();

  // One-hot encoding.
  // Do this whenever number of label channels is one.
  if (bottom[1]->channels() == 1) {
    tmp_.ReshapeLike(u_);
    ylabel = tmp_.mutable_cpu_data(); //switch to cpu
    caffe_set(count, Dtype(0), ylabel);
    for (int i = 0; i < num; ++i) {
      int label = static_cast<int>(bottom[1]->cpu_data()[i]);
      ylabel[i*dim + label] = Dtype(1);
    }
    ylabel = tmp_.mutable_gpu_data(); //back to gpu
  }

  v_.ReshapeLike(u_);
  Dtype* v = v_.mutable_gpu_data();
  Dtype* u = u_.mutable_gpu_data();
  float val = 1.0;
  caffe_gpu_set<Dtype>(count, val, v);
  caffe_gpu_set<Dtype>(count, val, u);

  uint32_t scaling_iter = this->layer_param_.wasserstein_param().scaling_iter();
  for (int i = 0; i < scaling_iter; i++) {
    // v = ylabel ./ K^t u
    caffe_gpu_gemm(CblasNoTrans, CblasNoTrans, num, dim, dim, Dtype(1.),
                   u, K, Dtype(0.), v);
    caffe_gpu_div(count, ylabel, v, v);

    // u = ypred ./ K v
    caffe_gpu_gemm(CblasNoTrans, CblasTrans, num, dim, dim, Dtype(1.),
                   v, K, Dtype(0.), u);
    caffe_gpu_div(count, ypred, u, u);
  }

  tmp_.ReshapeLike(u_);
  Dtype* tmp = tmp_.mutable_gpu_data();

  tmp2_.ReshapeLike(u_);
  Dtype* tmp2 = tmp2_.mutable_gpu_data();

  // Loss.
  Dtype loss;
  Dtype loss_tmp;

  const Dtype* KM = KM_.gpu_data();
  caffe_gpu_gemm(CblasNoTrans, CblasNoTrans, num, dim, dim, Dtype(1.),
                 u, KM, Dtype(0.), tmp);
  caffe_gpu_dot(count, v, tmp, &loss);

  // (u.logu)^t K v
  caffe_gpu_log(count, u, tmp);
  caffe_gpu_mul(count, u, tmp, tmp);

  caffe_gpu_gemm(CblasNoTrans, CblasNoTrans, num, dim, dim, Dtype(1.0/lambda),
                 tmp, K, Dtype(0.), tmp2);
  caffe_gpu_dot(count, tmp2, v, &loss_tmp);
  loss += loss_tmp;

  // u^t K (v.logv)
  caffe_gpu_log(count, v, tmp);
  caffe_gpu_mul(count, v, tmp, tmp);

  caffe_gpu_gemm(CblasNoTrans, CblasTrans, num, dim, dim, Dtype(1.0/lambda),
                 tmp, K, Dtype(0.), tmp2);
  //tmp2 = tmp2_.mutable_cpu_data();
  caffe_gpu_dot(count, tmp2, u, &loss_tmp);
  loss += loss_tmp;

  // u^t (K.logK) v
  caffe_gpu_gemm(CblasNoTrans, CblasNoTrans, num, dim, dim, Dtype(1.0/lambda),
                 u, KlogK_.gpu_data(), Dtype(0.), tmp);
  caffe_gpu_dot(count, tmp, v, &loss_tmp);
  loss += loss_tmp;

  top[0]->mutable_cpu_data()[0] = loss / num;
}

template <typename Dtype>
void WassersteinLossLayer<Dtype>::Backward_gpu(
    const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
    float lambda = this->layer_param_.wasserstein_param().lambda();
    int num = bottom[0]->num();
    int dim = bottom[0]->count() / num;

    // Gradient.
    // alpha = (1/(lambda*num)) log(u)
    Dtype* alpha = bottom[0]->mutable_gpu_diff();
    caffe_gpu_log(bottom[0]->count(), u_.gpu_data(), alpha);
    caffe_gpu_scal(bottom[0]->count(), Dtype(1.0/(lambda*num)), alpha);

    if (this->layer_param_.wasserstein_param().shift_gradient()) {
      caffe_gpu_gemm(CblasNoTrans, CblasNoTrans, num, dim, dim,
                     Dtype(-1.0/dim), alpha, one_.gpu_data(),
                     Dtype(1.), alpha);
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(WassersteinLossLayer);

}  // namespace caffe
