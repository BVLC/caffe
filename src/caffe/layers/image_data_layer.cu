// Copyright 2014 BVLC and contributors.

#include <hip/hip_runtime.h>
#include <stdint.h>
#include <leveldb/db.h>
#include <pthread.h>

#include <string>
#include <vector>
#include <iostream>  // NOLINT(readability/streams)
#include <fstream>  // NOLINT(readability/streams)

#include "caffe/blob.hpp"
#include "caffe/common.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/vision_layers.hpp"

using std::string;
using std::pair;

namespace caffe {

template <typename Dtype>
Dtype ImageDataLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      vector<Blob<Dtype>*>* top) {
  // First, join the thread
  JoinPrefetchThread();
  // Copy the data
  caffe_copy(prefetch_data_->count(), prefetch_data_->cpu_data(),
      (*top)[0]->mutable_gpu_data());
  caffe_copy(prefetch_label_->count(), prefetch_label_->cpu_data(),
      (*top)[1]->mutable_gpu_data());
  // Start a new prefetch thread
  CreatePrefetchThread();
  return Dtype(0.);
}

INSTANTIATE_CLASS(ImageDataLayer);

}  // namespace caffe
