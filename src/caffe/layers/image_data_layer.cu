// Copyright 2014 BVLC and contributors.

#include <hip/hip_runtime.h>
#include <stdint.h>
#include <leveldb/db.h>
#include <pthread.h>

#include <string>
#include <vector>
#include <iostream>  // NOLINT(readability/streams)
#include <fstream>  // NOLINT(readability/streams)

#include "caffe/blob.hpp"
#include "caffe/common.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/vision_layers.hpp"

using std::string;
using std::pair;

namespace caffe {

template <typename Dtype>
Dtype ImageDataLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      vector<Blob<Dtype>*>* top) {
  // First, join the thread
  JoinPrefetchThread();
  // Copy the data
  CUDA_CHECK(hipMemcpy((*top)[0]->mutable_gpu_data(),
      prefetch_data_->cpu_data(), sizeof(Dtype) * prefetch_data_->count(),
      hipMemcpyDefault));
  CUDA_CHECK(hipMemcpy((*top)[1]->mutable_gpu_data(),
      prefetch_label_->cpu_data(), sizeof(Dtype) * prefetch_label_->count(),
      hipMemcpyDefault));
  // Start a new prefetch thread
  CreatePrefetchThread();
  return Dtype(0.);
}

INSTANTIATE_CLASS(ImageDataLayer);

}  // namespace caffe
