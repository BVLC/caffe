#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layers/bias_layer.hpp"
#include "caffe/util/math_functions.hpp"


#ifdef USE_GREENTEA
#include "caffe/greentea/greentea.hpp"
#include "caffe/greentea/greentea_math_functions.hpp"
#endif

namespace caffe {

#ifdef USE_CUDA
template <typename Dtype>
__global__ void BiasForward(const int_tp n, const Dtype* in,
    const Dtype* bias, const int_tp bias_dim, const int_tp inner_dim,
    Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    const int_tp bias_index = (index / inner_dim) % bias_dim;
    out[index] = in[index] + bias[bias_index];
  }
}
#endif  // USE_CUDA

template <typename Dtype>
void BiasLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const int_tp count = top[0]->count();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* bias_data =
      ((bottom.size() > 1) ? bottom[1] : this->blobs_[0].get())->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();

  if (this->device_->backend() == BACKEND_CUDA) {
#ifdef USE_CUDA
    BiasForward<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
    CUDA_KERNEL(CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS)(
        count, bottom_data, bias_data, bias_dim_, inner_dim_, top_data);
#endif  // USE_CUDA
  } else {
#ifdef USE_GREENTEA
    viennacl::ocl::context &ctx = viennacl::ocl::get_context(
        this->device_->id());
    viennacl::ocl::program &program = this->device_->program();
    viennacl::ocl::kernel &oclk_bias_forward = program.get_kernel(
        CL_KERNEL_SELECT("bias_forward"));
    viennacl::ocl::enqueue(
        oclk_bias_forward(count, WrapHandle((cl_mem) bottom_data, &ctx),
                          WrapHandle((cl_mem) bias_data, &ctx), bias_dim_,
                          inner_dim_, WrapHandle((cl_mem) top_data, &ctx)),
        ctx.get_queue());
#endif  // USE_GREENTEA
  }
}

template<typename Dtype>
void BiasLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                    const vector<bool>& propagate_down,
                                    const vector<Blob<Dtype>*>& bottom) {
  if (this->device_->backend() == BACKEND_CUDA) {
#ifdef USE_CUDA
    if (propagate_down[0] && bottom[0] != top[0]) {
      const Dtype* top_diff = top[0]->gpu_diff();
      Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
      caffe_copy(bottom[0]->count(), top_diff, bottom_diff);
    }
    // in-place, we don't need to do anything with the data diff
    const bool bias_param = (bottom.size() == 1);
    if ((!bias_param && propagate_down[1])
        || (bias_param && this->param_propagate_down_[0])) {
      const Dtype* top_diff = top[0]->gpu_diff();
      Dtype* bias_diff = (bias_param ? this->blobs_[0].get() : bottom[1])
          ->mutable_gpu_diff();
      bool accum = bias_param;

      for (int_tp n = 0; n < outer_dim_; ++n) {
        caffe_gpu_gemv(CblasNoTrans, bias_dim_, inner_dim_, Dtype(1), top_diff,
                       bias_multiplier_.gpu_data(), Dtype(accum), bias_diff);
        top_diff += dim_;
        accum = true;
      }
    }
#endif  // USE_CUDA
  } else {
#ifdef USE_GREENTEA
    viennacl::ocl::context &ctx = viennacl::ocl::get_context(
        this->device_->id());

    if (propagate_down[0] && bottom[0] != top[0]) {
      const Dtype* top_diff = top[0]->gpu_diff();
      Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
      greentea_copy<Dtype>(bottom[0]->count(), (cl_mem) top_diff, 0,
                           (cl_mem) bottom_diff, 0, &ctx);
    }
    // in-place, we don't need to do anything with the data diff
    const bool bias_param = (bottom.size() == 1);
    if ((!bias_param && propagate_down[1])
        || (bias_param && this->param_propagate_down_[0])) {
      const Dtype* top_diff = top[0]->gpu_diff();
      Dtype* bias_diff = (bias_param ? this->blobs_[0].get() : bottom[1])
          ->mutable_gpu_diff();
      bool accum = bias_param;

      int_tp top_diff_off = 0;
      for (int_tp n = 0; n < outer_dim_; ++n) {
        greentea_gpu_gemv(this->device_->id(), CblasNoTrans, bias_dim_,
                          inner_dim_, Dtype(1), (cl_mem) top_diff, top_diff_off,
                          (cl_mem) (bias_multiplier_.gpu_data()), 0,
                          Dtype(accum), (cl_mem) bias_diff, 0);
        top_diff_off += dim_;
        accum = true;
      }
    }
#endif  // USE_GREENTEA
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(BiasLayer);

}  // namespace caffe
