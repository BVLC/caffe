#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/euclidean_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void EuclideanLossForwardGPU(const int n,
          const Dtype* label_data_, Dtype* bottom_data_,
          const int ignore_label_) {
  CUDA_KERNEL_LOOP(index, n) {
    const int label_value = static_cast<int>(label_data_[index]);
    if (label_value == ignore_label_) {
      bottom_data_[index] = label_data_[index];
    }
  }
}

template <typename Dtype>
void EuclideanLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  int count = bottom[0]->count();
  if (has_ignore_label_) {
    EuclideanLossForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(count),
        CAFFE_CUDA_NUM_THREADS>>>(count, bottom[1]->gpu_data(),
                                  bottom[0]->mutable_gpu_data(),
                                  ignore_label_);
  }
  caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),
      bottom[1]->gpu_data(),
      diff_.mutable_gpu_data());
  Dtype dot;
  caffe_gpu_dot(count, diff_.gpu_data(), diff_.gpu_data(), &dot);
  Dtype loss = dot / bottom[0]->num() / Dtype(2);
  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
void EuclideanLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  for (int i = 0; i < 2; ++i) {
    if (propagate_down[i]) {
      const Dtype sign = (i == 0) ? 1 : -1;
      const Dtype alpha = sign * top[0]->cpu_diff()[0] / bottom[i]->num();
      caffe_gpu_axpby(
          bottom[i]->count(),              // count
          alpha,                              // alpha
          diff_.gpu_data(),                   // a
          Dtype(0),                           // beta
          bottom[i]->mutable_gpu_diff());  // b
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(EuclideanLossLayer);

}  // namespace caffe
