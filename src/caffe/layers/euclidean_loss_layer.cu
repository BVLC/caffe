// Copyright 2014 BVLC and contributors.

#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
Dtype EuclideanLossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  int count = bottom[0]->count();
  caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),
      bottom[1]->gpu_data(),
      diff_.mutable_gpu_data());
  Dtype dot;
  caffe_gpu_dot(count, diff_.gpu_data(), diff_.gpu_data(), &dot);
  Dtype loss = dot / bottom[0]->num() / Dtype(2);
  return loss;
}

template <typename Dtype>
void EuclideanLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, vector<Blob<Dtype>*>* bottom) {
  for (int i = 0; i < 2; ++i) {
    if (propagate_down[i]) {
      const Dtype sign = (i == 0) ? 1 : -1;
      caffe_gpu_axpby(
          (*bottom)[i]->count(),              // count
          sign / (*bottom)[i]->num(),         // alpha
          diff_.gpu_data(),                   // a
          Dtype(0),                           // beta
          (*bottom)[i]->mutable_gpu_diff());  // b
    }
  }
}

INSTANTIATE_CLASS(EuclideanLossLayer);

}  // namespace caffe
