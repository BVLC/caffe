#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "thrust/device_vector.h"

#include "caffe/common_layers.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void kernel_channel_max(const int num, const int channels,
    const int spatial_dim, const Dtype* data, Dtype* out, const int slice) {
  CUDA_KERNEL_LOOP(index, num * spatial_dim) {
    int n = index / spatial_dim;
    int s = index % spatial_dim;
    Dtype maxval = -FLT_MAX;
    for (int c = 0; c < channels; ++c) {
      maxval = max(data[(n * channels * slice + c) * spatial_dim + s], maxval);
    }
    out[index] = maxval;
  }
}

template <typename Dtype>
__global__ void kernel_channel_subtract(const int count,
    const int num, const int channels,
    const int spatial_dim, const Dtype* channel_max, Dtype* data, int slice, int num_slice) {
  CUDA_KERNEL_LOOP(index, count) {
    int n = index / channels / spatial_dim;
    int s = index % spatial_dim;
    int start = channels * num_slice;
    int tot_channels = channels * slice;
    int row =  index/channels;
    int m = index % channels;
    int new_index = start + (row*tot_channels) + m;
    data[new_index] -= channel_max[n * spatial_dim + s];
  }
}

template <typename Dtype>
__global__ void kernel_exp(const int count, const Dtype* data, Dtype* out) {
  CUDA_KERNEL_LOOP(index, count) {
    out[index] = exp(data[index]);
  }
}

template <typename Dtype>
__global__ void kernel_channel_sum(const int num, const int channels,
    const int spatial_dim, const Dtype* data, Dtype* channel_sum, int slice) {
  CUDA_KERNEL_LOOP(index, num * spatial_dim) {
    int n = index / spatial_dim;
    int s = index % spatial_dim;
    Dtype sum = 0;
    for (int c = 0; c < channels; ++c) {
      sum += data[(n * channels * slice + c) * spatial_dim + s];
    }
    channel_sum[index] = sum;
  }
}

template <typename Dtype>
__global__ void kernel_channel_div(const int count,
    const int num, const int channels,
    const int spatial_dim, const Dtype* channel_sum, Dtype* data, int slice, int num_slice) {
  CUDA_KERNEL_LOOP(index, count) {
    int n = index / channels / spatial_dim;
    int s = index % spatial_dim;
    
    
    int start = channels * num_slice;
    int tot_channels = channels * slice;
    int row =  index/channels;
    int m = index % channels;
    int new_index = start + (row*tot_channels) + m;
    data[new_index] /= channel_sum[n * spatial_dim + s];
  }
}

template <typename Dtype>
__global__ void kernel_channel_dot(const int num, const int channels,
    const int spatial_dim, const Dtype* data_1, const Dtype* data_2,
    Dtype* channel_dot) {
  CUDA_KERNEL_LOOP(index, num * spatial_dim) {
    int n = index / spatial_dim;
    int s = index % spatial_dim;
    Dtype dot = 0;
    for (int c = 0; c < channels; ++c) {
      dot += (data_1[(n * channels + c) * spatial_dim + s]
          * data_2[(n * channels + c) * spatial_dim + s]);
    }
    channel_dot[index] = dot;
  }
}

template <typename Dtype>
void SoftmaxLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  Dtype* scale_data = scale_.mutable_gpu_data();
  int count = bottom[0]->count();
  int channels = top[0]->shape(softmax_axis_);
  caffe_copy(count, bottom_data, top_data);

  int step = channels/slice_;
  
  for(int fp = 0; fp < slice_; ++fp) {
    int offset_step = (fp * step) * (inner_num_);
    // We need to subtract the max to avoid numerical issues, compute the exp,
    // and then normalize.
    // compute max
    // NOLINT_NEXT_LINE(whitespace/operators)
    kernel_channel_max<Dtype><<<CAFFE_GET_BLOCKS(outer_num_ * inner_num_),
        CAFFE_CUDA_NUM_THREADS>>>(outer_num_, step, inner_num_, top_data + offset_step,
        scale_data + offset_step, slice_);
        
        
    // subtract
    // NOLINT_NEXT_LINE(whitespace/operators)
    kernel_channel_subtract<Dtype><<<CAFFE_GET_BLOCKS(count/slice_),
        CAFFE_CUDA_NUM_THREADS>>>(count/slice_, outer_num_, step, inner_num_,
        scale_data + offset_step, top_data + offset_step, slice_, fp);
        
  }
    // exponentiate
    // NOLINT_NEXT_LINE(whitespace/operators)
    kernel_exp<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_data, top_data);
        
  for(int fp = 0; fp < slice_; ++fp) {
    int offset_step = (fp * step) * (inner_num_);
    // sum after exp
    // NOLINT_NEXT_LINE(whitespace/operators)
    kernel_channel_sum<Dtype><<<CAFFE_GET_BLOCKS(outer_num_ * inner_num_),
        CAFFE_CUDA_NUM_THREADS>>>(outer_num_, step, inner_num_, top_data + offset_step,
        scale_data + offset_step, slice_);
    // divide
    // NOLINT_NEXT_LINE(whitespace/operators)
    kernel_channel_div<Dtype><<<CAFFE_GET_BLOCKS(count/slice_),
        CAFFE_CUDA_NUM_THREADS>>>(count/slice_, outer_num_, step, inner_num_,
        scale_data + offset_step, top_data + offset_step, slice_, fp);
  }
  
}

template <typename Dtype>
void SoftmaxLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* top_data = top[0]->gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  Dtype* scale_data = scale_.mutable_gpu_data();
  int count = top[0]->count();
  int channels = top[0]->shape(softmax_axis_);
  caffe_copy(count, top_diff, bottom_diff);
  // Compute inner1d(top_diff, top_data) and subtract them from the bottom diff.
  // NOLINT_NEXT_LINE(whitespace/operators)
  kernel_channel_dot<Dtype><<<CAFFE_GET_BLOCKS(outer_num_ * inner_num_),
      CAFFE_CUDA_NUM_THREADS>>>(outer_num_, channels, inner_num_,
      top_diff, top_data, scale_data);
  // NOLINT_NEXT_LINE(whitespace/operators)
  /*kernel_channel_subtract<Dtype><<<CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS>>>(count, outer_num_, channels, inner_num_,
      scale_data, bottom_diff);*/
  // elementwise multiplication
  caffe_gpu_mul<Dtype>(top[0]->count(), bottom_diff, top_data, bottom_diff);
}

INSTANTIATE_LAYER_GPU_FUNCS(SoftmaxLayer);


}  // namespace caffe
