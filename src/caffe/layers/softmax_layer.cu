#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#ifdef USE_CUDA
#include "thrust/device_vector.h"
#endif

#include "caffe/layers/softmax_layer.hpp"
#include "caffe/util/math_functions.hpp"

#ifdef USE_GREENTEA
#include "caffe/greentea/greentea_im2col.hpp"
#include "caffe/greentea/greentea_math_functions.hpp"
#endif

namespace caffe {

#ifdef USE_CUDA
template<typename Dtype>
__global__ void kernel_channel_max(const int_tp num, const int_tp channels,
                                   const int_tp spatial_dim, const Dtype* data,
                                   Dtype* out) {
  CUDA_KERNEL_LOOP(index, num * spatial_dim) {
    int_tp n = index / spatial_dim;
    int_tp s = index % spatial_dim;
    Dtype maxval = -FLT_MAX;
    for (int_tp c = 0; c < channels; ++c) {
      maxval = max(data[(n * channels + c) * spatial_dim + s], maxval);
    }
    out[index] = maxval;
  }
}

template<typename Dtype>
__global__ void kernel_channel_subtract(const int_tp count, const int_tp num,
                                        const int_tp channels,
                                        const int_tp spatial_dim,
                                        const Dtype* channel_max, Dtype* data) {
  CUDA_KERNEL_LOOP(index, count) {
    int_tp n = index / channels / spatial_dim;
    int_tp s = index % spatial_dim;
    data[index] -= channel_max[n * spatial_dim + s];
  }
}

template<typename Dtype>
__global__ void kernel_exp(const int_tp count, const Dtype* data, Dtype* out) {
  CUDA_KERNEL_LOOP(index, count) {
    out[index] = exp(data[index]);
  }
}

template<typename Dtype>
__global__ void kernel_channel_sum(const int_tp num, const int_tp channels,
                                   const int_tp spatial_dim, const Dtype* data,
                                   Dtype* channel_sum) {
  CUDA_KERNEL_LOOP(index, num * spatial_dim) {
    int_tp n = index / spatial_dim;
    int_tp s = index % spatial_dim;
    Dtype sum = 0;
    for (int_tp c = 0; c < channels; ++c) {
      sum += data[(n * channels + c) * spatial_dim + s];
    }
    channel_sum[index] = sum;
  }
}

template<typename Dtype>
__global__ void kernel_channel_div(const int_tp count, const int_tp num,
                                   const int_tp channels,
                                   const int_tp spatial_dim,
                                   const Dtype* channel_sum, Dtype* data) {
  CUDA_KERNEL_LOOP(index, count) {
    int_tp n = index / channels / spatial_dim;
    int_tp s = index % spatial_dim;
    data[index] /= channel_sum[n * spatial_dim + s];
  }
}

template<typename Dtype>
__global__ void kernel_channel_dot(const int_tp num, const int_tp channels,
                                   const int_tp spatial_dim,
                                   const Dtype* data_1, const Dtype* data_2,
                                   Dtype* channel_dot) {
  CUDA_KERNEL_LOOP(index, num * spatial_dim) {
    int_tp n = index / spatial_dim;
    int_tp s = index % spatial_dim;
    Dtype dot = 0;
    for (int_tp c = 0; c < channels; ++c) {
      dot += (data_1[(n * channels + c) * spatial_dim + s]
          * data_2[(n * channels + c) * spatial_dim + s]);
    }
    channel_dot[index] = dot;
  }
}
#endif

template<typename Dtype>
void SoftmaxLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  Dtype* scale_data = scale_.mutable_gpu_data();
  int_tp count = bottom[0]->count();
  int_tp channels = top[0]->shape(softmax_axis_);

  if (this->device_->backend() == BACKEND_CUDA) {
#ifdef USE_CUDA
    // CUDA backend code
    caffe_copy(count, bottom_data, top_data);
    // We need to subtract the max to avoid numerical issues, compute the exp,
    // and then normalize.
    // compute max
    // NOLINT_NEXT_LINE(whitespace/operators)
    kernel_channel_max<Dtype> CUDA_KERNEL(
        CAFFE_GET_BLOCKS(outer_num_ * inner_num_),
        CAFFE_CUDA_NUM_THREADS)(outer_num_, channels, inner_num_, top_data,
        scale_data);
    // subtract
    // NOLINT_NEXT_LINE(whitespace/operators)
    kernel_channel_subtract<Dtype> CUDA_KERNEL(CAFFE_GET_BLOCKS(count),
        CAFFE_CUDA_NUM_THREADS)(count, outer_num_, channels, inner_num_,
        scale_data, top_data);
    // exponentiate
    // NOLINT_NEXT_LINE(whitespace/operators)
    kernel_exp<Dtype> CUDA_KERNEL(
        CAFFE_GET_BLOCKS(count),
        CAFFE_CUDA_NUM_THREADS)(count, top_data,
        top_data);
    // sum after exp
    // NOLINT_NEXT_LINE(whitespace/operators)
    kernel_channel_sum<Dtype> CUDA_KERNEL(
        CAFFE_GET_BLOCKS(outer_num_ * inner_num_),
        CAFFE_CUDA_NUM_THREADS)(outer_num_, channels,
            inner_num_, top_data, scale_data);
    // divide
    // NOLINT_NEXT_LINE(whitespace/operators)
    kernel_channel_div<Dtype> CUDA_KERNEL(CAFFE_GET_BLOCKS(count),
        CAFFE_CUDA_NUM_THREADS)(count, outer_num_, channels, inner_num_,
        scale_data, top_data);
#endif
  } else {
#ifdef USE_GREENTEA
    viennacl::ocl::context &ctx = viennacl::ocl::get_context(
        this->device_->id());
    viennacl::ocl::program &program = this->device_->program();

    greentea_copy<Dtype>(count, (cl_mem) bottom_data, 0, (cl_mem) top_data, 0,
                         &ctx);

    viennacl::ocl::kernel &oclk_channel_max = program.get_kernel(
        CL_KERNEL_SELECT("kernel_channel_max"));
    viennacl::ocl::enqueue(
        oclk_channel_max(outer_num_, channels, inner_num_,
                         WrapHandle((cl_mem) top_data, &ctx),
                         WrapHandle((cl_mem) scale_data, &ctx)),
        ctx.get_queue());

    viennacl::ocl::kernel &oclk_channel_subtract = program.get_kernel(
        CL_KERNEL_SELECT("kernel_channel_subtract"));
    viennacl::ocl::enqueue(
        oclk_channel_subtract(count, outer_num_, channels, inner_num_,
                              WrapHandle((cl_mem) scale_data, &ctx),
                              WrapHandle((cl_mem) top_data, &ctx)),
        ctx.get_queue());

    viennacl::ocl::kernel &oclk_exp = program.get_kernel(
        CL_KERNEL_SELECT("kernel_exp"));
    viennacl::ocl::enqueue(
        oclk_exp(count,
                 WrapHandle((cl_mem) top_data, &ctx),
                 WrapHandle((cl_mem) top_data, &ctx)),
        ctx.get_queue());

    viennacl::ocl::kernel &oclk_channel_sum = program.get_kernel(
        CL_KERNEL_SELECT("kernel_channel_sum"));
    viennacl::ocl::enqueue(
        oclk_channel_sum(outer_num_, channels, inner_num_,
                         WrapHandle((cl_mem) top_data, &ctx),
                         WrapHandle((cl_mem) scale_data, &ctx)),
        ctx.get_queue());

    viennacl::ocl::kernel &oclk_channel_div = program.get_kernel(
        CL_KERNEL_SELECT("kernel_channel_div"));
    viennacl::ocl::enqueue(
        oclk_channel_div(count, outer_num_, channels, inner_num_,
                         WrapHandle((cl_mem) scale_data, &ctx),
                         WrapHandle((cl_mem) top_data, &ctx)),
        ctx.get_queue());

#endif
  }
}

template<typename Dtype>
void SoftmaxLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                       const vector<bool>& propagate_down,
                                       const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* top_data = top[0]->gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  Dtype* scale_data = scale_.mutable_gpu_data();
  int_tp count = top[0]->count();
  int_tp channels = top[0]->shape(softmax_axis_);

  if (this->device_->backend() == BACKEND_CUDA) {
#ifdef USE_CUDA
    caffe_copy(top[0]->count(), top_diff, bottom_diff);
    // Compute inner1d(top_diff, top_data) and
    // subtract them from the bottom diff.
    // NOLINT_NEXT_LINE(whitespace/operators)
    kernel_channel_dot<Dtype> CUDA_KERNEL(
        CAFFE_GET_BLOCKS(outer_num_ * inner_num_),
        CAFFE_CUDA_NUM_THREADS)(outer_num_, channels, inner_num_,
            top_diff, top_data, scale_data);
    // NOLINT_NEXT_LINE(whitespace/operators)
    kernel_channel_subtract<Dtype> CUDA_KERNEL(CAFFE_GET_BLOCKS(count),
        CAFFE_CUDA_NUM_THREADS)(count, outer_num_, channels, inner_num_,
        scale_data, bottom_diff);
    // elementwise multiplication
    caffe_gpu_mul<Dtype>(top[0]->count(), bottom_diff, top_data, bottom_diff);
#endif
  } else {
#ifdef USE_GREENTEA

    viennacl::ocl::context &ctx = viennacl::ocl::get_context(
        this->device_->id());
    viennacl::ocl::program &program = this->device_->program();

    greentea_copy<Dtype>(top[0]->count(), (cl_mem)top_diff,
                         0, (cl_mem)bottom_diff, 0, &ctx);

    viennacl::ocl::kernel &oclk_channel_dot = program.get_kernel(
        CL_KERNEL_SELECT("kernel_channel_dot"));
    viennacl::ocl::enqueue(
        oclk_channel_dot(outer_num_, channels, inner_num_,
                         WrapHandle((cl_mem)top_diff, &ctx),
                         WrapHandle((cl_mem)top_data, &ctx),
                         WrapHandle((cl_mem)scale_data, &ctx)),
        ctx.get_queue());

    viennacl::ocl::kernel &oclk_channel_subtract = program.get_kernel(
        CL_KERNEL_SELECT("kernel_channel_subtract"));
    viennacl::ocl::enqueue(
        oclk_channel_subtract(count, outer_num_, channels, inner_num_,
                              WrapHandle((cl_mem)scale_data, &ctx),
                              WrapHandle((cl_mem)bottom_diff, &ctx)),
        ctx.get_queue());

    greentea_gpu_mul<Dtype>(this->device_->id(), top[0]->count(),
                            (cl_mem)bottom_diff, 0,
                            (cl_mem)top_data, 0, (cl_mem)bottom_diff, 0);

#endif
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SoftmaxLayer);

}  // namespace caffe
