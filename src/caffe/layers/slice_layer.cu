// Copyright 2014 BVLC and contributors.

#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
Dtype SliceLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      vector<Blob<Dtype>*>* top) {
  const Dtype* bottom_data = bottom[0]->mutable_gpu_data();
  if (slice_dim_ == 0) {
    int offset_num = 0;
    for (int i = 0; i < top->size(); ++i) {
      Blob<Dtype>* blob = (*top)[i];
      Dtype* top_data = blob->mutable_gpu_data();
      caffe_copy(blob->count(), bottom_data + bottom[0]->offset(offset_num),
                 top_data);
      offset_num += blob->num();
    }
  } else if (slice_dim_ == 1) {
    int offset_channel = 0;
    for (int i = 0; i < top->size(); ++i) {
      Blob<Dtype>* blob = (*top)[i];
      Dtype* top_data = blob->mutable_gpu_data();
      const int num_elem = blob->channels() * blob->height() * blob->width();
      for (int n = 0; n < num_; ++n) {
        caffe_copy(num_elem, bottom_data + bottom[0]->offset(n, offset_channel),
                   top_data + blob->offset(n));
      }
      offset_channel += blob->channels();
    }
  }  // slice_dim_ is guaranteed to be 0 or 1 by SetUp.
  return Dtype(0.);
}

template <typename Dtype>
void SliceLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, vector<Blob<Dtype>*>* bottom) {
  if (!propagate_down[0]) { return; }
  Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff();
  if (slice_dim_ == 0) {
    int offset_num = 0;
    for (int i = 0; i < top.size(); ++i) {
      Blob<Dtype>* blob = top[i];
      const Dtype* top_diff = blob->gpu_diff();
      caffe_copy(blob->count(), top_diff,
                 bottom_diff + (*bottom)[0]->offset(offset_num));
      offset_num += blob->num();
    }
  } else if (slice_dim_ == 1) {
    int offset_channel = 0;
    for (int i = 0; i < top.size(); ++i) {
      Blob<Dtype>* blob = top[i];
      const Dtype* top_diff = blob->gpu_diff();
      const int num_elem = blob->channels() * blob->height() * blob->width();
      for (int n = 0; n < num_; ++n) {
        caffe_copy(num_elem, top_diff + blob->offset(n),
                   bottom_diff +  (*bottom)[0]->offset(n, offset_channel));
      }
      offset_channel += blob->channels();
    }
  }  // slice_dim_ is guaranteed to be 0 or 1 by SetUp.
}

INSTANTIATE_CLASS(SliceLayer);

}  // namespace caffe
