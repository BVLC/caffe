#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void Slice(const int nthreads, const Dtype* in_data,
    const bool forward, const int num_slices, const int slice_size,
    const int bottom_slice_axis, const int top_slice_axis,
    const int offset_slice_axis, Dtype* out_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int total_slice_size = slice_size * top_slice_axis;
    const int slice_num = index / total_slice_size;
    const int slice_index = index % total_slice_size;
    const int bottom_index = slice_index +
        (slice_num * bottom_slice_axis + offset_slice_axis) * slice_size;
    if (forward) {
      out_data[index] = in_data[bottom_index];
    } else {
      out_data[bottom_index] = in_data[index];
    }
  }
}

template <typename Dtype>
void SliceLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  int offset_slice_axis = 0;
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const int bottom_slice_axis = bottom[0]->shape(slice_axis_);
  const bool kForward = true;
  for (int i = 0; i < top.size(); ++i) {
    Dtype* top_data = top[i]->mutable_gpu_data();
    const int top_slice_axis = top[i]->shape(slice_axis_);
    const int top_slice_size = top_slice_axis * slice_size_;
    const int nthreads = top_slice_size * num_slices_;
    Slice<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
        nthreads, bottom_data, kForward, num_slices_, slice_size_,
        bottom_slice_axis, top_slice_axis, offset_slice_axis, top_data);
    offset_slice_axis += top_slice_axis;
  }
}

template <typename Dtype>
void SliceLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) { return; }
  int offset_slice_axis = 0;
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const int bottom_slice_axis = bottom[0]->shape(slice_axis_);
  const bool kForward = false;
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    const int top_slice_axis = top[i]->shape(slice_axis_);
    const int top_slice_size = top_slice_axis * slice_size_;
    const int nthreads = top_slice_size * num_slices_;
    Slice<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
        nthreads, top_diff, kForward, num_slices_, slice_size_,
        bottom_slice_axis, top_slice_axis, offset_slice_axis, bottom_diff);
    offset_slice_axis += top_slice_axis;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SliceLayer);

}  // namespace caffe
