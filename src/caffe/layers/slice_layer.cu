#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/slice_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void Slice(const int nthreads, const Dtype* in_data,
    const bool forward, const int num_slices, const int slice_size,
    const int bottom_slice_axis, const int top_slice_axis,
    const int offset_slice_axis, Dtype* out_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int total_slice_size = slice_size * top_slice_axis;
    const int slice_num = index / total_slice_size;
    const int slice_index = index % total_slice_size;
    const int bottom_index = slice_index +
        (slice_num * bottom_slice_axis + offset_slice_axis) * slice_size;
    if (forward) {
      out_data[index] = in_data[bottom_index];
    } else {
      out_data[bottom_index] = in_data[index];
    }
  }
}

template <typename Dtype>
void SliceLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  Forward_const_gpu(bottom,top);
}

template <typename Dtype>
void SliceLayer<Dtype>::Forward_const_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) const {
  if (top.size() == 1) { return; }
  int offset_slice_axis = 0;
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const int slice_axis=get_slice_axis(bottom);
  const int bottom_slice_axis = bottom[0]->shape(slice_axis);
  const bool kForward = true;
  const int num_slices = bottom[0]->count(0, slice_axis);
  const int slice_size = bottom[0]->count(slice_axis + 1);
  for (int i = 0; i < top.size(); ++i) {
    Dtype* top_data = top[i]->mutable_gpu_data();
    const int top_slice_axis = top[i]->shape(slice_axis);
    const int top_slice_size = top_slice_axis * slice_size;
    const int nthreads = top_slice_size * num_slices;
    Slice<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
        nthreads, bottom_data, kForward, num_slices, slice_size,
        bottom_slice_axis, top_slice_axis, offset_slice_axis, top_data);
    offset_slice_axis += top_slice_axis;
  }
}


INSTANTIATE_LAYER_GPU_FUNCS_CONST(SliceLayer);

}  // namespace caffe
