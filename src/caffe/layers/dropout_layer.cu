#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/dropout_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void DropoutForward(const int n, const Dtype* in,
    const unsigned int* mask, const unsigned int threshold, const float scale,
    Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] * (mask[index] > threshold) * scale;
  }
}

template <typename Dtype>
void DropoutLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  if (this->phase_ == TRAIN) {
    unsigned int* mask =
        static_cast<unsigned int*>(rand_vec_.mutable_gpu_data());
    caffe_gpu_rng_uniform(count, mask);
    // set thresholds
    if (scale_train_) {
      // NOLINT_NEXT_LINE(whitespace/operators)
      DropoutForward<Dtype><<<CAFFE_GET_BLOCKS(count),
        CAFFE_CUDA_NUM_THREADS>>>(
          count, bottom_data, mask, uint_thres_, scale_, top_data);
    } else {
      // NOLINT_NEXT_LINE(whitespace/operators)
      DropoutForward<Dtype><<<CAFFE_GET_BLOCKS(count),
        CAFFE_CUDA_NUM_THREADS>>>(
          count, bottom_data, mask, uint_thres_, 1.f, top_data);
    }
    CUDA_POST_KERNEL_CHECK;
  } else {
    caffe_copy(count, bottom_data, top_data);
    if (!scale_train_) {
      caffe_gpu_scal<Dtype>(count, 1. / scale_, top_data);
    }
  }
}

template <typename Dtype>
__global__ void DropoutBackward(const int n, const Dtype* in_diff,
    const unsigned int* mask, const unsigned int threshold, const float scale,
    Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * scale * (mask[index] > threshold);
  }
}

template <typename Dtype>
void DropoutLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    if (this->phase_ == TRAIN) {
      const unsigned int* mask =
          static_cast<const unsigned int*>(rand_vec_.gpu_data());
      const int count = bottom[0]->count();
      if (scale_train_) {
        // NOLINT_NEXT_LINE(whitespace/operators)
        DropoutBackward<Dtype><<<CAFFE_GET_BLOCKS(count),
          CAFFE_CUDA_NUM_THREADS>>>(
            count, top_diff, mask, uint_thres_, scale_, bottom_diff);
      } else {
        // NOLINT_NEXT_LINE(whitespace/operators)
        DropoutBackward<Dtype><<<CAFFE_GET_BLOCKS(count),
          CAFFE_CUDA_NUM_THREADS>>>(
           count, top_diff, mask, uint_thres_, 1.f, bottom_diff);
      }
      CUDA_POST_KERNEL_CHECK;
    } else {
      caffe_copy(top[0]->count(), top_diff, bottom_diff);
      if (!scale_train_) {
        caffe_gpu_scal<Dtype>(top[0]->count(), 1. / scale_, bottom_diff);
      }
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(DropoutLayer);

}  // namespace caffe
