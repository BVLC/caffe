#include "hip/hip_runtime.h"
// Copyright 2013 Yangqing Jia

#include <algorithm>
#include <limits>
#include <vector>

#include "caffe/common.hpp"
#include "caffe/layer.hpp"
#include "caffe/syncedmem.hpp"
#include "caffe/vision_layers.hpp"

using std::max;

namespace caffe {

template <typename Dtype>
void DropoutLayer<Dtype>::SetUp(const vector<Blob<Dtype>*>& bottom,
      vector<Blob<Dtype>*>* top) {
  NeuronLayer<Dtype>::SetUp(bottom, top);
  // Set up the cache for random number generation
  rand_vec_.reset(new SyncedMemory(bottom[0]->count() * sizeof(int)));
  threshold_ = this->layer_param_.dropout_ratio();
  DCHECK(threshold_ > 0.);
  DCHECK(threshold_ < 1.);
  scale_ = 1. / (1. - threshold_);
  uint_thres_ = (unsigned int)(UINT_MAX * threshold_);
}

template <typename Dtype>
void DropoutLayer<Dtype>::Forward_cpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  const Dtype* bottom_data = bottom[0]->cpu_data();
  Dtype* top_data = (*top)[0]->mutable_cpu_data();
  int* mask = reinterpret_cast<int*>(rand_vec_->mutable_cpu_data());
  const int count = bottom[0]->count();
  if (Caffe::phase() == Caffe::TRAIN) {
    // Create random numbers
    viRngBernoulli(VSL_RNG_METHOD_BERNOULLI_ICDF, Caffe::vsl_stream(),
        count, mask, 1. - threshold_);
    for (int i = 0; i < count; ++i) {
      top_data[i] = bottom_data[i] * mask[i] * scale_;
    }
  } else {
    memcpy(top_data, bottom_data, bottom[0]->count() * sizeof(Dtype));
  }
}

template <typename Dtype>
Dtype DropoutLayer<Dtype>::Backward_cpu(const vector<Blob<Dtype>*>& top,
    const bool propagate_down,
    vector<Blob<Dtype>*>* bottom) {
  CHECK(Caffe::phase() == Caffe::TRAIN);
  if (propagate_down) {
    const Dtype* top_diff = top[0]->cpu_diff();
    Dtype* bottom_diff = (*bottom)[0]->mutable_cpu_diff();
    const int* mask = reinterpret_cast<const int*>(rand_vec_->cpu_data());
    const int count = (*bottom)[0]->count();
    for (int i = 0; i < count; ++i) {
      bottom_diff[i] = top_diff[i] * mask[i] * scale_;
    }
  }
  return Dtype(0);
}

template <typename Dtype>
__global__ void DropoutForward(const int n, const Dtype* in,
    const unsigned int* mask, const unsigned int threshold, const float scale,
    Dtype* out) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < n) {
    out[index] = in[index] * (mask[index] > threshold) * scale;
  }
}

template <typename Dtype>
void DropoutLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = (*top)[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  if (Caffe::phase() == Caffe::TRAIN) {
    CURAND_CHECK(hiprandGenerate(Caffe::curand_generator(),
        (unsigned int*)(rand_vec_->mutable_gpu_data()), count));
    // set thresholds
    // NOLINT_NEXT_LINE(whitespace/operators)
    DropoutForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, (unsigned int*)rand_vec_->gpu_data(), uint_thres_,
        scale_, top_data);
    CUDA_POST_KERNEL_CHECK;
  } else {
    CUDA_CHECK(hipMemcpy(top_data, bottom_data,
        count * sizeof(Dtype), hipMemcpyDeviceToDevice));
  }
}

template <typename Dtype>
__global__ void DropoutBackward(const int n, const Dtype* in_diff,
    const unsigned int* mask, const unsigned int threshold, const float scale,
    Dtype* out_diff) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < n) {
    out_diff[index] = in_diff[index] * scale * (mask[index] > threshold);
  }
}

template <typename Dtype>
Dtype DropoutLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const bool propagate_down,
    vector<Blob<Dtype>*>* bottom) {
  CHECK(Caffe::phase() == Caffe::TRAIN);
  if (propagate_down) {
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff();
    const unsigned int* mask = (unsigned int*)rand_vec_->gpu_data();
    const int count = (*bottom)[0]->count();
    // NOLINT_NEXT_LINE(whitespace/operators)
    DropoutBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, mask, uint_thres_, scale_, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
  }
  return Dtype(0);
}

INSTANTIATE_CLASS(DropoutLayer);


}  // namespace caffe
