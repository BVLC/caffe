#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/dropout_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {


#ifdef USE_CUDA
template<typename Dtype>
__global__ void DropoutForward(const int_tp n, const Dtype* in,
                               const uint_tp* mask,
                               const uint_tp threshold, const float scale,
                               Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] * (mask[index] > threshold) * scale;
  }
}
#endif  // USE_CUDA

template<typename Dtype>
void DropoutLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int_tp count = bottom[0]->count();

  if (this->device_->backend() == BACKEND_CUDA) {
#ifdef USE_CUDA
    if (this->phase_ == TRAIN) {
      uint_tp* mask =
          static_cast<uint_tp*>(rand_vec_.mutable_gpu_data());
      caffe_gpu_rng_uniform(count, (uint_tpc*) (mask));  // NOLINT
      // set thresholds
      // NOLINT_NEXT_LINE(whitespace/operators)
      DropoutForward<Dtype> CUDA_KERNEL(CAFFE_GET_BLOCKS(count),
                                        CAFFE_CUDA_NUM_THREADS)(
          count, bottom_data, mask, uint_thres_, scale_, top_data);
      CUDA_POST_KERNEL_CHECK;
    } else {
      caffe_copy(count, bottom_data, top_data);
    }
#endif  // USE_CUDA
  } else {
#ifdef USE_GREENTEA
    viennacl::ocl::context &ctx = viennacl::ocl::get_context(
        this->device_->id());
    viennacl::ocl::program &program = this->device_->program();

    if (this->phase_ == TRAIN) {
      uint_tp* mask =
          static_cast<uint_tp*>(rand_vec_.mutable_gpu_data());
      caffe_gpu_rng_uniform(count, reinterpret_cast<uint_tpc*> (mask));
      // set thresholds
      viennacl::ocl::kernel &oclk_dropout = program.get_kernel(
          CL_KERNEL_SELECT("dropout_forward"));

      ClState& clState = Caffe::cl_state();
      ClMemOff<Dtype> buf_bottom = clState.get_buffer_mem(bottom_data);
      ClMemOff<unsigned int> buf_mask = clState.get_buffer_mem(mask);
      ClMemOff<Dtype> buf_top = clState.get_buffer_mem(top_data);

      viennacl::ocl::enqueue(
          oclk_dropout(count, WrapHandle(buf_bottom.memobj, &ctx),
                       WrapHandle(buf_mask.memobj, &ctx), uint_thres_, scale_,
                       WrapHandle(buf_top.memobj, &ctx)),
          ctx.get_queue());
    } else {
      caffe_copy(count, bottom_data, top_data);
    }
#endif  // USE_GREENTEA
  }
}

#ifdef USE_CUDA
template<typename Dtype>
__global__ void DropoutBackward(const int_tp n, const Dtype* in_diff,
                                const uint_tp* mask,
                                const uint_tp threshold, const float scale,
                                Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * scale * (mask[index] > threshold);
  }
}
#endif  // USE_CUDA

template<typename Dtype>
void DropoutLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                       const vector<bool>& propagate_down,
                                       const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();

    if (this->device_->backend() == BACKEND_CUDA) {
#ifdef USE_CUDA
      if (this->phase_ == TRAIN) {
        const uint_tp* mask = static_cast<const uint_tp*>(rand_vec_
            .gpu_data());
        const int_tp count = bottom[0]->count();
        // NOLINT_NEXT_LINE(whitespace/operators)
        DropoutBackward<Dtype> CUDA_KERNEL(CAFFE_GET_BLOCKS(count),
            CAFFE_CUDA_NUM_THREADS)(
            count, top_diff, mask, uint_thres_, scale_, bottom_diff);
        CUDA_POST_KERNEL_CHECK;
      } else {
        caffe_copy(top[0]->count(), top_diff, bottom_diff);
      }
#endif  // USE_CUDA
    } else {
#ifdef USE_GREENTEA
      viennacl::ocl::context &ctx = viennacl::ocl::get_context(
          this->device_->id());
      viennacl::ocl::program &program = this->device_->program();

      if (this->phase_ == TRAIN) {
        const uint_tp* mask = static_cast<const uint_tp*>(rand_vec_
            .gpu_data());
        const int_tp count = bottom[0]->count();
        viennacl::ocl::kernel &oclk_dropout = program.get_kernel(
            CL_KERNEL_SELECT("dropout_backward"));

        ClState& clState = Caffe::cl_state();
        ClMemOff<Dtype> buf_bottom = clState.get_buffer_mem(bottom_diff);
        ClMemOff<unsigned int> buf_mask = clState.get_buffer_mem(mask);
        ClMemOff<Dtype> buf_top = clState.get_buffer_mem(top_diff);

        viennacl::ocl::enqueue(
            oclk_dropout(count, WrapHandle(buf_top.memobj, &ctx),
                         WrapHandle(buf_mask.memobj, &ctx), uint_thres_, scale_,
                         WrapHandle(buf_bottom.memobj, &ctx)),
            ctx.get_queue());
      } else {
        caffe_copy(top[0]->count(), top_diff, bottom_diff);
      }
#endif  // USE_GREENTEA
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(DropoutLayer);

}  // namespace caffe
