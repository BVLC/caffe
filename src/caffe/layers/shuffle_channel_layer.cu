#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/shuffle_channel_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void ShuffleChannelKernel(const int nthreads, const int feature_map_size,
	Dtype *output, const Dtype *input, int group_row, int group_column, int len) {
	CUDA_KERNEL_LOOP(index, nthreads) {
		const int n = index / group_row / group_column;
		const int i = (index / group_column) % group_row;
		const int j = index % group_column;

		const Dtype* p_i = input + n * feature_map_size + (i * group_column + j) * len;
		Dtype* p_o = output + n * feature_map_size + (j * group_row + i) * len;

		for (int k = 0; k < len; k++)
			p_o[k] = p_i[k];
	}
}

template <typename Dtype>
void ShuffleChannelLayer<Dtype>::Resize_gpu(Dtype *output, const Dtype *input, int group_row, int group_column, int len)
{
    for (int i = 0; i < group_row; ++i) // 2
    {
        for(int j = 0; j < group_column ; ++j) // 3
        {
            const Dtype* p_i = input + (i * group_column + j ) * len;
            Dtype* p_o = output + (j * group_row + i ) * len;

            caffe_copy(len, p_i, p_o);
        }
    }
}

template <typename Dtype>
void ShuffleChannelLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();

    const int num = bottom[0]->num();
    const int feature_map_size = bottom[0]->count(1);
    const int sp_sz = bottom[0]->count(2);
    const int chs = bottom[0]->channels();

    int group_row = group_;
    int group_column = int(chs / group_row);
    CHECK_EQ(chs, (group_column * group_row)) << "Wrong group size.";
	int count = num * group_column * group_row;
	ShuffleChannelKernel<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >(
		count, feature_map_size, top_data, bottom_data, group_row, group_column, sp_sz);
    //Dtype* temp_data = temp_blob_.mutable_gpu_data();
    //for(int n = 0; n < num; ++n)
    //{
    //    Resize_gpu(top_data + n*feature_map_size, bottom_data + n*feature_map_size, group_row, group_column, sp_sz);
    //}
    //caffe_copy(bottom[0]->count(), temp_blob_.gpu_data(), top_data);
}

template <typename Dtype>
void ShuffleChannelLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
      const Dtype* top_diff = top[0]->gpu_diff();
      Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();

      const int num = bottom[0]->num();
      const int feature_map_size = bottom[0]->count(1);
      const int sp_sz = bottom[0]->count(2);
      const int chs = bottom[0]->channels();

      int group_row = int(chs / group_);
      int group_column = group_;
      int count = num * group_column * group_row;
	  ShuffleChannelKernel<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >(
		  count, feature_map_size, bottom_diff, top_diff, group_row, group_column, sp_sz);
      //Dtype* temp_diff = temp_blob_.mutable_gpu_diff();
    //  for(int n = 0; n < num; ++n)
    //  {
		  //Resize_gpu(bottom_diff + n * feature_map_size, top_diff + n*feature_map_size, group_row, group_column, sp_sz);
    //  }
      //caffe_copy(top[0]->count(), temp_blob_.gpu_diff(), bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(ShuffleChannelLayer);

}  // namespace caffe
