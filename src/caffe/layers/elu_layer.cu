#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/elu_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void ELUForward(const int n, const Dtype* in, Dtype* out,
    Dtype alpha) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] > 0 ? in[index] :
        alpha * (exp(in[index]) - 1);
  }
}

template <typename Dtype>
void ELULayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  Dtype alpha = this->layer_param_.elu_param().alpha();
  // NOLINT_NEXT_LINE(whitespace/operators)
  ELUForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data, alpha);
  CUDA_POST_KERNEL_CHECK;
}


INSTANTIATE_LAYER_GPU_FUNCS(ELULayer);


}  // namespace caffe
