#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/elu_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void ELUForward(const int n, const Dtype* in, Dtype* out,
    Dtype alpha) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] > 0 ? in[index] :
        alpha * (exp(in[index]) - 1);
  }
}

template <typename Dtype>
void ELULayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  Dtype alpha = this->layer_param_.elu_param().alpha();
  // NOLINT_NEXT_LINE(whitespace/operators)
  ELUForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data, alpha);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void ELUBackward(const int n, const Dtype* in_diff,
    const Dtype* out_data, const Dtype* in_data,
    Dtype* out_diff, Dtype alpha) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_data[index] > 0 ? in_diff[index] :
        in_diff[index] * (out_data[index] + alpha);
  }
}

template <typename Dtype>
void ELULayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* top_data = top[0]->gpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    Dtype alpha = this->layer_param_.elu_param().alpha();
    // NOLINT_NEXT_LINE(whitespace/operators)
    ELUBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, top_data, bottom_data, bottom_diff, alpha);
    CUDA_POST_KERNEL_CHECK;
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(ELULayer);


}  // namespace caffe
