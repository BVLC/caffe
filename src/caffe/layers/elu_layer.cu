#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/elu_layer.hpp"

namespace caffe {

#ifdef USE_CUDA
template <typename Dtype>
__global__ void ELUForward(const int n, const Dtype* in, Dtype* out,
    Dtype alpha) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] > 0 ? in[index] :
        alpha * (exp(in[index]) - 1);
  }
}
#endif  // USE_CUDA

template <typename Dtype>
void ELULayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  Dtype alpha = this->layer_param_.elu_param().alpha();

  if (this->device_->backend() == BACKEND_CUDA) {
#ifdef USE_CUDA
    // NOLINT_NEXT_LINE(whitespace/operators)
    ELUForward<Dtype> CUDA_KERNEL(CAFFE_GET_BLOCKS(count),
                                  CAFFE_CUDA_NUM_THREADS)(
        count, bottom_data, top_data, alpha);
    CUDA_POST_KERNEL_CHECK;
#endif  // USE_CUDA
  } else {
#ifdef USE_GREENTEA
    viennacl::ocl::context &ctx = viennacl::ocl::get_context(
        this->device_->id());
    viennacl::ocl::program &program = this->device_->program();

    viennacl::ocl::kernel &oclk_elu = program.get_kernel(
        CL_KERNEL_SELECT("elu_forward"));

    ClState& clState = Caffe::cl_state();
    ClMemOff<Dtype> buf_bottom = clState.get_buffer_mem(bottom_data);
    ClMemOff<Dtype> buf_top = clState.get_buffer_mem(top_data);

    viennacl::ocl::enqueue(
        oclk_elu(count, WrapHandle(buf_bottom.memobj, &ctx),
                  WrapHandle(buf_top.memobj, &ctx), alpha),
        ctx.get_queue());
#endif  // USE_GREENTEA
  }
}

#ifdef USE_CUDA
template <typename Dtype>
__global__ void ELUBackward(const int n, const Dtype* in_diff,
    const Dtype* out_data, const Dtype* in_data,
    Dtype* out_diff, Dtype alpha) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_data[index] > 0 ? in_diff[index] :
        in_diff[index] * (out_data[index] + alpha);
  }
}
#endif  // USE_CUDA

template <typename Dtype>
void ELULayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* bottom_data = bottom[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* top_data = top[0]->gpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    Dtype alpha = this->layer_param_.elu_param().alpha();

    if (this->device_->backend() == BACKEND_CUDA) {
#ifdef USE_CUDA
      // NOLINT_NEXT_LINE(whitespace/operators)
      ELUBackward<Dtype> CUDA_KERNEL(CAFFE_GET_BLOCKS(count),
                                     CAFFE_CUDA_NUM_THREADS)(
          count, top_diff, top_data, bottom_data, bottom_diff, alpha);
      CUDA_POST_KERNEL_CHECK;
#endif  // USE_CUDA
    } else {
#ifdef USE_GREENTEA
      viennacl::ocl::context &ctx = viennacl::ocl::get_context(
          this->device_->id());
      viennacl::ocl::program &program = this->device_->program();

      viennacl::ocl::kernel &oclk_elu = program.get_kernel(
          CL_KERNEL_SELECT("elu_backward"));
      ClState& clState = Caffe::cl_state();
      ClMemOff<Dtype> buf_bottom_diff = clState.get_buffer_mem(bottom_diff);
      ClMemOff<Dtype> buf_top_diff = clState.get_buffer_mem(top_diff);
      ClMemOff<Dtype> buf_bottom_data = clState.get_buffer_mem(bottom_data);
      ClMemOff<Dtype> buf_top_data = clState.get_buffer_mem(top_data);

      viennacl::ocl::enqueue(
          oclk_elu(count, WrapHandle(buf_top_diff.memobj, &ctx),
                   WrapHandle(buf_top_data.memobj, &ctx),
                   WrapHandle(buf_bottom_data.memobj, &ctx),
                   WrapHandle(buf_bottom_diff.memobj, &ctx), alpha),
          ctx.get_queue());
#endif  // USE_GREENTEA
    }
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(ELULayer);


}  // namespace caffe
