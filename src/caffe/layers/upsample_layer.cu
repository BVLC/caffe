#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/upsample_layer.hpp"

namespace caffe {

template <typename Dtype>
  __global__ void UpsampleForward(const int nthreads, int in_w, int in_h,
      int out_w, int out_h, const Dtype* bottom_data,
      const Dtype* bottom_mask, Dtype* top_data) {
    CUDA_KERNEL_LOOP(index, nthreads) {
      int offset = index / (in_w * in_h) * out_w * out_h;
      int upsample_idx = static_cast<int>(bottom_mask[index]);
      top_data[offset + upsample_idx] = bottom_data[index];
    }
  }

template <typename Dtype>
void UpsampleLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* bottom_mask = bottom[1]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  caffe_gpu_set(top[0]->count(), Dtype(0), top_data);
  int bottom_count = bottom[0]->count();
  UpsampleForward<Dtype><<<CAFFE_GET_BLOCKS(bottom_count), CAFFE_CUDA_NUM_THREADS>>>(
      bottom_count, bottom[0]->width(), bottom[0]->height(), 
      top[0]->width(), top[0]->height(), bottom_data, bottom_mask, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
  __global__ void UpsampleBackward(const int nthreads, int in_w, int in_h,
      int out_w, int out_h, const Dtype* top_diff,
      const Dtype* bottom_mask, Dtype* bottom_diff) {
    CUDA_KERNEL_LOOP(index, nthreads) {
      int offset = index / (in_w * in_h) * out_w * out_h;
      int upsample_idx = static_cast<int>(bottom_mask[index]);
      bottom_diff[index] = top_diff[offset + upsample_idx];
    }
  }

template <typename Dtype>
void UpsampleLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* bottom_mask = bottom[1]->gpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int bottom_count = bottom[0]->count();
    caffe_gpu_set(bottom_count, Dtype(0.), bottom_diff);
    UpsampleBackward<Dtype><<<CAFFE_GET_BLOCKS(bottom_count), CAFFE_CUDA_NUM_THREADS>>>(
        bottom_count, bottom[0]->width(), bottom[0]->height(), 
        top[0]->width(), top[0]->height(), top_diff, bottom_mask, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(UpsampleLayer);


}  // namespace caffe
