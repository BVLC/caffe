#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/util/local_update.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

/// @brief refer to CPU forward -- the BLAS implementation is the same.
template <typename Dtype>
void LocalWeightedConvolutionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {

  Dtype* x_data = col_buffer_.mutable_gpu_data();
  const Dtype* weight = this->blobs_[0]->gpu_data();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();

  Blob<Dtype> E;
  E.Reshape(1, 1, 1, K_);
  FillerParameter filler_param;
  filler_param.set_value(1);
  ConstantFiller<Dtype> filler(filler_param);
  filler.Fill(&E);

  Blob<Dtype> intermediate;
  intermediate.Reshape(1, 1, K_, N_);
  for (int n=0; n<num_; n++) {
    im2col_gpu(bottom_data + bottom[0]->offset(n), channels_, height_,
               width_, kernel_size_, kernel_size_, pad_, pad_, stride_, stride_, x_data);

    for (int m=0; m<num_output_; m++) {
      caffe_gpu_mul(K_*N_, x_data, weight+this->blobs_[0]->offset(m),
                    intermediate.mutable_gpu_data());

      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, 1, N_, K_,
                            (Dtype)1., E.gpu_data(), intermediate.gpu_data(),
                            (Dtype)0., top_data + top[0]->offset(n, m));
    }

    if (bias_term_) {
      caffe_gpu_add(M_ * N_, this->blobs_[1]->gpu_data(),
                    top_data + top[0]->offset(n),
                    top_data + top[0]->offset(n));
    }
  }

}

/// @brief refer to CPU backward -- the BLAS implementation is the same.
template <typename Dtype>
void LocalWeightedConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {


  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  Dtype* x_data = col_buffer_.mutable_gpu_data();
  Dtype* x_diff = col_buffer_.mutable_gpu_diff();
  const Dtype* weight = this->blobs_[0]->gpu_data();
  Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
  Dtype* bias_diff = NULL;

  Blob<Dtype> intermediate;
  intermediate.Reshape(1, 1, 1, N_);

  Blob<Dtype> xt;
  xt.Reshape(1, 1, K_, N_);
  Dtype* xt_data = xt.mutable_gpu_data();
  if (bias_term_) {
    bias_diff = this->blobs_[1]->mutable_gpu_diff();
    CUDA_CHECK(hipMemset(bias_diff, 0, sizeof(Dtype) * this->blobs_[1]->count()));
    for (int n = 0; n < num_; ++n) {
      caffe_gpu_add(M_ * N_, bias_diff,
                    top_diff + top[0]->offset(n),
                    bias_diff);
    }
  }

  Blob<Dtype> buf;
  buf.Reshape(1, 1, K_, N_);
  Dtype* buf_data = buf.mutable_gpu_data();
  CUDA_CHECK(hipMemset(weight_diff, 0, sizeof(Dtype) * this->blobs_[0]->count()));
  for (int n=0; n<num_; n++) {
    im2col_gpu(bottom_data + bottom[0]->offset(n), channels_, height_,
               width_, kernel_size_, kernel_size_, pad_, pad_, stride_, stride_, x_data);

    local_update1_gpu(top_diff+top[0]->offset(n), x_data, weight_diff, K_, N_, M_);

    if (propagate_down[0]) {
      CUDA_CHECK(hipMemset(x_diff, 0, col_buffer_.count() * sizeof(Dtype)));
      local_update2_gpu(top_diff+top[0]->offset(n), weight, x_diff, K_, N_, M_);

      // col2im back to the data
      col2im_gpu(x_diff, channels_, height_, width_, kernel_size_, kernel_size_,
                 pad_, pad_, stride_, stride_, bottom_diff + bottom[0]->offset(n));
    }



  }







/*
  const Dtype* weight = NULL;
  Dtype* weight_diff = NULL;
  if (this->param_propagate_down_[0]) {
    weight = this->blobs_[0]->gpu_data();
    weight_diff = this->blobs_[0]->mutable_gpu_diff();
    caffe_gpu_set(this->blobs_[0]->count(), Dtype(0), weight_diff);
  }
  Dtype* bias_diff = NULL;
  if (bias_term_ && this->param_propagate_down_[1]) {
    bias_diff = this->blobs_[1]->mutable_gpu_diff();
    caffe_gpu_set(this->blobs_[1]->count(), Dtype(0), bias_diff);
  }
  const int weight_offset = M_ * K_;
  const int col_offset = K_ * N_;
  const int top_offset = M_ * N_;
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = NULL;
    // Bias gradient, if necessary.
    if (bias_term_ && this->param_propagate_down_[1]) {
      top_diff = top[i]->gpu_diff();
      for (int n = 0; n < num_; ++n) {
        caffe_gpu_gemv<Dtype>(CblasNoTrans, num_output_, N_,
            1., top_diff + top[0]->offset(n),
            bias_multiplier_.gpu_data(), 1.,
            bias_diff);
      }
    }
    if (this->param_propagate_down_[0] || propagate_down[i]) {
      if (!top_diff) {
        top_diff = top[i]->gpu_diff();
      }
      Dtype* col_buff = NULL;
      if (!is_1x1_) {
        col_buff = col_buffer_.mutable_gpu_data();
      }
      const Dtype* bottom_data = bottom[i]->gpu_data();
      Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
      for (int n = 0; n < num_; ++n) {
        // Since we saved memory in the forward pass by not storing all col
        // data, we will need to recompute them.
        if (!is_1x1_) {
          im2col_gpu(bottom_data + bottom[i]->offset(n), channels_, height_,
                    width_, kernel_h_, kernel_w_, pad_h_, pad_w_,
                    stride_h_, stride_w_, col_buff);
        } else {
          col_buff = bottom[i]->mutable_gpu_data() + bottom[i]->offset(n);
        }
        // gradient w.r.t. weight. Note that we will accumulate diffs.
        if (this->param_propagate_down_[0]) {
          for (int g = 0; g < group_; ++g) {
            caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasTrans, M_, K_, N_,
                (Dtype)1., top_diff + top[i]->offset(n) + top_offset * g,
                col_buff + col_offset * g, (Dtype)1.,
                weight_diff + weight_offset * g);
          }
        }
        // gradient w.r.t. bottom data, if necessary
        if (propagate_down[i]) {
          if (weight == NULL) {
            weight = this->blobs_[0]->gpu_data();
          }
          if (is_1x1_) {
            col_buff = bottom[i]->mutable_gpu_diff() + bottom[i]->offset(n);
          }
          for (int g = 0; g < group_; ++g) {
            caffe_gpu_gemm<Dtype>(CblasTrans, CblasNoTrans, K_, N_, M_,
                (Dtype)1., weight + weight_offset * g,
                top_diff + top[i]->offset(n) + top_offset * g,
                (Dtype)0., col_buff + col_offset * g);
          }
          // col2im back to the data
          if (!is_1x1_) {
            col2im_gpu(col_buff, channels_, height_, width_,
                kernel_h_, kernel_w_, pad_h_, pad_w_, stride_h_, stride_w_,
                bottom_diff + bottom[i]->offset(n));
          }
        }
      }
    }
  }
*/
}


INSTANTIATE_LAYER_GPU_FUNCS(LocalWeightedConvolutionLayer);

}  // namespace caffe
