#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/util/local_update.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

/// @brief refer to CPU forward -- the BLAS implementation is the same.
template <typename Dtype>
void LocalWeightedConvolutionLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {

  Dtype* x_data = col_buffer_.mutable_gpu_data();
  const Dtype* weight = this->blobs_[0]->gpu_data();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();

  Blob<Dtype> E;
  E.Reshape(1, 1, 1, K_);
  FillerParameter filler_param;
  filler_param.set_value(1);
  ConstantFiller<Dtype> filler(filler_param);
  filler.Fill(&E);

  Blob<Dtype> intermediate;
  intermediate.Reshape(1, 1, K_, N_);
  for (int n=0; n<num_; n++) {
    im2col_gpu(bottom_data + bottom[0]->offset(n), channels_, height_,
               width_, kernel_size_, kernel_size_, pad_, pad_, stride_, stride_, x_data);

    for (int m=0; m<num_output_; m++) {
      caffe_gpu_mul(K_*N_, x_data, weight+this->blobs_[0]->offset(m),
                    intermediate.mutable_gpu_data());

      caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, 1, N_, K_,
                            (Dtype)1., E.gpu_data(), intermediate.gpu_data(),
                            (Dtype)0., top_data + top[0]->offset(n, m));
    }

    if (bias_term_) {
      caffe_gpu_add(M_ * N_, this->blobs_[1]->gpu_data(),
                    top_data + top[0]->offset(n),
                    top_data + top[0]->offset(n));
    }
  }

}

/// @brief refer to CPU backward -- the BLAS implementation is the same.
template <typename Dtype>
void LocalWeightedConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  Dtype* x_data = col_buffer_.mutable_gpu_data();
  Dtype* x_diff = col_buffer_.mutable_gpu_diff();
  const Dtype* weight = this->blobs_[0]->gpu_data();
  Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
  Dtype* bias_diff = NULL;

  Blob<Dtype> intermediate;
  intermediate.Reshape(1, 1, 1, N_);

  Blob<Dtype> xt;
  xt.Reshape(1, 1, K_, N_);
  Dtype* xt_data = xt.mutable_gpu_data();
  if (bias_term_) {
    bias_diff = this->blobs_[1]->mutable_gpu_diff();
    CUDA_CHECK(hipMemset(bias_diff, 0, sizeof(Dtype) * this->blobs_[1]->count()));
    for (int n = 0; n < num_; ++n) {
      caffe_gpu_add(M_ * N_, bias_diff,
                    top_diff + top[0]->offset(n),
                    bias_diff);
    }
  }

  Blob<Dtype> buf;
  buf.Reshape(1, 1, K_, N_);
  Dtype* buf_data = buf.mutable_gpu_data();
  CUDA_CHECK(hipMemset(weight_diff, 0, sizeof(Dtype) * this->blobs_[0]->count()));
  for (int n=0; n<num_; n++) {
    im2col_gpu(bottom_data + bottom[0]->offset(n), channels_, height_,
               width_, kernel_size_, kernel_size_, pad_, pad_, stride_, stride_, x_data);

    local_update1_gpu(top_diff+top[0]->offset(n), x_data, weight_diff, K_, N_, M_);

    if (propagate_down[0]) {
      CUDA_CHECK(hipMemset(x_diff, 0, col_buffer_.count() * sizeof(Dtype)));
      local_update2_gpu(top_diff+top[0]->offset(n), weight, x_diff, K_, N_, M_);

      // col2im back to the data
      col2im_gpu(x_diff, channels_, height_, width_, kernel_size_, kernel_size_,
                 pad_, pad_, stride_, stride_, bottom_diff + bottom[0]->offset(n));
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(LocalWeightedConvolutionLayer);

}  // namespace caffe
