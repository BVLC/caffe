#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/sigmoid_cross_entropy_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {


template <typename Dtype>
__global__ void SigmoidCrossEntropyLossForwardGPU(const int nthreads,
          const Dtype* input_data, const Dtype* target, Dtype* loss,
          const bool has_ignore_label_, const int ignore_label_,
          Dtype* counts) {
  CUDA_KERNEL_LOOP(i, nthreads) {
    const int target_value = static_cast<int>(target[i]);
    if (has_ignore_label_ && target_value == ignore_label_) {
      loss[i] = 0;
      counts[i] = 0;
    } else {
      loss[i] = input_data[i] * (target[i] - (input_data[i] >= 0)) -
          log(1 + exp(input_data[i] - 2 * input_data[i] *
          (input_data[i] >= 0)));
      counts[i] = 1;
    }
  }
}

template <typename Dtype>
__global__ void SigmoidCrossEntropyLossIgnoreDiffGPU(const int count,
    const int ignore_label, const Dtype* target, Dtype* diff) {
  CUDA_KERNEL_LOOP(i, count) {
    const int target_value = static_cast<int>(target[i]);
    if (target_value == ignore_label) {
      diff[i] = 0;
    }
  }
}


template <typename Dtype>
void SigmoidCrossEntropyLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  // The forward pass computes the sigmoid outputs.
  sigmoid_bottom_vec_[0] = bottom[0];
  sigmoid_layer_->Forward(sigmoid_bottom_vec_, sigmoid_top_vec_);
  // Compute the loss (negative log likelihood)
  const int count = bottom[0]->count();
  // Stable version of loss computation from input data
  const Dtype* input_data = bottom[0]->gpu_data();
  const Dtype* target = bottom[1]->gpu_data();
  // Since this memory is not used for anything, we use it here to avoid having
  // to allocate new GPU memory to accumulate intermediate results.
  Dtype* loss_data = bottom[0]->mutable_gpu_diff();
  Dtype* count_data = bottom[1]->mutable_gpu_diff();
  Dtype valid_count;
  // NOLINT_NEXT_LINE(whitespace/operators)
  SigmoidCrossEntropyLossForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(count),
      CAFFE_CUDA_NUM_THREADS>>>(count, input_data, target, loss_data,
      has_ignore_label_, ignore_label_, count_data);
  // Only launch another CUDA kernel if we actually need the valid count.
  if (normalization_ == LossParameter_NormalizationMode_VALID &&
      has_ignore_label_) {
    caffe_gpu_asum(count, count_data, &valid_count);
  } else {
    valid_count = count;
  }
  Dtype loss;
  caffe_gpu_asum(count, loss_data, &loss);
  normalizer_ = get_normalizer(normalization_, valid_count);
  top[0]->mutable_cpu_data()[0] = loss / normalizer_;

  // Clear scratch memory to prevent interfering with backward (see #6202).
  caffe_gpu_set(bottom[0]->count(), Dtype(0), bottom[0]->mutable_gpu_diff());
  caffe_gpu_set(bottom[1]->count(), Dtype(0), bottom[1]->mutable_gpu_diff());
}

template <typename Dtype>
void SigmoidCrossEntropyLossLayer<Dtype>::Backward_gpu(
    const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
    // First, compute the diff
    const int count = bottom[0]->count();
    const Dtype* sigmoid_output_data = sigmoid_output_->gpu_data();
    const Dtype* target = bottom[1]->gpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    caffe_copy(count, sigmoid_output_data, bottom_diff);
    caffe_gpu_axpy(count, Dtype(-1), target, bottom_diff);
    // Zero out gradient of ignored targets.
    if (has_ignore_label_) {
      // NOLINT_NEXT_LINE(whitespace/operators)
      SigmoidCrossEntropyLossIgnoreDiffGPU<Dtype><<<CAFFE_GET_BLOCKS(count),
        CAFFE_CUDA_NUM_THREADS>>>(count, ignore_label_, target, bottom_diff);
    }
    // Scale down gradient
    Dtype loss_weight = top[0]->cpu_diff()[0] / normalizer_;
    caffe_gpu_scal(count, loss_weight, bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SigmoidCrossEntropyLossLayer);

}  // namespace caffe
