#include "hip/hip_runtime.h"
#include <vector>

<<<<<<< HEAD
<<<<<<< HEAD
#include "caffe/layers/sigmoid_cross_entropy_loss_layer.hpp"
=======
#include "caffe/loss_layers.hpp"
>>>>>>> pod-caffe-pod.hpp-merge
=======
#include "caffe/loss_layers.hpp"
>>>>>>> pod/caffe-merge
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
=======
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
=======
>>>>>>> pod-caffe-pod.hpp-merge
<<<<<<< HEAD
=======
=======
>>>>>>> pod/caffe-merge
=======
>>>>>>> pod/caffe-merge
=======
=======
<<<<<<< HEAD
<<<<<<< HEAD
=======
>>>>>>> pod/device/blob.hpp
=======
=======
>>>>>>> pod-caffe-pod.hpp-merge
=======
=======
>>>>>>> pod/caffe-merge
=======
=======
<<<<<<< HEAD
<<<<<<< HEAD
=======
>>>>>>> pod/device/blob.hpp
void SigmoidCrossEntropyLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  // The forward pass computes the sigmoid outputs.
  sigmoid_bottom_vec_[0] = bottom[0];
  sigmoid_layer_->Forward(sigmoid_bottom_vec_, sigmoid_top_vec_);
  // Compute the loss (negative log likelihood)
  const int count = bottom[0]->count();
  const int num = bottom[0]->num();
  // Stable version of loss computation from input data
  const Dtype* input_data = bottom[0]->cpu_data();
  const Dtype* target = bottom[1]->cpu_data();
  Dtype loss = 0;
  for (int i = 0; i < count; ++i) {
    loss -= input_data[i] * (target[i] - (input_data[i] >= 0)) -
        log(1 + exp(input_data[i] - 2 * input_data[i] * (input_data[i] >= 0)));
  }
  top[0]->mutable_cpu_data()[0] = loss / num;
}

template <typename Dtype>
>>>>>>> origin/BVLC/parallel
=======
>>>>>>> caffe
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
>>>>>>> pod-caffe-pod.hpp-merge
=======
>>>>>>> pod-caffe-pod.hpp-merge
=======
>>>>>>> pod/caffe-merge
=======
>>>>>>> pod/caffe-merge
=======
>>>>>>> pod-caffe-pod.hpp-merge
>>>>>>> pod/device/blob.hpp
=======
>>>>>>> device-abstraction
=======
>>>>>>> pod-caffe-pod.hpp-merge
=======
>>>>>>> pod/caffe-merge
=======
>>>>>>> pod-caffe-pod.hpp-merge
>>>>>>> pod/device/blob.hpp
void SigmoidCrossEntropyLossLayer<Dtype>::Backward_gpu(
    const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
    // First, compute the diff
    const int count = bottom[0]->count();
    const int num = bottom[0]->num();
    const Dtype* sigmoid_output_data = sigmoid_output_->gpu_data();
    const Dtype* target = bottom[1]->gpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    caffe_copy(count, sigmoid_output_data, bottom_diff);
    caffe_gpu_axpy(count, Dtype(-1), target, bottom_diff);
    // Scale down gradient
    const Dtype loss_weight = top[0]->cpu_diff()[0];
    caffe_gpu_scal(count, loss_weight / num, bottom_diff);
  }
}

<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
=======
>>>>>>> pod/device/blob.hpp
=======
>>>>>>> pod/device/blob.hpp
INSTANTIATE_LAYER_GPU_BACKWARD(SigmoidCrossEntropyLossLayer);
=======
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
=======
>>>>>>> pod-caffe-pod.hpp-merge
<<<<<<< HEAD
INSTANTIATE_LAYER_GPU_BACKWARD(SigmoidCrossEntropyLossLayer);
=======
=======
>>>>>>> pod/caffe-merge
=======
>>>>>>> pod/caffe-merge
=======
INSTANTIATE_LAYER_GPU_BACKWARD(SigmoidCrossEntropyLossLayer);
=======
>>>>>>> pod/device/blob.hpp
=======
INSTANTIATE_LAYER_GPU_BACKWARD(SigmoidCrossEntropyLossLayer);
=======
>>>>>>> pod-caffe-pod.hpp-merge
=======
INSTANTIATE_LAYER_GPU_BACKWARD(SigmoidCrossEntropyLossLayer);
=======
>>>>>>> pod/caffe-merge
INSTANTIATE_LAYER_GPU_FUNCS(SigmoidCrossEntropyLossLayer);
>>>>>>> origin/BVLC/parallel
=======
INSTANTIATE_LAYER_GPU_BACKWARD(SigmoidCrossEntropyLossLayer);
>>>>>>> caffe
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
>>>>>>> pod-caffe-pod.hpp-merge
=======
>>>>>>> pod-caffe-pod.hpp-merge
=======
>>>>>>> pod/caffe-merge
=======
>>>>>>> pod/caffe-merge
=======
>>>>>>> pod-caffe-pod.hpp-merge
>>>>>>> pod/device/blob.hpp
=======
INSTANTIATE_LAYER_GPU_BACKWARD(SigmoidCrossEntropyLossLayer);
>>>>>>> device-abstraction
=======
>>>>>>> pod-caffe-pod.hpp-merge
=======
>>>>>>> pod/caffe-merge
=======
INSTANTIATE_LAYER_GPU_BACKWARD(SigmoidCrossEntropyLossLayer);
=======
INSTANTIATE_LAYER_GPU_FUNCS(SigmoidCrossEntropyLossLayer);
>>>>>>> origin/BVLC/parallel
=======
INSTANTIATE_LAYER_GPU_BACKWARD(SigmoidCrossEntropyLossLayer);
>>>>>>> caffe
>>>>>>> pod-caffe-pod.hpp-merge
>>>>>>> pod/device/blob.hpp


}  // namespace caffe
