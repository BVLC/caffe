#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/batch_norm_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template<typename Dtype>
void BatchNormLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                        const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int_tp num = bottom[0]->shape(0);
  int_tp spatial_dim = bottom[0]->count() / (channels_ * bottom[0]->shape(0));

  if (this->device_->backend() == BACKEND_CUDA) {
#ifdef USE_CUDA
    if (bottom[0] != top[0]) {
      caffe_copy(bottom[0]->count(), bottom_data, top_data);
    }

    if (use_global_stats_) {
      // use the stored mean/variance estimates.
      const Dtype scale_factor =
          this->blobs_[2]->cpu_data()[0] == 0 ?
              0 : 1 / this->blobs_[2]->cpu_data()[0];
      caffe_gpu_scale(variance_.count(), scale_factor,
                      this->blobs_[0]->gpu_data(), mean_.mutable_gpu_data());
      caffe_gpu_scale(variance_.count(), scale_factor,
                      this->blobs_[1]->gpu_data(),
                      variance_.mutable_gpu_data());
    } else {
      // compute mean
      caffe_gpu_gemv<Dtype>(CblasNoTrans, channels_ * num, spatial_dim,
                            1. / (num * spatial_dim), bottom_data,
                            spatial_sum_multiplier_.gpu_data(), 0.,
                            num_by_chans_.mutable_gpu_data());
      caffe_gpu_gemv<Dtype>(CblasTrans, num, channels_, 1.,
                            num_by_chans_.gpu_data(),
                            batch_sum_multiplier_.gpu_data(), 0.,
                            mean_.mutable_gpu_data());
    }

    // subtract mean
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num, channels_, 1, 1,
                          batch_sum_multiplier_.gpu_data(), mean_.gpu_data(),
                          0., num_by_chans_.mutable_gpu_data());
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, channels_ * num,
                          spatial_dim, 1, -1, num_by_chans_.gpu_data(),
                          spatial_sum_multiplier_.gpu_data(), 1., top_data);

    if (!use_global_stats_) {
      // compute variance using var(X) = E((X-EX)^2)
      caffe_gpu_powx(top[0]->count(), top_data, Dtype(2),
                     temp_.mutable_gpu_data());  // (X-EX)^2
      caffe_gpu_gemv<Dtype>(CblasNoTrans, channels_ * num, spatial_dim,
                            1. / (num * spatial_dim), temp_.gpu_data(),
                            spatial_sum_multiplier_.gpu_data(), 0.,
                            num_by_chans_.mutable_gpu_data());
      caffe_gpu_gemv<Dtype>(CblasTrans, num, channels_, 1.,
                            num_by_chans_.gpu_data(),
                            batch_sum_multiplier_.gpu_data(), 0.,
                            variance_.mutable_gpu_data());  // E((X_EX)^2)

      // compute and save moving average
      this->blobs_[2]->mutable_cpu_data()[0] *= moving_average_fraction_;
      this->blobs_[2]->mutable_cpu_data()[0] += 1;
      caffe_gpu_axpby(mean_.count(), Dtype(1), mean_.gpu_data(),
                      moving_average_fraction_,
                      this->blobs_[0]->mutable_gpu_data());
      int_tp m = bottom[0]->count() / channels_;
      Dtype bias_correction_factor = m > 1 ? Dtype(m) / (m - 1) : 1;
      caffe_gpu_axpby(variance_.count(), bias_correction_factor,
                      variance_.gpu_data(), moving_average_fraction_,
                      this->blobs_[1]->mutable_gpu_data());
    }

    // normalize variance
    caffe_gpu_add_scalar(variance_.count(), eps_, variance_.mutable_gpu_data());
    caffe_gpu_powx(variance_.count(), variance_.gpu_data(), Dtype(0.5),
                   variance_.mutable_gpu_data());

    // replicate variance to input size
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num, channels_, 1, 1,
                          batch_sum_multiplier_.gpu_data(),
                          variance_.gpu_data(), 0.,
                          num_by_chans_.mutable_gpu_data());
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, channels_ * num,
                          spatial_dim, 1, 1., num_by_chans_.gpu_data(),
                          spatial_sum_multiplier_.gpu_data(), 0.,
                          temp_.mutable_gpu_data());
    caffe_gpu_div(temp_.count(), top_data, temp_.gpu_data(), top_data);
    // TODO(cdoersch): The caching is only needed because later in-place layers
    //                 might clobber the data.  Can we skip this if they won't?
    caffe_copy(x_norm_.count(), top_data, x_norm_.mutable_gpu_data());
#endif  // USE_CUDA
  } else {
#ifdef USE_GREENTEA
    viennacl::ocl::context &ctx = viennacl::ocl::get_context(
        this->device_->id());

    if (bottom[0] != top[0]) {
      greentea_copy<Dtype>(bottom[0]->count(), (cl_mem) bottom_data, 0,
                           (cl_mem) top_data, 0, &ctx);
    }

    if (use_global_stats_) {
      // use the stored mean/variance estimates.
      const Dtype scale_factor =
          this->blobs_[2]->cpu_data()[0] == 0 ?
              0 : 1 / this->blobs_[2]->cpu_data()[0];
      greentea_gpu_scale<Dtype>(this->device_->id(), variance_.count(),
                                scale_factor,
                                (cl_mem) (this->blobs_[0]->gpu_data()), 0,
                                (cl_mem) (mean_.mutable_gpu_data()), 0);
      greentea_gpu_scale<Dtype>(this->device_->id(), variance_.count(),
                                scale_factor,
                                (cl_mem) (this->blobs_[1]->gpu_data()), 0,
                                (cl_mem) (variance_.mutable_gpu_data()), 0);
    } else {
      // compute mean
      greentea_gpu_gemv<Dtype>(this->device_->id(), CblasNoTrans,
                               channels_ * num, spatial_dim,
                               1. / (num * spatial_dim), (cl_mem) bottom_data,
                               0, (cl_mem) (spatial_sum_multiplier_.gpu_data()),
                               0, 0.,
                               (cl_mem) (num_by_chans_.mutable_gpu_data()), 0);
      greentea_gpu_gemv<Dtype>(this->device_->id(), CblasTrans, num, channels_,
                               1., (cl_mem) (num_by_chans_.gpu_data()), 0,
                               (cl_mem) (batch_sum_multiplier_.gpu_data()), 0,
                               0., (cl_mem) (mean_.mutable_gpu_data()), 0);
    }

    // subtract mean
    greentea_gpu_gemm<Dtype>(this->device_->id(), CblasNoTrans, CblasNoTrans,
                             num, channels_, 1, 1,
                             (cl_mem) (batch_sum_multiplier_.gpu_data()), 0,
                             (cl_mem) (mean_.gpu_data()), 0, 0.,
                             (cl_mem) (num_by_chans_.mutable_gpu_data()), 0);
    greentea_gpu_gemm<Dtype>(this->device_->id(), CblasNoTrans, CblasNoTrans,
                             channels_ * num, spatial_dim, 1, -1,
                             (cl_mem) (num_by_chans_.gpu_data()), 0,
                             (cl_mem) (spatial_sum_multiplier_.gpu_data()), 0,
                             1., (cl_mem) top_data, 0);

    if (!use_global_stats_) {
      // compute variance using var(X) = E((X-EX)^2)
      greentea_gpu_powx<Dtype>(this->device_->id(), top[0]->count(),
                               (cl_mem) top_data, 0, Dtype(2),
                               (cl_mem) (temp_.mutable_gpu_data()), 0);
      // (X-EX)^2
      greentea_gpu_gemv<Dtype>(this->device_->id(), CblasNoTrans,
                               channels_ * num, spatial_dim,
                               1. / (num * spatial_dim),
                               (cl_mem) (temp_.gpu_data()), 0,
                               (cl_mem) (spatial_sum_multiplier_.gpu_data()), 0,
                               0., (cl_mem) (num_by_chans_.mutable_gpu_data()),
                               0);
      greentea_gpu_gemv<Dtype>(this->device_->id(), CblasTrans, num, channels_,
                               1., (cl_mem) (num_by_chans_.gpu_data()), 0,
                               (cl_mem) (batch_sum_multiplier_.gpu_data()), 0,
                               0., (cl_mem) (variance_.mutable_gpu_data()), 0);
      // E((X_EX)^2)

      // compute and save moving average
      this->blobs_[2]->mutable_cpu_data()[0] *= moving_average_fraction_;
      this->blobs_[2]->mutable_cpu_data()[0] += 1;
      greentea_gpu_axpby<Dtype>(this->device_->id(), mean_.count(), Dtype(1),
                                (cl_mem) (mean_.gpu_data()), 0,
                                moving_average_fraction_,
                                (cl_mem) (this->blobs_[0]->mutable_gpu_data()),
                                0);
      int_tp m = bottom[0]->count() / channels_;
      Dtype bias_correction_factor = m > 1 ? Dtype(m) / (m - 1) : 1;
      greentea_gpu_axpby<Dtype>(this->device_->id(), variance_.count(),
                                bias_correction_factor,
                                (cl_mem) (variance_.gpu_data()), 0,
                                moving_average_fraction_,
                                (cl_mem) (this->blobs_[1]->mutable_gpu_data()),
                                0);
    }

    // normalize variance
    greentea_gpu_add_scalar<Dtype>(this->device_->id(), variance_.count(), eps_,
                                   (cl_mem) (variance_.mutable_gpu_data()), 0);
    greentea_gpu_powx<Dtype>(this->device_->id(), variance_.count(),
                             (cl_mem) (variance_.gpu_data()), 0, Dtype(0.5),
                             (cl_mem) (variance_.mutable_gpu_data()), 0);

    // replicate variance to input size
    greentea_gpu_gemm<Dtype>(this->device_->id(), CblasNoTrans, CblasNoTrans,
                             num, channels_, 1, 1,
                             (cl_mem) (batch_sum_multiplier_.gpu_data()), 0,
                             (cl_mem) (variance_.gpu_data()), 0, 0.,
                             (cl_mem) (num_by_chans_.mutable_gpu_data()), 0);
    greentea_gpu_gemm<Dtype>(this->device_->id(), CblasNoTrans, CblasNoTrans,
                             channels_ * num, spatial_dim, 1, 1.,
                             (cl_mem) (num_by_chans_.gpu_data()), 0,
                             (cl_mem) (spatial_sum_multiplier_.gpu_data()), 0,
                             0., (cl_mem) (temp_.mutable_gpu_data()), 0);
    greentea_gpu_div<Dtype>(this->device_->id(), temp_.count(),
                            (cl_mem) top_data, 0, (cl_mem) (temp_.gpu_data()),
                            0, (cl_mem) top_data, 0);
    // TODO(cdoersch): The caching is only needed because later in-place layers
    //                 might clobber the data.  Can we skip this if they won't?
    greentea_copy<Dtype>(x_norm_.count(), (cl_mem) top_data, 0,
                         (cl_mem) (x_norm_.mutable_gpu_data()), 0, &ctx);
#endif  // USE_GREENTEA
  }
}

template<typename Dtype>
void BatchNormLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                         const vector<bool>& propagate_down,
                                         const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_diff;

  if (this->device_->backend() == BACKEND_CUDA) {
#ifdef USE_CUDA
    if (bottom[0] != top[0]) {
      top_diff = top[0]->gpu_diff();
    } else {
      caffe_copy(x_norm_.count(), top[0]->gpu_diff(),
                 x_norm_.mutable_gpu_diff());
      top_diff = x_norm_.gpu_diff();
    }
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    if (use_global_stats_) {
      caffe_gpu_div(temp_.count(), top_diff, temp_.gpu_data(), bottom_diff);
      return;
    }
    const Dtype* top_data = x_norm_.gpu_data();
    int_tp num = bottom[0]->shape()[0];
    int_tp spatial_dim = bottom[0]->count() / (channels_ * bottom[0]->shape(0));
    // if Y = (X-mean(X))/(sqrt(var(X)+eps)), then
    //
    // dE(Y)/dX =
    //   (dE/dY - mean(dE/dY) - mean(dE/dY \cdot Y) \cdot Y)
    //     ./ sqrt(var(X) + eps)
    //
    // where \cdot and ./ are hadamard product and elementwise division,
    // respectively, dE/dY is the top diff, and mean/var/sum are all computed
    // along all dimensions except the channels dimension.  In the above
    // equation, the operations allow for expansion (i.e. broadcast) along all
    // dimensions except the channels dimension where required.

    // sum(dE/dY \cdot Y)
    caffe_gpu_mul<Dtype>(temp_.count(), top_data, top_diff, bottom_diff);
    caffe_gpu_gemv<Dtype>(CblasNoTrans, channels_ * num, spatial_dim, 1.,
                          bottom_diff, spatial_sum_multiplier_.gpu_data(), 0.,
                          num_by_chans_.mutable_gpu_data());
    caffe_gpu_gemv<Dtype>(CblasTrans, num, channels_, 1.,
                          num_by_chans_.gpu_data(),
                          batch_sum_multiplier_.gpu_data(), 0.,
                          mean_.mutable_gpu_data());

    // reshape (broadcast) the above
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num, channels_, 1, 1,
                          batch_sum_multiplier_.gpu_data(), mean_.gpu_data(),
                          0., num_by_chans_.mutable_gpu_data());
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, channels_ * num,
                          spatial_dim, 1, 1., num_by_chans_.gpu_data(),
                          spatial_sum_multiplier_.gpu_data(), 0., bottom_diff);

    // sum(dE/dY \cdot Y) \cdot Y
    caffe_gpu_mul<Dtype>(temp_.count(), top_data, bottom_diff, bottom_diff);

    // sum(dE/dY)-sum(dE/dY \cdot Y) \cdot Y
    caffe_gpu_gemv<Dtype>(CblasNoTrans, channels_ * num, spatial_dim, 1.,
                          top_diff, spatial_sum_multiplier_.gpu_data(), 0.,
                          num_by_chans_.mutable_gpu_data());
    caffe_gpu_gemv<Dtype>(CblasTrans, num, channels_, 1.,
                          num_by_chans_.gpu_data(),
                          batch_sum_multiplier_.gpu_data(), 0.,
                          mean_.mutable_gpu_data());
    // reshape (broadcast) the above to make
    // sum(dE/dY)-sum(dE/dY \cdot Y) \cdot Y
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num, channels_, 1, 1,
                          batch_sum_multiplier_.gpu_data(), mean_.gpu_data(),
                          0., num_by_chans_.mutable_gpu_data());
    caffe_gpu_gemm<Dtype>(CblasNoTrans, CblasNoTrans, num * channels_,
                          spatial_dim, 1, 1., num_by_chans_.gpu_data(),
                          spatial_sum_multiplier_.gpu_data(), 1., bottom_diff);

    // dE/dY - mean(dE/dY)-mean(dE/dY \cdot Y) \cdot Y
    caffe_gpu_axpby<Dtype>(temp_.count(), Dtype(1), top_diff,
                           Dtype(-1. / (num * spatial_dim)), bottom_diff);

    // note: temp_ still contains sqrt(var(X)+eps), computed during the forward
    // pass.
    caffe_gpu_div<Dtype>(temp_.count(), bottom_diff, temp_.gpu_data(),
                         bottom_diff);
#endif  // USE_CUDA
  } else {
#ifdef USE_GREENTEA
    viennacl::ocl::context &ctx = viennacl::ocl::get_context(
        this->device_->id());

    if (bottom[0] != top[0]) {
      top_diff = top[0]->gpu_diff();
    } else {
      greentea_copy<Dtype>(x_norm_.count(), (cl_mem) (top[0]->gpu_diff()), 0,
                           (cl_mem) (x_norm_.mutable_gpu_diff()), 0, &ctx);
      top_diff = x_norm_.gpu_diff();
    }
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    if (use_global_stats_) {
      greentea_gpu_div<Dtype>(this->device_->id(), temp_.count(),
                              (cl_mem) top_diff, 0, (cl_mem) (temp_.gpu_data()),
                              0, (cl_mem) bottom_diff, 0);
      return;
    }
    const Dtype* top_data = x_norm_.gpu_data();
    int_tp num = bottom[0]->shape()[0];
    int_tp spatial_dim = bottom[0]->count() / (channels_ * bottom[0]->shape(0));
    // if Y = (X-mean(X))/(sqrt(var(X)+eps)), then
    //
    // dE(Y)/dX =
    //   (dE/dY - mean(dE/dY) - mean(dE/dY \cdot Y) \cdot Y)
    //     ./ sqrt(var(X) + eps)
    //
    // where \cdot and ./ are hadamard product and elementwise division,
    // respectively, dE/dY is the top diff, and mean/var/sum are all computed
    // along all dimensions except the channels dimension.  In the above
    // equation, the operations allow for expansion (i.e. broadcast) along all
    // dimensions except the channels dimension where required.

    // sum(dE/dY \cdot Y)
    greentea_gpu_mul<Dtype>(this->device_->id(), temp_.count(),
                            (cl_mem) top_data, 0, (cl_mem) top_diff, 0,
                            (cl_mem) bottom_diff, 0);
    greentea_gpu_gemv<Dtype>(this->device_->id(), CblasNoTrans, channels_ * num,
                             spatial_dim, 1., (cl_mem) bottom_diff, 0,
                             (cl_mem) (spatial_sum_multiplier_.gpu_data()), 0,
                             0., (cl_mem) (num_by_chans_.mutable_gpu_data()),
                             0);
    greentea_gpu_gemv<Dtype>(this->device_->id(), CblasTrans, num, channels_,
                             1., (cl_mem) (num_by_chans_.gpu_data()), 0,
                             (cl_mem) (batch_sum_multiplier_.gpu_data()), 0, 0.,
                             (cl_mem) (mean_.mutable_gpu_data()), 0);

    // reshape (broadcast) the above
    greentea_gpu_gemm<Dtype>(this->device_->id(), CblasNoTrans, CblasNoTrans,
                             num, channels_, 1, 1,
                             (cl_mem) (batch_sum_multiplier_.gpu_data()), 0,
                             (cl_mem) (mean_.gpu_data()), 0, 0.,
                             (cl_mem) (num_by_chans_.mutable_gpu_data()), 0);
    greentea_gpu_gemm<Dtype>(this->device_->id(), CblasNoTrans, CblasNoTrans,
                             channels_ * num, spatial_dim, 1, 1.,
                             (cl_mem) (num_by_chans_.gpu_data()), 0,
                             (cl_mem) (spatial_sum_multiplier_.gpu_data()), 0,
                             0., (cl_mem) bottom_diff, 0);

    // sum(dE/dY \cdot Y) \cdot Y
    greentea_gpu_mul<Dtype>(this->device_->id(), temp_.count(),
                            (cl_mem) top_data, 0, (cl_mem) bottom_diff, 0,
                            (cl_mem) bottom_diff, 0);

    // sum(dE/dY)-sum(dE/dY \cdot Y) \cdot Y
    greentea_gpu_gemv<Dtype>(this->device_->id(), CblasNoTrans, channels_ * num,
                             spatial_dim, 1., (cl_mem) top_diff, 0,
                             (cl_mem) (spatial_sum_multiplier_.gpu_data()), 0,
                             0., (cl_mem) (num_by_chans_.mutable_gpu_data()),
                             0);
    greentea_gpu_gemv<Dtype>(this->device_->id(), CblasTrans, num, channels_,
                             1., (cl_mem) (num_by_chans_.gpu_data()), 0,
                             (cl_mem) (batch_sum_multiplier_.gpu_data()), 0, 0.,
                             (cl_mem) (mean_.mutable_gpu_data()), 0);
    // reshape (broadcast) the above to make
    // sum(dE/dY)-sum(dE/dY \cdot Y) \cdot Y
    greentea_gpu_gemm<Dtype>(this->device_->id(), CblasNoTrans, CblasNoTrans,
                             num, channels_, 1, 1,
                             (cl_mem) (batch_sum_multiplier_.gpu_data()), 0,
                             (cl_mem) (mean_.gpu_data()), 0, 0.,
                             (cl_mem) (num_by_chans_.mutable_gpu_data()), 0);
    greentea_gpu_gemm<Dtype>(this->device_->id(), CblasNoTrans, CblasNoTrans,
                             num * channels_, spatial_dim, 1, 1.,
                             (cl_mem) (num_by_chans_.gpu_data()), 0,
                             (cl_mem) (spatial_sum_multiplier_.gpu_data()), 0,
                             1., (cl_mem) bottom_diff, 0);

    // dE/dY - mean(dE/dY)-mean(dE/dY \cdot Y) \cdot Y
    greentea_gpu_axpby<Dtype>(this->device_->id(), temp_.count(), Dtype(1),
                              (cl_mem) top_diff, 0,
                              Dtype(-1. / (num * spatial_dim)),
                              (cl_mem) bottom_diff, 0);

    // note: temp_ still contains sqrt(var(X)+eps), computed during the forward
    // pass.
    greentea_gpu_div<Dtype>(this->device_->id(), temp_.count(),
                            (cl_mem) bottom_diff, 0,
                            (cl_mem) (temp_.gpu_data()), 0,
                            (cl_mem) bottom_diff, 0);
#endif  // USE_GREENTEA
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(BatchNormLayer);

}  // namespace caffe
