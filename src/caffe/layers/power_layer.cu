#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/power_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void PowerLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  // Special case where we can ignore the input: scale or power is 0.
  if (diff_scale_ == Dtype(0)) {
    Dtype value = (power_ == 0) ? Dtype(1) : pow(shift_, power_);
    caffe_gpu_set(count, value, top_data);
    return;
  }
  const Dtype* bottom_data = bottom[0]->gpu_data();
  caffe_copy(count, bottom_data, top_data);
  if (scale_ != Dtype(1)) {
    caffe_gpu_scal(count, scale_, top_data);
  }
  if (shift_ != Dtype(0)) {
    caffe_gpu_add_scalar(count, shift_, top_data);
  }
  if (power_ != Dtype(1)) {
    caffe_gpu_powx(count, top_data, power_, top_data);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(PowerLayer);


}  // namespace caffe
