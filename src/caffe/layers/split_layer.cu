// Copyright 2014 Jeff Donahue

#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
Dtype SplitLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      vector<Blob<Dtype>*>* top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  for (int i = 0; i < top->size(); ++i) {
    if (i == 0 && (*top)[i] == bottom[0]) {
      continue;
    }
    Dtype* top_data = (*top)[i]->mutable_gpu_data();
    caffe_gpu_copy(count_, bottom_data, top_data);
  }
  return Dtype(0.);
}

template <typename Dtype>
void SplitLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const bool propagate_down, vector<Blob<Dtype>*>* bottom) {
  if (propagate_down) {
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff();
    // Initialize by copying first top blob diff to our diff, unless we're
    // doing in-place computation for the first blob, in which case the diff is
    // already initialized.
    if (top[0] != (*bottom)[0]) {
      caffe_gpu_copy(count_, top_diff, bottom_diff);
    }
    // Add remaining top blob diffs.
    for (int i = 1; i < top.size(); ++i) {
      top_diff = top[i]->gpu_diff();
      caffe_gpu_axpy(count_, Dtype(1.), top_diff, bottom_diff);
    }
  }
}


INSTANTIATE_CLASS(SplitLayer);

}  // namespace caffe
