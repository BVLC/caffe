#include <vector>

#include "caffe/common_layers.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void SplitLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  for (int i = 0; i < top.size(); ++i) {
    top[i]->ShareData(*bottom[0]);
  }
}

template <typename Dtype>
void SplitLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) { return; }
  if (top.size() == 1) {
    caffe_copy(count_, top[0]->gpu_diff(), bottom[0]->mutable_gpu_diff());
    return;
  }
  caffe_gpu_add(count_, top[0]->gpu_diff(), top[1]->gpu_diff(),
                bottom[0]->mutable_gpu_diff());
  // Add remaining top blob diffs.
  for (int i = 2; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    caffe_gpu_axpy(count_, Dtype(1.), top_diff, bottom_diff);
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(SplitLayer);

}  // namespace caffe
