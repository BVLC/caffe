#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/permute_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void PermuteKernel(const int nthreads,
    Dtype* const bottom_data, const bool forward, const int* permute_order,
    const int* old_steps, const int* new_steps, const int num_axes,
    Dtype* const top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int temp_idx = index;
    int old_idx = 0;
    for (int i = 0; i < num_axes; ++i) {
      int order = permute_order[i];
      old_idx += (temp_idx / new_steps[i]) * old_steps[order];
      temp_idx %= new_steps[i];
    }
    if (forward) {
      top_data[index] = bottom_data[old_idx];
    } else {
      bottom_data[old_idx] = top_data[index];
    }
  }
}

template <typename Dtype>
void PermuteLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  if (need_permute_) {
    Dtype* bottom_data = bottom[0]->mutable_gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();
    int count = top[0]->count();
    const int* permute_order = permute_order_.gpu_data();
    const int* new_steps = new_steps_.gpu_data();
    const int* old_steps = old_steps_.gpu_data();
    bool foward = true;
    // NOLINT_NEXT_LINE(whitespace/operators)
    PermuteKernel<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, foward, permute_order, old_steps, new_steps,
        num_axes_, top_data);
    CUDA_POST_KERNEL_CHECK;
  } else {
    // If there is no need to permute, we share data to save memory.
    top[0]->ShareData(*bottom[0]);
  }
}


template <typename Dtype>
void PermuteLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (need_permute_) {
    Dtype* top_diff = top[0]->mutable_gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    const int* permute_order = permute_order_.gpu_data();
    const int* new_steps = new_steps_.gpu_data();
    const int* old_steps = old_steps_.gpu_data();
    bool foward = false;
    // NOLINT_NEXT_LINE(whitespace/operators)
    PermuteKernel<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_diff, foward, permute_order, old_steps, new_steps,
        num_axes_, top_diff);
    CUDA_POST_KERNEL_CHECK;
  } else {
    // If there is no need to permute, we share diff to save memory.
    bottom[0]->ShareDiff(*top[0]);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(PermuteLayer);

}  // namespace caffe
