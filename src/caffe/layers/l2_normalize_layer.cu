#include "hip/hip_runtime.h"
/*
Copyright (c) 2016. The Regents of the University of California (Regents). All
Rights Reserved. Permission to use, copy, modify, and distribute this software
and its documentation for educational, research, not-for-profit, and commercial
purposes (such rights not subject to transfer), without fee, and without a
signed licensing agreement, is hereby granted, provided that the above copyright
notice, this paragraph and the following two paragraphs appear in all copies,
modifications, and distributions. Contact The Office of Technology Licensing,
UC Berkeley, 2150 Shattuck Avenue, Suite 510, Berkeley, CA 94720-1620,
(510) 643-7201, for commercial licensing opportunities.

Yang Gao, University of California, Berkeley.


IN NO EVENT SHALL REGENTS BE LIABLE TO ANY PARTY FOR DIRECT, INDIRECT, SPECIAL,
INCIDENTAL, OR CONSEQUENTIAL DAMAGES, INCLUDING LOST PROFITS, ARISING OUT OF THE
USE OF THIS SOFTWARE AND ITS DOCUMENTATION, EVEN IF REGENTS HAS BEEN ADVISED OF
THE POSSIBILITY OF SUCH DAMAGE.

REGENTS SPECIFICALLY DISCLAIMS ANY WARRANTIES, INCLUDING, BUT NOT LIMITED TO,
THE IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE.
THE SOFTWARE AND ACCOMPANYING DOCUMENTATION, IF ANY, PROVIDED HEREUNDER IS
PROVIDED "AS IS". REGENTS HAS NO OBLIGATION TO PROVIDE MAINTENANCE, SUPPORT,
UPDATES, ENHANCEMENTS, OR MODIFICATIONS.
*/

#include <vector>

#include "caffe/layers/l2_normalize_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void L2NormalizeLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  Dtype* squared_data = squared_.mutable_gpu_data();
  Dtype normsqr;
  int n = bottom[0]->num();
  int d = bottom[0]->count() / n;
  caffe_gpu_powx(n * d, bottom_data, Dtype(2), squared_data);
  Dtype epsilon = 0.0000001;
  for (int i = 0;  i < n; ++i) {
    caffe_gpu_asum<Dtype>(d, squared_data + i * d, &normsqr);
    caffe_gpu_scale<Dtype>(d, pow(normsqr + epsilon, -0.5),
            bottom_data+i*d, top_data+i*d);
  }
}

template <typename Dtype>
void L2NormalizeLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  int n = top[0]->num();
  int d = top[0]->count() / n;
  Dtype a;
  Dtype epsilon = 0.0000001;
  for (int i = 0; i < n; ++i) {
    caffe_gpu_dot(d, top_data+i*d, top_diff+i*d, &a);
    caffe_gpu_scale(d, a, top_data+i*d, bottom_diff+i*d);
    caffe_gpu_sub(d, top_diff+i*d, bottom_diff+i*d, bottom_diff+i*d);
    caffe_gpu_dot(d, bottom_data+i*d, bottom_data+i*d, &a);
    caffe_gpu_scale(d, Dtype(pow(a + epsilon, -0.5)), bottom_diff+i*d,
            bottom_diff+i*d);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(L2NormalizeLayer);
}  // namespace caffe
