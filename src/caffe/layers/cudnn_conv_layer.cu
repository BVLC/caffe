
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

__global__ void sync_conv_groups() { }

template <typename Dtype>
void CuDNNConvolutionLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    Dtype* top_data = top[i]->mutable_gpu_data();
    const Dtype* weight = this->blobs_[0]->gpu_data();

    size_t workspace_limit_bytes = this->kernel_h_ *
                                   this->kernel_w_ *
                                   this->channels_ *
                                   sizeof(int) + 1;

    // Forward through cuDNN in parallel over groups.
    for (int g = 0; g < this->group_; g++) {
      cudnnConvolutionFwdAlgo_t algo;

      // pick the convolution algorithm
      // TODO(shelhamer) this should be done during reshape
      // TODO(shelhamer) the choice of automatic or manual algorithm picking
      // should be exposed in proto
      CUDNN_CHECK(cudnnGetConvolutionForwardAlgorithm(handle_[g],
        bottom_descs_[i],
        filter_desc_,
        conv_descs_[i],
        top_descs_[i],
        CUDNN_CONVOLUTION_FWD_SPECIFY_WORKSPACE_LIMIT,
        workspace_limit_bytes,  // memoryLimitInBytes,
        &algo));

      // get minimum size of the workspace needed for the desired algorithm
      size_t workspaceSizeInBytes_temp = 0;

      CUDNN_CHECK(cudnnGetConvolutionForwardWorkspaceSize(handle_[g],
        bottom_descs_[i],
        filter_desc_,
        conv_descs_[i],
        top_descs_[i],
        algo,
        &workspaceSizeInBytes_temp));

      if (workspaceSizeInBytes_temp > workspaceSizeInBytes) {
        workspaceSizeInBytes = workspaceSizeInBytes_temp;
        // free the existing workspace and allocate a new (larger) one
        cudaFree(this->workspace);
        cudaError_t err = cudaMalloc(&(this->workspace), workspaceSizeInBytes);
        if (err != cudaSuccess) {
          // force zero memory path
          algo = CUDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM;
          workspace = NULL;
          workspaceSizeInBytes = 0;
        }
      }

      // Filters.
      CUDNN_CHECK(cudnnConvolutionForward(handle_[g],
            cudnn::dataType<Dtype>::one,
            bottom_descs_[i], bottom_data + bottom_offset_ * g,
            filter_desc_, weight + weight_offset_ * g,
            conv_descs_[i],
            algo, workspace, workspaceSizeInBytes,
            cudnn::dataType<Dtype>::zero,
            top_descs_[i], top_data + top_offset_ * g));

      // Bias.
      if (this->bias_term_) {
        const Dtype* bias_data = this->blobs_[1]->gpu_data();
        CUDNN_CHECK(cudnnAddTensor(handle_[g], CUDNN_ADD_SAME_C,
              cudnn::dataType<Dtype>::one,
              bias_desc_, bias_data + bias_offset_ * g,
              cudnn::dataType<Dtype>::one,
              top_descs_[i], top_data + top_offset_ * g));
      }
    }

    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)
    sync_conv_groups<<<1, 1>>>();
  }
}

template <typename Dtype>
void CuDNNConvolutionLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* weight = NULL;
  Dtype* weight_diff = NULL;
  if (this->param_propagate_down_[0]) {
    weight = this->blobs_[0]->gpu_data();
    weight_diff = this->blobs_[0]->mutable_gpu_diff();
    caffe_gpu_set(this->blobs_[0]->count(), Dtype(0), weight_diff);
  }
  Dtype* bias_diff = NULL;
  if (this->bias_term_ && this->param_propagate_down_[1]) {
    bias_diff = this->blobs_[1]->mutable_gpu_diff();
    caffe_gpu_set(this->blobs_[1]->count(), Dtype(0), bias_diff);
  }
  for (int i = 0; i < top.size(); ++i) {
    const Dtype* top_diff = top[i]->gpu_diff();
    // Backward through cuDNN in parallel over groups and gradients.
    for (int g = 0; g < this->group_; g++) {
      // Gradient w.r.t. bias.
      if (this->bias_term_ && this->param_propagate_down_[1]) {
        CUDNN_CHECK(cudnnConvolutionBackwardBias(handle_[0*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              top_descs_[i],  top_diff + top_offset_ * g,
              cudnn::dataType<Dtype>::one,
              bias_desc_, bias_diff + bias_offset_ * g));
      }

      // Gradient w.r.t. weights.
      if (this->param_propagate_down_[0]) {
        const Dtype* bottom_data = bottom[i]->gpu_data();
        CUDNN_CHECK(cudnnConvolutionBackwardFilter(handle_[1*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              bottom_descs_[i], bottom_data + bottom_offset_ * g,
              top_descs_[i],    top_diff + top_offset_ * g,
              conv_descs_[i],
              cudnn::dataType<Dtype>::one,
              filter_desc_, weight_diff + weight_offset_ * g));
      }

      // Gradient w.r.t. bottom data.
      if (propagate_down[i]) {
        if (weight == NULL) {
          weight = this->blobs_[0]->gpu_data();
        }
        Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
        CUDNN_CHECK(cudnnConvolutionBackwardData(handle_[2*this->group_ + g],
              cudnn::dataType<Dtype>::one,
              filter_desc_, weight + weight_offset_ * g,
              top_descs_[i], top_diff + top_offset_ * g,
              conv_descs_[i],
              cudnn::dataType<Dtype>::zero,
              bottom_descs_[i], bottom_diff + bottom_offset_ * g));
      }
    }

    // Synchronize the work across groups, each of which went into its own
    // stream, by launching an empty kernel into the default (null) stream.
    // NOLINT_NEXT_LINE(whitespace/operators)
    sync_conv_groups<<<1, 1>>>();
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNConvolutionLayer);

}  // namespace caffe
#endif
