
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <vector>

#include "caffe/layers/cudnn_relu_layer.hpp"

namespace caffe {

template <typename Dtype>
void CuDNNReLULayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  // Fallback to standard Caffe for leaky ReLU.
  if (ReLULayer<Dtype>::layer_param_.relu_param().negative_slope() != 0) {
    return ReLULayer<Dtype>::Forward_gpu(bottom, top);
  }

  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
#if CUDNN_VERSION_MIN(5, 0, 0)
  CUDNN_CHECK(cudnnActivationForward(this->handle_,
        activ_desc_,
        cudnn::dataType<Dtype>::one,
        this->bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        this->top_desc_, top_data));
#else
  CUDNN_CHECK(cudnnActivationForward_v4(this->handle_,
        activ_desc_,
        cudnn::dataType<Dtype>::one,
        this->bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        this->top_desc_, top_data));
#endif
}

template <typename Dtype>
void CuDNNReLULayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }

  // Fallback to standard Caffe for leaky ReLU.
  if (ReLULayer<Dtype>::layer_param_.relu_param().negative_slope() != 0) {
    return ReLULayer<Dtype>::Backward_gpu(top, propagate_down, bottom);
  }

  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
#if CUDNN_VERSION_MIN(5, 0, 0)
  CUDNN_CHECK(cudnnActivationBackward(this->handle_,
        activ_desc_,
        cudnn::dataType<Dtype>::one,
        this->top_desc_, top_data, this->top_desc_, top_diff,
        this->bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        this->bottom_desc_, bottom_diff));
#else
  CUDNN_CHECK(cudnnActivationBackward_v4(this->handle_,
        activ_desc_,
        cudnn::dataType<Dtype>::one,
        this->top_desc_, top_data, this->top_desc_, top_diff,
        this->bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        this->bottom_desc_, bottom_diff));
#endif
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNReLULayer);

}  // namespace caffe
#endif
