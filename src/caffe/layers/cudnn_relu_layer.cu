
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void CuDNNReLULayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  // Fallback to standard Caffe for leaky ReLU.
  if (ReLULayer<Dtype>::layer_param_.relu_param().negative_slope() != 0) {
    return ReLULayer<Dtype>::Forward_gpu(bottom, top);
  }

  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = (*top)[0]->mutable_gpu_data();
  CUDNN_CHECK(cudnnActivationForward(this->handle_,
      CUDNN_ACTIVATION_RELU,
      this->bottom_desc_, bottom_data, this->top_desc_, top_data));
}

template <typename Dtype>
void CuDNNReLULayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    vector<Blob<Dtype>*>* bottom) {
  if (!propagate_down[0]) {
    return;
  }

  // Fallback to standard Caffe for leaky ReLU.
  if (ReLULayer<Dtype>::layer_param_.relu_param().negative_slope() != 0) {
    return ReLULayer<Dtype>::Backward_gpu(top, propagate_down, bottom);
  }

  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* bottom_data = (*bottom)[0]->gpu_data();
  Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff();
  CUDNN_CHECK(cudnnActivationBackward(this->handle_,
      CUDNN_ACTIVATION_RELU,
      this->top_desc_, top_data, this->top_desc_, top_diff,
      this->bottom_desc_, bottom_data, this->bottom_desc_, bottom_diff));
}

INSTANTIATE_CLASS(CuDNNReLULayer);

}  // namespace caffe
#endif
