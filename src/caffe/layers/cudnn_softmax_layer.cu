
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <algorithm>
#include <cfloat>
#include <vector>

#include "thrust/device_vector.h"

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void CuDNNSoftmaxLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = (*top)[0]->mutable_gpu_data();
  CUDNN_CHECK(cudnnSoftmaxForward(handle_, CUDNN_SOFTMAX_ACCURATE,
      CUDNN_SOFTMAX_MODE_CHANNEL,
      bottom_desc_, bottom_data, top_desc_, top_data));
}

template <typename Dtype>
void CuDNNSoftmaxLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, vector<Blob<Dtype>*>* bottom) {
  if (propagate_down[0]) {
    const Dtype* top_data = top[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* bottom_data = (*bottom)[0]->gpu_data();
    Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff();
    CUDNN_CHECK(cudnnSoftmaxBackward(handle_, CUDNN_SOFTMAX_ACCURATE,
        CUDNN_SOFTMAX_MODE_CHANNEL,
        top_desc_, top_data, top_desc_, top_diff, bottom_desc_, bottom_diff));
  }
}

INSTANTIATE_CLASS(CuDNNSoftmaxLayer);

}  // namespace caffe
#endif
