#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/bnll_layer.hpp"

namespace caffe {

const float kBNLL_THRESHOLD = 50.;

template <typename Dtype>
__global__ void BNLLForward(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] > 0 ?
        in[index] + log(1. + exp(-in[index])) :
        log(1. + exp(in[index]));
  }
}

template <typename Dtype>
void BNLLLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  BNLLForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void BNLLBackward(const int n, const Dtype* in_diff,
    const Dtype* in_data, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    Dtype expval = exp(min(in_data[index], Dtype(kBNLL_THRESHOLD)));
    out_diff[index] = in_diff[index] * expval / (expval + 1.);
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(BNLLLayer);


}  // namespace caffe
