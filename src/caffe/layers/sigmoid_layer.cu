#include "hip/hip_runtime.h"
#include <cmath>
#include <vector>

#include "caffe/layers/sigmoid_layer.hpp"

namespace caffe {

#ifdef USE_CUDA
template<typename Dtype>
__global__ void SigmoidForward(const int_tp n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = 1. / (1. + exp(-in[index]));
  }
}
#endif  // USE_CUDA

template<typename Dtype>
void SigmoidLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int_tp count = bottom[0]->count();

  if (this->device_->backend() == BACKEND_CUDA) {
#ifdef USE_CUDA
    // NOLINT_NEXT_LINE(whitespace/operators)
    SigmoidForward<Dtype> CUDA_KERNEL(CAFFE_GET_BLOCKS(count),
                                      CAFFE_CUDA_NUM_THREADS)(
        count, bottom_data, top_data);
    CUDA_POST_KERNEL_CHECK;
#endif  // USE_CUDA
  } else {
#ifdef USE_GREENTEA
    viennacl::ocl::context &ctx = viennacl::ocl::get_context(
        this->device_->id());
    viennacl::ocl::program &program = this->device_->program();

    viennacl::ocl::kernel &oclk_sigmoid = program.get_kernel(
        CL_KERNEL_SELECT("sigmoid_forward"));
    viennacl::ocl::enqueue(
        oclk_sigmoid(count, WrapHandle((cl_mem) bottom_data, &ctx),
                     WrapHandle((cl_mem) top_data, &ctx)),
        ctx.get_queue());
#endif  // USE_GREENTEA
  }

  // << " count: " << count << " bottom_data: "
  //     << (unsigned long)bottom_data
  //     << " top_data: " << (unsigned long)top_data
  //     << " blocks: " << CAFFE_GET_BLOCKS(count)
  //     << " threads: " << CAFFE_CUDA_NUM_THREADS;
}

#ifdef USE_CUDA
template<typename Dtype>
__global__ void SigmoidBackward(const int_tp n, const Dtype* in_diff,
                                const Dtype* out_data, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    const Dtype sigmoid_x = out_data[index];
    out_diff[index] = in_diff[index] * sigmoid_x * (1 - sigmoid_x);
  }
}
#endif  // USE_CUDA

template<typename Dtype>
void SigmoidLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                       const vector<bool>& propagate_down,
                                       const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_data = top[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int_tp count = bottom[0]->count();

    if (this->device_->backend() == BACKEND_CUDA) {
#ifdef USE_CUDA
      // NOLINT_NEXT_LINE(whitespace/operators)
      SigmoidBackward<Dtype> CUDA_KERNEL(CAFFE_GET_BLOCKS(count),
                                         CAFFE_CUDA_NUM_THREADS)(
          count, top_diff, top_data, bottom_diff);
      CUDA_POST_KERNEL_CHECK;
#endif  // USE_CUDA
    } else {
#ifdef USE_GREENTEA
      viennacl::ocl::context &ctx = viennacl::ocl::get_context(
          this->device_->id());
      viennacl::ocl::program &program = this->device_->program();

      viennacl::ocl::kernel &oclk_sigmoid = program.get_kernel(
          CL_KERNEL_SELECT("sigmoid_backward"));
      viennacl::ocl::enqueue(
          oclk_sigmoid(count, WrapHandle((cl_mem) top_diff, &ctx),
                       WrapHandle((cl_mem) top_data, &ctx),
                       WrapHandle((cl_mem) bottom_diff, &ctx)),
          ctx.get_queue());
#endif  // USE_GREENTEA
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SigmoidLayer);

}  // namespace caffe
