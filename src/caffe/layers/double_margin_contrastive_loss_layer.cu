#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/double_margin_contrastive_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void DoubleMarginContrastiveLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const int count = bottom[0]->count();
  caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),  // a
      bottom[1]->gpu_data(),  // b
      diff_.mutable_gpu_data());  // a_i-b_i
  caffe_gpu_powx(
      count,
      diff_.mutable_gpu_data(),  // a_i-b_i
      Dtype(2),
      diff_sq_.mutable_gpu_data());  // (a_i-b_i)^2
  caffe_gpu_gemv(
      CblasNoTrans,
      bottom[0]->num(),
      bottom[0]->channels(),
      Dtype(1.0),
      diff_sq_.gpu_data(),  // (a_i-b_i)^2
      summer_vec_.gpu_data(),
      Dtype(0.0),
      dist_sq_.mutable_gpu_data());  // \Sum (a_i-b_i)^2
  Dtype margin_gen =
      this->layer_param_.double_margin_contrastive_loss_param().margin_gen();
  Dtype margin_imp =
      this->layer_param_.double_margin_contrastive_loss_param().margin_imp();
  Dtype loss(0.0);
  for (int i = 0; i < bottom[0]->num(); ++i) {
    if (static_cast<int>(bottom[2]->cpu_data()[i])) {  // similar pairs
      Dtype dist = std::max(sqrt(dist_sq_.cpu_data()[i]) - margin_gen,
                            Dtype(0.0));
      loss += dist*dist;
    } else {  // dissimilar pairs
      Dtype dist = std::max(margin_imp - sqrt(dist_sq_.cpu_data()[i]),
                            Dtype(0.0));
      loss += dist*dist;
    }
  }
  loss = loss / static_cast<Dtype>(bottom[0]->num()) / Dtype(2);
  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
__global__ void CLLBackward(const int count, const int channels,
    const Dtype margin_gen, const Dtype margin_imp, const Dtype alpha,
    const Dtype* y, const Dtype* diff, const Dtype* dist_sq,
    Dtype *bottom_diff) {
  CUDA_KERNEL_LOOP(i, count) {
    int n = i / channels;  // the num index, to access y and dist_sq
    Dtype dist = sqrt(dist_sq[n]);
    Dtype mdist(0.0);
    Dtype beta(0.0);
    if (static_cast<int>(y[n])) {  // similar pairs
      mdist = (dist - margin_gen);
      beta = alpha * mdist / (dist + Dtype(1e-4)) * diff[i];
      if (mdist > 0.0) {
        bottom_diff[i] = beta;
      } else {
        bottom_diff[i] = 0;
      }
    } else {  // dissimilar pairs
      mdist = (margin_imp - dist);
      beta = -alpha * mdist / (dist + Dtype(1e-4)) * diff[i];
      if (mdist > 0.0) {
        bottom_diff[i] = beta;
      } else {
        bottom_diff[i] = 0;
      }
    }
  }
}

template <typename Dtype>
void DoubleMarginContrastiveLossLayer<Dtype>::Backward_gpu(
    const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  for (int i = 0; i < 2; ++i) {
    if (propagate_down[i]) {
      const int count = bottom[0]->count();
      const int channels = bottom[0]->channels();
      Dtype margin_gen =
        this->layer_param_.double_margin_contrastive_loss_param().margin_gen();
      Dtype margin_imp =
        this->layer_param_.double_margin_contrastive_loss_param().margin_imp();
      const Dtype sign = (i == 0) ? 1 : -1;
      const Dtype alpha = sign * top[0]->cpu_diff()[0] /
          static_cast<Dtype>(bottom[0]->num());
      // NOLINT_NEXT_LINE(whitespace/operators)
      CLLBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
          count, channels, margin_gen, margin_imp, alpha,
          bottom[2]->gpu_data(),  // pair similarity 0 or 1
          diff_.gpu_data(),  // the cached eltwise difference between a and b
          dist_sq_.gpu_data(),  // the cached square distance between a and b
          bottom[i]->mutable_gpu_diff());
      CUDA_POST_KERNEL_CHECK;
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(DoubleMarginContrastiveLossLayer);

}  // namespace caffe
