#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void ThresholdForward(const int n, const Dtype threshold,
    const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = in[index] > threshold ? 1 : 0;
  }
}

template <typename Dtype>
void ThresholdLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  ThresholdForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, threshold_, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
}


INSTANTIATE_LAYER_GPU_FORWARD(ThresholdLayer);


}  // namespace caffe
