#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/contrastive_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

#ifdef USE_GREENTEA
#include "caffe/greentea/greentea.hpp"
#include "caffe/greentea/greentea_math_functions.hpp"
#endif

namespace caffe {

template<typename Dtype>
void ContrastiveLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const int_tp count = bottom[0]->count();

  if (this->device_->backend() == BACKEND_CUDA) {
#ifdef USE_CUDA
    caffe_gpu_sub(count, bottom[0]->gpu_data(),  // a
                  bottom[1]->gpu_data(),  // b
                  diff_.mutable_gpu_data());  // a_i-b_i
    caffe_gpu_powx(count, diff_.mutable_gpu_data(),  // a_i-b_i
                   Dtype(2), diff_sq_.mutable_gpu_data());  // (a_i-b_i)^2
    caffe_gpu_gemv(CblasNoTrans, bottom[0]->num(), bottom[0]->channels(),
                   Dtype(1.0),
                   diff_sq_.gpu_data(),  // (a_i-b_i)^2
                   summer_vec_.gpu_data(), Dtype(0.0),
                   dist_sq_.mutable_gpu_data());  // \Sum (a_i-b_i)^2
#endif  // USE_CUDA
  } else {
#ifdef USE_GREENTEA
    greentea_gpu_sub<Dtype>(this->device_->id(), count,
                            (cl_mem) (bottom[0]->gpu_data()), 0,
                            (cl_mem) (bottom[1]->gpu_data()), 0,
                            (cl_mem) (diff_.mutable_gpu_data()), 0);
    greentea_gpu_powx<Dtype>(this->device_->id(), count,
                             (cl_mem) (diff_.mutable_gpu_data()),
                             0,  // a_i-b_i
                             Dtype(2), (cl_mem) (diff_sq_.mutable_gpu_data()),
                             0);  // (a_i-b_i)^2
    greentea_gpu_gemv<Dtype>(this->device_->id(), CblasNoTrans,
                             bottom[0]->num(), bottom[0]->channels(),
                             Dtype(1.0), (cl_mem) (diff_sq_.gpu_data()),
                             0,  // (a_i-b_i)^2
                             (cl_mem) (summer_vec_.gpu_data()), 0, Dtype(0.0),
                             (cl_mem) (dist_sq_.mutable_gpu_data()), 0);
#endif  // USE_GREENTEA
  }

  Dtype margin = this->layer_param_.contrastive_loss_param().margin();
  bool legacy_version = this->layer_param_.contrastive_loss_param()
      .legacy_version();
  Dtype loss(0.0);
  for (int_tp i = 0; i < bottom[0]->num(); ++i) {
    if (static_cast<int_tp>(bottom[2]->cpu_data()[i])) {  // similar pairs
      loss += dist_sq_.cpu_data()[i];
    } else {  // dissimilar pairs
      if (legacy_version) {
        loss += std::max(margin - dist_sq_.cpu_data()[i], Dtype(0.0));
      } else {
        Dtype dist = std::max(margin - (Dtype) sqrt(dist_sq_.cpu_data()[i]),
                              Dtype(0.0));
        loss += dist * dist;
      }
    }
  }
  loss = loss / static_cast<Dtype>(bottom[0]->num()) / Dtype(2);
  top[0]->mutable_cpu_data()[0] = loss;
}

#ifdef USE_CUDA
template<typename Dtype>
__global__ void CLLBackward(const int_tp count, const int_tp channels,
                            const Dtype margin, const bool legacy_version,
                            const Dtype alpha, const Dtype* y,
                            const Dtype* diff, const Dtype* dist_sq,
                            Dtype *bottom_diff) {
  CUDA_KERNEL_LOOP(i, count) {
    int_tp n = i / channels;  // the num index, to access y and dist_sq
    if (static_cast<int_tp>(y[n])) {  // similar pairs
      bottom_diff[i] = alpha * diff[i];
    } else {  // dissimilar pairs
      Dtype mdist(0.0);
      Dtype beta(0.0);
      if (legacy_version) {
        mdist = (margin - dist_sq[n]);
        beta = -alpha;
      } else {
        Dtype dist = sqrt(dist_sq[n]);
        mdist = (margin - dist);
        beta = -alpha * mdist / (dist + Dtype(1e-4)) * diff[i];
      }
      if (mdist > 0.0) {
        bottom_diff[i] = beta;
      } else {
        bottom_diff[i] = 0;
      }
    }
  }
}
#endif  // USE_CUDA

template<typename Dtype>
void ContrastiveLossLayer<Dtype>::Backward_gpu(
    const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  for (int_tp i = 0; i < 2; ++i) {
    if (propagate_down[i]) {
      const int_tp count = bottom[0]->count();
      const int_tp channels = bottom[0]->channels();
      Dtype margin = this->layer_param_.contrastive_loss_param().margin();
      const bool legacy_version = this->layer_param_.contrastive_loss_param()
          .legacy_version();
      const Dtype sign = (i == 0) ? 1 : -1;
      const Dtype alpha = sign * top[0]->cpu_diff()[0]
          / static_cast<Dtype>(bottom[0]->num());

      if (this->device_->backend() == BACKEND_CUDA) {
#ifdef USE_CUDA
        // NOLINT_NEXT_LINE(whitespace/operators)
        CLLBackward<Dtype> CUDA_KERNEL(CAFFE_GET_BLOCKS(count),
                                       CAFFE_CUDA_NUM_THREADS)(
            count, channels, margin, legacy_version, alpha,
            bottom[2]->gpu_data(),  // pair similarity 0 or 1
            diff_.gpu_data(),  // the cached eltwise difference between a and b
            dist_sq_.gpu_data(),  // the cached square distance between a and b
            bottom[i]->mutable_gpu_diff());
        CUDA_POST_KERNEL_CHECK;
#endif  // USE_CUDA
      } else {
#ifdef USE_GREENTEA
        viennacl::ocl::context &ctx = viennacl::ocl::get_context(
            this->device_->id());
        viennacl::ocl::program &program = this->device_->program();

        viennacl::ocl::kernel &oclk_cll = program.get_kernel(
            CL_KERNEL_SELECT("cll_backward"));
        viennacl::ocl::enqueue(
            oclk_cll(
                count, channels, margin, legacy_version ? 1 : 0, alpha,
                WrapHandle((cl_mem) (bottom[2]->gpu_data()), &ctx),
                WrapHandle((cl_mem) (diff_.gpu_data()), &ctx),
                WrapHandle((cl_mem) (dist_sq_.gpu_data()), &ctx),
                WrapHandle((cl_mem) (bottom[i]->mutable_gpu_diff()), &ctx)),
            ctx.get_queue());

#endif  // USE_GREENTEA
      }
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(ContrastiveLossLayer);

}  // namespace caffe
