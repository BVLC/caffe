#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/loss_layers.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void ContrastiveLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const int count = bottom[0]->count();
  caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),  // a
      bottom[1]->gpu_data(),  // b
      diff_.mutable_gpu_data());  // a_i-b_i
  caffe_gpu_powx(
      count,
      diff_.mutable_gpu_data(),  // a_i-b_i
      Dtype(2),
      diff_sq_.mutable_gpu_data());  // (a_i-b_i)^2
  caffe_gpu_gemv(
      CblasNoTrans,
      bottom[0]->num(),
      bottom[0]->channels(),
      Dtype(1.0),
      diff_sq_.gpu_data(),  // (a_i-b_i)^2
      summer_vec_.gpu_data(),
      Dtype(0.0),
      dist_sq_.mutable_gpu_data());  // \Sum (a_i-b_i)^2
  Dtype margin = this->layer_param_.contrastive_loss_param().margin();
  bool legacy_version =
      this->layer_param_.contrastive_loss_param().legacy_version();
  Dtype loss(0.0);
  for (int i = 0; i < bottom[0]->num(); ++i) {
    if (static_cast<int>(bottom[2]->cpu_data()[i])) {  // similar pairs
      loss += dist_sq_.cpu_data()[i];
    } else {  // dissimilar pairs
      if (legacy_version) {
        loss += std::max(margin - dist_sq_.cpu_data()[i], Dtype(0.0));
      } else {
        Dtype dist = std::max(margin - sqrt(dist_sq_.cpu_data()[i]),
                              Dtype(0.0));
        loss += dist*dist;
      }
    }
  }
  loss = loss / static_cast<Dtype>(bottom[0]->num()) / Dtype(2);
  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
__global__ void CLLBackward(const int count, const int channels,
    const Dtype margin, const bool legacy_version, const Dtype alpha,
    const Dtype* y, const Dtype* diff, const Dtype* dist_sq,
    Dtype *bottom_diff) {
  CUDA_KERNEL_LOOP(i, count) {
    int n = i / channels;  // the num index, to access y and dist_sq
    if (static_cast<int>(y[n])) {  // similar pairs
      bottom_diff[i] = alpha * diff[i];
    } else {  // dissimilar pairs
      Dtype mdist(0.0);
      Dtype beta(0.0);
      if (legacy_version) {
        mdist = (margin - dist_sq[n]);
        beta = -alpha;
      } else {
        Dtype dist = sqrt(dist_sq[n]);
        mdist = (margin - dist);
        beta = -alpha * mdist / (dist + Dtype(1e-4)) * diff[i];
      }
      if (mdist > 0.0) {
        bottom_diff[i] = beta;
      } else {
        bottom_diff[i] = 0;
      }
    }
  }
}

template <typename Dtype>
void ContrastiveLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  for (int i = 0; i < 2; ++i) {
    if (propagate_down[i]) {
      const int count = bottom[0]->count();
      const int channels = bottom[0]->channels();
      Dtype margin = this->layer_param_.contrastive_loss_param().margin();
      const bool legacy_version =
          this->layer_param_.contrastive_loss_param().legacy_version();
      const Dtype sign = (i == 0) ? 1 : -1;
      const Dtype alpha = sign * top[0]->cpu_diff()[0] /
          static_cast<Dtype>(bottom[0]->num());
      // NOLINT_NEXT_LINE(whitespace/operators)
      CLLBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
          count, channels, margin, legacy_version, alpha,
          bottom[2]->gpu_data(),  // pair similarity 0 or 1
          diff_.gpu_data(),  // the cached eltwise difference between a and b
          dist_sq_.gpu_data(),  // the cached square distance between a and b
          bottom[i]->mutable_gpu_diff());
      CUDA_POST_KERNEL_CHECK;
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(ContrastiveLossLayer);

}  // namespace caffe
