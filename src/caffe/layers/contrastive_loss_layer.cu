#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

<<<<<<< HEAD
<<<<<<< HEAD
#include "caffe/loss_layers.hpp"
#include "caffe/util/math_functions.hpp"
=======
<<<<<<< HEAD
=======
>>>>>>> pod-caffe-pod.hpp-merge
<<<<<<< HEAD
#include "caffe/loss_layers.hpp"
#include "caffe/util/math_functions.hpp"
=======
#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"
>>>>>>> origin/BVLC/parallel
=======
#include "caffe/loss_layers.hpp"
#include "caffe/util/math_functions.hpp"
>>>>>>> caffe
<<<<<<< HEAD
>>>>>>> pod-caffe-pod.hpp-merge
=======
>>>>>>> pod-caffe-pod.hpp-merge

namespace caffe {

template <typename Dtype>
void ContrastiveLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const int count = bottom[0]->count();
  caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),  // a
      bottom[1]->gpu_data(),  // b
      diff_.mutable_gpu_data());  // a_i-b_i
  caffe_gpu_powx(
      count,
      diff_.mutable_gpu_data(),  // a_i-b_i
      Dtype(2),
      diff_sq_.mutable_gpu_data());  // (a_i-b_i)^2
  caffe_gpu_gemv(
      CblasNoTrans,
      bottom[0]->num(),
      bottom[0]->channels(),
      Dtype(1.0),
      diff_sq_.gpu_data(),  // (a_i-b_i)^2
      summer_vec_.gpu_data(),
      Dtype(0.0),
      dist_sq_.mutable_gpu_data());  // \Sum (a_i-b_i)^2
  Dtype margin = this->layer_param_.contrastive_loss_param().margin();
<<<<<<< HEAD
<<<<<<< HEAD
  bool legacy_version =
      this->layer_param_.contrastive_loss_param().legacy_version();
=======
<<<<<<< HEAD
=======
>>>>>>> pod-caffe-pod.hpp-merge
<<<<<<< HEAD
  bool legacy_version =
      this->layer_param_.contrastive_loss_param().legacy_version();
=======
>>>>>>> origin/BVLC/parallel
=======
  bool legacy_version =
      this->layer_param_.contrastive_loss_param().legacy_version();
>>>>>>> caffe
<<<<<<< HEAD
>>>>>>> pod-caffe-pod.hpp-merge
=======
>>>>>>> pod-caffe-pod.hpp-merge
  Dtype loss(0.0);
  for (int i = 0; i < bottom[0]->num(); ++i) {
    if (static_cast<int>(bottom[2]->cpu_data()[i])) {  // similar pairs
      loss += dist_sq_.cpu_data()[i];
    } else {  // dissimilar pairs
<<<<<<< HEAD
<<<<<<< HEAD
=======
<<<<<<< HEAD
<<<<<<< HEAD
=======
>>>>>>> caffe
>>>>>>> pod-caffe-pod.hpp-merge
=======
<<<<<<< HEAD
=======
>>>>>>> caffe
>>>>>>> pod-caffe-pod.hpp-merge
      if (legacy_version) {
        loss += std::max(margin - dist_sq_.cpu_data()[i], Dtype(0.0));
      } else {
        Dtype dist = std::max(margin - sqrt(dist_sq_.cpu_data()[i]),
                              Dtype(0.0));
        loss += dist*dist;
      }
<<<<<<< HEAD
=======
<<<<<<< HEAD
<<<<<<< HEAD
=======
=======
>>>>>>> pod-caffe-pod.hpp-merge
      loss += std::max(margin-dist_sq_.cpu_data()[i], Dtype(0.0));
>>>>>>> origin/BVLC/parallel
=======
>>>>>>> caffe
<<<<<<< HEAD
>>>>>>> pod-caffe-pod.hpp-merge
=======
>>>>>>> pod-caffe-pod.hpp-merge
    }
  }
  loss = loss / static_cast<Dtype>(bottom[0]->num()) / Dtype(2);
  top[0]->mutable_cpu_data()[0] = loss;
}

template <typename Dtype>
<<<<<<< HEAD
<<<<<<< HEAD
__global__ void CLLBackward(const int count, const int channels,
    const Dtype margin, const bool legacy_version, const Dtype alpha,
=======
<<<<<<< HEAD
=======
>>>>>>> pod-caffe-pod.hpp-merge
<<<<<<< HEAD
__global__ void CLLBackward(const int count, const int channels,
    const Dtype margin, const bool legacy_version, const Dtype alpha,
=======
__global__ void CLLForward(const int count, const int channels,
    const Dtype margin, const Dtype alpha,
>>>>>>> origin/BVLC/parallel
=======
__global__ void CLLBackward(const int count, const int channels,
    const Dtype margin, const bool legacy_version, const Dtype alpha,
>>>>>>> caffe
<<<<<<< HEAD
>>>>>>> pod-caffe-pod.hpp-merge
=======
>>>>>>> pod-caffe-pod.hpp-merge
    const Dtype* y, const Dtype* diff, const Dtype* dist_sq,
    Dtype *bottom_diff) {
  CUDA_KERNEL_LOOP(i, count) {
    int n = i / channels;  // the num index, to access y and dist_sq
    if (static_cast<int>(y[n])) {  // similar pairs
      bottom_diff[i] = alpha * diff[i];
    } else {  // dissimilar pairs
<<<<<<< HEAD
<<<<<<< HEAD
=======
<<<<<<< HEAD
<<<<<<< HEAD
=======
>>>>>>> caffe
>>>>>>> pod-caffe-pod.hpp-merge
=======
<<<<<<< HEAD
=======
>>>>>>> caffe
>>>>>>> pod-caffe-pod.hpp-merge
      Dtype mdist(0.0);
      Dtype beta(0.0);
      if (legacy_version) {
        mdist = (margin - dist_sq[n]);
        beta = -alpha;
      } else {
        Dtype dist = sqrt(dist_sq[n]);
        mdist = (margin - dist);
        beta = -alpha * mdist / (dist + Dtype(1e-4)) * diff[i];
      }
      if (mdist > 0.0) {
        bottom_diff[i] = beta;
<<<<<<< HEAD
=======
<<<<<<< HEAD
<<<<<<< HEAD
=======
=======
>>>>>>> pod-caffe-pod.hpp-merge
      if ((margin-dist_sq[n]) > 0.0) {
        bottom_diff[i] = -alpha * diff[i];
>>>>>>> origin/BVLC/parallel
=======
>>>>>>> caffe
<<<<<<< HEAD
>>>>>>> pod-caffe-pod.hpp-merge
=======
>>>>>>> pod-caffe-pod.hpp-merge
      } else {
        bottom_diff[i] = 0;
      }
    }
  }
}

template <typename Dtype>
void ContrastiveLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  for (int i = 0; i < 2; ++i) {
    if (propagate_down[i]) {
      const int count = bottom[0]->count();
      const int channels = bottom[0]->channels();
      Dtype margin = this->layer_param_.contrastive_loss_param().margin();
<<<<<<< HEAD
<<<<<<< HEAD
      const bool legacy_version =
          this->layer_param_.contrastive_loss_param().legacy_version();
=======
<<<<<<< HEAD
=======
>>>>>>> pod-caffe-pod.hpp-merge
<<<<<<< HEAD
      const bool legacy_version =
          this->layer_param_.contrastive_loss_param().legacy_version();
=======
>>>>>>> origin/BVLC/parallel
=======
      const bool legacy_version =
          this->layer_param_.contrastive_loss_param().legacy_version();
>>>>>>> caffe
<<<<<<< HEAD
>>>>>>> pod-caffe-pod.hpp-merge
=======
>>>>>>> pod-caffe-pod.hpp-merge
      const Dtype sign = (i == 0) ? 1 : -1;
      const Dtype alpha = sign * top[0]->cpu_diff()[0] /
          static_cast<Dtype>(bottom[0]->num());
      // NOLINT_NEXT_LINE(whitespace/operators)
<<<<<<< HEAD
<<<<<<< HEAD
      CLLBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
          count, channels, margin, legacy_version, alpha,
=======
<<<<<<< HEAD
=======
>>>>>>> pod-caffe-pod.hpp-merge
<<<<<<< HEAD
      CLLBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
          count, channels, margin, legacy_version, alpha,
=======
      CLLForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
          count, channels, margin, alpha,
>>>>>>> origin/BVLC/parallel
=======
      CLLBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
          count, channels, margin, legacy_version, alpha,
>>>>>>> caffe
<<<<<<< HEAD
>>>>>>> pod-caffe-pod.hpp-merge
=======
>>>>>>> pod-caffe-pod.hpp-merge
          bottom[2]->gpu_data(),  // pair similarity 0 or 1
          diff_.gpu_data(),  // the cached eltwise difference between a and b
          dist_sq_.gpu_data(),  // the cached square distance between a and b
          bottom[i]->mutable_gpu_diff());
      CUDA_POST_KERNEL_CHECK;
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(ContrastiveLossLayer);

}  // namespace caffe
