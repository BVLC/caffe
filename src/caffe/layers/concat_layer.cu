#include "hip/hip_runtime.h"
#include <vector>

<<<<<<< HEAD
#include "caffe/layers/concat_layer.hpp"
=======
#include "caffe/common_layers.hpp"
>>>>>>> pod-caffe-pod.hpp-merge
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
=======
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
=======
>>>>>>> caffe
>>>>>>> pod-caffe-pod.hpp-merge
=======
<<<<<<< HEAD
=======
>>>>>>> caffe
>>>>>>> pod-caffe-pod.hpp-merge
=======
>>>>>>> caffe
>>>>>>> pod/caffe-merge
=======
>>>>>>> caffe
>>>>>>> pod/caffe-merge
=======
=======
<<<<<<< HEAD
<<<<<<< HEAD
=======
>>>>>>> caffe
>>>>>>> pod-caffe-pod.hpp-merge
>>>>>>> pod/device/blob.hpp
=======
>>>>>>> device-abstraction
=======
=======
>>>>>>> caffe
>>>>>>> pod-caffe-pod.hpp-merge
__global__ void Concat(const int nthreads, const Dtype* in_data,
    const bool forward, const int num_concats, const int concat_size,
    const int top_concat_axis, const int bottom_concat_axis,
    const int offset_concat_axis, Dtype* out_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int total_concat_size = concat_size * bottom_concat_axis;
    const int concat_num = index / total_concat_size;
    const int concat_index = index % total_concat_size;
    const int top_index = concat_index +
        (concat_num * top_concat_axis + offset_concat_axis) * concat_size;
    if (forward) {
      out_data[top_index] = in_data[index];
    } else {
      out_data[index] = in_data[top_index];
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
    }
  }
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
=======
>>>>>>> pod/device/blob.hpp
=======
<<<<<<< HEAD
    }
  }
<<<<<<< HEAD
=======
>>>>>>> pod-caffe-pod.hpp-merge
=======
>>>>>>> pod/caffe-merge
=======
>>>>>>> pod/caffe-merge
=======
>>>>>>> pod/device/blob.hpp
=======
    }
  }
>>>>>>> pod-caffe-pod.hpp-merge
}

template <typename Dtype>
void ConcatLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  if (bottom.size() == 1) { return; }
  Dtype* top_data = top[0]->mutable_gpu_data();
  int offset_concat_axis = 0;
  const int top_concat_axis = top[0]->shape(concat_axis_);
  const bool kForward = true;
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    const int bottom_concat_axis = bottom[i]->shape(concat_axis_);
    const int bottom_concat_size = bottom_concat_axis * concat_input_size_;
    const int nthreads = bottom_concat_size * num_concats_;
    Concat<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
        nthreads, bottom_data, kForward, num_concats_, concat_input_size_,
        top_concat_axis, bottom_concat_axis, offset_concat_axis, top_data);
    offset_concat_axis += bottom_concat_axis;
  }
=======
void ConcatLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  Dtype* top_data = top[0]->mutable_gpu_data();
  if (concat_dim_ == 0) {
    int offset_num = 0;
    for (int i = 0; i < bottom.size(); ++i) {
      const Dtype* bottom_data = bottom[i]->gpu_data();
      caffe_copy(bottom[i]->count(), bottom_data,
        top_data + top[0]->offset(offset_num));
      offset_num += bottom[i]->num();
    }
  } else if (concat_dim_ == 1) {
    int offset_channel = 0;
    for (int i = 0; i < bottom.size(); ++i) {
      const Dtype* bottom_data = bottom[i]->gpu_data();
      int num_elem =
        bottom[i]->channels() * bottom[i]->height() * bottom[i]->width();
      for (int n = 0; n < num_; ++n) {
        caffe_copy(num_elem, bottom_data+bottom[i]->offset(n),
          top_data + top[0]->offset(n, offset_channel));
      }
      offset_channel += bottom[i]->channels();
    }
  } else {
    LOG(FATAL) << "concat_dim along dim" << concat_dim_ <<
      " not implemented yet";
  }
>>>>>>> origin/BVLC/parallel
=======
    }
  }
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
>>>>>>> pod-caffe-pod.hpp-merge
=======
>>>>>>> pod-caffe-pod.hpp-merge
=======
>>>>>>> pod/caffe-merge
=======
>>>>>>> pod/caffe-merge
=======
>>>>>>> pod-caffe-pod.hpp-merge
>>>>>>> pod/device/blob.hpp
=======
    }
  }
>>>>>>> device-abstraction
=======
>>>>>>> pod-caffe-pod.hpp-merge
}

template <typename Dtype>
void ConcatLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  if (bottom.size() == 1) { return; }
  Dtype* top_data = top[0]->mutable_gpu_data();
  int offset_concat_axis = 0;
  const int top_concat_axis = top[0]->shape(concat_axis_);
  const bool kForward = true;
  for (int i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    const int bottom_concat_axis = bottom[i]->shape(concat_axis_);
    const int bottom_concat_size = bottom_concat_axis * concat_input_size_;
    const int nthreads = bottom_concat_size * num_concats_;
    Concat<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
        nthreads, bottom_data, kForward, num_concats_, concat_input_size_,
        top_concat_axis, bottom_concat_axis, offset_concat_axis, top_data);
    offset_concat_axis += bottom_concat_axis;
  }
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
=======
>>>>>>> caffe
>>>>>>> pod-caffe-pod.hpp-merge
=======
>>>>>>> caffe
>>>>>>> pod-caffe-pod.hpp-merge
=======
>>>>>>> caffe
>>>>>>> pod/caffe-merge
=======
>>>>>>> caffe
>>>>>>> pod/caffe-merge
=======
=======
>>>>>>> caffe
>>>>>>> pod-caffe-pod.hpp-merge
>>>>>>> pod/device/blob.hpp
=======
>>>>>>> device-abstraction
=======
>>>>>>> caffe
>>>>>>> pod-caffe-pod.hpp-merge
}

template <typename Dtype>
void ConcatLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
  if (bottom.size() == 1) { return; }
  const Dtype* top_diff = top[0]->gpu_diff();
<<<<<<< HEAD
<<<<<<< HEAD
=======
<<<<<<< HEAD
=======
>>>>>>> pod-caffe-pod.hpp-merge
<<<<<<< HEAD
  if (bottom.size() == 1) { return; }
  const Dtype* top_diff = top[0]->gpu_diff();
=======
>>>>>>> pod/caffe-merge
=======
>>>>>>> pod/caffe-merge
=======
  if (bottom.size() == 1) { return; }
  const Dtype* top_diff = top[0]->gpu_diff();
=======
<<<<<<< HEAD
<<<<<<< HEAD
  if (bottom.size() == 1) { return; }
  const Dtype* top_diff = top[0]->gpu_diff();
>>>>>>> pod/device/blob.hpp
=======
  if (bottom.size() == 1) { return; }
  const Dtype* top_diff = top[0]->gpu_diff();
>>>>>>> pod-caffe-pod.hpp-merge
  int offset_concat_axis = 0;
  const int top_concat_axis = top[0]->shape(concat_axis_);
  const bool kForward = false;
  for (int i = 0; i < bottom.size(); ++i) {
    const int bottom_concat_axis = bottom[i]->shape(concat_axis_);
    if (propagate_down[i]) {
      Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
      const int bottom_concat_size = bottom_concat_axis * concat_input_size_;
      const int nthreads = bottom_concat_size * num_concats_;
      Concat<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
          nthreads, top_diff, kForward, num_concats_, concat_input_size_,
          top_concat_axis, bottom_concat_axis, offset_concat_axis, bottom_diff);
=======
  const Dtype* top_diff = top[0]->gpu_diff();
  if (concat_dim_ == 0) {
    int offset_num = 0;
    for (int i = 0; i < bottom.size(); ++i) {
      Blob<Dtype>* blob = bottom[i];
      if (propagate_down[i]) {
        Dtype* bottom_diff = blob->mutable_gpu_diff();
        caffe_copy(blob->count(), top_diff + top[0]->offset(offset_num),
                       bottom_diff);
      }
      offset_num += blob->num();
    }
  } else if (concat_dim_ == 1) {
    int offset_channel = 0;
    for (int i = 0; i < bottom.size(); ++i) {
      Blob<Dtype>* blob = bottom[i];
      if (propagate_down[i]) {
        Dtype* bottom_diff = blob->mutable_gpu_diff();
        int num_elem = blob->channels()*blob->height()*blob->width();
        for (int n = 0; n < num_; ++n) {
          caffe_copy(num_elem, top_diff + top[0]->offset(n, offset_channel),
                         bottom_diff + blob->offset(n));
        }
      }
      offset_channel += blob->channels();
>>>>>>> origin/BVLC/parallel
=======
  if (bottom.size() == 1) { return; }
  const Dtype* top_diff = top[0]->gpu_diff();
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
>>>>>>> pod-caffe-pod.hpp-merge
=======
>>>>>>> pod-caffe-pod.hpp-merge
=======
>>>>>>> pod/caffe-merge
=======
>>>>>>> pod/caffe-merge
=======
>>>>>>> pod-caffe-pod.hpp-merge
>>>>>>> pod/device/blob.hpp
=======
  if (bottom.size() == 1) { return; }
  const Dtype* top_diff = top[0]->gpu_diff();
>>>>>>> device-abstraction
=======
>>>>>>> pod-caffe-pod.hpp-merge
  int offset_concat_axis = 0;
  const int top_concat_axis = top[0]->shape(concat_axis_);
  const bool kForward = false;
  for (int i = 0; i < bottom.size(); ++i) {
    const int bottom_concat_axis = bottom[i]->shape(concat_axis_);
    if (propagate_down[i]) {
      Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
      const int bottom_concat_size = bottom_concat_axis * concat_input_size_;
      const int nthreads = bottom_concat_size * num_concats_;
      Concat<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
          nthreads, top_diff, kForward, num_concats_, concat_input_size_,
          top_concat_axis, bottom_concat_axis, offset_concat_axis, bottom_diff);
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
=======
>>>>>>> caffe
>>>>>>> pod-caffe-pod.hpp-merge
=======
>>>>>>> caffe
>>>>>>> pod-caffe-pod.hpp-merge
=======
>>>>>>> caffe
>>>>>>> pod/caffe-merge
=======
>>>>>>> caffe
>>>>>>> pod/caffe-merge
=======
=======
>>>>>>> caffe
>>>>>>> pod-caffe-pod.hpp-merge
>>>>>>> pod/device/blob.hpp
=======
>>>>>>> device-abstraction
=======
>>>>>>> caffe
>>>>>>> pod-caffe-pod.hpp-merge
    }
    offset_concat_axis += bottom_concat_axis;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(ConcatLayer);

}  // namespace caffe
