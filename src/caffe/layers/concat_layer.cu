// Copyright 2014 BVLC and contributors.

#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
Dtype ConcatLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      vector<Blob<Dtype>*>* top) {
  Dtype* top_data = (*top)[0]->mutable_gpu_data();
  if (concat_dim_ == 0) {
    int offset_num = 0;
    for (int i = 0; i < bottom.size(); ++i) {
      const Dtype* bottom_data = bottom[i]->gpu_data();
      caffe_gpu_copy(bottom[i]->count(), bottom_data,
        top_data + (*top)[0]->offset(offset_num));
      offset_num += bottom[i]->num();
    }
  } else if (concat_dim_ == 1) {
    int offset_channel = 0;
    for (int i = 0; i < bottom.size(); ++i) {
      const Dtype* bottom_data = bottom[i]->gpu_data();
      int num_elem =
        bottom[i]->channels() * bottom[i]->height() * bottom[i]->width();
      for (int n = 0; n < num_; ++n) {
        caffe_gpu_copy(num_elem, bottom_data+bottom[i]->offset(n),
          top_data + (*top)[0]->offset(n, offset_channel));
      }
      offset_channel += bottom[i]->channels();
    }
  } else {
    LOG(FATAL) << "concat_dim along dim" << concat_dim_ <<
      " not implemented yet";
  }
  return Dtype(0.);
}

template <typename Dtype>
void ConcatLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, vector<Blob<Dtype>*>* bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  if (concat_dim_ == 0) {
    int offset_num = 0;
    for (int i = 0; i < bottom->size(); ++i) {
      Blob<Dtype>* blob = (*bottom)[i];
      if (propagate_down[i]) {
        Dtype* bottom_diff = blob->mutable_gpu_diff();
        caffe_gpu_copy(blob->count(), top_diff + top[0]->offset(offset_num),
                       bottom_diff);
      }
      offset_num += blob->num();
    }
  } else if (concat_dim_ == 1) {
    int offset_channel = 0;
    for (int i = 0; i < bottom->size(); ++i) {
      Blob<Dtype>* blob = (*bottom)[i];
      if (propagate_down[i]) {
        Dtype* bottom_diff = blob->mutable_gpu_diff();
        int num_elem = blob->channels()*blob->height()*blob->width();
        for (int n = 0; n < num_; ++n) {
          caffe_gpu_copy(num_elem, top_diff + top[0]->offset(n, offset_channel),
                         bottom_diff + blob->offset(n));
        }
      }
      offset_channel += blob->channels();
    }
  } else {
    LOG(FATAL) << "concat_dim along dim" << concat_dim_ <<
      " not implemented yet";
  }
}

INSTANTIATE_CLASS(ConcatLayer);

}  // namespace caffe
