#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/concat_layer.hpp"
#include "caffe/util/math_functions.hpp"

#ifdef USE_GREENTEA
#include "caffe/greentea/greentea.hpp"
#include "caffe/greentea/greentea_math_functions.hpp"
#endif

namespace caffe {

#ifdef USE_CUDA
template<typename Dtype>
__global__ void Concat(const int_tp nthreads, const Dtype* in_data,
                       const bool forward, const int_tp num_concats,
                       const int_tp concat_size, const int_tp top_concat_axis,
                       const int_tp bottom_concat_axis,
                       const int_tp offset_concat_axis, Dtype* out_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int_tp total_concat_size = concat_size * bottom_concat_axis;
    const int_tp concat_num = index / total_concat_size;
    const int_tp concat_index = index % total_concat_size;
    const int_tp top_index = concat_index
        + (concat_num * top_concat_axis + offset_concat_axis) * concat_size;
    if (forward) {
      out_data[top_index] = in_data[index];
    } else {
      out_data[index] = in_data[top_index];
    }
  }
}
#endif  // USE_CUDA

template<typename Dtype>
void ConcatLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  if (bottom.size() == 1) { return; }
  Dtype* top_data = top[0]->mutable_gpu_data();
  int_tp offset_concat_axis = 0;
  const int_tp top_concat_axis = top[0]->shape(concat_axis_);
  const bool kForward = true;
  for (int_tp i = 0; i < bottom.size(); ++i) {
    const Dtype* bottom_data = bottom[i]->gpu_data();
    const int_tp bottom_concat_axis = bottom[i]->shape(concat_axis_);
    const int_tp bottom_concat_size = bottom_concat_axis * concat_input_size_;
    const int_tp nthreads = bottom_concat_size * num_concats_;

    if (this->device_->backend() == BACKEND_CUDA) {
#ifdef USE_CUDA
      // NOLINT_NEXT_LINE(whitespace/operators)
      Concat<Dtype> CUDA_KERNEL(CAFFE_GET_BLOCKS(nthreads),
                                CAFFE_CUDA_NUM_THREADS)(
          nthreads, bottom_data, kForward, num_concats_, concat_input_size_,
          top_concat_axis, bottom_concat_axis, offset_concat_axis, top_data);
#endif  // USE_CUDA
    } else {
#ifdef USE_GREENTEA

      viennacl::ocl::context &ctx = viennacl::ocl::get_context(
          this->device_->id());
      viennacl::ocl::program &program = this->device_->program();

      viennacl::ocl::kernel &oclk_concat = program.get_kernel(
          CL_KERNEL_SELECT("concat"));
      viennacl::ocl::enqueue(
          oclk_concat(nthreads, WrapHandle((cl_mem) bottom_data, &ctx),
                      kForward ? 1 : 0, num_concats_, concat_input_size_,
                      top_concat_axis, bottom_concat_axis, offset_concat_axis,
                      WrapHandle((cl_mem) top_data, &ctx)),
          ctx.get_queue());
#endif  // USE_GREENTEA
    }
    offset_concat_axis += bottom_concat_axis;
  }
}

template<typename Dtype>
void ConcatLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (bottom.size() == 1) { return; }
  const Dtype* top_diff = top[0]->gpu_diff();
  int_tp offset_concat_axis = 0;
  const int_tp top_concat_axis = top[0]->shape(concat_axis_);
  const bool kForward = false;
  for (int_tp i = 0; i < bottom.size(); ++i) {
    const int_tp bottom_concat_axis = bottom[i]->shape(concat_axis_);
    if (propagate_down[i]) {
      Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
      const int_tp bottom_concat_axis = bottom[i]->shape(concat_axis_);
      const int_tp bottom_concat_size = bottom_concat_axis * concat_input_size_;
      const int_tp nthreads = bottom_concat_size * num_concats_;

      if (this->device_->backend() == BACKEND_CUDA) {
#ifdef USE_CUDA
        // NOLINT_NEXT_LINE(whitespace/operators)
        Concat<Dtype> CUDA_KERNEL(CAFFE_GET_BLOCKS(nthreads),
            CAFFE_CUDA_NUM_THREADS)(
            nthreads, top_diff, kForward, num_concats_, concat_input_size_,
            top_concat_axis, bottom_concat_axis,
            offset_concat_axis, bottom_diff);
#endif  // USE_CUDA
      } else {
#ifdef USE_GREENTEA

        viennacl::ocl::context &ctx = viennacl::ocl::get_context(
            this->device_->id());
        viennacl::ocl::program &program = this->device_->program();

        viennacl::ocl::kernel &oclk_concat = program.get_kernel(
            CL_KERNEL_SELECT("concat"));
        viennacl::ocl::enqueue(
            oclk_concat(nthreads, WrapHandle((cl_mem) top_diff, &ctx),
                        kForward ? 1 : 0, num_concats_, concat_input_size_,
                        top_concat_axis, bottom_concat_axis, offset_concat_axis,
                        WrapHandle((cl_mem) bottom_diff, &ctx)),
            ctx.get_queue());
#endif  // USE_GREENTEA
      }
    }
    offset_concat_axis += bottom_concat_axis;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(ConcatLayer);

}  // namespace caffe
