#include "hip/hip_runtime.h"
// TanH neuron activation function layer.
// Adapted from ReLU layer code written by Yangqing Jia

#include <vector>

#include "caffe/layers/tanh_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void TanHForward(const int n, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    out[index] = tanh(in[index]);
  }
}

template <typename Dtype>
void TanHLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  TanHForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data);
  CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_LAYER_GPU_FUNCS(TanHLayer);


}  // namespace caffe
