#include "hip/hip_runtime.h"
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void MaxForward(const int nthreads, const Dtype* bottom_data_a,
    const Dtype* bottom_data_b, const int blob_idx, Dtype* top_data,
    int* mask) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    Dtype maxval = -FLT_MAX;
    int maxidx = -1;
    if (bottom_data_a[index] > bottom_data_b[index]) {
      // only update for very first bottom_data blob (blob_idx == 0)
      if (blob_idx == 0) {
        maxval = bottom_data_a[index];
        top_data[index] = maxval;
        maxidx = blob_idx;
        mask[index] = maxidx;
      }
    } else {
      maxval = bottom_data_b[index];
      top_data[index] = maxval;
      maxidx = blob_idx + 1;
      mask[index] = maxidx;
    }
  }
}

template <typename Dtype>
__global__ void CoeffSum(const int count, const int dim,
    const int num_offset, const Dtype coeff, const Dtype* coeff_data,
    const bool backward, const Dtype* in, Dtype* out) {
  CUDA_KERNEL_LOOP(index, count) {
    const int n = num_offset + index / dim;
    const Dtype other_coeff = coeff_data ? coeff_data[n] : Dtype(1);
    const Dtype final_coeff = coeff * other_coeff;
    const Dtype result = in[index] * final_coeff;
    if (num_offset == 0 || backward) {
      out[index] = result;
    } else {
      out[index] += result;
    }
  }
}

template <typename Dtype>
void EltwiseLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  int* mask = NULL;
  const int count = top[0]->count();
  const int num = top[0]->num();
  const int dim = count / num;
  Dtype* top_data = top[0]->mutable_gpu_data();
  const Dtype* coeff_data = NULL;
  const bool kBackward = false;
  switch (op_) {
  case EltwiseParameter_EltwiseOp_PROD:
    caffe_gpu_mul(count, bottom[0]->gpu_data(), bottom[1]->gpu_data(),
        top_data);
    for (int i = 2; i < bottom.size(); ++i) {
      caffe_gpu_mul(count, top_data, bottom[i]->gpu_data(), top_data);
    }
    break;
  case EltwiseParameter_EltwiseOp_SUM:
    // TODO(shelhamer) does cuBLAS optimize to sum for coeff = 1?
    if (coeff_blob_) {
      coeff_data = bottom[bottom.size() - 1]->gpu_data();
    }
    for (int i = 0; i < bottom.size() - coeff_blob_; ++i) {
      const Dtype* bottom_data = bottom[i]->gpu_data();
      CoeffSum<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
          count, dim, i * num, coeffs_[i], coeff_data,
          kBackward, bottom_data, top_data);
      CUDA_POST_KERNEL_CHECK;
    }
    break;
  case EltwiseParameter_EltwiseOp_MAX:
    mask = max_idx_.mutable_gpu_data();
    // NOLINT_NEXT_LINE(whitespace/operators)
    MaxForward<Dtype> <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom[0]->gpu_data(), bottom[1]->gpu_data(), 0, top_data, mask);
    for (int i = 2; i < bottom.size(); ++i) {
      // NOLINT_NEXT_LINE(whitespace/operators)
      MaxForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
          count, top_data, bottom[i]->gpu_data(), i-1, top_data, mask);
    }
    break;
  default:
    LOG(FATAL) << "Unknown elementwise operation.";
  }
}

template <typename Dtype>
__global__ void MaxBackward(const int nthreads, const Dtype* top_diff,
    const int blob_idx, const int* mask, Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    Dtype gradient = 0;
    if (mask[index] == blob_idx) {
      gradient += top_diff[index];
    }
    bottom_diff[index] = gradient;
  }
}

template <typename Dtype>
void EltwiseLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const int* mask = NULL;
  const int count = top[0]->count();
  const int num = top[0]->num();
  const int dim = count / num;
  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* coeff_data = NULL;
  if (coeff_blob_) {
    coeff_data = bottom[bottom.size() - 1]->gpu_data();
  }
  const bool kBackward = true;
  for (int i = 0; i < bottom.size() - coeff_blob_; ++i) {
    if (propagate_down[i]) {
      const Dtype* bottom_data = bottom[i]->gpu_data();
      const Dtype* top_diff = top[0]->gpu_diff();
      Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
      switch (op_) {
      case EltwiseParameter_EltwiseOp_PROD:
        if (stable_prod_grad_) {
          bool initialized = false;
          for (int j = 0; j < bottom.size(); ++j) {
            if (i == j) { continue; }
            if (!initialized) {
              caffe_copy(count, bottom[j]->gpu_data(), bottom_diff);
              initialized = true;
            } else {
              caffe_gpu_mul(count, bottom[j]->gpu_data(), bottom_diff,
                            bottom_diff);
            }
          }
        } else {
          caffe_gpu_div(count, top_data, bottom_data, bottom_diff);
        }
        caffe_gpu_mul(count, bottom_diff, top_diff, bottom_diff);
        break;
      case EltwiseParameter_EltwiseOp_SUM:
        CoeffSum<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
            <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
            count, dim, i * num, coeffs_[i], coeff_data,
            kBackward, top_diff, bottom_diff);
        CUDA_POST_KERNEL_CHECK;
        break;
      case EltwiseParameter_EltwiseOp_MAX:
        mask = max_idx_.gpu_data();
        MaxBackward<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
            <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
            count, top_diff, i, mask, bottom_diff);
        break;
      default:
        LOG(FATAL) << "Unknown elementwise operation.";
      }
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(EltwiseLayer);

}  // namespace caffe
