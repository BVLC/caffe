#include "hip/hip_runtime.h"
#include <cfloat>
#include <vector>

#include "caffe/layers/eltwise_layer.hpp"
#include "caffe/util/math_functions.hpp"

#ifdef USE_GREENTEA
#include "caffe/greentea/greentea.hpp"
#endif

namespace caffe {

#ifdef USE_CUDA
template<typename Dtype>
__global__ void MaxForward(const int_tp nthreads, const Dtype* bottom_data_a,
                           const Dtype* bottom_data_b, const int_tp blob_idx,
                           Dtype* top_data, int_tp* mask) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    Dtype maxval = -FLT_MAX;
    int_tp maxidx = -1;
    if (bottom_data_a[index] > bottom_data_b[index]) {
      // only update for very first bottom_data blob (blob_idx == 0)
      if (blob_idx == 0) {
        maxval = bottom_data_a[index];
        top_data[index] = maxval;
        maxidx = blob_idx;
        mask[index] = maxidx;
      }
    } else {
      maxval = bottom_data_b[index];
      top_data[index] = maxval;
      maxidx = blob_idx + 1;
      mask[index] = maxidx;
    }
  }
}
#endif  // USE_CUDA

template<typename Dtype>
void EltwiseLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                      const vector<Blob<Dtype>*>& top) {
  int_tp* mask = NULL;
  const int_tp count = top[0]->count();
  Dtype* top_data = top[0]->mutable_gpu_data();

  if (this->device_->backend() == BACKEND_CUDA) {
#ifdef USE_CUDA
    switch (op_) {
      case EltwiseParameter_EltwiseOp_PROD:
        caffe_gpu_mul(count, bottom[0]->gpu_data(), bottom[1]->gpu_data(),
                      top_data);
        for (int_tp i = 2; i < bottom.size(); ++i) {
          caffe_gpu_mul(count, top_data, bottom[i]->gpu_data(), top_data);
        }
        break;
      case EltwiseParameter_EltwiseOp_SUM:
        caffe_gpu_set(count, Dtype(0.), top_data);
        // TODO(shelhamer) does cuBLAS optimize to sum for coeff = 1?
        for (int_tp i = 0; i < bottom.size(); ++i) {
          caffe_gpu_axpy(count, coeffs_[i], bottom[i]->gpu_data(), top_data);
        }
        break;
      case EltwiseParameter_EltwiseOp_MAX:
        mask = max_idx_.mutable_gpu_data();
        // NOLINT_NEXT_LINE(whitespace/operators)
        MaxForward<Dtype> CUDA_KERNEL(CAFFE_GET_BLOCKS(count),
                                      CAFFE_CUDA_NUM_THREADS)(
            count, bottom[0]->gpu_data(), bottom[1]->gpu_data(),
            0, top_data, mask);
        for (int_tp i = 2; i < bottom.size(); ++i) {
          // NOLINT_NEXT_LINE(whitespace/operators)
          MaxForward<Dtype> CUDA_KERNEL(CAFFE_GET_BLOCKS(count),
                                        CAFFE_CUDA_NUM_THREADS)(
              count, top_data, bottom[i]->gpu_data(), i-1, top_data, mask);
        }
        break;
      default: {
        LOG(FATAL)<< "Unknown elementwise operation.";
      }
    }
#endif  // USE_CUDA
  } else {
#ifdef USE_GREENTEA
    viennacl::ocl::context &ctx = viennacl::ocl::get_context(
        this->device_->id());
    viennacl::ocl::program &program = this->device_->program();

    switch (op_) {
      case EltwiseParameter_EltwiseOp_PROD:
        caffe_gpu_mul(count, bottom[0]->gpu_data(), bottom[1]->gpu_data(),
                      top_data);
        for (int_tp i = 2; i < bottom.size(); ++i) {
          caffe_gpu_mul(count, top_data, bottom[i]->gpu_data(), top_data);
        }
        break;
      case EltwiseParameter_EltwiseOp_SUM:
        caffe_gpu_set(count, Dtype(0.), top_data);
        // TODO(shelhamer) does cuBLAS optimize to sum for coeff = 1?
        for (int_tp i = 0; i < bottom.size(); ++i) {
          caffe_gpu_axpy(count, coeffs_[i], bottom[i]->gpu_data(), top_data);
        }
        break;
      case EltwiseParameter_EltwiseOp_MAX: {
        mask = max_idx_.mutable_gpu_data();

        viennacl::ocl::kernel &oclk_max_forward = program.get_kernel(
            CL_KERNEL_SELECT("eltwise_max_forward"));

        ClState& clState = Caffe::cl_state();
        ClMemOff<Dtype> buf_bottom0 =
            clState.get_buffer_mem(bottom[0]->gpu_data());
        ClMemOff<Dtype> buf_bottom1 =
            clState.get_buffer_mem(bottom[1]->gpu_data());
        ClMemOff<Dtype> buf_top =
            clState.get_buffer_mem(top_data);
        ClMemOff<int_tp> buf_mask =
            clState.get_buffer_mem(mask);

        viennacl::ocl::enqueue(
            oclk_max_forward(count,
                WrapHandle(buf_bottom0.memobj, &ctx),
                WrapHandle(buf_bottom1.memobj, &ctx), (int_tp)0,
                WrapHandle(buf_top.memobj, &ctx),
                WrapHandle(buf_mask.memobj, &ctx)),
            ctx.get_queue());

        for (int_tp i = 2; i < bottom.size(); ++i) {
          ClMemOff<Dtype> buf_bottomi =
              clState.get_buffer_mem(bottom[i]->gpu_data());
          viennacl::ocl::enqueue(
              oclk_max_forward(count, WrapHandle(buf_top.memobj, &ctx),
                  WrapHandle(buf_bottomi.memobj, &ctx), i-1,
                  WrapHandle(buf_top.memobj, &ctx),
                  WrapHandle(buf_mask.memobj, &ctx)),
              ctx.get_queue());
        }
      }
      break;
      default: {
        LOG(FATAL)<< "Unknown elementwise operation.";
      }
    }
#endif  // USE_GREENTEA
  }
}

#ifdef USE_CUDA
template<typename Dtype>
__global__ void MaxBackward(const int_tp nthreads, const Dtype* top_diff,
                            const int_tp blob_idx, const int_tp* mask,
                            Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    Dtype gradient = 0;
    if (mask[index] == blob_idx) {
      gradient += top_diff[index];
    }
    bottom_diff[index] = gradient;
  }
}
#endif  // USE_CUDA

template<typename Dtype>
void EltwiseLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
                                       const vector<bool>& propagate_down,
                                       const vector<Blob<Dtype>*>& bottom) {
  const int_tp* mask = NULL;
  const int_tp count = top[0]->count();
  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* top_diff = top[0]->gpu_diff();

  if (this->device_->backend() == BACKEND_CUDA) {
#ifdef USE_CUDA
    for (int_tp i = 0; i < bottom.size(); ++i) {
      if (propagate_down[i]) {
        const Dtype* bottom_data = bottom[i]->gpu_data();
        Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
        switch (op_) {
          case EltwiseParameter_EltwiseOp_PROD:
            if (stable_prod_grad_) {
              bool initialized = false;
              for (int_tp j = 0; j < bottom.size(); ++j) {
                if (i == j) {
                  continue;
                }
                if (!initialized) {
                  caffe_copy(count, bottom[j]->gpu_data(), bottom_diff);
                  initialized = true;
                } else {
                  caffe_gpu_mul(count, bottom[j]->gpu_data(), bottom_diff,
                                bottom_diff);
                }
              }
            } else {
              caffe_gpu_div(count, top_data, bottom_data, bottom_diff);
            }
            caffe_gpu_mul(count, bottom_diff, top_diff, bottom_diff);
            break;
          case EltwiseParameter_EltwiseOp_SUM:
            if (coeffs_[i] == Dtype(1.)) {
              caffe_copy(count, top_diff, bottom_diff);
            } else {
              caffe_gpu_scale(count, coeffs_[i], top_diff, bottom_diff);
            }
            break;
          case EltwiseParameter_EltwiseOp_MAX:
            mask = max_idx_.gpu_data();
            MaxBackward<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
            CUDA_KERNEL(CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS)(
                count, top_diff, i, mask, bottom_diff);
            break;
          default: {
            LOG(FATAL)<< "Unknown elementwise operation.";
          }
        }
      }
    }
#endif  // USE_CUDA
  } else {
#ifdef USE_GREENTEA
    viennacl::ocl::context &ctx = viennacl::ocl::get_context(
        this->device_->id());
    viennacl::ocl::program &program = this->device_->program();

    for (int_tp i = 0; i < bottom.size(); ++i) {
      if (propagate_down[i]) {
        const Dtype* bottom_data = bottom[i]->gpu_data();
        Dtype* bottom_diff = bottom[i]->mutable_gpu_diff();
        switch (op_) {
          case EltwiseParameter_EltwiseOp_PROD:
            if (stable_prod_grad_) {
              bool initialized = false;
              for (int_tp j = 0; j < bottom.size(); ++j) {
                if (i == j) {
                  continue;
                }
                if (!initialized) {
                  caffe_copy(count, bottom[j]->gpu_data(), bottom_diff);
                  initialized = true;
                } else {
                  caffe_gpu_mul(count, bottom[j]->gpu_data(), bottom_diff,
                                bottom_diff);
                }
              }
            } else {
              caffe_gpu_div(count, top_data, bottom_data, bottom_diff);
            }
            caffe_gpu_mul(count, bottom_diff, top_diff, bottom_diff);
            break;
          case EltwiseParameter_EltwiseOp_SUM:
            if (coeffs_[i] == Dtype(1.)) {
              caffe_copy(count, top_diff, bottom_diff);
            } else {
              caffe_gpu_scale(count, coeffs_[i], top_diff, bottom_diff);
            }
            break;
          case EltwiseParameter_EltwiseOp_MAX: {
            mask = max_idx_.gpu_data();

            ClState& clState = Caffe::cl_state();
            ClMemOff<Dtype> buf_bottom = clState.get_buffer_mem(bottom_diff);
            ClMemOff<Dtype> buf_top = clState.get_buffer_mem(top_diff);
            ClMemOff<int_tp> buf_mask = clState.get_buffer_mem(mask);

            viennacl::ocl::kernel &oclk_max_backward = program.get_kernel(
                CL_KERNEL_SELECT("eltwise_max_backward"));

            viennacl::ocl::enqueue(
                oclk_max_backward(count, WrapHandle(buf_top.memobj, &ctx), i,
                    WrapHandle(buf_mask.memobj, &ctx),
                    WrapHandle(buf_bottom.memobj, &ctx)),
                ctx.get_queue());
            }
            break;
          default: {
            LOG(FATAL)<< "Unknown elementwise operation.";
          }
        }
      }
    }
#endif
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(EltwiseLayer);

}  // namespace caffe
