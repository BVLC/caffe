// Copyright 2014 BVLC and contributors.

#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
Dtype EltwiseLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, vector<Blob<Dtype>*>* top) {
  const int count = (*top)[0]->count();
  Dtype* top_data = (*top)[0]->mutable_gpu_data();
  switch (op_) {
  case EltwiseParameter_EltwiseOp_PROD:
    caffe_gpu_mul(count, bottom[0]->gpu_data(),
        bottom[1]->gpu_data(), top_data);
    for (int i = 2; i < bottom.size(); ++i) {
      caffe_gpu_mul(count, top_data, bottom[i]->gpu_data(), top_data);
    }
    break;
  case EltwiseParameter_EltwiseOp_SUM:
    caffe_gpu_set(count, Dtype(0.), top_data);
    // TODO(shelhamer) does cuBLAS optimize to sum for coeff = 1?
    for (int i = 0; i < bottom.size(); ++i) {
      caffe_gpu_axpy(count, coeffs_[i], bottom[i]->gpu_data(), top_data);
    }
    break;
  default:
    LOG(FATAL) << "Unknown elementwise operation.";
  }
  return Dtype(0.);
}

template <typename Dtype>
void EltwiseLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const bool propagate_down, vector<Blob<Dtype>*>* bottom) {
  if (propagate_down) {
    const int count = top[0]->count();
    const Dtype* top_data = top[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    for (int i = 0; i < bottom->size(); ++i) {
      const Dtype* bottom_data = (*bottom)[i]->gpu_data();
      Dtype* bottom_diff = (*bottom)[i]->mutable_gpu_diff();
      switch (op_) {
      case EltwiseParameter_EltwiseOp_PROD:
        caffe_gpu_div(count, top_data, bottom_data, bottom_diff);
        caffe_gpu_mul(count, bottom_diff, top_diff, bottom_diff);
        break;
      case EltwiseParameter_EltwiseOp_SUM:
        if (coeffs_[i] == Dtype(1.)) {
          caffe_gpu_copy(count, top_diff, bottom_diff);
        } else {
          caffe_gpu_scale(count, coeffs_[i], top_diff, bottom_diff);
        }
        break;
      default:
        LOG(FATAL) << "Unknown elementwise operation.";
      }
    }
  }
}

INSTANTIATE_CLASS(EltwiseLayer);


}  // namespace caffe
