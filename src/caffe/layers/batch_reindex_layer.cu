#include "hip/hip_runtime.h"
#include <algorithm>
#include <utility>
#include <vector>

#include "caffe/common_layers.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template<typename Dtype>
__global__ void BRForward(const int count, const int inner_dim, const Dtype* in,
                          const Dtype* permut, Dtype* out) {
  CUDA_KERNEL_LOOP(index, count) {
    int n = index / (inner_dim);
    int in_n = static_cast<int>(permut[n]);
    out[index] = in[in_n * (inner_dim) + index % (inner_dim)];
  }
}

template<typename Dtype>
void BatchReindexLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
                                           const vector<Blob<Dtype>*>& top) {
  check_batch_reindex(bottom[0]->shape(0), bottom[1]->count(),
                      bottom[1]->cpu_data());
  if (top[0]->count() == 0) {
    return;
  }
  int threads = top[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  BRForward<Dtype> <<<CAFFE_GET_BLOCKS(threads), CAFFE_CUDA_NUM_THREADS>>>(
      top[0]->count(), bottom[0]->count() / bottom[0]->shape(0),
      bottom[0]->gpu_data(), bottom[1]->gpu_data(), top[0]->mutable_gpu_data());
  CUDA_POST_KERNEL_CHECK;
}

template<typename Dtype>
__global__ void BRBackward(const int count, const int inner_dim,
                           const Dtype* in, const Dtype* top_indexes,
                           const Dtype* begins, const Dtype* counts,
                           Dtype* out) {
  CUDA_KERNEL_LOOP(index, count) {
    int n = index / (inner_dim);
    out[index] = 0;
    int lower = static_cast<int>(begins[n]);
    int upper = lower + static_cast<int>(counts[n]);
    for (int i = lower; i < upper; ++i) {
      int in_n = static_cast<int>(top_indexes[i]);
      out[index] += in[in_n * (inner_dim) + index % (inner_dim)];
    }
  }
}

template<typename Dtype>
void BatchReindexLayer<Dtype>::Backward_gpu(
    const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  CHECK(!propagate_down[1]) << "Cannot backprop to index.";
  if (!propagate_down[0]) {
    return;
  }

  vector<std::pair<int, int> > mapping;
  const Dtype* perm = bottom[1]->cpu_data();
  for (int i = 0; i < bottom[1]->count(); ++i) {
    mapping.push_back(pair<int, int>(static_cast<int>(perm[i]), i));
  }
  std::sort(mapping.begin(), mapping.end(), pair_sort_first());

  // Each element of the bottom diff is potentially the sum of many top diffs.
  // However, we'd like each CUDA thread to handle exactly one output.  Hence,
  // we first pre-compute a list of lists of indices that need to be summed for
  // each output. `top_indexes` holds the data of this list of lists.  The
  // k'th element of `begins` points to the location in `top_indexes` where the
  // list for the k'th example begin, and the k'th element of `counts` is the
  // length of that list.
  vector<int> shape;
  shape.push_back(bottom[1]->count());
  Blob<Dtype> top_indexes(shape);
  shape[0] = bottom[0]->shape(0);
  Blob<Dtype> counts(shape);
  Blob<Dtype> begins(shape);
  Dtype* t_i_data = top_indexes.mutable_cpu_data();
  Dtype* c_data = counts.mutable_cpu_data();
  Dtype* b_data = begins.mutable_cpu_data();
  caffe_set(begins.count(), Dtype(-1), b_data);
  caffe_set(counts.count(), Dtype(0), c_data);
  for (int i = 0; i < mapping.size(); ++i) {
    t_i_data[i] = mapping[i].second;
    if (b_data[mapping[i].first] == -1) {
      b_data[mapping[i].first] = i;
    }
    c_data[mapping[i].first] += 1;
  }

  int threads = bottom[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  BRBackward<Dtype> <<<CAFFE_GET_BLOCKS(threads), CAFFE_CUDA_NUM_THREADS>>>(
      bottom[0]->count(), bottom[0]->count() / bottom[0]->shape(0),
      top[0]->gpu_diff(), top_indexes.gpu_data(), begins.gpu_data(),
      counts.gpu_data(), bottom[0]->mutable_gpu_diff());
  CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_LAYER_GPU_FUNCS(BatchReindexLayer);

}  // namespace caffe
