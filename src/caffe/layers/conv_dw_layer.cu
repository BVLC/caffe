#include "hip/hip_runtime.h"
#include <vector>
#include "caffe/layers/conv_dw_layer.hpp"
#include "caffe/util/gpu_util.cuh"

namespace caffe {

template <typename Dtype>
__global__ void ConvolutionDepthwiseWeightForward(const int nthreads,
    const Dtype* const bottom_data, const Dtype* const weight_data, const int num, const int channels,
    const int top_height, const int top_width, const int bottom_height, const int bottom_width,
    const int kernel_h, const int kernel_w, const int stride_h, const int stride_w,
    const int pad_h, const int pad_w, const int dilation_h, const int dilation_w,
    Dtype* const top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / channels / top_height / top_width;
    const int c = (index / top_height / top_width) % channels;
    const int h = (index / top_width) % top_height;
    const int w = index % top_width;
    const Dtype* weight = weight_data + c * kernel_h * kernel_w;
    Dtype value = 0;
    for (int kh = 0; kh < kernel_h; ++kh)
    {
      for (int kw = 0; kw < kernel_w; ++kw)
      {
        const int h_in = -pad_h + h * stride_h + kh * dilation_h;
        const int w_in = -pad_w + w * stride_w + kw * dilation_w;
        if ((h_in >= 0) && (h_in < bottom_height) && (w_in >= 0) && (w_in < bottom_width))
        {
          const int offset = ((n * channels + c) * bottom_height + h_in) * bottom_width + w_in;
          value += (*weight) * bottom_data[offset];
        }
        ++weight;
      }
    }
    top_data[index] = value;
  }
}

template <typename Dtype>
__global__ void ConvolutionDepthwiseBiasForward(const int nthreads,
    const Dtype* const bias_data, const int num, const int channels,
    const int top_height, const int top_width, Dtype* const top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int c = (index / top_height / top_width) % channels;
    top_data[index] += bias_data[c];
  }
}

template <typename Dtype>
void ConvolutionDepthwiseLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const Dtype* weight_data = this->blobs_[0]->gpu_data();
  const int count = top[0]->count();
  const int num = top[0]->num();
  const int channels = top[0]->channels();
  const int top_height = top[0]->height();
  const int top_width = top[0]->width();
  const int bottom_height = bottom[0]->height();
  const int bottom_width = bottom[0]->width();
  ConvolutionDepthwiseWeightForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, weight_data, num, channels,
      top_height, top_width, bottom_height, bottom_width,
      kernel_h_, kernel_w_, stride_h_, stride_w_,
      pad_h_, pad_w_, dilation_h_, dilation_w_, top_data);
  if (this->layer_param_.convolution_param().bias_term())
  {
    const Dtype* bias_data = this->blobs_[1]->gpu_data();
    ConvolutionDepthwiseBiasForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bias_data, num, channels,
        top_height, top_width, top_data);
  }
}

template <typename Dtype>
__global__ void ConvolutionDepthwiseWeightBackward(const int nthreads,
    const Dtype* const top_diff, const Dtype* const bottom_data, const int num, const int channels,
    const int top_height, const int top_width, const int bottom_height, const int bottom_width,
    const int kernel_h, const int kernel_w, const int stride_h, const int stride_w,
    const int pad_h, const int pad_w, const int dilation_h, const int dilation_w,
    Dtype* const buffer_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int h = (index / top_width) % top_height;
    const int w = index % top_width;
    const int kh = (index / kernel_w / num / top_height / top_width) % kernel_h;
    const int kw = (index / num / top_height / top_width) % kernel_w;
    const int h_in = -pad_h + h * stride_h + kh * dilation_h;
    const int w_in = -pad_w + w * stride_w + kw * dilation_w;
    if ((h_in >= 0) && (h_in < bottom_height) && (w_in >= 0) && (w_in < bottom_width))
    {
      const int c = index / kernel_h / kernel_w / num / top_height / top_width;
      const int n = (index / top_height / top_width) % num;
      const int top_offset = ((n * channels + c) * top_height + h) * top_width + w;
      const int bottom_offset = ((n * channels + c) * bottom_height + h_in) * bottom_width + w_in;
      buffer_data[index] = top_diff[top_offset] * bottom_data[bottom_offset];
    }
    else
    {
      buffer_data[index] = 0;
    }
  }
}

template <typename Dtype>
__global__ void ConvolutionDepthwiseBottomBackward(const int nthreads,
    const Dtype* const top_diff, const Dtype* const weight_data, const int num, const int channels,
    const int top_height, const int top_width, const int bottom_height, const int bottom_width,
    const int kernel_h, const int kernel_w, const int stride_h, const int stride_w,
    const int pad_h, const int pad_w, const int dilation_h, const int dilation_w,
    Dtype* const bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / channels / bottom_height / bottom_width;
    const int c = (index / bottom_height / bottom_width) % channels;
    const int h = (index / bottom_width) % bottom_height;
    const int w = index % bottom_width;
    const Dtype* weight = weight_data + c * kernel_h * kernel_w;
    Dtype value = 0;
    for (int kh = 0; kh < kernel_h; ++kh)
    {
      for (int kw = 0; kw < kernel_w; ++kw)
      {
        const int h_out_s = h + pad_h - kh * dilation_h;
        const int w_out_s = w + pad_w - kw * dilation_w;
        if (((h_out_s % stride_h) == 0) && ((w_out_s % stride_w) == 0))
        {
          const int h_out = h_out_s / stride_h;
          const int w_out = w_out_s / stride_w;
          if ((h_out >= 0) && (h_out < top_height) && (w_out >= 0) && (w_out < top_width))
          {
            const int offset = ((n * channels + c) * top_height + h_out) * top_width + w_out;
            value += (*weight) * top_diff[offset];
          }
        }
        ++weight;
      }
    }
    bottom_diff[index] += value;
  }
}

template <typename Dtype>
__global__ void ConvolutionDepthwiseBiasBackward(const int nthreads,
    const Dtype* const top_diff, const int num, const int channels,
    const int top_height, const int top_width, Dtype* const buffer_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int c = index / num / top_height / top_width;
    const int n = (index / top_height / top_width) % num;
    const int h = (index / top_width) % top_height;
    const int w = index % top_width;
    const int offset = ((n * channels + c) * top_height + h) * top_width + w;
    buffer_data[index] = top_diff[offset];
  }
}

template <typename Dtype>
void ConvolutionDepthwiseLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  const int bottom_count = bottom[0]->count();
  const int num = top[0]->num();
  const int channels = top[0]->channels();
  const int top_height = top[0]->height();
  const int top_width = top[0]->width();
  const int bottom_height = bottom[0]->height();
  const int bottom_width = bottom[0]->width();
  const int length = num * top_height * top_width;
  caffe_gpu_set(bottom_count, Dtype(0), bottom[0]->mutable_gpu_diff());
  if (this->layer_param_.convolution_param().bias_term() && this->param_propagate_down_[1])
  {
    const int bias_buffer_count = bias_buffer_.count();
    Dtype* bias_buffer_mutable_data = bias_buffer_.mutable_gpu_data();
    ConvolutionDepthwiseBiasBackward<Dtype><<<CAFFE_GET_BLOCKS(bias_buffer_count), CAFFE_CUDA_NUM_THREADS>>>(
        bias_buffer_count, top_diff, num, channels,
        top_height, top_width, bias_buffer_mutable_data);
    const int bias_count = this->blobs_[1]->count();
    const Dtype* bias_buffer_data = bias_buffer_.gpu_data();
    Dtype* bias_diff = this->blobs_[1]->mutable_gpu_diff();
    const Dtype* bias_multiplier_data = bias_multiplier_.gpu_data();
    caffe_gpu_gemv(CblasNoTrans, bias_count, length, Dtype(1), bias_buffer_data, bias_multiplier_data, Dtype(1), bias_diff);
  }
  if (this->param_propagate_down_[0])
  {
    const int weight_buffer_count = weight_buffer_.count();
    const Dtype* bottom_data = bottom[0]->gpu_data();
    Dtype* weight_buffer_mutable_data = weight_buffer_.mutable_gpu_data();
    ConvolutionDepthwiseWeightBackward<Dtype><<<CAFFE_GET_BLOCKS(weight_buffer_count), CAFFE_CUDA_NUM_THREADS>>>(
        weight_buffer_count, top_diff, bottom_data, num, channels,
        top_height, top_width, bottom_height, bottom_width,
        kernel_h_, kernel_w_, stride_h_, stride_w_,
        pad_h_, pad_w_, dilation_h_, dilation_w_, weight_buffer_mutable_data);
    const int weight_count = this->blobs_[0]->count();
    const Dtype* weight_buffer_data = weight_buffer_.gpu_data();
    Dtype* weight_diff = this->blobs_[0]->mutable_gpu_diff();
    const Dtype* weight_multiplier_data = weight_multiplier_.gpu_data();
    caffe_gpu_gemv(CblasNoTrans, weight_count, length, Dtype(1), weight_buffer_data, weight_multiplier_data, Dtype(1), weight_diff);
  }
  if (propagate_down[0])
  {
    const Dtype* weight_data = this->blobs_[0]->gpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    ConvolutionDepthwiseBottomBackward<Dtype><<<CAFFE_GET_BLOCKS(bottom_count), CAFFE_CUDA_NUM_THREADS>>>(
        bottom_count, top_diff, weight_data, num, channels,
        top_height, top_width, bottom_height, bottom_width,
        kernel_h_, kernel_w_, stride_h_, stride_w_,
        pad_h_, pad_w_, dilation_h_, dilation_w_, bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(ConvolutionDepthwiseLayer);

}  // namespace caffe
