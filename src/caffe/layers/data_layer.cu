#include <string>
#include <vector>

#include "leveldb/db.h"
#include "pthread.h"
#include "stdint.h"

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void DataLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      vector<Blob<Dtype>*>* top) {
  // First, join the thread
  JoinPrefetchThread();
  // Copy the data
  caffe_copy(prefetch_data_.count(), prefetch_data_.cpu_data(),
      (*top)[0]->mutable_gpu_data());
  if (output_labels_) {
    caffe_copy(prefetch_label_.count(), prefetch_label_.cpu_data(),
        (*top)[1]->mutable_gpu_data());
  }
  // Start a new prefetch thread
  CreatePrefetchThread();
}

INSTANTIATE_CLASS(DataLayer);

}  // namespace caffe
