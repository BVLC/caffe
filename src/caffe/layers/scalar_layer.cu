#include "hip/hip_runtime.h"
#include <cfloat>
#include <vector>

#include "caffe/layers/scalar_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void ScalarForward(const int n, const Dtype* in,
    const Dtype* scalar, const int scalar_dim, const int inner_dim,
    Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    const int scalar_index = (index / inner_dim) % scalar_dim;
    out[index] = in[index] * scalar[scalar_index];
  }
}

template <typename Dtype>
void ScalarLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const int count = top[0]->count();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  if (bottom[0] == top[0]) {
    // in-place computation; need to store bottom data before overwriting it.
    // Note that this is only necessary for Backward; we could skip this if not
    // doing Backward, but Caffe currently provides no way of knowing whether
    // we'll need to do Backward at the time of the Forward call.
    caffe_copy(bottom[0]->count(), bottom[0]->gpu_data(),
               temp_.mutable_gpu_data());
  }
  const Dtype* scalar_data =
      ((bottom.size() > 1) ? bottom[1] : this->blobs_[0].get())->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  ScalarForward<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, scalar_data, scalar_dim_, inner_dim_, top_data);
}

template <typename Dtype>
void ScalarLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const bool scalar_param = (bottom.size() == 1);
  Blob<Dtype>* scalar = scalar_param ? this->blobs_[0].get() : bottom[1];
  if ((!scalar_param && propagate_down[1]) ||
      (scalar_param && this->param_propagate_down_[0])) {
    const Dtype* top_diff = top[0]->gpu_diff();
    const bool in_place = (bottom[0] == top[0]);
    const Dtype* bottom_data = (in_place ? &temp_ : bottom[0])->gpu_data();
    // Hack: store big eltwise product in bottom[0] diff, except in the special
    // case where this layer itself does the eltwise product, in which case we
    // can store it directly in the scalar diff, and we're done.
    // If we're computing in-place (and not doing eltwise computation), this
    // hack doesn't work and we store the product in temp_.
    const bool is_eltwise = (bottom[0]->count() == scalar->count());
    Dtype* product = (is_eltwise ? scalar->mutable_gpu_diff() :
        (in_place ? temp_.mutable_gpu_data() : bottom[0]->mutable_gpu_diff()));
    caffe_gpu_mul(top[0]->count(), top_diff, bottom_data, product);
    if (!is_eltwise) {
      Dtype* sum_result = NULL;
      if (inner_dim_ == 1) {
        sum_result = product;
      } else if (sum_result_.count() == 1) {
        const Dtype* sum_mult = sum_multiplier_.gpu_data();
        Dtype* scalar_diff = scalar->mutable_cpu_diff();
        if (scalar_param) {
          Dtype result;
          caffe_gpu_dot(inner_dim_, product, sum_mult, &result);
          *scalar_diff += result;
        } else {
          caffe_gpu_dot(inner_dim_, product, sum_mult, scalar_diff);
        }
      } else {
        const Dtype* sum_mult = sum_multiplier_.gpu_data();
        sum_result = (outer_dim_ == 1) ?
            scalar->mutable_gpu_diff() : sum_result_.mutable_gpu_data();
        caffe_gpu_gemv(CblasNoTrans, sum_result_.count(), inner_dim_,
                       Dtype(1), product, sum_mult, Dtype(0), sum_result);
      }
      if (outer_dim_ != 1) {
        const Dtype* sum_mult = sum_multiplier_.gpu_data();
        if (scalar_dim_ == 1) {
          Dtype* scalar_diff = scalar->mutable_cpu_diff();
          if (scalar_param) {
            Dtype result;
            caffe_gpu_dot(outer_dim_, sum_mult, sum_result, &result);
            *scalar_diff += result;
          } else {
            caffe_gpu_dot(outer_dim_, sum_mult, sum_result, scalar_diff);
          }
        } else {
          Dtype* scalar_diff = scalar->mutable_gpu_diff();
          caffe_gpu_gemv(CblasTrans, outer_dim_, scalar_dim_,
                         Dtype(1), sum_result, sum_mult, Dtype(scalar_param),
                         scalar_diff);
        }
      }
    }
  }
  if (propagate_down[0]) {
    const int count = top[0]->count();
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* scalar_data = scalar->gpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    ScalarForward<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, scalar_data, scalar_dim_, inner_dim_, bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(ScalarLayer);

}  // namespace caffe
