#include "hip/hip_runtime.h"
#include <cmath>
#include <vector>

#include "caffe/layers/swish_layer.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SwishForward(const int n, const Dtype* in, Dtype* sigmoid_x,
    Dtype* out) {
  CUDA_KERNEL_LOOP(index, n) {
    sigmoid_x[index] = 0.5 * tanh(0.5 * in[index]) + 0.5;
    out[index] = in[index] * sigmoid_x[index];
  }
}

template <typename Dtype>
void SwishLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  Dtype* sigmoid_x_data = this->sigmoid_x_.mutable_gpu_data();
  const int count = bottom[0]->count();
  // NOLINT_NEXT_LINE(whitespace/operators)
  SwishForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, sigmoid_x_data, top_data);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void SwishBackward(const int n, const Dtype* in_diff,
    const Dtype* out_data, const Dtype* sigmoid_x_data, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    const Dtype swish_x = out_data[index];
    out_diff[index] = in_diff[index] * (swish_x + sigmoid_x_data[index]
        * (1 - swish_x));
  }
}

template <typename Dtype>
void SwishLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_data = top[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* sigmoid_x_data = this->sigmoid_x_.gpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    // NOLINT_NEXT_LINE(whitespace/operators)
    SwishBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, top_data, sigmoid_x_data, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SwishLayer);

}  // namespace caffe
