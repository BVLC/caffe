#include "hip/hip_runtime.h"
#include <cmath>
#include <vector>

#include "caffe/layers/swish_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
void SwishLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* sigmoid_input_data = sigmoid_input_->mutable_gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  Dtype beta = this->layer_param_.swish_param().beta();
  caffe_copy(count, bottom_data, sigmoid_input_data);
  caffe_gpu_scal(count, beta, sigmoid_input_data);
  sigmoid_layer_->Forward(sigmoid_bottom_vec_, sigmoid_top_vec_);
  caffe_gpu_mul(count, bottom_data, sigmoid_output_->gpu_data(), top_data);
}

template <typename Dtype>
__global__ void SwishBackward(const int n, const Dtype* in_diff,
    const Dtype* out_data, const Dtype* sigmoid_output_data, Dtype* out_diff,
    const Dtype beta) {
  CUDA_KERNEL_LOOP(index, n) {
    const Dtype swish_x = out_data[index];
    out_diff[index] = in_diff[index] * (beta * swish_x
        + sigmoid_output_data[index] * (1 - beta * swish_x));
  }
}

template <typename Dtype>
void SwishLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[0]) {
    const Dtype* top_data = top[0]->gpu_data();
    const Dtype* top_diff = top[0]->gpu_diff();
    const Dtype* sigmoid_output_data = sigmoid_output_->gpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const int count = bottom[0]->count();
    Dtype beta = this->layer_param_.swish_param().beta();
    // NOLINT_NEXT_LINE(whitespace/operators)
    SwishBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, top_data, sigmoid_output_data, bottom_diff, beta);
    CUDA_POST_KERNEL_CHECK;
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SwishLayer);

}  // namespace caffe
