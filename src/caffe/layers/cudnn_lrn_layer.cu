
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <vector>

#include "caffe/layers/cudnn_lrn_layer.hpp"

namespace caffe {

template <typename Dtype>
void CuDNNLRNLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();

  CUDNN_CHECK(cudnnLRNCrossChannelForward(
        handle_, norm_desc_, CUDNN_LRN_CROSS_CHANNEL_DIM1,
        cudnn::dataType<Dtype>::one,
        bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        top_desc_, top_data) );
}

template <typename Dtype>
void CuDNNLRNLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();

  CUDNN_CHECK(cudnnLRNCrossChannelBackward(
        handle_, norm_desc_, CUDNN_LRN_CROSS_CHANNEL_DIM1,
        cudnn::dataType<Dtype>::one,
        top_desc_, top_data,
        top_desc_, top_diff,
        bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        bottom_desc_, bottom_diff) );
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNLRNLayer);

};  // namespace caffe

#endif
