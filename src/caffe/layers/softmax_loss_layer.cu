#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/softmax_loss_layer.hpp"
#include "caffe/util/math_functions.hpp"

#ifdef USE_GREENTEA
#include "caffe/greentea/greentea_im2col.hpp"
#include "caffe/greentea/greentea_math_functions.hpp"
#endif

namespace caffe {

#ifdef USE_CUDA
template<typename Dtype>
__global__ void SoftmaxLossForwardGPU(const int_tp nthreads,
                                      const Dtype* prob_data,
                                      const Dtype* label, Dtype* loss,
                                      const int_tp num, const int_tp dim,
                                      const int_tp spatial_dim,
                                      const bool has_ignore_label_,
                                      const int_tp ignore_label_,
                                      Dtype* counts) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int_tp n = index / spatial_dim;
    const int_tp s = index % spatial_dim;
    const int_tp label_value = static_cast<int_tp>(label[n * spatial_dim + s]);
    if (has_ignore_label_ && label_value == ignore_label_) {
      loss[index] = 0;
      counts[index] = 0;
    } else {
      loss[index] = -log(
          max(prob_data[n * dim + label_value * spatial_dim + s],
              Dtype(FLT_MIN)));
      counts[index] = 1;
    }
  }
}
#endif  // USE_CUDA

template<typename Dtype>
void SoftmaxWithLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  softmax_layer_->Forward(softmax_bottom_vec_, softmax_top_vec_);
  if (this->device_->backend() == BACKEND_CUDA) {
#ifdef USE_CUDA
    const Dtype* prob_data = prob_.gpu_data();
    const Dtype* label = bottom[1]->gpu_data();
    const int_tp dim = prob_.count() / outer_num_;
    const int_tp nthreads = outer_num_ * inner_num_;
    // Since this memory is not used for anything until it is overwritten
    // on the backward pass, we use it here to avoid having to allocate new GPU
    // memory to accumulate intermediate results in the kernel.
    Dtype* loss_data = bottom[0]->mutable_gpu_diff();
    // Similarly, this memory is never used elsewhere, and thus we can use it
    // to avoid having to allocate additional GPU memory.
    Dtype* counts = prob_.mutable_gpu_diff();
    // NOLINT_NEXT_LINE(whitespace/operators)
    SoftmaxLossForwardGPU<Dtype> CUDA_KERNEL(CAFFE_GET_BLOCKS(nthreads),
        CAFFE_CUDA_NUM_THREADS)(nthreads, prob_data,
            label, loss_data, outer_num_,
            dim, inner_num_, has_ignore_label_, ignore_label_, counts);
    Dtype loss;
    caffe_gpu_asum(nthreads, loss_data, &loss);
    Dtype valid_count = -1;
    // Only launch another CUDA kernel if we actually need the count of valid
    // outputs.
    if (normalization_ == LossParameter_NormalizationMode_VALID
        && has_ignore_label_) {
      caffe_gpu_asum(nthreads, counts, &valid_count);
    }
    top[0]->mutable_cpu_data()[0] = loss
        / get_normalizer(normalization_, valid_count);
    if (top.size() >= 2) {
      top[1]->ShareData(prob_);
    }

#endif  // USE_CUDA
  } else {
#ifdef USE_GREENTEA
    viennacl::ocl::context &ctx = viennacl::ocl::get_context(
        this->device_->id());
    viennacl::ocl::program &program = this->device_->program();

    cl_mem prob_data = (cl_mem) (prob_.gpu_data());
    cl_mem label = (cl_mem) (bottom[1]->gpu_data());
    const int_tp dim = prob_.count() / outer_num_;
    const int_tp nthreads = outer_num_ * inner_num_;
    cl_mem loss_data = (cl_mem) (bottom[0]->mutable_gpu_diff());
    cl_mem counts = (cl_mem) (prob_.mutable_gpu_diff());

    viennacl::ocl::kernel &oclk_softmax_loss_forward = program.get_kernel(
        CL_KERNEL_SELECT("softmax_loss_forward"));
    viennacl::ocl::enqueue(
        oclk_softmax_loss_forward(nthreads, WrapHandle(prob_data, &ctx),
                                  WrapHandle(label, &ctx),
                                  WrapHandle(loss_data, &ctx), outer_num_, dim,
                                  inner_num_, has_ignore_label_ ? 1 : 0,
                                  ignore_label_, WrapHandle(counts, &ctx)),
        ctx.get_queue());

    Dtype loss;
    greentea_gpu_asum<Dtype>(this->device_->id(), nthreads, loss_data, 0,
                             &loss);
    Dtype valid_count = -1;
    // Only launch another CUDA kernel if we actually need the count of valid
    // outputs.
    if (normalization_ == LossParameter_NormalizationMode_VALID
        && has_ignore_label_) {
      greentea_gpu_asum<Dtype>(this->device_->id(), nthreads, counts, 0,
                               &valid_count);
    }
    top[0]->mutable_cpu_data()[0] = loss
        / get_normalizer(normalization_, valid_count);
    if (top.size() >= 2) {
      top[1]->ShareData(prob_);
    }
#endif  // USE_GREENTEA
  }
}

#ifdef USE_CUDA
template<typename Dtype>
__global__ void SoftmaxLossBackwardGPU(const int_tp nthreads, const Dtype* top,
                                       const Dtype* label, Dtype* bottom_diff,
                                       const int_tp num, const int_tp dim,
                                       const int_tp spatial_dim,
                                       const bool has_ignore_label_,
                                       const int_tp ignore_label_,
                                       Dtype* counts) {
  const int_tp channels = dim / spatial_dim;

  CUDA_KERNEL_LOOP(index, nthreads) {
    const int_tp n = index / spatial_dim;
    const int_tp s = index % spatial_dim;
    const int_tp label_value = static_cast<int_tp>(label[n * spatial_dim + s]);

    if (has_ignore_label_ && label_value == ignore_label_) {
      for (int_tp c = 0; c < channels; ++c) {
        bottom_diff[n * dim + c * spatial_dim + s] = 0;
      }
      counts[index] = 0;
    } else {
      bottom_diff[n * dim + label_value * spatial_dim + s] -= 1;
      counts[index] = 1;
    }
  }
}
#endif  // USE_CUDA

template<typename Dtype>
void SoftmaxWithLossLayer<Dtype>::Backward_gpu(
    const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) <<
        this->type() << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
    if (this->device_->backend() == BACKEND_CUDA) {
#ifdef USE_CUDA
      Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
      const Dtype* prob_data = prob_.gpu_data();
      const Dtype* top_data = top[0]->gpu_data();
      caffe_gpu_memcpy(prob_.count() * sizeof(Dtype), prob_data, bottom_diff);
      const Dtype* label = bottom[1]->gpu_data();
      const int_tp dim = prob_.count() / outer_num_;
      const int_tp nthreads = outer_num_ * inner_num_;
      // Since this memory is never used for anything else,
      // we use to to avoid allocating new GPU memory.
      Dtype* counts = prob_.mutable_gpu_diff();
      // NOLINT_NEXT_LINE(whitespace/operators)
      SoftmaxLossBackwardGPU<Dtype> CUDA_KERNEL(CAFFE_GET_BLOCKS(nthreads),
          CAFFE_CUDA_NUM_THREADS) (nthreads, top_data, label, bottom_diff,
          outer_num_, dim, inner_num_, has_ignore_label_,
          ignore_label_, counts);

      Dtype valid_count = -1;
      if (normalization_ == LossParameter_NormalizationMode_VALID &&
          has_ignore_label_) {
        caffe_gpu_asum(nthreads, counts, &valid_count);
      }
      const Dtype loss_weight = top[0]->cpu_diff()[0] /
      get_normalizer(normalization_, valid_count);
      caffe_gpu_scal(prob_.count(), loss_weight , bottom_diff);
#endif  // USE_CUDA
    } else {
#ifdef USE_GREENTEA
      viennacl::ocl::context &ctx = viennacl::ocl::get_context(
          this->device_->id());
      viennacl::ocl::program &program = this->device_->program();

      cl_mem bottom_diff = (cl_mem)(bottom[0]->mutable_gpu_diff());
      cl_mem prob_data = (cl_mem)(prob_.gpu_data());
      cl_mem top_data = (cl_mem)(top[0]->gpu_data());
      greentea_gpu_memcpy(prob_.count() * sizeof(Dtype),
          prob_data, 0, bottom_diff, 0, &ctx);
      cl_mem label = (cl_mem)(bottom[1]->gpu_data());
      const int_tp dim = prob_.count() / outer_num_;
      const int_tp nthreads = outer_num_ * inner_num_;
      cl_mem counts = (cl_mem)(prob_.mutable_gpu_diff());

      viennacl::ocl::kernel &oclk_softmax_loss_backward = program.get_kernel(
          CL_KERNEL_SELECT("softmax_loss_backward"));
      viennacl::ocl::enqueue(
          oclk_softmax_loss_backward(nthreads, WrapHandle(top_data, &ctx),
              WrapHandle(label, &ctx), WrapHandle(bottom_diff, &ctx),
              outer_num_, dim, inner_num_, has_ignore_label_ ? 1 : 0,
              ignore_label_, WrapHandle(counts, &ctx)),
          ctx.get_queue());

      Dtype valid_count = -1;
      if (normalization_ == LossParameter_NormalizationMode_VALID &&
          has_ignore_label_) {
        greentea_gpu_asum<Dtype>(this->device_->id(),
            nthreads, counts, 0, &valid_count);
      }
      const Dtype loss_weight = top[0]->cpu_diff()[0] /
      get_normalizer(normalization_, valid_count);
      greentea_gpu_scal<Dtype>(this->device_->id(),
          prob_.count(), loss_weight, bottom_diff, 0);
#endif  // USE_GREENTEA
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SoftmaxWithLossLayer);

}  // namespace caffe
