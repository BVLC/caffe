#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void SoftmaxWithLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  Forward_cpu(bottom, top);
}

template <typename Dtype>
void SoftmaxWithLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  // TODO(Yangqing): implement the GPU version of softmax.
  Backward_cpu(top, propagate_down, bottom);
}

INSTANTIATE_CLASS(SoftmaxWithLossLayer);


}  // namespace caffe
