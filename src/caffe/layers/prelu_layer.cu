#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layers/neuron_layer.hpp"
#include "caffe/layers/prelu_layer.hpp"

namespace caffe {

// CUDA kernele for forward
template <typename Dtype>
__global__ void PReLUForward(const int n, const int channels, const int dim,
    const Dtype* in, Dtype* out, const Dtype* slope_data,
    const int div_factor) {
  CUDA_KERNEL_LOOP(index, n) {
    int c = (index / dim) % channels / div_factor;
    out[index] = in[index] > 0 ? in[index] : in[index] * slope_data[c];
  }
}

// CUDA kernel for bottom backward
template <typename Dtype>
__global__ void PReLUBackward(const int n, const int channels, const int dim,
    const Dtype* in_diff, const Dtype* in_data, Dtype* out_diff,
    const Dtype* slope_data, const int div_factor) {
  CUDA_KERNEL_LOOP(index, n) {
    int c = (index / dim) % channels / div_factor;
    out_diff[index] = in_diff[index] * ((in_data[index] > 0)
        + (in_data[index] <= 0) * slope_data[c]);
  }
}

// CUDA kernel for element-wise parameter backward
template <typename Dtype>
__global__ void PReLUParamBackward(const int n,
    const int rows, const int rowPitch, const Dtype* in_diff,
    const Dtype* in_data, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * in_data[index] * (in_data[index] <= 0);
    for ( int k = 1; k < rows; k++ ) {
        out_diff[index] += in_diff[index + k*rowPitch]
           * in_data[index + k*rowPitch] * (in_data[index + k*rowPitch] <= 0);
    }
  }
}

template <typename Dtype>
void PReLULayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  const int dim = bottom[0]->count(2);
  const int channels = bottom[0]->channels();
  const Dtype* slope_data = this->blobs_[0]->gpu_data();
  const int div_factor = channel_shared_ ? channels : 1;

  // For in-place computation
  if (top[0] == bottom[0]) {
    caffe_copy(count, bottom_data, bottom_memory_.mutable_gpu_data());
  }

  // NOLINT_NEXT_LINE(whitespace/operators)
  PReLUForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, channels, dim, bottom_data, top_data, slope_data, div_factor);
  CUDA_POST_KERNEL_CHECK;
}


INSTANTIATE_LAYER_GPU_FUNCS(PReLULayer);


}  // namespace caffe
