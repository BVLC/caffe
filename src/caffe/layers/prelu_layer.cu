#include "hip/hip_runtime.h"
#include <algorithm>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

// CUDA kernele for forward
template <typename Dtype>
__global__ void PReLUForward(const int n, const int channels, const int dim,
    const Dtype* in, Dtype* out, const Dtype* slope_data,
    const int div_factor) {
  CUDA_KERNEL_LOOP(index, n) {
    int c = (index / dim) % channels / div_factor;
    out[index] = in[index] > 0 ? in[index] : in[index] * slope_data[c];
  }
}

// CUDA kernel for bottom backward
template <typename Dtype>
__global__ void PReLUBackward(const int n, const int channels, const int dim,
    const Dtype* in_diff, const Dtype* in_data, Dtype* out_diff,
    const Dtype* slope_data, const int div_factor) {
  CUDA_KERNEL_LOOP(index, n) {
    int c = (index / dim) % channels / div_factor;
    out_diff[index] = in_diff[index] * ((in_data[index] > 0)
        + (in_data[index] <= 0) * slope_data[c]);
  }
}

// CUDA kernel for element-wise parameter backward
template <typename Dtype>
__global__ void PReLUParamBackward(const int n, const Dtype* in_diff,
    const Dtype* in_data, Dtype* out_diff) {
  CUDA_KERNEL_LOOP(index, n) {
    out_diff[index] = in_diff[index] * in_data[index] * (in_data[index] <= 0);
  }
}

template <typename Dtype>
void PReLULayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  const int dim = bottom[0]->count(2);
  const int channels = bottom[0]->channels();
  const Dtype* slope_data = this->blobs_[0]->gpu_data();
  const int div_factor = channel_shared_ ? channels : 1;

  // For in-place computation
  if (top[0] == bottom[0]) {
    caffe_copy(count, bottom_data, bottom_memory_.mutable_gpu_data());
  }

  // NOLINT_NEXT_LINE(whitespace/operators)
  PReLUForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, channels, dim, bottom_data, top_data, slope_data, div_factor);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
void PReLULayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  const Dtype* top_diff = top[0]->gpu_diff();
  const int count = bottom[0]->count();
  const int dim = bottom[0]->count(2);
  const int channels = bottom[0]->channels();

  // For in-place computation
  if (top[0] == bottom[0]) {
    bottom_data = bottom_memory_.gpu_data();
  }

  // Propagte to param
  // Since to write bottom diff will affect top diff if top and bottom blobs
  // are identical (in-place computaion), we first compute param backward to
  // keep top_diff unchanged.
  if (this->param_propagate_down_[0]) {
    Dtype* slope_diff = this->blobs_[0]->mutable_gpu_diff();
    // slope_diff is set as 0, then accumulated over batches
    caffe_gpu_set<Dtype>(this->blobs_[0]->count(), Dtype(0), slope_diff);
    int cdim = channels * dim;
    Dtype dsum = 0.;
    for (int n = 0; n < bottom[0]->num(); ++n) {
      Dtype* temp_buff = multiplier_.mutable_gpu_diff();
      // compute element-wise diff
      // NOLINT_NEXT_LINE(whitespace/operators)
      PReLUParamBackward<Dtype><<<CAFFE_GET_BLOCKS(count),
          CAFFE_CUDA_NUM_THREADS>>>(
          cdim, top_diff + top[0]->offset(n),
          bottom_data + bottom[0]->offset(n), multiplier_.mutable_gpu_diff());
      CUDA_POST_KERNEL_CHECK;
      if (channel_shared_) {
        Dtype d;
        caffe_gpu_dot<Dtype>(channels * dim, multiplier_.gpu_diff(),
            multiplier_.gpu_data(), &d);
        dsum += d;
      } else {
        caffe_gpu_gemv<Dtype>(CblasNoTrans, channels, dim, 1.,
            multiplier_.gpu_diff(), multiplier_.gpu_data(), 1.,
            slope_diff);
      }
    }
    if (channel_shared_) {
      caffe_gpu_set(this->blobs_[0]->count(), Dtype(dsum), slope_diff);
    }
  }
  // Propagate to bottom
  if (propagate_down[0]) {
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const Dtype* slope_data = this->blobs_[0]->gpu_data();
    int div_factor = channel_shared_ ? channels : 1;
    // NOLINT_NEXT_LINE(whitespace/operators)
    PReLUBackward<Dtype><<<CAFFE_GET_BLOCKS(count),
        CAFFE_CUDA_NUM_THREADS>>>(
        count, channels, dim, top_diff, bottom_data, bottom_diff, slope_data,
        div_factor);
    CUDA_POST_KERNEL_CHECK;
  }
}


INSTANTIATE_LAYER_GPU_FUNCS(PReLULayer);


}  // namespace caffe
