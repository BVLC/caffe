#include "hip/hip_runtime.h"
#include <sstream>
#include <string>
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/layers/conv_spatial_layer.hpp"
#include "caffe/util/benchmark.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/util/math_functions.hpp"

#ifdef USE_GREENTEA
#include "caffe/greentea/cl_kernels.hpp"
#include "caffe/greentea/greentea.hpp"
#endif

#include <boost/filesystem.hpp>

namespace caffe {
#ifndef CPU_ONLY
#ifdef USE_GREENTEA

//  #define dbg
#ifdef dbg
#define dbgPrint(x) (x)
#else
#define dbgPrint(x)
#endif

#define CACHE_DIRECTORY ".spatialkernels/"

template<>
void ConvolutionLayerSpatial<float>::generate_key() {
  std::stringstream keyBuilder;
  keyBuilder << kernel_w_ << "_" << kernel_h_ << "_" << channels_ << "_"
             << group_ << "_" << stride_h_ << "_" << stride_w_ << "_"
             << bias_term_ << "_" << padded_width_ << "_" << padded_height_
             << "_" << num_ << "_" << group_ << "_" << M_;
  key_ = keyBuilder.str();
}

template<>
std::string ConvolutionLayerSpatial<float>::generate_unique_key() {
  std::stringstream keyBuilder;
  keyBuilder << key_ << "" << kernel_uid_;
  kernel_uid_++;
  return keyBuilder.str();
}

template<>
std::string ConvolutionLayerSpatial<float>::generate_specific_key(
    int_tp type, int_tp blockWidth, int_tp blockHeight, int_tp blockDepth) {
  std::stringstream keyBuilder;
  keyBuilder << key_ << "_" << type << "_" << blockWidth << "_" << blockHeight
             << "_" << blockDepth;
  return keyBuilder.str();
}

template<>
bool ConvolutionLayerSpatial<float>::generate_kernel(
    const vector<Blob<float>*>& bottom, const vector<Blob<float>*>& top,
    int_tp blockWidth,
    int_tp blockHeight, int_tp blockDepth) {
  // Standard spatial setup is done here
  std::string kernelDef = "MULTI";
  std::string stringBuilder;
  std::stringstream optionsString;

  int_tp workItemOutput[3];
  int_tp yDim = blockHeight;
  int_tp zDim = blockDepth;

  std::string kernelUKey = generate_specific_key(1, blockWidth, blockHeight,
                                                 blockDepth);
  std::stringstream multFunctionBuilder;
  workItemOutput[0] = 4;
  workItemOutput[1] = yDim;
  workItemOutput[2] = zDim;

  std::string multiplication_func = "floatDotV4(V1,V2)=(V1.s0123*V2.s0123)";

  if (kernel_w_ <= 11) {
    multFunctionBuilder << "floatDotV4(V1,V2)=" << "(";
    for (int_tp kw = 0; kw < kernel_w_; kw++) {
      multFunctionBuilder << "V1.s" << std::hex << kw << kw + 1 * stride_w_
                          << kw + 2 * stride_w_ << kw + 3 * stride_w_
                          << std::dec;
      multFunctionBuilder << "*";
      multFunctionBuilder << "V2.s" << std::hex << kw << std::dec;

      if (kw == kernel_w_ - 1)
        multFunctionBuilder << ")";
      else
        multFunctionBuilder << "+";
    }
    multiplication_func = multFunctionBuilder.str();
  }

  int_tp lineSize = kernel_w_ + (workItemOutput[0] - 1) * stride_w_;

  kernel_name_ = "U";
  kernel_name_ += kernelUKey.c_str();
  if (kernel_h_ == 11 && stride_h_ == 4) {
    kernel_name_ += "_1";
    kernelDef = "MULTI_11";
    workItemOutput[1] = 1;
  } else if (kernel_w_ <= 11 && lineSize <= 16 && stride_h_ == 1) {
    kernel_name_ += "_2";
    kernelDef = "MULTI_GEN";
  } else {
    kernel_name_ += "_5";
    kernelDef = "MULTI";
    workItemOutput[1] = 1;
    workItemOutput[0] = 1;
  }

  // Build list of options and defines
  optionsString.str("");
  optionsString << "-cl-fast-relaxed-math " << " -D KERNELSIZE="
                << kernel_w_ * kernel_h_ << " -D KERNEL_W=" << kernel_w_
                << " -D KERNEL_H=" << kernel_h_ << " -D CHANNELS="
                << channels_ / group_ << " -D STRIDE_H=" << stride_h_
                << " -D STRIDE_W=" << stride_w_ << " -D APPLY_BIAS="
                << bias_term_ << " -D OUTPUT_W=" << output_w_ << " -D OUTPUT_H="
                << output_h_ << " -D OUTPUT_Z=" << M_ << " -D WIDTH="
                << padded_width_ << " -D HEIGHT=" << padded_height_ << " -D "
                << multiplication_func.c_str() << " -D XPAR="
                << workItemOutput[0] << " -D YPAR=" << workItemOutput[1]
                << " -D ZPAR=" << workItemOutput[2] << " -D "
                << kernelDef.c_str() << " -D CFMulti_11_11_4=U"
                << kernelUKey.c_str() << "_1" << " -D CFMulti_6=U"
                << kernelUKey.c_str() << "_2" << " -D CFMulti=U"
                << kernelUKey.c_str() << "_5";

  if (lineSize <= 4)
    optionsString << " -D DTImage=" << "Dtype4";
  else if (lineSize <= 8)
    optionsString << " -D DTImage=" << "Dtype8";
  else
    optionsString << " -D DTImage=" << "Dtype16";

  if (kernel_w_ <= 4)
    optionsString << " -D DTKernel=" << "Dtype4";
  else if (kernel_w_ <= 8)
    optionsString << " -D DTKernel=" << "Dtype8";
  else
    optionsString << " -D DTKernel=" << "Dtype16";

  string options = optionsString.str();
  viennacl::ocl::context &ctx = viennacl::ocl::get_context(
      this->device_->id());

  try {
    viennacl::ocl::program & program = submit_conv_spatial_program(&ctx,
                                                                   kernel_name_,
                                                                   options);
    cl_ulong privateMemUsed;
    viennacl::ocl::kernel & kernel = program.get_kernel(kernel_name_);
    clGetKernelWorkGroupInfo(kernel.handle().get(),
                             viennacl::ocl::current_device().id(),
                             CL_KERNEL_PRIVATE_MEM_SIZE,
                             sizeof(cl_ulong), &privateMemUsed,
                             NULL);
    size_t workSize[3] = { 1, 1, 1 };
    if (privateMemUsed == 0) {
      kernelQueue.push_back(
          new kernelConfig(kernel_name_, workSize, workSize, workItemOutput,
                           true, false, false, false, 1));
      dbgPrint(std::cout <<
          "successfully generated kernel using generate Kernel"
          << std::endl);
    } else {
      ctx.delete_program(kernel_name_);
    }
  } catch (std::exception & e) {
    dbgPrint(std::cout << e.what() << std::endl);
    return false;
  }

  return true;
}

template<>
bool ConvolutionLayerSpatial<float>::generate_batched_kernel(
    const vector<Blob<float>*>& bottom, const vector<Blob<float>*>& top,
    int_tp blockWidth,
    int_tp blockHeight, int_tp blockDepth) {
  std::string kernelDef = "MULTI";
  std::stringstream multFunctionBuilder;
  std::string stringBuilder;
  std::stringstream optionsString;
  int_tp workItemOutput[3];
  std::string kernelUKey = generate_specific_key(3, blockWidth, blockHeight,
                                                 blockDepth);

  workItemOutput[0] = 4;
  workItemOutput[1] = 1;
  workItemOutput[2] = 1;

  std::string multiplication_func = "floatDotV4(V1,V2)=(V1.s0123*V2.s0123)";

  if (kernel_w_ <= 11) {
    multFunctionBuilder << "floatDotV4(V1,V2)=" << "(";
    for (int_tp kw = 0; kw < kernel_w_; kw++) {
      multFunctionBuilder << "V1.s" << std::hex << kw << kw + 1 * stride_w_
                          << kw + 2 * stride_w_ << kw + 3 * stride_w_
                          << std::dec;
      multFunctionBuilder << "*";
      multFunctionBuilder << "V2.s" << std::hex << kw << std::dec;

      if (kw == kernel_w_ - 1)
        multFunctionBuilder << ")";
      else
        multFunctionBuilder << "+";
    }
    multiplication_func = multFunctionBuilder.str();
  }

  if (stride_h_ > 1)
    workItemOutput[1] = 1;
  else
    workItemOutput[1] = blockHeight;

  workItemOutput[2] = blockDepth;

  int_tp lineSize = kernel_w_ + (workItemOutput[0] - 1) * stride_w_;

  kernel_name_ = "U";
  kernel_name_ += kernelUKey.c_str();
  if (lineSize <= 16) {
    kernel_name_ += "_2";
    kernelDef = "MULTI_BATCHED";
  } else {
    return false;
  }

  // Build list of options and defines
  optionsString.str("");
  optionsString << " -cl-fast-relaxed-math " << " -D KERNELSIZE="
                << kernel_w_ * kernel_h_ << " -D KERNEL_W=" << kernel_w_
                << " -D KERNEL_H=" << kernel_h_ << " -D CHANNELS="
                << channels_ / group_ << " -D STRIDE_H=" << stride_h_
                << " -D STRIDE_W=" << stride_w_ << " -D APPLY_BIAS="
                << bias_term_ << " -D OUTPUT_W=" << output_w_ << " -D OUTPUT_H="
                << output_h_ << " -D OUTPUT_Z=" << M_ << " -D IMG_OFFSET="
                << padded_width_ * padded_height_ * channels_
                << " -D OUTPUT_OFFSET=" << this->top_dim_ << " -D WIDTH="
                << padded_width_ << " -D HEIGHT=" << padded_height_ << " -D "
                << multiplication_func.c_str() << " -D XPAR="
                << workItemOutput[0] << " -D YPAR=" << workItemOutput[1]
                << " -D ZPAR=" << workItemOutput[2] << " -D "
                << kernelDef.c_str() << " -D CFMulti_6=U" << kernelUKey.c_str()
                << "_2";

  if (lineSize <= 4)
    optionsString << " -D DTImage=" << "Dtype4";
  else if (lineSize <= 8)
    optionsString << " -D DTImage=" << "Dtype8";
  else
    optionsString << " -D DTImage=" << "Dtype16";

  if (kernel_w_ <= 4)
    optionsString << " -D DTKernel=" << "Dtype4";
  else if (kernel_w_ <= 8)
    optionsString << " -D DTKernel=" << "Dtype8";
  else
    optionsString << " -D DTKernel=" << "Dtype16";

  string options = optionsString.str();

  viennacl::ocl::context &ctx = viennacl::ocl::get_context(
      this->device_->id());

  try {
    viennacl::ocl::program & program = submit_conv_spatial_program(&ctx,
                                                                   kernel_name_,
                                                                   options);
    cl_ulong privateMemUsed;
    viennacl::ocl::kernel & kernel = program.get_kernel(kernel_name_);

    clGetKernelWorkGroupInfo(kernel.handle().get(),
                             viennacl::ocl::current_device().id(),
                             CL_KERNEL_PRIVATE_MEM_SIZE,
                             sizeof(cl_ulong), &privateMemUsed,
                             NULL);
    size_t workSize[3] = { 1, 1, 1 };
    if (privateMemUsed == 0) {
      kernelQueue.push_back(
          new kernelConfig(kernel_name_, workSize, workSize, workItemOutput,
                           true, false, false, false, 1));
      dbgPrint(std::cout <<
          "successfully generated kernel using generate Kernel" << std::endl);
    } else {
      ctx.delete_program(kernel_name_);
    }
  } catch (std::exception& e) {
    dbgPrint(std::cout << e.what() << std::endl);
    return false;
  }

  return true;
}

template<typename Dtype>
void ConvolutionLayerSpatial<Dtype>::swizzleWeights(
    const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top,
    int_tp swizzled_factor) {

  viennacl::ocl::context &ctx = viennacl::ocl::get_context(
      this->device_->id());
  viennacl::ocl::program &program = this->device_->program();
  viennacl::ocl::kernel &oclk_copy_weight = program.get_kernel(
      CL_KERNEL_SELECT("copyWeightsSwizzled"));
  cl_uint argIdx = 0;

  int_tp channels = this->channels_ / this->group_;

  ClState& clState = Caffe::cl_state();
  ClMemOff<Dtype> buf_weight = clState.get_buffer_mem(weight);
  ClMemOff<Dtype> buf_swizzled = clState.get_buffer_mem(swizzled_weights);

  oclk_copy_weight.arg(argIdx++, WrapHandle(buf_weight.memobj, &ctx));
  oclk_copy_weight.arg(argIdx++, WrapHandle(buf_swizzled.memobj, &ctx));
  oclk_copy_weight.arg(argIdx++, kernel_w_);
  oclk_copy_weight.arg(argIdx++, kernel_h_);
  oclk_copy_weight.arg(argIdx++, channels);
  oclk_copy_weight.arg(argIdx++, this->num_output_);
  oclk_copy_weight.arg(argIdx++, swizzled_factor);
  const size_t global_work_size_Copy[3] = { (size_t) (this->num_output_
      * channels * kernel_w_ * kernel_h_), 1, 1 };

  OCL_CHECK(clEnqueueNDRangeKernel(ctx.get_queue().handle().get(),
                                       oclk_copy_weight.handle().get(), 3, NULL,
                                       global_work_size_Copy, NULL, 0, NULL,
                                       NULL));
}

template<>
void ConvolutionLayerSpatial<float>::calculate_global_size(int_tp batch,
                                  int_tp* wio,  // work item output size
                                  size_t* lSize,  // local size
                                  size_t* gSize) {  // global size
  gSize[0] = ceil(
      (fmax(static_cast<float>(output_w_) / wio[0], 1.0)) / lSize[0])
      * lSize[0];
  gSize[1] = ceil(
      (fmax(static_cast<float>(output_h_) / wio[1], 1.0)) / lSize[1])
      * lSize[1];
  gSize[2] = ceil(
      static_cast<float>((ceil(static_cast<float>(M_) * batch / wio[2])))
          / lSize[2]) * lSize[2];
}

template<typename Dtype>
void ConvolutionLayerSpatial<Dtype>::pad_image(
    const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top,
    int_tp image_offset,
    kernelConfig* config,
    int_tp imgNum) {
#ifdef USE_GREENTEA
  viennacl::ocl::context &ctx = viennacl::ocl::get_context(
      this->device_->id());
  // Copy kernel
  viennacl::ocl::program &program = this->device_->program();
  viennacl::ocl::kernel &oclk_copy = program.get_kernel(
                                       CL_KERNEL_SELECT("copyImage"));
  cl_uint argIdx = 0;
  int_tp col_data_offset = 0;
  int_tp channels = this->channels_;

  ClState& clState = Caffe::cl_state();
  ClMemOff<Dtype> buf_bottom = clState.get_buffer_mem(bottom_data);
  ClMemOff<Dtype> buf_col = clState.get_buffer_mem(col_data);

  oclk_copy.arg(argIdx++, WrapHandle(buf_bottom.memobj, &ctx));
  oclk_copy.arg(argIdx++, image_offset);
  oclk_copy.arg(argIdx++, channels);
  oclk_copy.arg(argIdx++, height_);
  oclk_copy.arg(argIdx++, width_);
  oclk_copy.arg(argIdx++, padded_height_);
  oclk_copy.arg(argIdx++, padded_width_);
  oclk_copy.arg(argIdx++, pad_h_);
  oclk_copy.arg(argIdx++, pad_w_);
  oclk_copy.arg(argIdx++, WrapHandle(buf_col.memobj, &ctx));
  oclk_copy.arg(argIdx++, col_data_offset);
  oclk_copy.arg(argIdx++, imgNum);
  const size_t global_work_size_Copy[3] = { (size_t) padded_width_,
      (size_t) padded_height_, (size_t) channels };

  clEnqueueNDRangeKernel(ctx.get_queue().handle().get(),
                         oclk_copy.handle().get(), 3, NULL,
                         global_work_size_Copy, NULL, 0, NULL, NULL);
#endif
}

template<>
bool ConvolutionLayerSpatial<float>::create_basic_kernel(
    const vector<Blob<float>*>& bottom, const vector<Blob<float>*>& top,
    int_tp blockWidth,
    int_tp blockHeight, int_tp blockDepth) {
  // Standard spatial setup is done here
  std::stringstream keyBuilder;
  std::stringstream multFunctionBuilder;
  std::string stringBuilder;
  std::stringstream optionsString;
  std::string kernelDef = "MULTI";
  std::string kernelUKey = generate_specific_key(1, blockWidth, blockHeight,
                                                 blockDepth);

  int_tp workItemOutput[3];
  workItemOutput[0] = 1;
  workItemOutput[1] = 1;
  workItemOutput[2] = 1;

  kernel_name_ = "U";
  kernel_name_ += kernelUKey.c_str();
  kernel_name_ += "_BASIC";

  // Build list of options and defines
  optionsString.str("");
  optionsString << "-cl-fast-relaxed-math " << " -D KERNELSIZE="
                << kernel_w_ * kernel_h_ << " -D KERNEL_W=" << kernel_w_
                << " -D KERNEL_H=" << kernel_h_ << " -D CHANNELS="
                << channels_ / group_ << " -D STRIDE_H=" << stride_h_
                << " -D STRIDE_W=" << stride_w_ << " -D APPLY_BIAS="
                << bias_term_ << " -D OUTPUT_W=" << output_w_ << " -D OUTPUT_H="
                << output_h_ << " -D OUTPUT_Z=" << M_ << " -D WIDTH="
                << padded_width_ << " -D HEIGHT=" << padded_height_
                << " -D XPAR=" << workItemOutput[0] << " -D YPAR="
                << workItemOutput[1] << " -D ZPAR=" << workItemOutput[2]
                << " -D " << kernelDef.c_str() << " -D CFMulti=U"
                << kernelUKey.c_str() << "_BASIC";

  string options = optionsString.str();

  viennacl::ocl::context &ctx = viennacl::ocl::get_context(this->device_->id());
  try {
    submit_conv_spatial_program(&ctx, kernel_name_, options);
  } catch (std::exception& e) {
    dbgPrint(std::cout << "Basic kernel generation failed" << std::endl);
    return false;
  }

  size_t localSize[3] = { 1, 1, 1 };
  size_t globalSize[3];
  calculate_global_size(1, workItemOutput, localSize, globalSize);

  kernelQueue.push_back(
      new kernelConfig(kernel_name_, globalSize, localSize, workItemOutput,
                       false, false, false, true, 4));

  return true;
}

template<>
cl_int ConvolutionLayerSpatial<float>::convolve(
    const vector<Blob<float>*>& bottom, const vector<Blob<float>*>& top,
    int_tp index,
    int_tp numImages, kernelConfig* config) {

  if (config->swizzle_weights)
    swizzleWeights(bottom, top, 16);

  viennacl::ocl::context &ctx = viennacl::ocl::get_context(this->device_->id());
  viennacl::ocl::program & program = ctx.get_program(config->kernelName);
  viennacl::ocl::kernel &kernel = program.get_kernel(config->kernelName);
  cl_int err = 0;

  for (int_tp n = 0; n < numImages; ++n) {
    for (int_tp g = 0; g < group_; ++g) {
      bias_offset_ = M_ * g;
      int_tp image_offset = n * this->bottom_dim_
          + width_ * height_ * (channels_ / group_) * g;
      int_tp output_image_offset = n * this->top_dim_
          + output_w_ * output_h_ * M_ * g;

      cl_uint argIdx = 0;
      int_tp kernel_offset = kernel_h_ * kernel_w_ * (channels_ / group_) * M_
          * g;

      // Copy image
      ClState& clState = Caffe::cl_state();
      ClMemOff<float> buf_col = clState.get_buffer_mem(col_data);
      ClMemOff<float> buf_bottom = clState.get_buffer_mem(bottom_data);
      ClMemOff<float> buf_swizzled = clState.get_buffer_mem(swizzled_weights);
      ClMemOff<float> buf_weight = clState.get_buffer_mem(weight);
      ClMemOff<float> buf_bias = clState.get_buffer_mem(bias_);
      ClMemOff<float> buf_top = clState.get_buffer_mem(top_data);

      if (pad_w_ > 0 || pad_h_ > 0) {
        pad_image(bottom, top, image_offset, config, numImages);
        image_offset = 0;
        kernel.arg(argIdx++, WrapHandle(buf_col.memobj, &ctx));
      } else {
        kernel.arg(argIdx++, WrapHandle(buf_bottom.memobj, &ctx));
      }
      kernel.arg(argIdx++, image_offset);
      if (config->swizzle_weights)
        kernel.arg(argIdx++, WrapHandle(buf_swizzled.memobj, &ctx));
      else
        kernel.arg(argIdx++, WrapHandle(buf_weight.memobj, &ctx));
      kernel.arg(argIdx++, kernel_offset);
      kernel.arg(argIdx++, WrapHandle(buf_bias.memobj, &ctx));
      kernel.arg(argIdx++, bias_offset_);
      kernel.arg(argIdx++, WrapHandle(buf_top.memobj, &ctx));
      kernel.arg(argIdx++, output_image_offset);

      if (config->use_null_local) {
        err = clEnqueueNDRangeKernel(ctx.get_queue().handle().get(),
                                     kernel.handle().get(), 3,
                                     NULL,
                                     config->global_work_size, NULL, 0, NULL,
                                     NULL);
      } else {
        err = clEnqueueNDRangeKernel(ctx.get_queue().handle().get(),
                                     kernel.handle().get(), 3,
                                     NULL,
                                     config->global_work_size,
                                     config->local_work_size, 0, NULL,
                                     NULL);
      }

      if (err != CL_SUCCESS)
        return err;
      viennacl::backend::finish();
    }
    if (config->kernelType == 2)
      break;
  }

  return err;
}

template<>
cl_int ConvolutionLayerSpatial<float>::batched_convolve(
    const vector<Blob<float>*>& bottom, const vector<Blob<float>*>& top,
    int_tp index,
    int_tp numImages, kernelConfig* config) {

  if (config->swizzle_weights)
    swizzleWeights(bottom, top, 16);

  viennacl::ocl::context &ctx = viennacl::ocl::get_context(this->device_->id());
  viennacl::ocl::program & program = ctx.get_program(config->kernelName);
  viennacl::ocl::kernel &kernel = program.get_kernel(config->kernelName);
  cl_int err = 0;

  for (int_tp g = 0; g < group_; ++g) {
    bias_offset_ = M_ * g;
    int_tp image_offset = width_ * height_ * (channels_ / group_) * g;
    int_tp output_image_offset = output_w_ * output_h_ * M_ * g;

    cl_uint argIdx = 0;
    int_tp kernel_offset = kernel_h_ * kernel_w_ * (channels_ / group_) * M_
        * g;

    pad_image(bottom, top, image_offset, config, numImages);

    ClState& clState = Caffe::cl_state();
    ClMemOff<float> buf_col = clState.get_buffer_mem(col_data);
    ClMemOff<float> buf_swizzled = clState.get_buffer_mem(swizzled_weights);
    ClMemOff<float> buf_weight = clState.get_buffer_mem(weight);
    ClMemOff<float> buf_bias = clState.get_buffer_mem(bias_);
    ClMemOff<float> buf_top = clState.get_buffer_mem(top_data);

    kernel.arg(argIdx++, WrapHandle(buf_col.memobj, &ctx));
    kernel.arg(argIdx++, image_offset);
    if (config->swizzle_weights)
      kernel.arg(argIdx++, WrapHandle(buf_swizzled.memobj, &ctx));
    else
      kernel.arg(argIdx++, WrapHandle(buf_weight.memobj, &ctx));
    kernel.arg(argIdx++, kernel_offset);
    kernel.arg(argIdx++, WrapHandle(buf_bias.memobj, &ctx));
    kernel.arg(argIdx++, bias_offset_);
    kernel.arg(argIdx++, WrapHandle(buf_top.memobj, &ctx));
    kernel.arg(argIdx++, output_image_offset);
    kernel.arg(argIdx++, numImages);

    if (config->use_null_local) {
      err = clEnqueueNDRangeKernel(ctx.get_queue().handle().get(),
                                   kernel.handle().get(), 3,
                                   NULL,
                                   config->global_work_size, NULL, 0, NULL,
                                   NULL);
    } else {
      err = clEnqueueNDRangeKernel(ctx.get_queue().handle().get(),
                                   kernel.handle().get(), 3,
                                   NULL,
                                   config->global_work_size,
                                   config->local_work_size, 0, NULL,
                                   NULL);
    }
    if (err != CL_SUCCESS)
      return err;
  }
  return err;
}

template<>
float ConvolutionLayerSpatial<float>::timed_convolve(
    const vector<Blob<float>*>& bottom, const vector<Blob<float>*>& top,
    int_tp index,
    int_tp numImages, kernelConfig* config) {
  Timer timer;
  timer.initted();
  timer.Start();
  cl_int err;
  dbgPrint(std::cout << "Bechmarking kernel: " << config->kernelName
           << std::endl);
  if (config->batched_execute)
    err = batched_convolve(bottom, top, index, num_, config);
  else
    err = convolve(bottom, top, index, num_, config);
  timer.Stop();
  if (err != CL_SUCCESS) {
    config->tested = true;
    config->verified = false;
  }

  float elapsedTime = timer.MilliSeconds();
#ifdef dbg
  double out_w = output_w_;
  double out_h = output_h_;
  double out_z = M_;
  double k_w = kernel_w_;
  double k_h = kernel_h_;
  double k_z = channels_;
  double totalFlops = ((k_w*k_h*k_z -1)*2)*(out_w*out_h*out_z)*num_;
  std::cout << "\tEstimated Gflops:" << ((totalFlops/1000)/1000)/1000
  << std::endl;
  std::cout << "\tEstimated GFLOPS/S: " <<
  (((totalFlops/1000)/1000)/1000)*(1000.0/elapsedTime) << std::endl;
#if 0
  std::cout << "Estimated utilization: " <<
  ((((totalFlops/1000)/1000)/1000)*(1000.0/elapsedTime))/880.0
  << std::endl;
#endif
#endif
  return elapsedTime;
}

template<>
bool ConvolutionLayerSpatial<float>::verify_result(
    const vector<Blob<float>*>& bottom, const vector<Blob<float>*>& top,
    int_tp index,
    int_tp numImages, const Blob<float> &verify_blob, kernelConfig* config) {

  uint_tp verificationFail = 0;

  if (config->verified)
    return true;
  else if (config->tested)
    return false;

  config->executionTime = timed_convolve(bottom, top, index, numImages,
                                         config);
  const float *verify_data = verify_blob.cpu_data();
  const float *data = top[index]->cpu_data();

  for (int_tp n = 0; n < numImages; ++n) {
    for (int_tp g = 0; g < group_; ++g) {
      int_tp output_image_offset = n * this->top_dim_
          + output_w_ * output_h_ * M_ * g;
      for (int out_ch = 0; out_ch < M_ && !verificationFail; out_ch++)
        for (int h = 0; h < output_h_ && !verificationFail; h++)
          for (int w = 0; w < output_w_; w++) {
            size_t offset = output_image_offset + out_ch * output_w_ * output_h_
                            + h * output_w_ + w;
            if (fabs(data[offset] - verify_data[offset]) >
                       0.1 * fabs(verify_data[offset]) &&
                !(fabs(verify_data[offset]) < 1.e-3
                  && fabs(data[offset] - verify_data[offset]) < 1.e-4)) {
              dbgPrint(printf("test verification failed @ image %d out_ch %d h "
                              "%d w %d got %G expected %G\n",
                      n, out_ch, h, w, data[offset], verify_data[offset]));
              verificationFail = 1;
              break;
            }
          }
      if (verificationFail)
        return false;
    }
  }
  return true;
}

template<>
bool ConvolutionLayerSpatial<float>::setup_IDLF(
    const vector<Blob<float>*>& bottom, const vector<Blob<float>*>& top,
    int_tp blockWidth,
    int_tp blockHeight, int_tp blockDepth) {
  std::stringstream multFunctionBuilder;
  std::string stringBuilder;
  std::stringstream optionsString;
  std::string kernelUKey = generate_specific_key(2, blockWidth, blockHeight,
                                                 blockDepth);
  int_tp workItemOutput[3] = { blockWidth, blockHeight, blockDepth };
  std::string kernelDef = "MULTI";

  const int_tp num_output_maps = M_;
  int_tp output_width = output_w_;
  int_tp output_height = output_h_;
  int_tp output_block_width = blockWidth;
  int_tp output_block_height = blockHeight;
  int_tp simd_size = 16;
  int_tp num_batches = num_;

  kernel_name_ = "U";
  kernel_name_ += kernelUKey.c_str();
  kernel_name_ += "_SIMD16";
  kernelDef = "SIMD16";

  // Build list of options and defines
  optionsString.str("");
  optionsString << "-cl-fast-relaxed-math " << " -D IDLF" << " -D "
                << kernelDef.c_str() << " -D convolve_simd16=U"
                << kernelUKey.c_str() << "_SIMD16";

  const int_tp in_buffer_size = (output_block_height - 1) * stride_h_
                                 + kernel_h_;
  const int_tp last_block_width =
      (output_width % output_block_width == 0) ?
          output_block_width : output_width % output_block_width;
  const int_tp last_block_height =
      (output_height % output_block_height == 0) ?
          output_block_height : output_height % output_block_height;

  size_t global_size[3] = { (size_t) (output_width + output_block_width - 1)
      / output_block_width, (size_t) (output_height + output_block_height - 1)
      / output_block_height, (size_t) num_batches * num_output_maps };

  size_t local_size[3] = { 1, 1, static_cast<size_t>(simd_size) };

  optionsString << " -D SIMD_SIZE=" << simd_size
                << " -D filter_qualifier=__global" << " -D OUT_BLOCK_WIDTH="
                << output_block_width << " -D OUT_BLOCK_HEIGHT="
                << output_block_height << " -D IN_BUFFER_SIZE="
                << in_buffer_size << " -D LAST_BLOCK_WIDTH=" << last_block_width
                << " -D LAST_BLOCK_HEIGHT=" << last_block_height
                << " -D INPUT_WIDTH=" << padded_width_ << " -D INPUT_HEIGHT="
                << padded_height_ << " -D INPUT_DEPTH=" << channels_ / group_
                << " -DTOTAL_INPUT_DEPTH_SIZE=" << channels_ / group_
                << " -DTOTAL_OUTPUT_DEPTH=" << M_ / group_
                << " -DINPUT_START_X=" << 0 << " -DINPUT_START_Y=" << 0
                << " -DINPUT_START_Z=" << 0 << " -DOUTPUT_WIDTH=" << output_w_
                << " -DOUTPUT_HEIGHT=" << output_h_ << " -DFILTER_WIDTH="
                << kernel_w_ << " -DFILTER_HEIGHT=" << kernel_h_
                << " -DNUM_FILTERS=" << M_ << " -DSTRIDEX=" << stride_w_
                << " -DSTRIDEY=" << stride_h_ << " -DOWPAD=" << 0 << " -DOHPAD="
                << 0 << " -DOUT_BUFF_OFFSET=" << 0;

  string options = optionsString.str();
  viennacl::ocl::context &ctx = viennacl::ocl::get_context(this->device_->id());

  viennacl::ocl::program & program = submit_conv_spatial_program(&ctx,
                                                                 kernel_name_,
                                                                 options);

  // ClKernel kernel;
  size_t workgroupSize_used;
  viennacl::ocl::kernel & kernel = program.get_kernel(kernel_name_);
  cl_int err = clGetKernelWorkGroupInfo(
      kernel.handle().get(), viennacl::ocl::current_device().id(),
      CL_KERNEL_PREFERRED_WORK_GROUP_SIZE_MULTIPLE,
      sizeof(size_t), &workgroupSize_used,
      NULL);

  if (workgroupSize_used != simd_size) {
    ctx.delete_program(kernel_name_);
    return false;
  }

  if (err == CL_SUCCESS || err == true) {
    kernelQueue.push_back(
        new kernelConfig(kernel_name_, global_size, local_size, workItemOutput,
                         false, true, false, false, 2));
    return true;
  } else {
    ctx.delete_program(kernel_name_);
    return false;
  }
}

template<>
bool ConvolutionLayerSpatial<float>::tune_local_size(
    const vector<Blob<float>*>& bottom, const vector<Blob<float>*>& top,
    kernelConfig* config) {
  if (config->use_null_local || !config->autoTune)
    return true;

  float fastestTime = 999999990000000000000000000.0f;
  uint_tp multiplier = 4;
  uint_tp localSize[3] = { 1, 1, 1 };

  int_tp skip = 0;
  Timer timer;
  timer.initted();
  bool allFailed = true;
  for (int_tp z = 0; z <= 16; z++) {
    for (int_tp y = 0; y <= 16; y++) {
      for (int_tp x = 1; x <= 16; x++) {
        timer.Start();
        skip = 0;

        if (config->autoTune) {
          config->local_work_size[0] =
              (multiplier * x == 0) ? 1 : multiplier * x;
          config->local_work_size[1] =
              (multiplier * y == 0) ? 1 : multiplier * y;
          config->local_work_size[2] =
              (multiplier * z == 0) ? 1 : multiplier * z;

          if (config->batched_execute) {
            calculate_global_size(2, config->workItem_output,
                                  config->local_work_size,
                                  config->global_work_size);
          } else {
            calculate_global_size(1, config->workItem_output,
                                  config->local_work_size,
                                  config->global_work_size);
          }
        }
        if (config->workItem_output[2] *
            config->global_work_size[2] != M_)
          break;

        if (config->swizzle_weights)
          z = 32;

        int_tp err = 0;
        if (config->batched_execute)
          err = batched_convolve(bottom, top, 0, 1, config);
        else
          err = convolve(bottom, top, 0, 1, config);

        if (err != CL_SUCCESS)
          skip = 1;

        if (skip) {
          timer.Stop();
          break;
        }
        timer.Stop();
        allFailed = false;
        float elapsedTime = timer.MilliSeconds();

        if (elapsedTime < fastestTime) {
          fastestTime = elapsedTime;
          localSize[0] = config->local_work_size[0];
          localSize[1] = config->local_work_size[1];
          localSize[2] = config->local_work_size[2];
        }
      }
    }
  }
  if (allFailed) {
    // 1,1,1 is never a good local size and no need to test at all.
    dbgPrint(std::cout << "Can't find good local size for "
                       << config->kernelName << std::endl);
    return false;
  }

  dbgPrint(std::cout << "Best local size[" << localSize[0] << "][" <<
      localSize[1] << "]["<< localSize[2] << "]: " << fastestTime <<
      " Kernel_h: " << kernel_h_ << " kernel_w_: " << kernel_w_ <<
      " stride_w: " << stride_w_ << " pad_w_: " << pad_w_ << std::endl);

  if (config->autoTune) {
    for (int_tp li = 0; li < 3; li++)
      config->local_work_size[li] = localSize[li];

    if (config->batched_execute) {
      calculate_global_size(num_, config->workItem_output,
                            config->local_work_size, config->global_work_size);
    } else {
      calculate_global_size(1, config->workItem_output, config->local_work_size,
                            config->global_work_size);
    }
  }
  return true;
}

template<>
void ConvolutionLayerSpatial<float>::create_convolution_kernel(
    const vector<Blob<float>*>& bottom, const vector<Blob<float>*>& top,
    int_tp kernelType,
    int_tp blockWidth, int_tp blockHeight,
    int_tp blockDepth) {
  if (kernelType == 1)
    generate_kernel(bottom, top, blockWidth, blockHeight, blockDepth);
  else if (kernelType == 2)
    setup_IDLF(bottom, top, blockWidth, blockHeight, blockDepth);
  else if (kernelType == 3)
    generate_batched_kernel(bottom, top, blockWidth, blockHeight, blockDepth);
  else if (kernelType == 4)
    create_basic_kernel(bottom, top, blockWidth, blockHeight, blockDepth);
}

template<>
void ConvolutionLayerSpatial<float>::setup_convolution(
    const vector<Blob<float>*>& bottom, const vector<Blob<float>*>& top,
    const Blob<float> &verify_blob) {
  // Generates static key_
  generate_key();
  // Initializes unique kernel ID
  kernel_uid_ = 0;

  viennacl::ocl::context &ctx = viennacl::ocl::get_context(this->device_->id());
  const viennacl::ocl::device &device = ctx.current_device();
  if (device.vendor().find("Intel") != std::string::npos &&
    M_ % 16 == 0) {
    /* IDLF kernels are using Intel specific extension which make
       them intel only. */
    int kernelCnt = 0;
    for (uint32_t width = 14; width > 0; width--) {
      int candidate = 0;
      if (width > output_w_)
        continue;
      for (uint32_t height = 14; height > 0; height--) {
        if (height * width > 32 || height > output_h_)
          continue;
        int tile_x = kernel_w_ + (width - 1) * stride_w_;
        int tile_y = kernel_h_ + (height - 1) * stride_h_;
        int tile_y_stride = 64 / tile_x;

        if (tile_x % 4 != 0 && tile_x <= 16) {
          create_convolution_kernel(bottom, top, 2, width, height, 1);
          candidate++;
        } else if ((tile_x % 4 == 0) &&
                 ((tile_y + tile_y_stride - 1) / tile_y_stride < 4)) {
          create_convolution_kernel(bottom, top, 2, width, height, 1);
          candidate++;
        }
        if (candidate >= 4 && height == 2)
          break;
      }
      kernelCnt += candidate;
      if (kernelCnt >= 12 && width == 2)
        break;
    }
  } else {
    for (int_tp y = 1; y < 4; y += 1)
      for (int_tp z = 1; z < 16 && z < M_; z += 1) {
        if (4 * y * z > 32) continue;
        create_convolution_kernel(bottom, top, 1, 4, y, z);
      }
  }
  for (int_tp x = 0; x < kernelQueue.size(); x++)
    if (tune_local_size(bottom, top, kernelQueue[x])) {
      kernelQueue[x]->executionTime = timed_convolve(bottom, top, bottom_index_,
                                                     num_, kernelQueue[x]);
    } else {
      // skip those kernels without a good local size.
      kernelQueue[x]->verified = false;
      kernelQueue[x]->tested = true;
    }

  int_tp failures = 0;
  bool verification = false;
  if (kernelQueue.size()) {
    while (failures < kernelQueue.size()) {
      int_tp fastestKernel = -1;
      float fastestTime = 999999990000000000000000000.0f;

      for (int_tp x = 0; x < kernelQueue.size(); x++) {
        if (kernelQueue[x]->executionTime < fastestTime
            && kernelQueue[x]->tested == false) {
          fastestKernel = x;
          fastestTime = kernelQueue[x]->executionTime;
        }
      }
      if (fastestKernel < 0) break;
      // Test fastest kernel
      bool verified = verify_result(bottom, top, bottom_index_, num_,
                                    verify_blob, kernelQueue[fastestKernel]);
      if (verified == true) {
        kernelQueue[fastestKernel]->verified = true;
        kernel_index_ = fastestKernel;
        verification = true;
        break;
      } else {
        kernelQueue[fastestKernel]->tested = true;
        dbgPrint(std::cout << "Kernel "
                           << kernelQueue[fastestKernel]->kernelName
                           << " failed verification" << std::endl);
        failures++;
      }
    }
  }
  if (verification) {
    dbgPrint(std::cout << "Kernel <" << kernelQueue[kernel_index_]->kernelName
                       << "> passed verification" << std::endl);
  } else {
    dbgPrint(std::cout << "Verification was not successful, "
                       << "fallback to basic kernel" << std::endl);
    create_basic_kernel(bottom, top, 1, 1, 1);
    kernel_index_ = kernelQueue.size() - 1;
    verification = verify_result(bottom, top, bottom_index_, num_,
                                 verify_blob, kernelQueue[kernel_index_]);
    CHECK_EQ(verification, true) << "Basic kernel failed verification."
                                 << std::endl;
  }

  dbgPrint(std::cout << "Convolution Time:"
                     << kernelQueue[kernel_index_]->executionTime << std::endl);

  for (int_tp x = 0; x < kernelQueue.size(); x++) {
    if (x != kernel_index_)
      viennacl::ocl::current_context().delete_program(
          kernelQueue[x]->kernelName);
  }

  tuned_ = true;

  const boost::filesystem::path& path = CACHE_DIRECTORY;
  const boost::filesystem::path& dir =
                   boost::filesystem::unique_path(path).string();
  bool hasCacheDir = false;
  if (!boost::filesystem::exists(dir))
    hasCacheDir = boost::filesystem::create_directory(dir);
  else
    hasCacheDir = boost::filesystem::is_directory(dir);

  if (hasCacheDir != true) {
    std::cout << "Failed to create cache directory,"
              << "will tune again for next running" << std::endl;
    return;
  }

  string outputFile;
  outputFile = CACHE_DIRECTORY + key_;
  std::ifstream cachedKernel(outputFile.c_str());
  std::ofstream outputKernel;
  outputKernel.open(outputFile.c_str());
  outputKernel << kernelQueue[kernel_index_]->workItem_output[0] << " "
               << kernelQueue[kernel_index_]->workItem_output[1] << " "
               << kernelQueue[kernel_index_]->workItem_output[2] << " "
               << kernelQueue[kernel_index_]->kernelType << " "
               << kernelQueue[kernel_index_]->global_work_size[0] << " "
               << kernelQueue[kernel_index_]->global_work_size[1] << " "
               << kernelQueue[kernel_index_]->global_work_size[2] << " "
               << kernelQueue[kernel_index_]->local_work_size[0] << " "
               << kernelQueue[kernel_index_]->local_work_size[1] << " "
               << kernelQueue[kernel_index_]->local_work_size[2] << " "
               << kernelQueue[kernel_index_]->swizzle_weights << " "
               << kernelQueue[kernel_index_]->batched_execute << " "
               << kernelQueue[kernel_index_]->use_null_local << " ";
  outputKernel.close();
}

template<>
void ConvolutionLayerSpatial<float>::Forward_gpu(
    const vector<Blob<float>*>& bottom, const vector<Blob<float>*>& top) {

  for (int_tp i = 0; i < bottom.size(); ++i) {
    bottom_index_ = i;
    bottom_data = bottom[i]->gpu_data();
    top_data = top[i]->mutable_gpu_data();
    col_data = spatial_col_buffer_.mutable_gpu_data();
    weight = this->blobs_[0]->gpu_data();
    swizzled_weights = swizzled_weights_.mutable_gpu_data();

    weight_offset = M_ * K_;
    col_offset = K_ * N_;
    top_offset = M_ * N_;

    bias_ = NULL;
    bias_offset_ = 0;

    if (bias_term_)
      bias_ = this->blobs_[1]->gpu_data();

    if (!tuned_) {
      Blob<float> verify_blob;
      verify_blob.ReshapeLike(*top[i]);
      float *verify_data = verify_blob.mutable_gpu_data();
      const float *weight_gpu_data = this->blobs_[0]->gpu_data();
      const float *bottom_gpu_data = bottom[i]->gpu_data();
      for (int_tp n = 0; n < this->num_; ++n) {
        this->forward_gpu_gemm(bottom_gpu_data, n * this->bottom_dim_,
                               weight_gpu_data, verify_data,
                               n * this->top_dim_);
        if (this->bias_term_) {
          const float* bias = this->blobs_[1]->gpu_data();
          this->forward_gpu_bias(verify_data, n * this->top_dim_, bias);
        }
      }
      setup_convolution(bottom, top, verify_blob);
      CHECK_EQ(tuned_, true) << "Spatial convolution auto-tuning failed.";
    }

    if (kernelQueue[kernel_index_]->batched_execute)
      batched_convolve(bottom, top, i, num_, kernelQueue[kernel_index_]);
    else
      convolve(bottom, top, i, num_, kernelQueue[kernel_index_]);
  }
  viennacl::backend::finish();
}

template<>
void ConvolutionLayerSpatial<float>::Backward_gpu(
    const vector<Blob<float>*>& top, const vector<bool>& propagate_down,
    const vector<Blob<float>*>& bottom) {
  const float* weight = this->blobs_[0]->gpu_data();
  float* weight_diff = this->blobs_[0]->mutable_gpu_diff();
  for (int_tp i = 0; i < top.size(); ++i) {
    const float* top_diff = top[i]->gpu_diff();
    // Bias gradient, if necessary.
    if (this->bias_term_ && this->param_propagate_down_[1]) {
      float* bias_diff = this->blobs_[1]->mutable_gpu_diff();
      for (int_tp n = 0; n < this->num_; ++n) {
        this->backward_gpu_bias(bias_diff, top_diff, n * this->top_dim_);
      }
    }
    if (this->param_propagate_down_[0] || propagate_down[i]) {
      const float* bottom_data = bottom[i]->gpu_data();
      float* bottom_diff = bottom[i]->mutable_gpu_diff();
      for (int_tp n = 0; n < this->num_; ++n) {
        // gradient w.r.t. weight. Note that we will accumulate diffs.
        if (this->param_propagate_down_[0]) {
          this->weight_gpu_gemm(bottom_data, n * this->bottom_dim_,
              top_diff, n * this->top_dim_, weight_diff);
        }
      }
      // gradient w.r.t. bottom data, if necessary.
      if (propagate_down[i]) {
        // Multi queue execution, all previous work needs to be done first
        this->device_->FinishQueues();
        for (int_tp n = 0; n < this->num_; ++n) {
          // Multi queue execution, go through work queues
          this->device_->SwitchQueue(n);
          this->backward_gpu_gemm(top_diff, n * this->top_dim_, weight,
                                  bottom_diff, n * this->bottom_dim_);
        }
        // Multi queue execution, finish all queues
        this->device_->FinishQueues();
      }
    }
  }
}

template<typename Dtype>
void ConvolutionLayerSpatial<Dtype>::load_cached_kernels(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  // Generates static key_
  if (tuned_)
    return;
  generate_key();
  // Initializes unique kernel ID
  kernel_uid_ = 0;

  string outputFile;
  outputFile = CACHE_DIRECTORY + key_;
  std::ifstream cachedKernel(outputFile.c_str());

  if (cachedKernel) {
    int_tp x, y, z, type;
    cachedKernel >> x;
    cachedKernel >> y;
    cachedKernel >> z;
    cachedKernel >> type;
    create_convolution_kernel(bottom, top, type, x, y, z);
    kernel_index_ = kernelQueue.size() - 1;
  if (kernel_index_ == -1) {
    std::cerr << "Failed to get kernel from cached configurations."
              << std::endl;
    std::cerr << "Deleting broken cache file and try tuning again..."
              << std::endl;
    string bakFile = outputFile + ".bak";
    std::rename(outputFile.c_str(), bakFile.c_str());
    return;
  }
    cachedKernel >> kernelQueue[kernel_index_]->global_work_size[0];
    cachedKernel >> kernelQueue[kernel_index_]->global_work_size[1];
    cachedKernel >> kernelQueue[kernel_index_]->global_work_size[2];
    cachedKernel >> kernelQueue[kernel_index_]->local_work_size[0];
    cachedKernel >> kernelQueue[kernel_index_]->local_work_size[1];
    cachedKernel >> kernelQueue[kernel_index_]->local_work_size[2];
    cachedKernel >> kernelQueue[kernel_index_]->swizzle_weights;
    cachedKernel >> kernelQueue[kernel_index_]->batched_execute;
    cachedKernel >> kernelQueue[kernel_index_]->use_null_local;

    tuned_ = true;
  }
  return;
}

template<typename Dtype>
void ConvolutionLayerSpatial<Dtype>::SetUp(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top,
    caffe::Backend backend) {
  if (backend == caffe::BACKEND_OpenCL) {
    load_cached_kernels(bottom, top);
  }
}

template<>
bool ConvolutionLayerSpatial<double>::generate_kernel(
    const vector<Blob<double>*>& bottom, const vector<Blob<double>*>& top,
    int_tp blockWidth,
    int_tp blockHeight, int_tp blockDepth) {
  NOT_IMPLEMENTED;
  return false;
}

template void ConvolutionLayerSpatial<float>::SetUp(
    const vector<Blob<float>*>& bottom, const vector<Blob<float>*>& top,
    caffe::Backend backend);

template void ConvolutionLayerSpatial<double>::SetUp(
    const vector<Blob<double>*>& bottom, const vector<Blob<double>*>& top,
    caffe::Backend backend);

template void ConvolutionLayerSpatial<float>::swizzleWeights(
    const vector<Blob<float>*>& bottom,
    const vector<Blob<float>*>& top,
    int_tp swizzle_factor);
template void ConvolutionLayerSpatial<double>::swizzleWeights(
    const vector<Blob<double>*>& bottom,
    const vector<Blob<double>*>& top,
    int_tp swizzle_factor);
template void ConvolutionLayerSpatial<float>::pad_image(
    const vector<Blob<float>*>& bottom,
    const vector<Blob<float>*>& top,
    int_tp image_offset, kernelConfig* config,
    int_tp imgNum);
template void ConvolutionLayerSpatial<double>::pad_image(
    const vector<Blob<double>*>& bottom,
    const vector<Blob<double>*>& top,
    int_tp image_offset, kernelConfig* config,
    int_tp imgNum);

template<>
void ConvolutionLayerSpatial<double>::create_convolution_kernel(
    const vector<Blob<double>*>& bottom, const vector<Blob<double>*>& top,
    int_tp kernelType,
    int_tp blockWidth, int_tp blockHeight,
    int_tp blockDepth) {
  NOT_IMPLEMENTED;
  return;
}
template<>
bool ConvolutionLayerSpatial<double>::generate_batched_kernel(
    const vector<Blob<double>*>& bottom, const vector<Blob<double>*>& top,
    int_tp blockWidth,
    int_tp blockHeight, int_tp blockDepth) {
  NOT_IMPLEMENTED;
  return false;
}
template<>
bool ConvolutionLayerSpatial<double>::setup_IDLF(
    const vector<Blob<double>*>& bottom, const vector<Blob<double>*>& top,
    int_tp blockWidth,
    int_tp blockHeight, int_tp blockDepth) {
  NOT_IMPLEMENTED;
  return false;
}

template<>
bool ConvolutionLayerSpatial<double>::verify_result(
    const vector<Blob<double>*>& bottom, const vector<Blob<double>*>& top,
    int_tp index,
    int_tp numImages, const Blob<double> &verify_blob, kernelConfig* config) {
  NOT_IMPLEMENTED;
  return false;
}

template<>
bool ConvolutionLayerSpatial<double>::create_basic_kernel(
    const vector<Blob<double>*>& bottom, const vector<Blob<double>*>& top,
    int_tp blockWidth,
    int_tp blockHeight, int_tp blockDepth) {
  NOT_IMPLEMENTED;
  return false;
}

template<>
bool ConvolutionLayerSpatial<double>::tune_local_size(
    const vector<Blob<double>*>& bottom, const vector<Blob<double>*>& top,
    kernelConfig* config) {
  NOT_IMPLEMENTED;
  return false;
}

template<>
cl_int ConvolutionLayerSpatial<double>::convolve(
    const vector<Blob<double>*>& bottom, const vector<Blob<double>*>& top,
    int_tp index,
    int_tp numImages, kernelConfig* config) {
  NOT_IMPLEMENTED;
  return false;
}

template<>
cl_int ConvolutionLayerSpatial<double>::batched_convolve(
    const vector<Blob<double>*>& bottom, const vector<Blob<double>*>& top,
    int_tp index,
    int_tp numImages, kernelConfig* config) {
  NOT_IMPLEMENTED;
  return 0;
}

template<>
float ConvolutionLayerSpatial<double>::timed_convolve(
    const vector<Blob<double>*>& bottom, const vector<Blob<double>*>& top,
    int_tp index,
    int_tp numImages, kernelConfig* config) {
  NOT_IMPLEMENTED;
  return 0.f;
}

template<>
void ConvolutionLayerSpatial<double>::setup_convolution(
    const vector<Blob<double>*>& bottom, const vector<Blob<double>*>& top,
    const Blob<double> &verify_blob) {
  NOT_IMPLEMENTED;
}

template<>
void ConvolutionLayerSpatial<double>::calculate_global_size(
    int_tp batch,
    int_tp* workItemOutput,
    size_t* localSizes, size_t* globalSizes) {
  NOT_IMPLEMENTED;
}

template<>
void ConvolutionLayerSpatial<double>::generate_key() {
  NOT_IMPLEMENTED;
}
template<>
std::string ConvolutionLayerSpatial<double>::generate_unique_key() {
  NOT_IMPLEMENTED;
  return "";
}

template<>
std::string ConvolutionLayerSpatial<double>::generate_specific_key(
    int_tp type, int_tp blockWidth, int_tp blockHeight, int_tp blockDepth) {
  NOT_IMPLEMENTED;
  return "";
}

template<>
void ConvolutionLayerSpatial<double>::Forward_gpu(
    const vector<Blob<double>*>& bottom, const vector<Blob<double>*>& top) {
  NOT_IMPLEMENTED;
}

template<>
void ConvolutionLayerSpatial<double>::Backward_gpu(
    const vector<Blob<double>*>& top, const vector<bool>& propagate_down,
    const vector<Blob<double>*>& bottom) {
  NOT_IMPLEMENTED;
}
#else
template<typename Dtype>
void ConvolutionLayerSpatial<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  NOT_IMPLEMENTED;
}

template<typename Dtype>
void ConvolutionLayerSpatial<Dtype>::Backward_gpu(
    const vector<Blob<Dtype>*>& top, const vector<bool>& propagate_down,
    const vector<Blob<Dtype>*>& bottom) {
  NOT_IMPLEMENTED;
}
#endif
INSTANTIATE_LAYER_GPU_FUNCS(ConvolutionLayerSpatial);
#endif

}  // namespace caffe
