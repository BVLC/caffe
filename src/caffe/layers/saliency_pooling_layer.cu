#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>
#include <string>

#include <unistd.h>
#include <stdio.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>


#include "caffe/layers/saliency_pooling_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SalPoolForward_SaliencyWeighting(const int nthreads,
    const Dtype* const image_data, const Dtype* const saliency_data,
    const int num, const int channels, const int height, const int width,
    const int pooled_height, const int pooled_width, const int kernel_h,
    const int kernel_w, const int stride_h, const int stride_w, const int pad_h,
    const int pad_w, Dtype* const top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int pw = index % pooled_width;
    const int ph = (index / pooled_width) % pooled_height;
    const int c = (index / pooled_width / pooled_height) % channels;
    const int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h - pad_h;
    int wstart = pw * stride_w - pad_w;
    int hend = min(hstart + kernel_h, height + pad_h);
    int wend = min(wstart + kernel_w, width + pad_w);
    //const int pool_size = (hend - hstart) * (wend - wstart);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    hend = min(hend, height);
    wend = min(wend, width);
    float aveval = 0;
    float salval = 0;
    const Dtype* const bottom_slice = image_data + (n * channels + c) * height * width;
    const Dtype* const saliency_bottom_slice = saliency_data + (n * channels + c) * height * width;

    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        aveval += bottom_slice[h * width + w] * saliency_bottom_slice[h * width + w];
        //aveval += bottom_slice[h * width + w];
        salval += saliency_bottom_slice[h * width + w];
      }
    }
    //printf("Index:%d \t Eval:%f \t Salval:%f\n", index, aveval, salval);
    if (salval == 0) {
      top_data[index] = 0;
    } else {
      top_data[index] = aveval / salval;
    }
  }
}

template <typename Dtype>
__global__ void SalPoolForward_Random_Sampling(const int nthreads,
  const Dtype* const image_data, const Dtype* const saliency_data,
  const int num, float*numbers, const int channels, const int height, const int width,
  const int pooled_height, const int pooled_width, const int kernel_h,
  const int kernel_w, const int stride_h, const int stride_w, const int pad_h,
  const int pad_w, Dtype* const top_data) {
CUDA_KERNEL_LOOP(index, nthreads) {
  const int pw = index % pooled_width;
  const int ph = (index / pooled_width) % pooled_height;
  const int c = (index / pooled_width / pooled_height) % channels;
  const int n = index / pooled_width / pooled_height / channels;
  int hstart = ph * stride_h - pad_h;
  int wstart = pw * stride_w - pad_w;
  int hend = min(hstart + kernel_h, height + pad_h);
  int wend = min(wstart + kernel_w, width + pad_w);
  //const int pool_size = (hend - hstart) * (wend - wstart);
  hstart = max(hstart, 0);
  wstart = max(wstart, 0);
  hend = min(hend, height);
  wend = min(wend, width);

  const Dtype* const bottom_slice = image_data + (n * channels + c) * height * width;
  const Dtype* const saliency_bottom_slice = saliency_data + (n * channels + c) * height * width;

  // Weibull distribution
  float lambda = 0.5;
  float k = 4.0;

  float Ps = lambda * pow(-log(1-numbers[index]), (1/k));

  // Saliency value at index position
  float salval = saliency_bottom_slice[((hend-hstart)/2)+hstart * width + ((wend-wstart)/2)+wstart];

  if (Ps < salval){
    // Compute MaxPooling
    float maxval = -FLT_MAX;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        if (bottom_slice[h * width + w] > maxval){
          maxval = bottom_slice[h * width + w];
        }
      }
    }
    top_data[index] = maxval;
    top_data[index] = 1.0;
    //printf("Index:%d \t MaxVal:%f \t Salval:%d, Rand:%f\t (Max)\n", index, maxval, salval, Ps);
  }
  else{
    // Compute min val
    float minval = FLT_MAX;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        if (bottom_slice[h * width + w] < minval){
          minval = bottom_slice[h * width + w];
        }
      }
    }
    top_data[index] = minval;
    top_data[index] = 0.0;
    //printf("Index:%d \t AveVal:%f \t Salval:%f, Rand:%f\t (Average)\n", index, minval, salval, Ps);
    }
  }
}

template <typename Dtype>
__global__ void SalPoolForward_Random_Sampling_Weighting(const int nthreads,
  const Dtype* const image_data, const Dtype* const saliency_data,
  const int num, float* numbers, const int channels, const int height, const int width,
  const int pooled_height, const int pooled_width, const int kernel_h,
  const int kernel_w, const int stride_h, const int stride_w, const int pad_h,
  const int pad_w, Dtype* const top_data) {
CUDA_KERNEL_LOOP(index, nthreads) {
  const int pw = index % pooled_width;
  const int ph = (index / pooled_width) % pooled_height;
  const int c = (index / pooled_width / pooled_height) % channels;
  const int n = index / pooled_width / pooled_height / channels;
  int hstart = ph * stride_h - pad_h;
  int wstart = pw * stride_w - pad_w;
  int hend = min(hstart + kernel_h, height + pad_h);
  int wend = min(wstart + kernel_w, width + pad_w);
  //const int pool_size = (hend - hstart) * (wend - wstart);
  hstart = max(hstart, 0);
  wstart = max(wstart, 0);
  hend = min(hend, height);
  wend = min(wend, width);

  const Dtype* const bottom_slice = image_data + (n * channels + c) * height * width;
  const Dtype* const saliency_bottom_slice = saliency_data + (n * channels + c) * height * width;

  // Weibull distribution
  float lambda = 0.5;
  float k = 4.0;

  float Ps = lambda * pow(-log(1-numbers[index]), (1/k));

  // Saliency value at index position
  float salval = saliency_bottom_slice[((hend-hstart)/2)+hstart * width + ((wend-wstart)/2)+wstart];

  if (Ps <= salval){
    // Compute MaxPooling
    float maxval = -FLT_MAX;
    int maxvalidx = -1;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        if (bottom_slice[h * width + w] > maxval){
          maxval = bottom_slice[h * width + w];
          maxvalidx = h * width + w;
        }
      }
    }
    top_data[index] = maxval * saliency_bottom_slice[maxvalidx];
    //top_data[index] = 1.0;
    //printf("Index:%d \t MaxVal:%f \t Salval:%d, Rand:%f\t (Max)\n", index, maxval, salval, Ps);
  }
  else{
    // Compute min val
    float minval = FLT_MAX;
    int minvalidx = -1;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        if (bottom_slice[h * width + w] < minval){
          minval = bottom_slice[h * width + w];
          minvalidx = h * width + w;
        }
      }
    }
    top_data[index] = minval * saliency_bottom_slice[minvalidx];
    //top_data[index] = 0.0;
    //printf("Index:%d \t AveVal:%f \t Salval:%f, Rand:%f\t (Average)\n", index, minval, salval, Ps);
    }
  }
}

/* this GPU kernel function is used to initialize the random states */
__global__ void init(unsigned int seed, hiprandState_t* states) {

  /* we have to initialize the state */
  hiprand_init(seed, /* the seed can be the same for each core, here we pass the time in from the CPU */
              blockIdx.x, /* the sequence number should be different for each core (unless you want all
                             cores to get the same sequence of numbers for some reason - use thread id! */
              0, /* the offset is how much extra we advance in the sequence for each call, can be 0 */
              &states[blockIdx.x]);
}

/* this GPU kernel takes an array of states, and an array of ints, and puts a random int into each */
__global__ void randoms(hiprandState_t* states, float* numbers) {
  /* hiprand works like rand - except that it takes a state as a parameter */
  numbers[blockIdx.x] = hiprand_uniform(&states[blockIdx.x]);
}

template <typename Dtype>
void SaliencyPoolingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
  const vector<Blob<Dtype>*>& top){
    const Dtype* image_data = bottom[0]->gpu_data();
    const Dtype* saliency_data = bottom[1]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();
    int count = top[0]->count();
    hipDeviceSynchronize();

    /* Random numbers stuff */
    hiprandState_t* states;
    /* allocate space on the GPU for the random states */
    hipMalloc(&states, count * sizeof(hiprandState_t));
    /* invoke the GPU to initialize all of the random states */
    init<<<count, 1>>>(time(0), states);

    /* allocate an array of unsigned ints on the GPU */
    float* gpu_nums;
    float cpu_nums[count];
    hipMalloc(&gpu_nums, count * sizeof(unsigned int));
    /* invoke the kernel to get some random numbers */
    randoms<<<count, 1>>>(states, gpu_nums);
    /* copy the random numbers back */
    //hipMemcpy(cpu_nums, gpu_nums, count * sizeof(unsigned int), hipMemcpyDeviceToHost);


    int PoolingMethod = 1;
    // 0 = Saliency Weighting     (SAL: SalWeighting,           NON-SAL: Zero)
    // 1 = RandomSampling         (SAL: MaxPooling,             NON-SAL: Mean value) - Using Ps (Weibull distribution)
    // 2 = MaxPooling*Weighting   (SAL: MaxValue*SalientValue   NON-SAL: Mean value)

    switch (PoolingMethod) {
      case 0:
        // CUDA Routine for SalPoolForward_SaliencyWeighting
        SalPoolForward_SaliencyWeighting<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
          count, image_data, saliency_data, bottom[0]->num(), channels_,
          height_, width_, pooled_height_, pooled_width_, kernel_h_,
          kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, top_data);
      break;
      case 1:
        // CUDA Routine for SalPoolForward_Random_Sampling
        SalPoolForward_Random_Sampling<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
          count, image_data, saliency_data, bottom[0]->num(), gpu_nums, channels_,
          height_, width_, pooled_height_, pooled_width_, kernel_h_,
          kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, top_data);
      break;
      case 2:
        // CUDA Routine for SalPoolForward_Random_Sampling_Weighting
        SalPoolForward_Random_Sampling_Weighting<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
          count, image_data, saliency_data, bottom[0]->num(), gpu_nums, channels_,
          height_, width_, pooled_height_, pooled_width_, kernel_h_,
          kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, top_data);
        break;
      default:
        LOG(FATAL) << "Unknown pooling method.";
    }
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void SalPoolBackward(const int nthreads, const Dtype* const top_diff,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, const int pad_h, const int pad_w,
    Dtype* const bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int w = index % width + pad_w;
    const int h = (index / width) % height + pad_h;
    const int c = (index / width / height) % channels;
    const int n = index / width / height / channels;
    const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int phend = min(h / stride_h + 1, pooled_height);
    const int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    const int pwend = min(w / stride_w + 1, pooled_width);
    Dtype gradient = 0;
    const Dtype* const top_diff_slice =
        top_diff + (n * channels + c) * pooled_height * pooled_width;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        // figure out the pooling size
        int hstart = ph * stride_h - pad_h;
        int wstart = pw * stride_w - pad_w;
        int hend = min(hstart + kernel_h, height + pad_h);
        int wend = min(wstart + kernel_w, width + pad_w);
        int pool_size = (hend - hstart) * (wend - wstart);
        gradient += top_diff_slice[ph * pooled_width + pw] / pool_size;
      }
    }
    bottom_diff[index] = gradient;
  }
}

template <typename Dtype>
void SaliencyPoolingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const int count = bottom[0]->count();
  caffe_gpu_set(count, Dtype(0.), bottom_diff);

  // CUDA Routine
  SalPoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
    count, top_diff, top[0]->num(), channels_,
    height_, width_, pooled_height_, pooled_width_, kernel_h_,
    kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_LAYER_GPU_FUNCS(SaliencyPoolingLayer);

}  // namespace caffe
