#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>
#include <string>

#include <unistd.h>
#include <stdio.h>
#include "caffe/blob.hpp"

#include "caffe/layers/saliency_pooling_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SalPoolForward_SaliencyWeighting(const int nthreads,
    const Dtype* const image_data, const Dtype* const saliency_data,
    const int num, const int channels, const int height, const int width,
    const int pooled_height, const int pooled_width, const int kernel_h,
    const int kernel_w, const int stride_h, const int stride_w, const int pad_h,
    const int pad_w, Dtype* const top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int pw = index % pooled_width;
    const int ph = (index / pooled_width) % pooled_height;
    const int c = (index / pooled_width / pooled_height) % channels;
    const int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h - pad_h;
    int wstart = pw * stride_w - pad_w;
    int hend = min(hstart + kernel_h, height + pad_h);
    int wend = min(wstart + kernel_w, width + pad_w);
    //const int pool_size = (hend - hstart) * (wend - wstart);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    hend = min(hend, height);
    wend = min(wend, width);
    float aveval = 0;
    float salval = 0;
    const Dtype* const bottom_slice = image_data + (n * channels + c) * height * width;
    const Dtype* const saliency_bottom_slice = saliency_data + (n * channels + c) * height * width;

    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        aveval += bottom_slice[h * width + w] * saliency_bottom_slice[h * width + w];
        //aveval += bottom_slice[h * width + w];
        salval += saliency_bottom_slice[h * width + w];
      }
    }
    //printf("Index:%d \t Eval:%f \t Salval:%f\n", index, aveval, salval);
    if (salval == 0) {
      top_data[index] = 0;
    } else {
      top_data[index] = aveval / salval;
    }
  }
}

template <typename Dtype>
__global__ void SalPoolForward_Random_Sampling(const int nthreads,
  const Dtype* const image_data, const Dtype* const saliency_data,
  const int num, float* numbers, const int channels, const int height, const int width,
  const int pooled_height, const int pooled_width, const int kernel_h,
  const int kernel_w, const int stride_h, const int stride_w, const int pad_h,
  const int pad_w, Dtype* const top_data) {
CUDA_KERNEL_LOOP(index, nthreads) {
  const int pw = index % pooled_width;
  const int ph = (index / pooled_width) % pooled_height;
  const int c = (index / pooled_width / pooled_height) % channels;
  const int n = index / pooled_width / pooled_height / channels;
  int hstart = ph * stride_h - pad_h;
  int wstart = pw * stride_w - pad_w;
  int hend = min(hstart + kernel_h, height + pad_h);
  int wend = min(wstart + kernel_w, width + pad_w);
  //const int pool_size = (hend - hstart) * (wend - wstart);
  hstart = max(hstart, 0);
  wstart = max(wstart, 0);
  hend = min(hend, height);
  wend = min(wend, width);

  const Dtype* const bottom_slice = image_data + (n * channels + c) * height * width;
  const Dtype* const saliency_bottom_slice = saliency_data + (n * channels + c) * height * width;

  // Weibull distribution
  float lambda = 0.5;
  float k = 4.0;

  float Ps = lambda * pow(-log(1-numbers[index]), (1/k));

  // Saliency value at index position
  float salval = saliency_bottom_slice[((hend-hstart)/2)+hstart * width + ((wend-wstart)/2)+wstart];

  if (Ps < salval){
    // Compute MaxPooling
    float maxval = -FLT_MAX;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        if (bottom_slice[h * width + w] > maxval){
          maxval = bottom_slice[h * width + w];
        }
      }
    }
    top_data[index] = maxval;
    //top_data[index] = 1.0;
    //printf("Index:%d \t MaxVal:%f \t Salval:%d, Rand:%f\t (Max)\n", index, maxval, salval, Ps);
  }
  else{
    // Compute min val
    float minval = FLT_MAX;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        if (bottom_slice[h * width + w] < minval){
          minval = bottom_slice[h * width + w];
        }
      }
    }
    top_data[index] = minval;
    //top_data[index] = 0.0;
    //printf("Index:%d \t AveVal:%f \t Salval:%f, Rand:%f\t (Average)\n", index, minval, salval, Ps);
    }
  }
}

template <typename Dtype>
__global__ void SalPoolForward_Random_Sampling_Weighting(const int nthreads,
  const Dtype* const image_data, const Dtype* const saliency_data,
  const int num, float* numbers, const int channels, const int height, const int width,
  const int pooled_height, const int pooled_width, const int kernel_h,
  const int kernel_w, const int stride_h, const int stride_w, const int pad_h,
  const int pad_w, Dtype* const top_data) {
CUDA_KERNEL_LOOP(index, nthreads) {
  const int pw = index % pooled_width;
  const int ph = (index / pooled_width) % pooled_height;
  const int c = (index / pooled_width / pooled_height) % channels;
  const int n = index / pooled_width / pooled_height / channels;
  int hstart = ph * stride_h - pad_h;
  int wstart = pw * stride_w - pad_w;
  int hend = min(hstart + kernel_h, height + pad_h);
  int wend = min(wstart + kernel_w, width + pad_w);
  //const int pool_size = (hend - hstart) * (wend - wstart);
  hstart = max(hstart, 0);
  wstart = max(wstart, 0);
  hend = min(hend, height);
  wend = min(wend, width);

  const Dtype* const bottom_slice = image_data + (n * channels + c) * height * width;
  const Dtype* const saliency_bottom_slice = saliency_data + (n * channels + c) * height * width;

  // Weibull distribution
  float lambda = 0.5;
  float k = 4.0;

  float Ps = lambda * pow(-log(1-numbers[index]), (1/k));

  // Saliency value at index position
  float salval = saliency_bottom_slice[((hend-hstart)/2)+hstart * width + ((wend-wstart)/2)+wstart];

  if (Ps <= salval){
    // Compute MaxPooling
    float maxval = -FLT_MAX;
    int maxvalidx = -1;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        if (bottom_slice[h * width + w] > maxval){
          maxval = bottom_slice[h * width + w];
          maxvalidx = h * width + w;
        }
      }
    }
    top_data[index] = maxval * saliency_bottom_slice[maxvalidx];
    //top_data[index] = 1.0;
    //printf("Index:%d \t MaxVal:%f \t Salval:%d, Rand:%f\t (Max)\n", index, maxval, salval, Ps);
  }
  else{
    // Compute min val
    float minval = FLT_MAX;
    int minvalidx = -1;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        if (bottom_slice[h * width + w] < minval){
          minval = bottom_slice[h * width + w];
          minvalidx = h * width + w;
        }
      }
    }
    top_data[index] = minval * saliency_bottom_slice[minvalidx];
    //top_data[index] = 0.0;
    //printf("Index:%d \t AveVal:%f \t Salval:%f, Rand:%f\t (Average)\n", index, minval, salval, Ps);
    }
  }
}

template <typename Dtype>
__global__ void AveragePoolForward(const int nthreads,
    const Dtype* const image_data, const int num, const int channels,
    const int height, const int width, const int pooled_height,
    const int pooled_width, const int kernel_h, const int kernel_w,
    const int stride_h, const int stride_w, const int pad_h, const int pad_w,
    Dtype* const top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int pw = index % pooled_width;
    const int ph = (index / pooled_width) % pooled_height;
    const int c = (index / pooled_width / pooled_height) % channels;
    const int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h - pad_h;
    int wstart = pw * stride_w - pad_w;
    int hend = min(hstart + kernel_h, height + pad_h);
    int wend = min(wstart + kernel_w, width + pad_w);
    const int pool_size = (hend - hstart) * (wend - wstart);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    hend = min(hend, height);
    wend = min(wend, width);
    Dtype aveval = 0;
    const Dtype* const bottom_slice =
        image_data + (n * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        aveval += bottom_slice[h * width + w];
      }
    }
    top_data[index] = aveval / pool_size;
  }
}

template <typename Dtype>
void SaliencyPoolingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
  const vector<Blob<Dtype>*>& top){
    const Dtype* image_data = bottom[0]->gpu_data();
    const Dtype* saliency_data = bottom[1]->gpu_data();
    Dtype* top_data = top[0]->mutable_gpu_data();
    int count = top[0]->count();

    // Generate random numbers
    const float lower = 0.0;
    const float upper = 1.0;
    Blob<float>* rands = new Blob<float>(bottom[0]->num(), bottom[0]->channels(), bottom[0]->height(), bottom[0]->width());
    caffe_gpu_rng_uniform(bottom[0]->count(), lower, upper, rands->mutable_gpu_data());

    int PoolingMethod = 1;
    // 0 = Saliency Weighting     (SAL: SalWeighting,                         NON-SAL: Zero)
    // 1 = RandomSampling         (SAL: MaxPooling,                           NON-SAL: Mean value) - Using Ps (Weibull distribution)
    // 2 = MaxPooling*Weighting   (SAL: MaxValue*SalientValue                 NON-SAL: Mean value)
    // 3 = Average Pooling        (Average Pooling for each value)

    switch (PoolingMethod) {
      case 0:
        // CUDA Routine for SalPoolForward_SaliencyWeighting
        SalPoolForward_SaliencyWeighting<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
          count, image_data, saliency_data, bottom[0]->num(), channels_,
          height_, width_, pooled_height_, pooled_width_, kernel_h_,
          kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, top_data);
      break;
      case 1:
        // CUDA Routine for SalPoolForward_Random_Sampling
        SalPoolForward_Random_Sampling<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
          count, image_data, saliency_data, bottom[0]->num(), rands->mutable_gpu_data(), channels_,
          height_, width_, pooled_height_, pooled_width_, kernel_h_,
          kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, top_data);
      break;
      case 2:
        // CUDA Routine for SalPoolForward_Random_Sampling_Weighting
        SalPoolForward_Random_Sampling_Weighting<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
          count, image_data, saliency_data, bottom[0]->num(), rands->mutable_gpu_data(), channels_,
          height_, width_, pooled_height_, pooled_width_, kernel_h_,
          kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, top_data);
      break;
      case 3:
      // CUDA Routine for Average Pooling
        AveragePoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
          count, image_data, bottom[0]->num(), channels_,
          height_, width_, pooled_height_, pooled_width_, kernel_h_,
          kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, top_data);
      break;
      default:
        LOG(FATAL) << "Unknown pooling method.";
    }
  CUDA_POST_KERNEL_CHECK;
  delete rands;
}

template <typename Dtype>
__global__ void SalPoolBackward(const int nthreads, const Dtype* const top_diff,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, const int pad_h, const int pad_w,
    Dtype* const bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int w = index % width + pad_w;
    const int h = (index / width) % height + pad_h;
    const int c = (index / width / height) % channels;
    const int n = index / width / height / channels;
    const int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int phend = min(h / stride_h + 1, pooled_height);
    const int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    const int pwend = min(w / stride_w + 1, pooled_width);
    Dtype gradient = 0;
    const Dtype* const top_diff_slice =
        top_diff + (n * channels + c) * pooled_height * pooled_width;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        // figure out the pooling size
        int hstart = ph * stride_h - pad_h;
        int wstart = pw * stride_w - pad_w;
        int hend = min(hstart + kernel_h, height + pad_h);
        int wend = min(wstart + kernel_w, width + pad_w);
        int pool_size = (hend - hstart) * (wend - wstart);
        gradient += top_diff_slice[ph * pooled_width + pw] / pool_size;
      }
    }
    bottom_diff[index] = gradient;
  }
}

template <typename Dtype>
void SaliencyPoolingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  return;
  /*
  if (!propagate_down[0]) {
    return;
  }
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const int count = bottom[0]->count();
  caffe_gpu_set(count, Dtype(0.), bottom_diff);

  // CUDA Routine
  SalPoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
    count, top_diff, top[0]->num(), channels_,
    height_, width_, pooled_height_, pooled_width_, kernel_h_,
    kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, bottom_diff);
    CUDA_POST_KERNEL_CHECK;
  */
}

INSTANTIATE_LAYER_GPU_FUNCS(SaliencyPoolingLayer);

}  // namespace caffe
