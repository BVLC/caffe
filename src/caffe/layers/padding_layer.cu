#include "hip/hip_runtime.h"
// Copyright 2013 Yangqing Jia

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"

#include <iostream>

namespace caffe {

template <typename Dtype>
void PaddingLayer<Dtype>::SetUp(const vector<Blob<Dtype>*>& bottom,
      vector<Blob<Dtype>*>* top) {
  PAD_ = this->layer_param_.pad();
  CHECK_EQ(bottom.size(), 1) << "Padding Layer takes a single blob as input.";
  CHECK_EQ(top->size(), 1) << "Padding Layer takes a single blob as output.";
  NUM_ = bottom[0]->num();
  CHANNEL_ = bottom[0]->channels();
  HEIGHT_IN_ = bottom[0]->height();
  WIDTH_IN_ = bottom[0]->width();
  HEIGHT_OUT_ = HEIGHT_IN_ + PAD_ * 2;
  WIDTH_OUT_ = WIDTH_IN_ + PAD_ * 2;
  (*top)[0]->Reshape(NUM_, CHANNEL_, HEIGHT_OUT_, WIDTH_OUT_);

};

template <typename Dtype>
void PaddingLayer<Dtype>::Forward_cpu(const vector<Blob<Dtype>*>& bottom,
      vector<Blob<Dtype>*>* top) {
  Dtype* top_data = (*top)[0]->mutable_cpu_data();
  const Dtype* bottom_data = bottom[0]->cpu_data();
  memset(top_data, 0, sizeof(Dtype) * (*top)[0]->count());
  // In short, top[n, c, h, w] = bottom[n, c, h-pad, w-pad] if in range
  for (int n = 0; n < NUM_; ++n) {
    for (int c = 0; c < CHANNEL_; ++c) {
      for (int h = 0; h < HEIGHT_IN_; ++h) {
        // copy the width part
        memcpy(
            top_data + ((n * CHANNEL_ + c) * HEIGHT_OUT_ + h + PAD_)
                * WIDTH_OUT_ + PAD_,
            bottom_data + ((n * CHANNEL_ + c) * HEIGHT_IN_ + h) * WIDTH_IN_,
            sizeof(Dtype) * WIDTH_IN_);
      }
    }
  }
}

template <typename Dtype>
Dtype PaddingLayer<Dtype>::Backward_cpu(const vector<Blob<Dtype>*>& top,
      const bool propagate_down, vector<Blob<Dtype>*>* bottom) {
  const Dtype* top_diff = top[0]->cpu_diff();
  Dtype* bottom_diff = (*bottom)[0]->mutable_cpu_diff();
  //memset(bottom_data, 0, sizeof(Dtype) * (*bottom)[0]->count());
  for (int n = 0; n < NUM_; ++n) {
    for (int c = 0; c < CHANNEL_; ++c) {
      for (int h = 0; h < HEIGHT_IN_; ++h) {
        // copy the width part
        memcpy(
            bottom_diff + ((n * CHANNEL_ + c) * HEIGHT_IN_ + h) * WIDTH_IN_,
            top_diff + ((n * CHANNEL_ + c) * HEIGHT_OUT_ + h + PAD_)
                * WIDTH_OUT_ + PAD_,
            sizeof(Dtype) * WIDTH_IN_);
      }
    }
  }
  return Dtype(0.);
}

template <typename Dtype>
__global__ void PaddingForward(const int count, const Dtype* in, Dtype* out,
    const int num, const int channel, const int height_in, const int width_in,
    const int pad) {
  CUDA_KERNEL_LOOP(index, count) {
    int height_out = height_in + pad + pad;
    int width_out = width_in + pad + pad;
    int w = index % width_in;
    index /= width_in;
    int h = index % height_in;
    index /= height_in;
    int c = index % channel;
    index /= channel;
    out[((index * channel + c) * height_out + h + pad) * width_out + pad + w] =
        in[((index * channel + c) * height_in + h) * width_in + w];
  }  
}

template <typename Dtype>
void PaddingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    vector<Blob<Dtype>*>* top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = (*top)[0]->mutable_gpu_data();
  const int count = bottom[0]->count();
  // First, set all data to be zero for the boundary pixels
  CUDA_CHECK(hipMemset(top_data, 0, sizeof(Dtype) * (*top)[0]->count()));
  PaddingForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, bottom_data, top_data, NUM_, CHANNEL_, HEIGHT_IN_, WIDTH_IN_,
      PAD_);
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void PaddingBackward(const int count, const Dtype* in, Dtype* out,
    const int num, const int channel, const int height_in, const int width_in,
    const int pad) {
  CUDA_KERNEL_LOOP(index, count) {
    int height_out = height_in + pad + pad;
    int width_out = width_in + pad + pad;
    int w = index % width_in;
    index /= width_in;
    int h = index % height_in;
    index /= height_in;
    int c = index % channel;
    index /= channel;
    out[((index * channel + c) * height_in + h) * width_in + w] =
        in[((index * channel + c) * height_out + h + pad) * width_out + pad + w];	
  }
}

template <typename Dtype>
Dtype PaddingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const bool propagate_down,
    vector<Blob<Dtype>*>* bottom) {
  if (propagate_down) {
    const Dtype* top_diff = top[0]->gpu_diff();
    Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff();
    const int count = (*bottom)[0]->count();
    PaddingBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, bottom_diff, NUM_, CHANNEL_, HEIGHT_IN_, WIDTH_IN_,
        PAD_);
    CUDA_POST_KERNEL_CHECK;
  }
  return Dtype(0);
}

INSTANTIATE_CLASS(PaddingLayer);


}  // namespace caffe
