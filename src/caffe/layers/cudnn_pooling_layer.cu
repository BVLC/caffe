
#include <hip/hip_runtime.h>
#ifdef USE_CUDNN
#include <vector>

#include "caffe/filler.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
void CuDNNPoolingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  CUDNN_CHECK(cudnnPoolingForward(handle_, pooling_desc_,
        cudnn::dataType<Dtype>::one,
        bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        top_desc_, top_data));
}

template <typename Dtype>
void CuDNNPoolingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }
  const Dtype* top_diff = top[0]->gpu_diff();
  const Dtype* top_data = top[0]->gpu_data();
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  CUDNN_CHECK(cudnnPoolingBackward(handle_, pooling_desc_,
        cudnn::dataType<Dtype>::one,
        top_desc_, top_data, top_desc_, top_diff,
        bottom_desc_, bottom_data,
        cudnn::dataType<Dtype>::zero,
        bottom_desc_, bottom_diff));
}

INSTANTIATE_LAYER_GPU_FUNCS(CuDNNPoolingLayer);

}  // namespace caffe
#endif
