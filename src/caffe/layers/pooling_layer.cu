#include "hip/hip_runtime.h"
// Copyright 2013 Yangqing Jia

#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"
#include "caffe/util/math_functions.hpp"

using std::max;
using std::min;

namespace caffe {

template <typename Dtype>
__global__ void MaxPoolForward(const int nthreads, const Dtype* bottom_data,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int ksize, const int stride, Dtype* top_data) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < nthreads) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride;
    int hend = min(hstart + ksize, height);
    int wstart = pw * stride;
    int wend = min(wstart + ksize, width);
    Dtype maxval = -FLT_MAX;
    bottom_data += (n * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        maxval = max(maxval, bottom_data[h * width + w]);
      }
    }
    top_data[index] = maxval;
  }  // (if index < nthreads)
}

template <typename Dtype>
__global__ void AvePoolForward(const int nthreads, const Dtype* bottom_data,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int ksize, const int stride, Dtype* top_data) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < nthreads) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride;
    int hend = min(hstart + ksize, height);
    int wstart = pw * stride;
    int wend = min(wstart + ksize, width);
    Dtype aveval = 0;
    bottom_data += (n * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        aveval += bottom_data[h * width + w];
      }
    }
    top_data[index] = aveval / (hend - hstart) / (wend - wstart);
  }  // (if index < nthreads)
}

template <typename Dtype>
__global__ void StoPoolForwardTrain(const int nthreads,
    const Dtype* bottom_data,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int ksize, const int stride, float* rand_idx, Dtype* top_data) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < nthreads) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride;
    int hend = min(hstart + ksize, height);
    int wstart = pw * stride;
    int wend = min(wstart + ksize, width);
    Dtype cumsum = 0.;
    bottom_data += (n * channels + c) * height * width;
    // First pass: get sum
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        cumsum += bottom_data[h * width + w];
      }
    }
    float thres = rand_idx[index] * cumsum;
    // Second pass: get value, and set index.
    cumsum = 0;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        cumsum += bottom_data[h * width + w];
        if (cumsum >= thres) {
          rand_idx[index] = ((n * channels + c) * height + h) * width + w;
          top_data[index] = bottom_data[h * width + w];
          return;
        }
      }
    }
  }  // (if index < nthreads)
}


template <typename Dtype>
__global__ void StoPoolForwardTest(const int nthreads,
    const Dtype* bottom_data,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int ksize, const int stride, Dtype* top_data) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < nthreads) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride;
    int hend = min(hstart + ksize, height);
    int wstart = pw * stride;
    int wend = min(wstart + ksize, width);
    // We set cumsum to be 0 to avoid divide-by-zero problems
    Dtype cumsum = FLT_MIN;
    Dtype cumvalues = 0.;
    bottom_data += (n * channels + c) * height * width;
    // First pass: get sum
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        cumsum += bottom_data[h * width + w];
        cumvalues += bottom_data[h * width + w] * bottom_data[h * width + w];
      }
    }
    top_data[index] = cumvalues / cumsum;
  }  // (if index < nthreads)
}


template <typename Dtype>
void PoolingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      vector<Blob<Dtype>*>* top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = (*top)[0]->mutable_gpu_data();
  int count = (*top)[0]->count();
  switch (this->layer_param_.pool()) {
  case LayerParameter_PoolMethod_MAX:
    // NOLINT_NEXTLINE(whitespace/operators)
    MaxPoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, bottom[0]->num(), CHANNELS_,
        HEIGHT_, WIDTH_, POOLED_HEIGHT_, POOLED_WIDTH_, KSIZE_, STRIDE_,
        top_data);
    break;
  case LayerParameter_PoolMethod_AVE:
    // NOLINT_NEXTLINE(whitespace/operators)
    AvePoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, bottom[0]->num(), CHANNELS_,
        HEIGHT_, WIDTH_, POOLED_HEIGHT_, POOLED_WIDTH_, KSIZE_, STRIDE_,
        top_data);
    break;
  case LayerParameter_PoolMethod_STOCHASTIC:
    if (Caffe::phase() == Caffe::TRAIN) {
      // We need to create the random index as well.
      CURAND_CHECK(hiprandGenerateUniform(Caffe::curand_generator(),
          rand_idx_.mutable_gpu_data(), count));
      // NOLINT_NEXTLINE(whitespace/operators)
      StoPoolForwardTrain<Dtype><<<CAFFE_GET_BLOCKS(count),
                                   CAFFE_CUDA_NUM_THREADS>>>(
          count, bottom_data, bottom[0]->num(), CHANNELS_,
          HEIGHT_, WIDTH_, POOLED_HEIGHT_, POOLED_WIDTH_, KSIZE_, STRIDE_,
          rand_idx_.mutable_gpu_data(), top_data);
    } else {
      // NOLINT_NEXTLINE(whitespace/operators)
      StoPoolForwardTest<Dtype><<<CAFFE_GET_BLOCKS(count),
                                  CAFFE_CUDA_NUM_THREADS>>>(
          count, bottom_data, bottom[0]->num(), CHANNELS_,
          HEIGHT_, WIDTH_, POOLED_HEIGHT_, POOLED_WIDTH_, KSIZE_, STRIDE_,
          top_data);
    }
    break;
  default:
    LOG(FATAL) << "Unknown pooling method.";
  }
  CUDA_POST_KERNEL_CHECK;
}

template <typename Dtype>
__global__ void MaxPoolBackward(const int nthreads, const Dtype* bottom_data,
    const Dtype* top_data, const Dtype* top_diff,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int ksize, const int stride, Dtype* bottom_diff) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < nthreads) {
    // find out the local index
    // find out the local offset
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;
    int phstart = (h < ksize) ? 0 : (h - ksize) / stride + 1;
    int phend = min(h / stride + 1, pooled_height);
    int pwstart = (w < ksize) ? 0 : (w - ksize) / stride + 1;
    int pwend = min(w / stride + 1, pooled_width);
    Dtype gradient = 0;
    Dtype bottom_datum =
        bottom_data[((n * channels + c) * height + h) * width + w];
    top_data += (n * channels + c) * pooled_height * pooled_width;
    top_diff += (n * channels + c) * pooled_height * pooled_width;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        gradient += top_diff[ph * pooled_width + pw] *
            (bottom_datum == top_data[ph * pooled_width + pw]);
      }
    }
    bottom_diff[index] = gradient;
  }  // (if index < nthreads)
}


template <typename Dtype>
__global__ void AvePoolBackward(const int nthreads, const Dtype* top_diff,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int ksize, const int stride, Dtype* bottom_diff) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < nthreads) {
    // find out the local index
    // find out the local offset
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;
    int phstart = (h < ksize) ? 0 : (h - ksize) / stride + 1;
    int phend = min(h / stride + 1, pooled_height);
    int pwstart = (w < ksize) ? 0 : (w - ksize) / stride + 1;
    int pwend = min(w / stride + 1, pooled_width);
    Dtype gradient = 0;
    top_diff += (n * channels + c) * pooled_height * pooled_width;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        // figure out the pooling size
        int poolsize = (min(ph * stride + ksize, height) - ph * stride) *
            (min(pw * stride + ksize, width) - pw * stride);
        gradient += top_diff[ph * pooled_width + pw] / poolsize;
      }
    }
    bottom_diff[index] = gradient;
  }  // (if index < nthreads)
}


template <typename Dtype>
__global__ void StoPoolBackward(const int nthreads,
    const float* rand_idx, const Dtype* top_diff,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int ksize, const int stride, Dtype* bottom_diff) {
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < nthreads) {
    // find out the local index
    // find out the local offset
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;
    int phstart = (h < ksize) ? 0 : (h - ksize) / stride + 1;
    int phend = min(h / stride + 1, pooled_height);
    int pwstart = (w < ksize) ? 0 : (w - ksize) / stride + 1;
    int pwend = min(w / stride + 1, pooled_width);
    Dtype gradient = 0;
    rand_idx += (n * channels + c) * pooled_height * pooled_width;
    top_diff += (n * channels + c) * pooled_height * pooled_width;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        gradient += top_diff[ph * pooled_width + pw] *
            (index == static_cast<int>(rand_idx[ph * pooled_width + pw]));
      }
    }
    bottom_diff[index] = gradient;
  }  // (if index < nthreads)
}


template <typename Dtype>
Dtype PoolingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const bool propagate_down, vector<Blob<Dtype>*>* bottom) {
  if (!propagate_down) {
    return Dtype(0.);
  }
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff();
  int count = (*bottom)[0]->count();
  switch (this->layer_param_.pool()) {
  case LayerParameter_PoolMethod_MAX:
    // NOLINT_NEXTLINE(whitespace/operators)
    MaxPoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, (*bottom)[0]->gpu_data(), top[0]->gpu_data(), top_diff,
        top[0]->num(), CHANNELS_, HEIGHT_, WIDTH_, POOLED_HEIGHT_,
        POOLED_WIDTH_, KSIZE_, STRIDE_, bottom_diff);
    break;
  case LayerParameter_PoolMethod_AVE:
    // NOLINT_NEXTLINE(whitespace/operators)
    AvePoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, top[0]->num(), CHANNELS_,
        HEIGHT_, WIDTH_, POOLED_HEIGHT_, POOLED_WIDTH_, KSIZE_, STRIDE_,
        bottom_diff);
    break;
  case LayerParameter_PoolMethod_STOCHASTIC:
    // NOLINT_NEXTLINE(whitespace/operators)
    StoPoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, rand_idx_.gpu_data(), top_diff,
        top[0]->num(), CHANNELS_, HEIGHT_, WIDTH_, POOLED_HEIGHT_,
        POOLED_WIDTH_, KSIZE_, STRIDE_, bottom_diff);
    break;
  default:
    LOG(FATAL) << "Unknown pooling method.";
  }
  CUDA_POST_KERNEL_CHECK;
  return Dtype(0.);
}


INSTANTIATE_CLASS(PoolingLayer);


}  // namespace caffe
