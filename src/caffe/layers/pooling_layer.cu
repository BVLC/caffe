#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/pooling_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void MaxPoolForward(const int nthreads,
    const Dtype* const bottom_data, const int num, const int channels,
    const int height, const int width, const int pooled_height,
    const int pooled_width, const int kernel_h, const int kernel_w,
    const int stride_h, const int stride_w, const int pad_h, const int pad_w,
    Dtype* const top_data, int* mask, Dtype* top_mask) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int pw = index % pooled_width;
    const int ph = (index / pooled_width) % pooled_height;
    const int c = (index / pooled_width / pooled_height) % channels;
    const int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h - pad_h;
    int wstart = pw * stride_w - pad_w;
    const int hend = min(hstart + kernel_h, height);
    const int wend = min(wstart + kernel_w, width);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    Dtype maxval = -FLT_MAX;
    int maxidx = -1;
    const Dtype* const bottom_slice =
        bottom_data + (n * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        if (bottom_slice[h * width + w] > maxval) {
          maxidx = h * width + w;
          maxval = bottom_slice[maxidx];
        }
      }
    }
    top_data[index] = maxval;
    if (mask) {
      mask[index] = maxidx;
    } else {
      top_mask[index] = maxidx;
    }
  }
}

template <typename Dtype>
__global__ void AvePoolForward(const int nthreads,
    const Dtype* const bottom_data, const int num, const int channels,
    const int height, const int width, const int pooled_height,
    const int pooled_width, const int kernel_h, const int kernel_w,
    const int stride_h, const int stride_w, const int pad_h, const int pad_w,
    Dtype* const top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int pw = index % pooled_width;
    const int ph = (index / pooled_width) % pooled_height;
    const int c = (index / pooled_width / pooled_height) % channels;
    const int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h - pad_h;
    int wstart = pw * stride_w - pad_w;
    int hend = min(hstart + kernel_h, height + pad_h);
    int wend = min(wstart + kernel_w, width + pad_w);
    const int pool_size = (hend - hstart) * (wend - wstart);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    hend = min(hend, height);
    wend = min(wend, width);
    Dtype aveval = 0;
    const Dtype* const bottom_slice =
        bottom_data + (n * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        aveval += bottom_slice[h * width + w];
      }
    }
    top_data[index] = aveval / pool_size;
  }
}

template <typename Dtype>
__global__ void StoPoolForwardTrain(const int nthreads,
    const Dtype* const bottom_data,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, Dtype* const rand_idx, Dtype* const top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int pw = index % pooled_width;
    const int ph = (index / pooled_width) % pooled_height;
    const int c = (index / pooled_width / pooled_height) % channels;
    const int n = index / pooled_width / pooled_height / channels;
    const int hstart = ph * stride_h;
    const int hend = min(hstart + kernel_h, height);
    const int wstart = pw * stride_w;
    const int wend = min(wstart + kernel_w, width);
    Dtype cumsum = 0.;
    const Dtype* const bottom_slice =
        bottom_data + (n * channels + c) * height * width;
    // First pass: get sum
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        cumsum += bottom_slice[h * width + w];
      }
    }
    const float thres = rand_idx[index] * cumsum;
    // Second pass: get value, and set index.
    cumsum = 0;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        cumsum += bottom_slice[h * width + w];
        if (cumsum >= thres) {
          rand_idx[index] = ((n * channels + c) * height + h) * width + w;
          top_data[index] = bottom_slice[h * width + w];
          return;
        }
      }
    }
  }
}


template <typename Dtype>
__global__ void StoPoolForwardTest(const int nthreads,
    const Dtype* const bottom_data,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, Dtype* const top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int pw = index % pooled_width;
    const int ph = (index / pooled_width) % pooled_height;
    const int c = (index / pooled_width / pooled_height) % channels;
    const int n = index / pooled_width / pooled_height / channels;
    const int hstart = ph * stride_h;
    const int hend = min(hstart + kernel_h, height);
    const int wstart = pw * stride_w;
    const int wend = min(wstart + kernel_w, width);
    // We set cumsum to be 0 to avoid divide-by-zero problems
    Dtype cumsum = 0.;
    Dtype cumvalues = 0.;
    const Dtype* const bottom_slice =
        bottom_data + (n * channels + c) * height * width;
    // First pass: get sum
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        cumsum += bottom_slice[h * width + w];
        cumvalues += bottom_slice[h * width + w] * bottom_slice[h * width + w];
      }
    }
    top_data[index] = (cumsum > 0.) ? cumvalues / cumsum : 0.;
  }
}


template <typename Dtype>
void PoolingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int count = top[0]->count();
  // We'll output the mask to top[1] if it's of size >1.
  const bool use_top_mask = top.size() > 1;
  int* mask = NULL;
  Dtype* top_mask = NULL;
  switch (this->layer_param_.pooling_param().pool()) {
  case PoolingParameter_PoolMethod_MAX:
    if (use_top_mask) {
      top_mask = top[1]->mutable_gpu_data();
    } else {
      mask = max_idx_.mutable_gpu_data();
    }
    // NOLINT_NEXT_LINE(whitespace/operators)
    MaxPoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, bottom[0]->num(), channels_,
        height_, width_, pooled_height_, pooled_width_, kernel_h_,
        kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, top_data,
        mask, top_mask);
    break;
  case PoolingParameter_PoolMethod_AVE:
    // NOLINT_NEXT_LINE(whitespace/operators)
    AvePoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, bottom[0]->num(), channels_,
        height_, width_, pooled_height_, pooled_width_, kernel_h_,
        kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, top_data);
    break;
  case PoolingParameter_PoolMethod_STOCHASTIC:
    /*
    if (this->phase_ == TRAIN) {
      // We need to create the random index as well.
      caffe_gpu_rng_uniform(count, Dtype(0), Dtype(1),
                            rand_idx_.mutable_gpu_data());
      // NOLINT_NEXT_LINE(whitespace/operators)
      StoPoolForwardTrain<Dtype><<<CAFFE_GET_BLOCKS(count),
                                   CAFFE_CUDA_NUM_THREADS>>>(
          count, bottom_data, bottom[0]->num(), channels_,
          height_, width_, pooled_height_, pooled_width_, kernel_h_,
          kernel_w_, stride_h_, stride_w_,
          rand_idx_.mutable_gpu_data(), top_data);
    } else*/ {
      // NOLINT_NEXT_LINE(whitespace/operators)
      StoPoolForwardTest<Dtype><<<CAFFE_GET_BLOCKS(count),
                                  CAFFE_CUDA_NUM_THREADS>>>(
          count, bottom_data, bottom[0]->num(), channels_,
          height_, width_, pooled_height_, pooled_width_, kernel_h_,
          kernel_w_, stride_h_, stride_w_, top_data);
    }
    break;
  default:
    LOG(FATAL) << "Unknown pooling method.";
  }
  CUDA_POST_KERNEL_CHECK;
}


INSTANTIATE_LAYER_GPU_FUNCS(PoolingLayer);


}  // namespace caffe
