#include "hip/hip_runtime.h"
// Copyright 2014 BVLC and contributors.

#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/vision_layers.hpp"
#include "caffe/util/math_functions.hpp"

using std::max;
using std::min;

namespace caffe {
 
template <typename Dtype>
__global__ void MaxPoolForward(const int nthreads, const Dtype* bottom_data,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_size, const int stride, Dtype* top_data, int* mask) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride;
    int hend = min(hstart + kernel_size, height);
    int wstart = pw * stride;
    int wend = min(wstart + kernel_size, width);
    Dtype maxval = -FLT_MAX;
    int maxidx = -1;
    bottom_data += (n * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        if (bottom_data[h * width + w] > maxval) {
          maxidx = h * width + w;
          maxval = bottom_data[maxidx];
        }
        
      }
    }
    top_data[index] = maxval;
    mask[index] = maxidx;
  }
}

template <typename Dtype>
__global__ void AvePoolForward(const int nthreads, const Dtype* bottom_data,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_size, const int stride, const int pad, Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride - pad;
    int wstart = pw * stride - pad;
    int hend = min(hstart + kernel_size, height + pad);
    int wend = min(wstart + kernel_size, width + pad);
    int pool_size = (hend - hstart) * (wend - wstart);
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    hend = min(hend, height);
    wend = min(wend, width);
    Dtype aveval = 0;
    bottom_data += (n * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        aveval += bottom_data[h * width + w];
      }
    }
    top_data[index] = aveval / pool_size;
  }
}

template <typename Dtype>
__global__ void StoPoolForwardTrain(const int nthreads,
    const Dtype* bottom_data,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_size, const int stride, Dtype* rand_idx, Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride;
    int hend = min(hstart + kernel_size, height);
    int wstart = pw * stride;
    int wend = min(wstart + kernel_size, width);
    Dtype cumsum = 0.;
    bottom_data += (n * channels + c) * height * width;
    // First pass: get sum
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        cumsum += bottom_data[h * width + w];
      }
    }
    float thres = rand_idx[index] * cumsum;
    // Second pass: get value, and set index.
    cumsum = 0;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        cumsum += bottom_data[h * width + w];
        if (cumsum >= thres) {
          rand_idx[index] = ((n * channels + c) * height + h) * width + w;
          top_data[index] = bottom_data[h * width + w];
          return;
        }
      }
    }
  }
}


template <typename Dtype>
__global__ void StoPoolForwardTest(const int nthreads,
    const Dtype* bottom_data,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_size, const int stride, Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride;
    int hend = min(hstart + kernel_size, height);
    int wstart = pw * stride;
    int wend = min(wstart + kernel_size, width);
    // We set cumsum to be 0 to avoid divide-by-zero problems
    Dtype cumsum = FLT_MIN;
    Dtype cumvalues = 0.;
    bottom_data += (n * channels + c) * height * width;
    // First pass: get sum
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        cumsum += bottom_data[h * width + w];
        cumvalues += bottom_data[h * width + w] * bottom_data[h * width + w];
      }
    }
    top_data[index] = cumvalues / cumsum;
  }
}


template <typename Dtype>
Dtype PoolingLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      vector<Blob<Dtype>*>* top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = (*top)[0]->mutable_gpu_data();
  int count = (*top)[0]->count();
  int* mask;
  switch (this->layer_param_.pooling_param().pool()) {
  case PoolingParameter_PoolMethod_MAX:
    // NOLINT_NEXT_LINE(whitespace/operators)
    mask = (int*)max_idx_->mutable_gpu_data();
    MaxPoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, bottom[0]->num(), channels_,
        height_, width_, pooled_height_, pooled_width_, kernel_size_, stride_,
        top_data, mask);
    break;
  case PoolingParameter_PoolMethod_AVE:
    // NOLINT_NEXT_LINE(whitespace/operators)
    AvePoolForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, bottom[0]->num(), channels_,
        height_, width_, pooled_height_, pooled_width_, kernel_size_, stride_,
        pad_, top_data);
    break;
  case PoolingParameter_PoolMethod_STOCHASTIC:
    if (Caffe::phase() == Caffe::TRAIN) {
      // We need to create the random index as well.
      caffe_gpu_rng_uniform(count, Dtype(0), Dtype(1),
                            rand_idx_.mutable_gpu_data());
      // NOLINT_NEXT_LINE(whitespace/operators)
      StoPoolForwardTrain<Dtype><<<CAFFE_GET_BLOCKS(count),
                                   CAFFE_CUDA_NUM_THREADS>>>(
          count, bottom_data, bottom[0]->num(), channels_,
          height_, width_, pooled_height_, pooled_width_, kernel_size_, stride_,
          rand_idx_.mutable_gpu_data(), top_data);
    } else {
      // NOLINT_NEXT_LINE(whitespace/operators)
      StoPoolForwardTest<Dtype><<<CAFFE_GET_BLOCKS(count),
                                  CAFFE_CUDA_NUM_THREADS>>>(
          count, bottom_data, bottom[0]->num(), channels_,
          height_, width_, pooled_height_, pooled_width_, kernel_size_, stride_,
          top_data);
    }
    break;
  default:
    LOG(FATAL) << "Unknown pooling method.";
  }
  CUDA_POST_KERNEL_CHECK;
  return Dtype(0.);
}


template <typename Dtype>
__global__ void MaxPoolBackward(const int nthreads, const Dtype* top_diff,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_size, const int stride, Dtype* bottom_diff, int* mask) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;
    int phstart = (h < kernel_size) ? 0 : (h - kernel_size) / stride + 1;
    int phend = min(h / stride + 1, pooled_height);
    int pwstart = (w < kernel_size) ? 0 : (w - kernel_size) / stride + 1;
    int pwend = min(w / stride + 1, pooled_width);
    Dtype gradient = 0;
    top_diff += (n * channels + c) * pooled_height * pooled_width;
    mask += (n * channels + c) * pooled_height * pooled_width;
    //bottom_diff[index] += top_diff[mask[index]];
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        if (mask[ph * pooled_width + pw] == h * width + w)
          gradient += top_diff[ph * pooled_width + pw];
      }
    }
    bottom_diff[index] = gradient;
  }  
}

/*
__device__ double atomicAdd(double* address, double val)
{
    unsigned long long int* address_as_ull =
                              (unsigned long long int*)address;
    unsigned long long int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));
    } while (assumed != old);
    return __longlong_as_double(old);
}

__device__ float atomicAdd(float* address, float val)
{
    unsigned int* address_as_ull = (unsigned int*)address;
    unsigned int old = *address_as_ull, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __float_as_int(val +
                               __int_as_float(assumed)));
    } while (assumed != old);
    return __int_as_float(old);
}

template <typename Dtype>
__global__ void MaxPoolBackward(const int nthreads, const Dtype* top_diff,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int ksize, const int stride, Dtype* bottom_diff, int* mask) {
  // __shared__ Dtype cache[CAFFE_CUDA_NUM_THREADS];
  // cache[threadIdx.x] = Dtype(0.);
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;
    bottom_diff += (n * channels + c) * height * width;
    // read from global bottom_diff into cache
    // atomicAdd(cache + mask[index],bottom_diff[mask[index]]);
    atomicAdd(bottom_diff + mask[index],top_diff[index]);
    // __syncthreads();
    // write from cache to global bottom_diff
    // bottom_diff[mask[index]] = cache[mask[index]];
  }  // (if index < nthreads)
}
*/

template <typename Dtype>
__global__ void AvePoolBackward(const int nthreads, const Dtype* top_diff,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_size, const int stride, const int pad,
    Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    int w = index % width + pad;
    int h = (index / width) % height + pad;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;
    int phstart = (h < kernel_size) ? 0 : (h - kernel_size) / stride + 1;
    int phend = min(h / stride + 1, pooled_height);
    int pwstart = (w < kernel_size) ? 0 : (w - kernel_size) / stride + 1;
    int pwend = min(w / stride + 1, pooled_width);
    Dtype gradient = 0;
    top_diff += (n * channels + c) * pooled_height * pooled_width;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        // figure out the pooling size
        int hstart = ph * stride - pad;
        int wstart = pw * stride - pad;
        int hend = min(hstart + kernel_size, height + pad);
        int wend = min(wstart + kernel_size, width + pad);
        int pool_size = (hend - hstart) * (wend - wstart);
        gradient += top_diff[ph * pooled_width + pw] / pool_size;
      }
    }
    bottom_diff[index] = gradient;
  }
}


template <typename Dtype>
__global__ void StoPoolBackward(const int nthreads,
    const Dtype* rand_idx, const Dtype* top_diff,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_size, const int stride, Dtype* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;
    int phstart = (h < kernel_size) ? 0 : (h - kernel_size) / stride + 1;
    int phend = min(h / stride + 1, pooled_height);
    int pwstart = (w < kernel_size) ? 0 : (w - kernel_size) / stride + 1;
    int pwend = min(w / stride + 1, pooled_width);
    Dtype gradient = 0;
    rand_idx += (n * channels + c) * pooled_height * pooled_width;
    top_diff += (n * channels + c) * pooled_height * pooled_width;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        gradient += top_diff[ph * pooled_width + pw] *
            (index == static_cast<int>(rand_idx[ph * pooled_width + pw]));
      }
    }
    bottom_diff[index] = gradient;
  }
}


template <typename Dtype>
void PoolingLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const bool propagate_down, vector<Blob<Dtype>*>* bottom) {
  if (!propagate_down) {
    return;
  }
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff();
  int count = (*bottom)[0]->count();
  CUDA_CHECK(hipMemset(bottom_diff, 0, sizeof(Dtype) * count));
  int* mask;
  
  switch (this->layer_param_.pooling_param().pool()) {
  case PoolingParameter_PoolMethod_MAX:
    mask = (int*)max_idx_->gpu_data();
    // Since we have the mask we only need count top_diff
    count = top[0]->count(); 
    // NOLINT_NEXT_LINE(whitespace/operators)
    caffe_gpu_set(count,Dtype(0.),bottom_diff);
    MaxPoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, top[0]->num(), channels_,
        height_, width_, pooled_height_, pooled_width_,
        kernel_size_, stride_, bottom_diff, mask);
    break;
  case PoolingParameter_PoolMethod_AVE:
    // NOLINT_NEXT_LINE(whitespace/operators)
    AvePoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, top_diff, top[0]->num(), channels_,
        height_, width_, pooled_height_, pooled_width_, kernel_size_, stride_,
        pad_, bottom_diff);
    break;
  case PoolingParameter_PoolMethod_STOCHASTIC:
    // NOLINT_NEXT_LINE(whitespace/operators)
    StoPoolBackward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, rand_idx_.gpu_data(), top_diff,
        top[0]->num(), channels_, height_, width_, pooled_height_,
        pooled_width_, kernel_size_, stride_, bottom_diff);
    break;
  default:
    LOG(FATAL) << "Unknown pooling method.";
  }
  CUDA_POST_KERNEL_CHECK;
}


INSTANTIATE_CLASS(PoolingLayer);


}  // namespace caffe
