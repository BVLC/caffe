#include <vector>

#include "caffe/data_layers.hpp"

namespace caffe {

template <typename Dtype>
void BasePrefetchingDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  // First, join the thread
  JoinPrefetchThread();
  // Reshape to loaded data.
  top[0]->ReshapeLike(this->prefetch_data_);
  // Copy the data
  caffe_copy(prefetch_data_.count(), prefetch_data_.cpu_data(),
      top[0]->mutable_gpu_data());
  if (this->output_labels_) {
    // Reshape to loaded labels.
    top[1]->ReshapeLike(prefetch_label_);
    // Copy the labels.
    caffe_copy(prefetch_label_.count(), prefetch_label_.cpu_data(),
        top[1]->mutable_gpu_data());
  }
  // Start a new prefetch thread
  CreatePrefetchThread();
}

INSTANTIATE_LAYER_GPU_FORWARD(BasePrefetchingDataLayer);

template <typename Dtype>
void BasePrefetchingMultiDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  // First, join the thread
  JoinPrefetchThread();
  // Reshape to loaded data.
  for (int data_id = 0; data_id < input_data_size_; data_id++) {
    top[data_id]->ReshapeLike(*prefetch_data_[data_id]);
    // Copy the data
    caffe_copy(
        prefetch_data_[data_id]->count(),
        prefetch_data_[data_id]->cpu_data(),
        top[data_id]->mutable_gpu_data());
    DLOG(INFO) << "Prefetch copied";
  }
  if (this->output_labels_) {
    // Reshape to loaded labels.
    top[input_data_size_]->ReshapeLike(prefetch_label_);
    // Copy the labels.
    caffe_copy(prefetch_label_.count(), prefetch_label_.cpu_data(),
               top[input_data_size_]->mutable_gpu_data());
  }
  // Start a new prefetch thread
  DLOG(INFO) << "CreatePrefetchThread";
  CreatePrefetchThread();
}

INSTANTIATE_LAYER_GPU_FORWARD(BasePrefetchingMultiDataLayer);

}  // namespace caffe
