#include <vector>

#include "caffe/layers/base_data_layer.hpp"

namespace caffe {

template <typename Dtype>
void BasePrefetchingDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  Batch<Dtype>* batch = prefetch_full_.pop("Data layer prefetch queue empty");
  for (size_t ib = 0; ib < batch->data_.size(); ib++) {
    // Reshape to loaded data.
    top[ib]->ReshapeLike(*batch->data_[ib].get());
    // Copy the data
    caffe_copy(batch->data_[ib]->count(), batch->data_[ib]->gpu_data(),
      top[ib]->mutable_gpu_data());
  }
  // Ensure the copy is synchronous wrt the host, so that the next batch isn't
  // copied in meanwhile.
  CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
  prefetch_free_.push(batch);
}

INSTANTIATE_LAYER_GPU_FORWARD(BasePrefetchingDataLayer);

}  // namespace caffe
