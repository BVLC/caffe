#include <vector>

#include "caffe/data_layers.hpp"

namespace caffe {

template <typename Dtype>
void BasePrefetchingDataLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  Batch<Dtype>* batch = prefetch_full_.pop("Data layer prefetch queue empty");
<<<<<<< HEAD
<<<<<<< HEAD
<<<<<<< HEAD
  // Reshape to loaded data.
  top[0]->ReshapeLike(batch->data_);
  // Copy the data
  caffe_copy(batch->data_.count(), batch->data_.gpu_data(),
      top[0]->mutable_gpu_data());
  if (this->output_labels_) {
    // Reshape to loaded labels.
    top[1]->ReshapeLike(batch->label_);
    // Copy the labels.
    caffe_copy(batch->label_.count(), batch->label_.gpu_data(),
        top[1]->mutable_gpu_data());
  }
  // Ensure the copy is synchronous wrt the host, so that the next batch isn't
  // copied in meanwhile.
  CUDA_CHECK(hipStreamSynchronize(hipStreamDefault));
=======

  caffe_copy(batch->data_.count(), batch->data_.gpu_data(),
      top[0]->mutable_gpu_data());
  if (this->output_labels_) {
    caffe_copy(batch->label_.count(), batch->label_.gpu_data(),
        top[1]->mutable_gpu_data());
  }

>>>>>>> origin/BVLC/parallel
=======

  caffe_copy(batch->data_.count(), batch->data_.gpu_data(),
      top[0]->mutable_gpu_data());
  if (this->output_labels_) {
    caffe_copy(batch->label_.count(), batch->label_.gpu_data(),
        top[1]->mutable_gpu_data());
  }

>>>>>>> origin/BVLC/parallel
=======

  caffe_copy(batch->data_.count(), batch->data_.gpu_data(),
      top[0]->mutable_gpu_data());
  if (this->output_labels_) {
    caffe_copy(batch->label_.count(), batch->label_.gpu_data(),
        top[1]->mutable_gpu_data());
  }

>>>>>>> origin/BVLC/parallel
  prefetch_free_.push(batch);
}

INSTANTIATE_LAYER_GPU_FORWARD(BasePrefetchingDataLayer);

}  // namespace caffe
