// Copyright 2014 BVLC and contributors.

#include <vector>

#include "caffe/common.hpp"
#include "caffe/layer.hpp"
#include "caffe/util/im2col.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
Dtype Im2colLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      vector<Blob<Dtype>*>* top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = (*top)[0]->mutable_gpu_data();
  for (int n = 0; n < bottom[0]->num(); ++n) {
    im2col_gpu(bottom_data + bottom[0]->offset(n), channels_, height_,
        width_, kernel_h_, kernel_w_, pad_h_, pad_w_,
        stride_h_, stride_w_, top_data + (*top)[0]->offset(n));
  }
  return Dtype(0.);
}

template <typename Dtype>
void Im2colLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, vector<Blob<Dtype>*>* bottom) {
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = (*bottom)[0]->mutable_gpu_diff();
  for (int n = 0; n < top[0]->num(); ++n) {
    col2im_gpu(top_diff + top[0]->offset(n), channels_, height_, width_,
        kernel_h_, kernel_w_, pad_h_, pad_w_,
        stride_h_, stride_w_, bottom_diff + (*bottom)[0]->offset(n));
  }
}


INSTANTIATE_CLASS(Im2colLayer);

}  // namespace caffe
