#include "hip/hip_runtime.h"
// ------------------------------------------------------------------
// Fast R-CNN
// Copyright (c) 2015 Microsoft
// Licensed under The MIT License [see fast-rcnn/LICENSE for details]
// Written by Ross Girshick
// ------------------------------------------------------------------

#include "caffe/layers/fast_rcnn_layers.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SmoothL1Forward(const int n, const Dtype* in, Dtype* out,
    Dtype sigma2) {
  // f(x) = 0.5 * (sigma * x)^2          if |x| < 1 / sigma / sigma
  //        |x| - 0.5 / sigma / sigma    otherwise
  CUDA_KERNEL_LOOP(index, n) {
    Dtype val = in[index];
    Dtype abs_val = abs(val);
    if (abs_val < 1.0 / sigma2) {
      out[index] = 0.5 * val * val * sigma2;
    } else {
      out[index] = abs_val - 0.5 / sigma2;
    }
  }
}

template <typename Dtype>
void SmoothL1LossLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
    const vector<Blob<Dtype>*>& top) {
  int count = bottom[0]->count();
  caffe_gpu_sub(
      count,
      bottom[0]->gpu_data(),
      bottom[1]->gpu_data(),
      diff_.mutable_gpu_data());    // d := b0 - b1
  if (has_weights_) {
    // apply "inside" weights
    caffe_gpu_mul(
        count,
        bottom[2]->gpu_data(),
        diff_.gpu_data(),
        diff_.mutable_gpu_data());  // d := w_in * (b0 - b1)
  }
  SmoothL1Forward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
      count, diff_.gpu_data(), errors_.mutable_gpu_data(), sigma2_);
  CUDA_POST_KERNEL_CHECK;

  if (has_weights_) {
    // apply "outside" weights
    caffe_gpu_mul(
        count,
        bottom[3]->gpu_data(),
        errors_.gpu_data(),
        errors_.mutable_gpu_data());  // d := w_out * SmoothL1(w_in * (b0 - b1))
  }

  Dtype loss;
  caffe_gpu_dot(count, ones_.gpu_data(), errors_.gpu_data(), &loss);
  top[0]->mutable_cpu_data()[0] = loss / bottom[0]->num();
}

template <typename Dtype>
__global__ void SmoothL1Backward(const int n, const Dtype* in, Dtype* out,
    Dtype sigma2) {
  // f'(x) = sigma * sigma * x         if |x| < 1 / sigma / sigma
  //       = sign(x)                   otherwise
  CUDA_KERNEL_LOOP(index, n) {
    Dtype val = in[index];
    Dtype abs_val = abs(val);
    if (abs_val < 1.0 / sigma2) {
      out[index] = sigma2 * val;
    } else {
      out[index] = (Dtype(0) < val) - (val < Dtype(0));
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(SmoothL1LossLayer);

}  // namespace caffe
