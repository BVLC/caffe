#include "hip/hip_runtime.h"
#include <vector>

#include "caffe/layers/tile_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void Tile(const int nthreads, const Dtype* bottom_data,
    const int tile_size, const int num_tiles, const int bottom_tile_axis,
    Dtype* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int d = index % tile_size;
    const int b = (index / tile_size / num_tiles) % bottom_tile_axis;
    const int n = index / tile_size / num_tiles / bottom_tile_axis;
    const int bottom_index = (n * bottom_tile_axis + b) * tile_size + d;
    top_data[index] = bottom_data[bottom_index];
  }
}

template <typename Dtype>
void TileLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  const int bottom_tile_axis = bottom[0]->shape(axis_);
  const int nthreads = top[0]->count();
  Tile<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(nthreads), CAFFE_CUDA_NUM_THREADS>>>(
      nthreads, bottom_data, inner_dim_, tiles_, bottom_tile_axis, top_data);
}

INSTANTIATE_LAYER_GPU_FUNCS(TileLayer);

}  // namespace caffe
