// Copyright 2014 BVLC and contributors.
//
// Based on data_layer.cpp by Yangqing Jia.

#include <stdint.h>
#include <pthread.h>

#include <string>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/io.hpp"
#include "caffe/vision_layers.hpp"

// caffe.proto > LayerParameter > WindowDataParameter
//   'source' field specifies the window_file
//   'crop_size' indicates the desired warped size

namespace caffe {

template <typename Dtype>
Dtype WindowDataLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      vector<Blob<Dtype>*>* top) {
  // First, join the thread
  JoinPrefetchThread();
  // Copy the data
  caffe_copy(prefetch_data_.count(), prefetch_data_.cpu_data(),
      (*top)[0]->mutable_gpu_data());
  caffe_copy(prefetch_label_.count(), prefetch_label_.cpu_data(),
      (*top)[1]->mutable_gpu_data());
  // Start a new prefetch thread
  CreatePrefetchThread();
  return Dtype(0.);
}

INSTANTIATE_CLASS(WindowDataLayer);

}  // namespace caffe
