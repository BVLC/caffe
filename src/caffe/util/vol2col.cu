#include "hip/hip_runtime.h"
#include <algorithm>
#include <cmath>
#include <cstdlib>
#include <cstring>

#include "caffe/common.hpp"
#include "caffe/util/vol2col.hpp"

namespace caffe {

template <typename Dtype>
__global__ void vol2col_gpu_kernel(const int n, const Dtype* data_im,
    const int height, const int width, const int depth, const int kernel_h,
    const int kernel_w, const int kernel_d, const int pad_h, const int pad_w,
    const int pad_d, const int stride_h, const int stride_w, const int stride_d,
    const int height_col, const int width_col, const int depth_col,
    Dtype* data_col) {
  CUDA_KERNEL_LOOP(index, n) {
    int d_out = index % depth_col;
    int w_index = index / depth_col;
    int w_out = w_index % width_col;
    int h_index = w_index / width_col;
    int h_out = h_index % height_col;
    int channel_in = h_index / height_col;
    int channel_out = channel_in * kernel_h * kernel_w * kernel_d;
    int h_in = h_out * stride_h - pad_h;
    int w_in = w_out * stride_w - pad_w;
    int d_in = d_out * stride_d - pad_d;
    Dtype* data_col_ptr = data_col;
    data_col_ptr += ((channel_out * height_col + h_out) * width_col + w_out)
        * depth_col + d_out;
    const Dtype* data_im_ptr = data_im;
    data_im_ptr += ((channel_in * height + h_in) * width + w_in) * depth + d_in;
    for (int i = 0; i < kernel_h; ++i) {
      for (int j = 0; j < kernel_w; ++j) {
        for (int k = 0; k < kernel_d; ++k) {
          int h = h_in + i;
          int w = w_in + j;
          int d = d_in + k;
          *data_col_ptr = (h >= 0 && w >= 0 && d >=0 && h < height && w < width
              && d < depth) ? data_im_ptr[(i * width + j) * depth + k] : 0;
          data_col_ptr += height_col * width_col * depth_col;
        }
      }
    }
  }
}

template <typename Dtype>
void vol2col_gpu(const Dtype* data_im, const int channels, const int height,
    const int width, const int depth, const int kernel_h, const int kernel_w,
    const int kernel_d, const int pad_h, const int pad_w, const int pad_d,
    const int stride_h, const int stride_w, const int stride_d,
    Dtype* data_col) {
  // We are going to launch channels * height_col * width_col * depth_col
  // kernels, each kernel responsible for copying a single-channel grid.
  int height_col = (height + 2 * pad_h - kernel_h) / stride_h + 1;
  int width_col = (width + 2 * pad_w - kernel_w) / stride_w + 1;
  int depth_col = (depth + 2 * pad_d - kernel_d) / stride_d + 1;
  int num_kernels = channels * height_col * width_col * depth_col;
  // NOLINT_NEXT_LINE(whitespace/operators)
  vol2col_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
                             CAFFE_CUDA_NUM_THREADS>>>(
      num_kernels, data_im, height, width, depth, kernel_h, kernel_w, kernel_d,
      pad_h, pad_w, pad_d, stride_h, stride_w, stride_d, height_col, width_col,
      depth_col, data_col);
  CUDA_POST_KERNEL_CHECK;
}


// Explicit instantiation
template void vol2col_gpu<float>(const float* data_im, const int channels,
    const int height, const int width, const int depth, const int kernel_h,
    const int kernel_w, const int kernel_d, const int pad_h, const int pad_w,
    const int pad_d, const int stride_h, const int stride_w, const int stride_d,
    float* data_col);
template void vol2col_gpu<double>(const double* data_im, const int channels,
    const int height, const int width, const int depth, const int kernel_h,
    const int kernel_w, const int kernel_d, const int pad_h, const int pad_w,
    const int pad_d, const int stride_h, const int stride_w, const int stride_d,
    double* data_col);


template <typename Dtype>
__global__ void col2vol_gpu_kernel(const int n, const Dtype* data_col,
    const int height, const int width, const int depth,
    const int channels, const int patch_h, const int patch_w, const int patch_d,
    const int pad_h, const int pad_w, const int pad_d,
    const int stride_h, const int stride_w, const int stride_d,
    const int height_col, const int width_col, const int depth_col,
    Dtype* data_im) {
  CUDA_KERNEL_LOOP(index, n) {
    Dtype val = 0;
    int d = index % depth + pad_d;
    int w = (index / depth) % width + pad_w;
    int h = ((index / depth) / width) % height + pad_h;
    int c = index / (width * height * depth);
    // compute the start and end of the output
    int d_col_start = (d < patch_d) ? 0 : (d - patch_d) / stride_d + 1;
    int d_col_end = min(d / stride_d + 1, depth_col);
    int w_col_start = (w < patch_w) ? 0 : (w - patch_w) / stride_w + 1;
    int w_col_end = min(w / stride_w + 1, width_col);
    int h_col_start = (h < patch_h) ? 0 : (h - patch_h) / stride_h + 1;
    int h_col_end = min(h / stride_h + 1, height_col);
    /*
    for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
      for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
        // the col location: [c * width * height + h_out, w_out]
        int c_col = c * patch_h * patch_w + (h - h_col * stride_h) * ksize
            + (w - w_col * stride_w);
        val += data_col[(c_col * height_col + h_col) * width_col + w_col];
      }
    }
    */
    // equivalent implementation
    int offset =
        ((c * patch_h * patch_w * patch_d) + h * patch_w * patch_d +
            w * patch_d + d) * height_col * width_col * depth_col;
    int coeff_h_col = (1 - stride_h * (patch_d * patch_w) * height_col) *
        width_col * depth_col;
    int coeff_w_col = (1 - stride_w * patch_d * height_col * width_col) *
        depth_col;
    int coeff_d_col = (1 - stride_d * height_col * width_col * depth_col);
    for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
      for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
        for (int d_col = d_col_start; d_col < d_col_end; ++d_col) {
          val += data_col[offset + h_col * coeff_h_col + w_col * coeff_w_col +
              d_col * coeff_d_col];
        }
      }
    }
    data_im[index] = val;
  }
}

template <typename Dtype>
void col2vol_gpu(const Dtype* data_col, const int channels, const int height,
    const int width, const int depth, const int patch_h, const int patch_w,
    const int patch_d, const int pad_h, const int pad_w, const int pad_d,
    const int stride_h, const int stride_w, const int stride_d,
    Dtype* data_im) {
  int height_col = (height + 2 * pad_h - patch_h) / stride_h + 1;
  int width_col = (width + 2 * pad_w - patch_w) / stride_w + 1;
  int depth_col = (depth + 2 * pad_d - patch_d) / stride_d + 1;
  int num_kernels = channels * height * width * depth;
  // To avoid involving atomic operations, we will launch one kernel per
  // bottom dimension, and then in the kernel add up the top dimensions.
  // NOLINT_NEXT_LINE(whitespace/operators)
  col2vol_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
                             CAFFE_CUDA_NUM_THREADS>>>(
      num_kernels, data_col, height, width, depth, channels, patch_h, patch_w,
      patch_d, pad_h, pad_w, pad_d, stride_h, stride_w, stride_d, height_col,
      width_col, depth_col, data_im);
  CUDA_POST_KERNEL_CHECK;
}

// Explicit instantiation
template void col2vol_gpu<float>(const float* data_col, const int channels,
    const int height, const int width, const int depth, const int patch_h,
    const int patch_w, const int patch_d, const int pad_h, const int pad_w,
    const int pad_d, const int stride_h, const int stride_w, const int stride_d,
    float* data_im);
template void col2vol_gpu<double>(const double* data_col, const int channels,
    const int height, const int width, const int depth, const int patch_h,
    const int patch_w, const int patch_d, const int pad_h, const int pad_w,
    const int pad_d, const int stride_h, const int stride_w, const int stride_d,
    double* data_im);

}  // namespace caffe
