#include "hip/hip_runtime.h"
#include <algorithm>
#include <cmath>
#include <cstdlib>
#include <cstring>

#include "caffe/common.hpp"
#include "caffe/util/im2col.hpp"

namespace caffe {

template <typename Dtype>
__global__ void im2col_gpu_kernel(const int n, const Dtype* data_im,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    const int height_col, const int width_col,
    Dtype* data_col) {
  CUDA_KERNEL_LOOP(index, n) {
    int w_out = index % width_col;
    int h_index = index / width_col;
    int h_out = h_index % height_col;
    int channel_in = h_index / height_col;
    int channel_out = channel_in * kernel_h * kernel_w;
    int h_in = h_out * stride_h - pad_h;
    int w_in = w_out * stride_w - pad_w;
    Dtype* data_col_ptr = data_col;
    data_col_ptr += (channel_out * height_col + h_out) * width_col + w_out;
    const Dtype* data_im_ptr = data_im;
    data_im_ptr += (channel_in * height + h_in) * width + w_in;
    for (int i = 0; i < kernel_h; ++i) {
      for (int j = 0; j < kernel_w; ++j) {
        int h = h_in + i;
        int w = w_in + j;
        *data_col_ptr = (h >= 0 && w >= 0 && h < height && w < width) ?
            data_im_ptr[i * width + j] : 0;
        data_col_ptr += height_col * width_col;
      }
    }
  }
}

template <typename Dtype>
void im2col_gpu(const Dtype* data_im, const int channels,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    Dtype* data_col) {
  // We are going to launch channels * height_col * width_col kernels, each
  // kernel responsible for copying a single-channel grid.
  int height_col = (height + 2 * pad_h - kernel_h) / stride_h + 1;
  int width_col = (width + 2 * pad_w - kernel_w) / stride_w + 1;
  int num_kernels = channels * height_col * width_col;
  // NOLINT_NEXT_LINE(whitespace/operators)
  im2col_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
                             CAFFE_CUDA_NUM_THREADS>>>(
      num_kernels, data_im, height, width, kernel_h, kernel_w, pad_h,
      pad_w, stride_h, stride_w, height_col,
      width_col, data_col);
  CUDA_POST_KERNEL_CHECK;
}

// Explicit instantiation
template void im2col_gpu<float>(const float* data_im, const int channels,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w, const int stride_h, const int stride_w,
    float* data_col);
template void im2col_gpu<double>(const double* data_im, const int channels,
    const int height, const int width, const int kernel_h, const int kernel_w,
    const int pad_h, const int pad_w, const int stride_h, const int stride_w,
    double* data_col);

template <typename Dtype, int num_axes>
__global__ void im2col_nd_gpu_kernel(const int n, const Dtype* data_im,
    const int* im_shape, const int* col_shape,
    const int* kernel_shape, const int* pad, const int* stride,
    Dtype* data_col) {
  int d_temp[num_axes];  // NOLINT(runtime/arrays)
  int d_iter[num_axes];  // NOLINT(runtime/arrays)
  int i;
  CUDA_KERNEL_LOOP(index, n) {
    // Initialize channel_in, computed in the loop below, with intermediate
    // computations used to compute the spatial indices.
    int channel_in = index;
    int channel_out = 1;
    for (i = num_axes - 1; i >= 0; --i) {
      d_temp[i] = channel_in % col_shape[i + 1];
      channel_in /= col_shape[i + 1];
      channel_out *= kernel_shape[i];
    }
    channel_out *= channel_in;
    int data_col_inc = 1;
    for (i = 0; i < num_axes; ++i) {
      channel_out *= col_shape[i + 1];
      channel_out += d_temp[i];
      d_temp[i] = d_temp[i] * stride[i] - pad[i];
      channel_in *= im_shape[i + 1];
      channel_in += d_temp[i];
      data_col_inc *= col_shape[i + 1];
      d_iter[i] = 0;
    }
    Dtype* data_col_ptr = data_col + channel_out;
    const Dtype* data_im_ptr = data_im + channel_in;
    bool incremented;
    do {
      bool in_range = true;
      for (i = 0; i < num_axes; ++i) {
        const int d_iter_im = d_iter[i] + d_temp[i];
        in_range &= d_iter_im >= 0 && d_iter_im < im_shape[i + 1];
        if (!in_range) { break; }
      }
      if (in_range) {
        int data_im_offset = d_iter[0];
        for (i = 1; i < num_axes; ++i) {
          data_im_offset *= im_shape[i + 1];
          data_im_offset += d_iter[i];
        }
        *data_col_ptr = data_im_ptr[data_im_offset];
      } else {
        *data_col_ptr = 0;
      }
      data_col_ptr += data_col_inc;
      incremented = false;
      for (i = num_axes - 1; i >= 0; --i) {
        const int d_max = kernel_shape[i];
        if (d_iter[i] == d_max - 1) {
          d_iter[i] = 0;
        } else {  // d_iter[i] < d_max - 1
          ++d_iter[i];
          incremented = true;
          break;
        }
      }  // for (int i = num_axes - 1; i >= 0; --i)
    } while (incremented);  // do
  }  // CUDA_KERNEL_LOOP(index, n)
}

template <typename Dtype>
void im2col_nd_gpu(const Dtype* data_im, const int num_spatial_axes,
    const int num_kernels, const int* im_shape, const int* col_shape,
    const int* kernel_shape, const int* pad, const int* stride,
    Dtype* data_col) {
  switch (num_spatial_axes) {
  case 1:
    im2col_nd_gpu_kernel<Dtype, 1>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, data_im, im_shape, col_shape,
        kernel_shape, pad, stride, data_col);
    break;
  case 2:
    im2col_nd_gpu_kernel<Dtype, 2>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, data_im, im_shape, col_shape,
        kernel_shape, pad, stride, data_col);
    break;
  case 3:
    im2col_nd_gpu_kernel<Dtype, 3>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, data_im, im_shape, col_shape,
        kernel_shape, pad, stride, data_col);
    break;
  case 4:
    im2col_nd_gpu_kernel<Dtype, 4>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, data_im, im_shape, col_shape,
        kernel_shape, pad, stride, data_col);
    break;
  case 5:
    im2col_nd_gpu_kernel<Dtype, 5>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, data_im, im_shape, col_shape,
        kernel_shape, pad, stride, data_col);
    break;
  case 6:
    im2col_nd_gpu_kernel<Dtype, 6>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, data_im, im_shape, col_shape,
        kernel_shape, pad, stride, data_col);
    break;
  case 7:
    im2col_nd_gpu_kernel<Dtype, 7>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, data_im, im_shape, col_shape,
        kernel_shape, pad, stride, data_col);
    break;
  case 8:
    im2col_nd_gpu_kernel<Dtype, 8>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, data_im, im_shape, col_shape,
        kernel_shape, pad, stride, data_col);
    break;
  case 9:
    im2col_nd_gpu_kernel<Dtype, 9>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, data_im, im_shape, col_shape,
        kernel_shape, pad, stride, data_col);
    break;
  case 10:
    im2col_nd_gpu_kernel<Dtype, 10>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(num_kernels), CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, data_im, im_shape, col_shape,
        kernel_shape, pad, stride, data_col);
    break;
  default:
    LOG(FATAL) << "im2col_nd_gpu does not support computation with "
               << num_spatial_axes << " spatial axes";
  }
  CUDA_POST_KERNEL_CHECK;
}

// Explicit instantiation
template void im2col_nd_gpu<float>(const float* data_im,
    const int num_spatial_axes, const int col_size,
    const int* im_shape, const int* col_shape,
    const int* kernel_shape, const int* pad, const int* stride,
    float* data_col);
template void im2col_nd_gpu<double>(const double* data_im,
    const int num_spatial_axes, const int col_size,
    const int* im_shape, const int* col_shape,
    const int* kernel_shape, const int* pad, const int* stride,
    double* data_col);

template <typename Dtype>
__global__ void col2im_gpu_kernel(const int n, const Dtype* data_col,
    const int height, const int width, const int channels,
    const int patch_h, const int patch_w,
    const int pad_h, const int pad_w,
    const int stride_h, const int stride_w,
    const int height_col, const int width_col,
    Dtype* data_im) {
  CUDA_KERNEL_LOOP(index, n) {
    Dtype val = 0;
    int w = index % width + pad_w;
    int h = (index / width) % height + pad_h;
    int c = index / (width * height);
    // compute the start and end of the output
    int w_col_start = (w < patch_w) ? 0 : (w - patch_w) / stride_w + 1;
    int w_col_end = min(w / stride_w + 1, width_col);
    int h_col_start = (h < patch_h) ? 0 : (h - patch_h) / stride_h + 1;
    int h_col_end = min(h / stride_h + 1, height_col);
    /*
    for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
      for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
        // the col location: [c * width * height + h_out, w_out]
        int c_col = c * patch_h * patch_w + (h - h_col * stride_h) * ksize
            + (w - w_col * stride_w);
        val += data_col[(c_col * height_col + h_col) * width_col + w_col];
      }
    }
    */
    // equivalent implementation
    int offset =
        (c * patch_h * patch_w + h * patch_w + w) * height_col * width_col;
    int coeff_h_col = (1 - stride_h * patch_w * height_col) * width_col;
    int coeff_w_col = (1 - stride_w * height_col * width_col);
    for (int h_col = h_col_start; h_col < h_col_end; ++h_col) {
      for (int w_col = w_col_start; w_col < w_col_end; ++w_col) {
        val += data_col[offset + h_col * coeff_h_col + w_col * coeff_w_col];
      }
    }
    data_im[index] = val;
  }
}

template <typename Dtype>
void col2im_gpu(const Dtype* data_col, const int channels,
    const int height, const int width, const int patch_h, const int patch_w,
    const int pad_h, const int pad_w, const int stride_h,
    const int stride_w, Dtype* data_im) {
  int height_col = (height + 2 * pad_h - patch_h) / stride_h + 1;
  int width_col = (width + 2 * pad_w - patch_w) / stride_w + 1;
  int num_kernels = channels * height * width;
  // To avoid involving atomic operations, we will launch one kernel per
  // bottom dimension, and then in the kernel add up the top dimensions.
  // NOLINT_NEXT_LINE(whitespace/operators)
  col2im_gpu_kernel<Dtype><<<CAFFE_GET_BLOCKS(num_kernels),
                             CAFFE_CUDA_NUM_THREADS>>>(
      num_kernels, data_col, height, width, channels, patch_h, patch_w,
      pad_h, pad_w, stride_h, stride_w,
      height_col, width_col, data_im);
  CUDA_POST_KERNEL_CHECK;
}

// Explicit instantiation
template void col2im_gpu<float>(const float* data_col, const int channels,
    const int height, const int width, const int patch_h, const int patch_w,
    const int pad_h, const int pad_w, const int stride_h,
    const int stride_w, float* data_im);
template void col2im_gpu<double>(const double* data_col, const int channels,
    const int height, const int width, const int patch_h, const int patch_w,
    const int pad_h, const int pad_w, const int stride_h,
    const int stride_w, double* data_im);

template <typename Dtype, int num_axes>
__global__ void col2im_nd_gpu_kernel(const int n, const Dtype* data_col,
    const int* im_shape, const int* col_shape,
    const int* kernel_shape, const int* pad, const int* stride,
    Dtype* data_im) {
  int d_im[num_axes];  // NOLINT(runtime/arrays)
  int d_col_iter[num_axes];  // NOLINT(runtime/arrays)
  int d_col_start[num_axes];  // NOLINT(runtime/arrays)
  int d_col_end[num_axes];  // NOLINT(runtime/arrays)
  CUDA_KERNEL_LOOP(index, n) {
    // Initialize channel_in, computed in the loop below, with intermediate
    // computations used to compute the spatial indices.
    int channel_im = index;
    // Calculate d_im (image dimensions).
    for (int i = num_axes - 1; i >= 0; --i) {
      d_im[i] = channel_im % im_shape[i + 1] + pad[i];
      channel_im /= im_shape[i + 1];
    }
    // Calculate col start/end indices.
    bool done = false;
    for (int i = 0; i < num_axes; ++i) {
      d_col_start[i] = d_col_iter[i] =
          (d_im[i] < kernel_shape[i]) ?
          0 : (d_im[i] - kernel_shape[i]) / stride[i] + 1;
      d_col_end[i] = min(d_im[i] / stride[i] + 1, col_shape[i + 1]);
      if (d_col_start[i] >= d_col_end[i]) {
        // Skip computation if the dimension is 0 at any spatial axis --
        // final val will be 0.
        data_im[index] = 0;
        done = true;
        break;  // for (int i = 0; i < num_axes; ++i)
      }
    }
    if (done) {
      continue;  // CUDA_KERNEL_LOOP(index, n)
    }
    // Loop over the col to compute the output val.
    Dtype val = 0;
    bool incremented = true;
    do {
      // Compute the final offset.
      int final_offset = 0;
      int kernel_shape_prod = 1;
      for (int i = num_axes - 1; i >= 0; --i) {
        final_offset +=
            (d_im[i] - d_col_iter[i] * stride[i]) * kernel_shape_prod;
        kernel_shape_prod *= kernel_shape[i];
      }
      final_offset += kernel_shape_prod * channel_im;
      for (int i = 0; i < num_axes; ++i) {
        final_offset *= col_shape[i + 1];
        final_offset += d_col_iter[i];
      }
      val += data_col[final_offset];
      incremented = false;
      for (int i = num_axes - 1; i >= 0; --i) {
        const int d_max = d_col_end[i];
        if (d_col_iter[i] == d_max - 1) {
          d_col_iter[i] = d_col_start[i];
        } else {  // d_col_iter[i] < d_max - 1
          ++d_col_iter[i];
          incremented = true;
          break;  // for (int i = num_axes - 1; i >= 0; --i)
        }
      }  // for (int i = num_axes - 1; i >= 0; --i)
    }  while (incremented);
    data_im[index] = val;
  }  // CUDA_KERNEL_LOOP(index, n)
}

template <typename Dtype>
void col2im_nd_gpu(const Dtype* data_col, const int num_spatial_axes,
    const int im_size, const int* im_shape, const int* col_shape,
    const int* kernel_shape, const int* pad, const int* stride,
    Dtype* data_im) {
  switch (num_spatial_axes) {
  case 1:
    col2im_nd_gpu_kernel<Dtype, 1>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, data_col, im_shape, col_shape,
          kernel_shape, pad, stride, data_im);
    break;
  case 2:
    col2im_nd_gpu_kernel<Dtype, 2>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, data_col, im_shape, col_shape,
          kernel_shape, pad, stride, data_im);
    break;
  case 3:
    col2im_nd_gpu_kernel<Dtype, 3>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, data_col, im_shape, col_shape,
          kernel_shape, pad, stride, data_im);
    break;
  case 4:
    col2im_nd_gpu_kernel<Dtype, 4>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, data_col, im_shape, col_shape,
          kernel_shape, pad, stride, data_im);
    break;
  case 5:
    col2im_nd_gpu_kernel<Dtype, 5>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, data_col, im_shape, col_shape,
          kernel_shape, pad, stride, data_im);
    break;
  case 6:
    col2im_nd_gpu_kernel<Dtype, 6>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, data_col, im_shape, col_shape,
          kernel_shape, pad, stride, data_im);
    break;
  case 7:
    col2im_nd_gpu_kernel<Dtype, 7>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, data_col, im_shape, col_shape,
          kernel_shape, pad, stride, data_im);
    break;
  case 8:
    col2im_nd_gpu_kernel<Dtype, 8>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, data_col, im_shape, col_shape,
          kernel_shape, pad, stride, data_im);
    break;
  case 9:
    col2im_nd_gpu_kernel<Dtype, 9>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, data_col, im_shape, col_shape,
          kernel_shape, pad, stride, data_im);
    break;
  case 10:
    col2im_nd_gpu_kernel<Dtype, 10>  // NOLINT_NEXT_LINE(whitespace/operators)
          <<<CAFFE_GET_BLOCKS(im_size), CAFFE_CUDA_NUM_THREADS>>>(
          im_size, data_col, im_shape, col_shape,
          kernel_shape, pad, stride, data_im);
    break;
  default:
    LOG(FATAL) << "col2im_nd_gpu does not support computation with "
               << num_spatial_axes << " spatial axes";
  }
  CUDA_POST_KERNEL_CHECK;
}

// Explicit instantiation
template void col2im_nd_gpu<float>(const float* data_col,
    const int num_spatial_axes, const int im_size,
    const int* im_shape, const int* col_shape,
    const int* kernel_shape, const int* pad, const int* stride,
    float* data_im);
template void col2im_nd_gpu<double>(const double* data_col,
    const int num_spatial_axes, const int im_size,
    const int* im_shape, const int* col_shape,
    const int* kernel_shape, const int* pad, const int* stride,
    double* data_im);

}  // namespace caffe
