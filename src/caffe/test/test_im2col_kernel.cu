
#include <hip/hip_runtime.h>
#ifdef USE_CUDA
#include <cstring>
#include <vector>

#include "gtest/gtest.h"

#include "caffe/blob.hpp"
#include "caffe/common.hpp"
#include "caffe/filler.hpp"
#include "caffe/layers/im2col_layer.hpp"
#include "caffe/util/im2col.hpp"

#include "caffe/test/test_caffe_main.hpp"

namespace caffe {

// Forward declare kernel functions
template<typename Dtype>
__global__ void im2col_gpu_kernel(const int_tp n, const Dtype* data_im,
                                  const int_tp height, const int_tp width,
                                  const int_tp kernel_h, const int_tp kernel_w,
                                  const int_tp pad_h, const int_tp pad_w,
                                  const int_tp stride_h, const int_tp stride_w,
                                  const int_tp dilation_h,
                                  const int_tp dilation_w,
                                  const int_tp height_col,
                                  const int_tp width_col, Dtype* data_col);

template<typename Dtype, int_tp num_axes>
__global__ void im2col_nd_gpu_kernel(const int_tp n, const Dtype* data_im,
                                     const int_tp* im_shape,
                                     const int_tp* col_shape,
                                     const int_tp* kernel_shape,
                                     const int_tp* pad, const int_tp* stride,
                                     const int_tp* dilation, Dtype* data_col);

template <typename Dtype>
class Im2colKernelTest : public GPUDeviceTest<Dtype> {
 protected:
  Im2colKernelTest()
      // big so launches > 1024 threads
      : blob_bottom_(new Blob<Dtype>(5, 500, 15, 15)),
        blob_kernel_shape_(new Blob<int_tp>()),
        blob_stride_(new Blob<int_tp>()), blob_pad_(new Blob<int_tp>()),
        blob_dilation_(new Blob<int_tp>()), blob_top_(new Blob<Dtype>()),
        blob_top_cpu_(new Blob<Dtype>()) {
    FillerParameter filler_param;
    GaussianFiller<Dtype> filler(filler_param);
    filler.Fill(this->blob_bottom_);
    vector<int_tp> dim_blob_shape(1, 2);
    blob_kernel_shape_->Reshape(dim_blob_shape);
    blob_stride_->Reshape(dim_blob_shape);
    blob_pad_->Reshape(dim_blob_shape);
    blob_dilation_->Reshape(dim_blob_shape);

    height_ = blob_bottom_->height();
    width_ = blob_bottom_->width();
    channels_ = blob_bottom_->channels();
    pad_ = 0;
    stride_ = 2;
    dilation_ = 3;
    kernel_size_ = 3;
    height_col_ = (height_ + 2 * pad_ - (dilation_ * (kernel_size_ - 1) + 1))
        / stride_ + 1;
    width_col_ = (width_ + 2 * pad_ - (dilation_ * (kernel_size_ - 1) + 1))
        / stride_ + 1;

    for (int_tp i = 0; i < 2; ++i) {
      blob_kernel_shape_->mutable_cpu_data()[i] = kernel_size_;
      blob_stride_->mutable_cpu_data()[i] = stride_;
      blob_pad_->mutable_cpu_data()[i] = pad_;
      blob_dilation_->mutable_cpu_data()[i] = dilation_;
    }
  }

  virtual ~Im2colKernelTest() {
    delete blob_bottom_;
    delete blob_top_;
    delete blob_top_cpu_;
    delete blob_kernel_shape_;
    delete blob_stride_;
    delete blob_pad_;
    delete blob_dilation_;
  }

  Blob<int_tp>* const blob_kernel_shape_;
  Blob<int_tp>* const blob_stride_;
  Blob<int_tp>* const blob_pad_;
  Blob<int_tp>* const blob_dilation_;
  Blob<Dtype>* const blob_bottom_;
  Blob<Dtype>* const blob_top_;
  Blob<Dtype>* const blob_top_cpu_;
  int_tp height_;
  int_tp width_;
  int_tp channels_;
  int_tp pad_;
  int_tp stride_;
  int_tp dilation_;
  int_tp kernel_size_;
  int_tp height_col_;
  int_tp width_col_;
};

TYPED_TEST_CASE(Im2colKernelTest, TestDtypes);

TYPED_TEST(Im2colKernelTest, Test2D) {
  if (Caffe::GetDefaultDevice()->backend() == BACKEND_CUDA) {
    // Reshape the blobs to correct size for im2col output
    this->blob_top_->Reshape(this->blob_bottom_->num(),
        this->channels_ * this->kernel_size_ * this->kernel_size_,
        this->height_col_,
        this->width_col_);

    this->blob_top_cpu_->Reshape(this->blob_bottom_->num(),
        this->channels_ * this->kernel_size_ * this->kernel_size_,
        this->height_col_,
        this->width_col_);

    const TypeParam* bottom_data = this->blob_bottom_->gpu_data();
    TypeParam* top_data = this->blob_top_->mutable_gpu_data();
    TypeParam* cpu_data = this->blob_top_cpu_->mutable_cpu_data();

    // CPU Version
    for (int_tp n = 0; n < this->blob_bottom_->num(); ++n) {
      im2col_cpu(this->blob_bottom_->cpu_data() + this->blob_bottom_->offset(n),
          this->channels_, this->height_, this->width_,
          this->kernel_size_, this->kernel_size_, this->pad_, this->pad_,
          this->stride_, this->stride_, this->dilation_, this->dilation_,
          cpu_data + this->blob_top_cpu_->offset(n));
    }

    // GPU version
    int_tp num_kernels = this->channels_ * this->height_col_ * this->width_col_;
    int_tp default_grid_dim = CAFFE_GET_BLOCKS(num_kernels);

    // Launch with different grid sizes
    for (int_tp grid_div = 2; grid_div <= 8; grid_div++) {
      for (int_tp n = 0; n < this->blob_bottom_->num(); ++n) {
        int_tp grid_dim = default_grid_dim/grid_div;
        // NOLINT_NEXT_LINE(whitespace/operators)
        im2col_gpu_kernel<TypeParam>
        CUDA_KERNEL(grid_dim, CAFFE_CUDA_NUM_THREADS)(
            num_kernels, bottom_data + this->blob_bottom_->offset(n),
            this->height_, this->width_, this->kernel_size_, this->kernel_size_,
            this->pad_, this->pad_, this->stride_, this->stride_,
            this->dilation_, this->dilation_,
            this->height_col_, this->width_col_,
            top_data + this->blob_top_->offset(n));
        CUDA_POST_KERNEL_CHECK;
      }

      // Compare results against CPU version
      for (int_tp i = 0; i < this->blob_top_->count(); ++i) {
        TypeParam cpuval = cpu_data[i];
        TypeParam gpuval = this->blob_top_->cpu_data()[i];
        EXPECT_EQ(cpuval, gpuval);
        if (cpuval != gpuval) {
          break;
        }
      }
    }
  }
}

TYPED_TEST(Im2colKernelTest, TestND) {
  if (Caffe::GetDefaultDevice()->backend() == BACKEND_CUDA) {
    // Reshape the blobs to correct size for im2col output
    this->blob_top_->Reshape(this->blob_bottom_->num(),
        this->channels_ * this->kernel_size_ * this->kernel_size_,
        this->height_col_,
        this->width_col_);

    this->blob_top_cpu_->ReshapeLike(*this->blob_top_);

    const TypeParam* bottom_data_cpu = this->blob_bottom_->cpu_data();
    TypeParam* top_data_cpu = this->blob_top_cpu_->mutable_cpu_data();

    // CPU Version
    for (int_tp n = 0; n < this->blob_bottom_->num(); ++n) {
      im2col_nd_cpu(bottom_data_cpu + this->blob_bottom_->offset(n), 2,
          this->blob_bottom_->shape().data() + 1,
          this->blob_top_cpu_->shape().data() + 1,
          this->blob_kernel_shape_->cpu_data(),
          this->blob_pad_->cpu_data(), this->blob_stride_->cpu_data(),
          this->blob_dilation_->cpu_data(),
          top_data_cpu + this->blob_top_cpu_->offset(n));
    }

    // GPU version
    int_tp num_kernels = this->channels_ * this->height_col_ * this->width_col_;
    int_tp default_grid_dim = CAFFE_GET_BLOCKS(num_kernels);
    const TypeParam* bottom_data_gpu = this->blob_bottom_->gpu_data();

    // Launch with different grid sizes
    for (int_tp grid_div = 2; grid_div <= 8; grid_div++) {
      for (int_tp n = 0; n < this->blob_bottom_->num(); ++n) {
        const int_tp grid_dim = default_grid_dim / grid_div;
        TypeParam* top_data_gpu = this->blob_top_->mutable_gpu_data();
        // NOLINT_NEXT_LINE(whitespace/operators)
        im2col_nd_gpu_kernel<TypeParam, 2>
        CUDA_KERNEL(grid_dim, CAFFE_CUDA_NUM_THREADS)(
            num_kernels, bottom_data_gpu + this->blob_bottom_->offset(n),
            this->blob_bottom_->gpu_shape() + 1,
            this->blob_top_->gpu_shape() + 1,
            this->blob_kernel_shape_->gpu_data(), this->blob_pad_->gpu_data(),
            this->blob_stride_->gpu_data(), this->blob_dilation_->gpu_data(),
            top_data_gpu + this->blob_top_->offset(n));
        CUDA_POST_KERNEL_CHECK;
      }

      // Compare results against CPU version
      for (int_tp i = 0; i < this->blob_top_->count(); ++i) {
        TypeParam cpuval = top_data_cpu[i];
        TypeParam gpuval = this->blob_top_->cpu_data()[i];
        EXPECT_EQ(cpuval, gpuval);
        if (cpuval != gpuval) {
          break;
        }
      }
    }
  }
}

}  // namespace caffe
#endif  // USE_CUDA
