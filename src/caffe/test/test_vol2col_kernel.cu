#include "hip/hip_runtime.h"
#include <cstring>
#include <vector>

#include "gtest/gtest.h"

#include "caffe/blob.hpp"
#include "caffe/common.hpp"
#include "caffe/filler.hpp"
#include "caffe/util/vol2col.hpp"
#include "caffe/vision_layers.hpp"

#include "caffe/test/test_caffe_main.hpp"

namespace caffe {

// Forward declare kernel functions
template <typename Dtype>
__global__ void vol2col_gpu_kernel(const int n, const Dtype* data_im,
    const int height, const int width, const int depth, const int kernel_h,
    const int kernel_w, const int kernel_d, const int pad_h, const int pad_w,
    const int pad_d, const int stride_h, const int stride_w, const int stride_d,
    const int height_col, const int width_col, const int depth_col,
    Dtype* data_col);

extern hipDeviceProp_t CAFFE_TEST_CUDA_PROP;

template <typename Dtype>
class Vol2colKernelTest : public ::testing::Test {
 protected:
  Vol2colKernelTest()
        // big so launches > 1024 threads
      : blob_bottom_(new Blob<Dtype>(vector<int>())),
        blob_top_(new Blob<Dtype>()),
        blob_top_cpu_(new Blob<Dtype>()) {
    // update the blob shape
    int bot_shape_arr[] = {5, 500, 10, 10, 10};
    vector<int> bot_shape(bot_shape_arr, bot_shape_arr +
        sizeof(bot_shape_arr) / sizeof(int));
    blob_bottom_->Reshape(bot_shape);

    FillerParameter filler_param;
    GaussianFiller<Dtype> filler(filler_param);
    filler.Fill(this->blob_bottom_);

    vector<int> shape = blob_bottom_->shape();
    channels_ = shape[1];
    height_ = shape[2];
    width_ = shape[3];
    depth_ = shape[4];
    pad_ = 0;
    stride_ = 2;
    kernel_size_ = 3;
    height_col_ = (height_ + 2 * pad_ - kernel_size_) / stride_ + 1;
    width_col_ = (width_ + 2 * pad_ - kernel_size_) / stride_ + 1;
    depth_col_ = (depth_ + 2 * pad_ - kernel_size_) / stride_ + 1;
  }

  virtual ~Vol2colKernelTest() {
      delete blob_bottom_;
      delete blob_top_;
      delete blob_top_cpu_;
  }

  Blob<Dtype>* const blob_bottom_;
  Blob<Dtype>* const blob_top_;
  Blob<Dtype>* const blob_top_cpu_;
  int height_;
  int width_;
  int depth_;
  int channels_;
  int pad_;
  int stride_;
  int kernel_size_;
  int height_col_;
  int width_col_;
  int depth_col_;
};

TYPED_TEST_CASE(Vol2colKernelTest, TestDtypes);

TYPED_TEST(Vol2colKernelTest, TestGPU) {
  Caffe::set_mode(Caffe::GPU);

  // Reshape the blobs to correct size for vol2col output
  vector<int> top_shape(5);
  top_shape[0] = this->blob_bottom_->shape(0);
  top_shape[1] = this->channels_ * this->kernel_size_ * this->kernel_size_ *
      this->kernel_size_;
  top_shape[2] = this->height_col_;
  top_shape[3] = this->width_col_;
  top_shape[4] = this->depth_col_;
  this->blob_top_->Reshape(top_shape);
  this->blob_top_cpu_->Reshape(top_shape);

  const TypeParam* bottom_data = this->blob_bottom_->gpu_data();
  TypeParam* top_data = this->blob_top_->mutable_gpu_data();
  TypeParam* cpu_data = this->blob_top_cpu_->mutable_cpu_data();

  // CPU Version
  for (int n = 0; n < this->blob_bottom_->shape(0); ++n) {
    vol2col_cpu(this->blob_bottom_->cpu_data() +
      this->blob_bottom_->offset(vector<int>(1, n)), this->channels_,
      this->height_, this->width_, this->depth_, this->kernel_size_,
      this->kernel_size_, this->kernel_size_, this->pad_, this->pad_,
      this->pad_, this->stride_, this->stride_, this->stride_, cpu_data
      + this->blob_top_cpu_->offset(vector<int>(1, n)));
  }

  // GPU version
  int num_kernels = this->channels_ * this->height_col_ * this->width_col_ *
      this->depth_col_;
  int default_grid_dim = CAFFE_GET_BLOCKS(num_kernels);

  // Launch with different grid sizes
  for (int grid_div = 2; grid_div <= 8; grid_div++) {
    for (int n = 0; n < this->blob_bottom_->shape(0); ++n) {
      int grid_dim = default_grid_dim/grid_div;
      // NOLINT_NEXT_LINE(whitespace/operators)
      vol2col_gpu_kernel<TypeParam><<<grid_dim, CAFFE_CUDA_NUM_THREADS>>>(
        num_kernels, bottom_data +
        this->blob_bottom_->offset(vector<int>(1, n)), this->height_,
        this->width_, this->depth_, this->kernel_size_, this->kernel_size_,
        this->kernel_size_, this->pad_, this->pad_, this->pad_, this->stride_,
        this->stride_, this->stride_, this->height_col_, this->width_col_,
        this->depth_col_, top_data +
        this->blob_top_->offset(vector<int>(1, n)));
      CUDA_POST_KERNEL_CHECK;
    }

    // Compare results against CPU version
    for (int i = 0; i < this->blob_top_->count(); ++i) {
      TypeParam cpuval = cpu_data[i];
      TypeParam gpuval = this->blob_top_->cpu_data()[i];
      EXPECT_EQ(cpuval, gpuval);
      if (cpuval != gpuval) {
        break;
      }
    }
  }
}

}  // namespace caffe
