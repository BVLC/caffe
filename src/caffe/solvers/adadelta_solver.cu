#include "hip/hip_runtime.h"
#include "caffe/util/math_functions.hpp"


namespace caffe {

template <typename Dtype>
__global__ void AdaDeltaUpdate(int N, Dtype* g, Dtype* h, Dtype* h2,
    Dtype momentum, Dtype delta, Dtype local_rate) {
  CUDA_KERNEL_LOOP(i, N) {
    float gi = g[i];
    float hi = h[i] = momentum * h[i] + (1-momentum) * gi * gi;
    gi = gi * sqrt((h2[i] + delta) / (hi + delta));
    h2[i] = momentum * h2[i] + (1-momentum) * gi * gi;
    g[i] = local_rate * gi;
  }
}
template <typename Dtype>
void adadelta_update_gpu(int N, Dtype* g, Dtype* h, Dtype* h2, Dtype momentum,
    Dtype delta, Dtype local_rate) {
  AdaDeltaUpdate<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, g, h, h2, momentum, delta, local_rate);
  CUDA_POST_KERNEL_CHECK;
}
template void adadelta_update_gpu<float>(int , float*, float*, float*,
    float, float, float);
template void adadelta_update_gpu<double>(int, double*, double*, double*,
    double, double, double);

}  // namespace caffe
