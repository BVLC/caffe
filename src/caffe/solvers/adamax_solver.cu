#include "hip/hip_runtime.h"
#include <algorithm>

#include "caffe/util/math_functions.hpp"


namespace caffe {

template <typename Dtype>
__global__ void AdaMaxUpdate(int N, Dtype* g, Dtype* m, Dtype* v,
    Dtype beta1, Dtype beta2, Dtype eps_hat, Dtype corrected_local_rate) {
  CUDA_KERNEL_LOOP(i, N) {
    float gi = g[i];
    float mi = m[i] = m[i]*beta1 + gi*(1-beta1);
    float vi = v[i] = max(v[i]*beta2 + eps_hat, abs(gi));
    g[i] = corrected_local_rate * mi / vi;
  }
}
template <typename Dtype>
void adamax_update_gpu(int N, Dtype* g, Dtype* m, Dtype* v, Dtype beta1,
    Dtype beta2, Dtype eps_hat, Dtype corrected_local_rate) {
  AdaMaxUpdate<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, g, m, v, beta1, beta2, eps_hat, corrected_local_rate);
  CUDA_POST_KERNEL_CHECK;
}
template void adamax_update_gpu<float>(int, float*, float*, float*,
    float, float, float, float);
template void adamax_update_gpu<double>(int, double*, double*, double*,
    double, double, double, double);

}  // namespace caffe
