#include "hip/hip_runtime.h"
#include "caffe/util/math_functions.hpp"


namespace caffe {

template <typename Dtype>
__global__ void AdaGradUpdate(int N, Dtype* g, Dtype* h, Dtype delta,
    Dtype local_rate) {
  CUDA_KERNEL_LOOP(i, N) {
    float gi = g[i];
    float hi = h[i] = h[i] + gi*gi;
    g[i] = local_rate * gi / (sqrt(hi) + delta);
  }
}
template <typename Dtype>
void adagrad_update_gpu(int N, Dtype* g, Dtype* h, Dtype delta,
    Dtype local_rate) {
  AdaGradUpdate<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, g, h, delta, local_rate);
  CUDA_POST_KERNEL_CHECK;
}
template void adagrad_update_gpu<float>(int, float*, float*, float, float);
template void adagrad_update_gpu<double>(int, double*, double*, double, double);

}  // namespace caffe
