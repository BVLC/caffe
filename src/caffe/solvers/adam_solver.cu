#include "hip/hip_runtime.h"
#include "caffe/util/math_functions.hpp"


namespace caffe {

template <typename Dtype>
__global__ void AdamUpdate(int N, int t, Dtype* g, Dtype* m, Dtype* v,
                           Dtype beta1, Dtype beta2, Dtype eps_hat, Dtype corrected_local_rate,
                           bool amsgrad, bool rectified) {
  CUDA_KERNEL_LOOP(i, N) {
    float gi = g[i];
    float mi = m[i] = m[i]*beta1 + gi*(1-beta1);
    float vi_old = v[i];
    float vi = v[i] = v[i]*beta2 + gi*gi*(1-beta2);
    if (amsgrad) {
      if (vi < vi_old)
        v[i] = vi = vi_old;
    }
    if (!rectified)
      g[i] = corrected_local_rate * mi / (sqrt(vi) + eps_hat);
    else
      {
        Dtype rho_inf = 2.0/(1.0-beta2) - 1.0;
        Dtype rho_t = rho_inf - 2.0 * t * pow(beta2,t)/(1.0-pow(beta2,t)) ;
        if (rho_t > 4.0)
          {
            Dtype r_t = sqrt( (rho_t-4.0) * (rho_t-2.0) * rho_inf
                              / (rho_inf - 4.0) / (rho_inf - 2.0) / rho_t);

            g[i] = corrected_local_rate * mi * r_t / (sqrt(vi) + eps_hat);
          }
        else
          {
            g[i] = corrected_local_rate * mi;
          }
      }

  }
}


  template <typename Dtype>
  __global__ void AdamUpdateDecoupledWD(int N, int t, Dtype* g, Dtype* m, Dtype* v, const Dtype* param,
                                        Dtype beta1, Dtype beta2, Dtype eps_hat,
                                        Dtype corrected_local_rate,  Dtype nu_lambda, bool amsgrad,
                                        bool rectified) {
    CUDA_KERNEL_LOOP(i, N) {
      float gi = g[i];
      float mi = m[i] = m[i]*beta1 + gi*(1-beta1);
      float vi_old = v[i];
      float vi = v[i] = v[i]*beta2 + gi*gi*(1-beta2);
      if (amsgrad) {
        if (vi < vi_old)
          v[i] = vi = vi_old;
      }
      if (!rectified)
        g[i] = corrected_local_rate * mi / (sqrt(vi) + eps_hat) + param[i] * nu_lambda;
      else
        {
          Dtype rho_inf = 2.0/(1.0-beta2) - 1.0;
          Dtype rho_t = rho_inf - 2.0 * t * pow(beta2,t)/(1.0-pow(beta2,t)) ;
          if (rho_t > 4.0)
            {
              Dtype r_t = sqrt( (rho_t-4.0) * (rho_t-2.0) * rho_inf
                                / (rho_inf - 4.0) / (rho_inf - 2.0) / rho_t);
              g[i] = corrected_local_rate * mi * r_t / (sqrt(vi) + eps_hat) + param[i] * nu_lambda;
            }
          else
            {
              g[i] = corrected_local_rate * mi + param[i] * nu_lambda;
            }
        }
    }
  }

  template <typename Dtype>
  void adam_update_gpu(int N, int t, Dtype* g, Dtype* m, Dtype* v, const Dtype* param, Dtype beta1,
                     Dtype beta2, Dtype eps_hat, Dtype corrected_local_rate, Dtype nu_lambda,
                     bool amsgrad, bool decoupled_wd, bool rectified) {
  if (!decoupled_wd)
    {
    AdamUpdate<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, t, g, m, v, beta1, beta2, eps_hat, corrected_local_rate, amsgrad, rectified);
  CUDA_POST_KERNEL_CHECK;
    }
  else{
    AdamUpdateDecoupledWD<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
     N, t, g, m, v, param, beta1, beta2, eps_hat, corrected_local_rate, nu_lambda, amsgrad, rectified);
  CUDA_POST_KERNEL_CHECK;
  }
}
  template void adam_update_gpu<float>(int, int, float*, float*, float*, const float*,
                                       float, float, float, float, float, bool, bool, bool);
  template void adam_update_gpu<double>(int, int, double*, double*, double*, const double*,
                                        double, double, double, double, double, bool, bool, bool);

}  // namespace caffe
