#include "hip/hip_runtime.h"
#include "caffe/util/math_functions.hpp"


namespace caffe {

template <typename Dtype>
__global__ void AdamUpdate(int N, Dtype* g, Dtype* m, Dtype* v,
                           Dtype beta1, Dtype beta2, Dtype eps_hat, Dtype corrected_local_rate,
                           bool amsgrad) {
  CUDA_KERNEL_LOOP(i, N) {
    float gi = g[i];
    float mi = m[i] = m[i]*beta1 + gi*(1-beta1);
    float vi_old = v[i];
    float vi = v[i] = v[i]*beta2 + gi*gi*(1-beta2);
    if (amsgrad) {
      if (vi < vi_old)
        v[i] = vi = vi_old;
    }
    g[i] = corrected_local_rate * mi / (sqrt(vi) + eps_hat);
  }
}


  template <typename Dtype>
  __global__ void AdamUpdateDecoupledWD(int N, Dtype* g, Dtype* m, Dtype* v, const Dtype* params,
                                        Dtype beta1, Dtype beta2, Dtype eps_hat, Dtype local_rate,
                                        Dtype correction, Dtype lambda, bool amsgrad) {
    CUDA_KERNEL_LOOP(i, N) {
      float gi = g[i];
      float mi = m[i] = m[i]*beta1 + gi*(1-beta1);
      float vi_old = v[i];
      float vi = v[i] = v[i]*beta2 + gi*gi*(1-beta2);
      if (amsgrad) {
        if (vi < vi_old)
          v[i] = vi = vi_old;
      }
      g[i] = local_rate * (correction * mi / (sqrt(vi) + eps_hat) + params[i] * lambda);
    }
  }

  template <typename Dtype>
void adam_update_gpu(int N, Dtype* g, Dtype* m, Dtype* v, const Dtype* params, Dtype beta1,
                     Dtype beta2, Dtype eps_hat, Dtype local_rate, Dtype correction, Dtype lambda,
                     bool amsgrad, bool decoupled_wd) {
  if (!decoupled_wd)
    {
    AdamUpdate<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
     N, g, m, v, beta1, beta2, eps_hat, local_rate * correction, amsgrad);
  CUDA_POST_KERNEL_CHECK;
    }
  else{
    AdamUpdateDecoupledWD<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
  N, g, m, v, params, beta1, beta2, eps_hat, local_rate, correction, lambda, amsgrad);
  CUDA_POST_KERNEL_CHECK;
  }
}
  template void adam_update_gpu<float>(int, float*, float*, float*, const float*,
                                       float, float, float, float, float, float, bool, bool);
  template void adam_update_gpu<double>(int, double*, double*, double*, const double*,
                                        double, double, double, double, double, double, bool, bool);

}  // namespace caffe
