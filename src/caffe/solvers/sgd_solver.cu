#include "hip/hip_runtime.h"
#include "caffe/device.hpp"
#include "caffe/util/math_functions.hpp"

#ifdef USE_GREENTEA
#include "caffe/greentea/greentea.hpp"
#endif

namespace caffe {

#ifdef USE_CUDA
template <typename Dtype>
__global__ void SGDUpdate(int N, Dtype* g, Dtype* h,
    Dtype momentum, Dtype local_rate) {
  CUDA_KERNEL_LOOP(i, N) {
    g[i] = h[i] = momentum*h[i] + local_rate*g[i];
  }
}
#endif

template <typename Dtype>
void sgd_update_gpu(device* dev, int_tp N, Dtype* g, Dtype* h, Dtype momentum,
    Dtype local_rate) {
  if (dev->backend() == BACKEND_CUDA) {
#ifdef USE_CUDA
    SGDUpdate<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
        CUDA_KERNEL(CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS) (
        N, g, h, momentum, local_rate);
    CUDA_POST_KERNEL_CHECK;
#endif  // USE_CUDA
  } else {
#ifdef USE_GREENTEA
    viennacl::ocl::context &ctx = viennacl::ocl::get_context(dev->id());
    viennacl::ocl::program &program = dev->program();
    viennacl::ocl::kernel &oclk_sgd_update = program.get_kernel(
        CL_KERNEL_SELECT("sgd_update"));

    ClState& clState = Caffe::cl_state();
    ClMemOff<Dtype> bufg = clState.get_buffer_mem(g);
    ClMemOff<Dtype> bufh = clState.get_buffer_mem(h);

    viennacl::ocl::enqueue(
        oclk_sgd_update(N, WrapHandle(bufg.memobj, &ctx),
                        WrapHandle(bufh.memobj, &ctx), momentum, local_rate),
        ctx.get_queue());

#endif  // USE_GREENTEA
  }
}
template void sgd_update_gpu<float>(device*, int_tp, float*, float*, float,
                                    float);
template void sgd_update_gpu<double>(device*, int_tp, double*, double*, double,
                                     double);

}  // namespace caffe
