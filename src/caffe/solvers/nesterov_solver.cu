#include "hip/hip_runtime.h"
#include "caffe/util/math_functions.hpp"


namespace caffe {

template <typename Dtype>
__global__ void NesterovUpdate(int N, Dtype* g, Dtype* h,
    Dtype momentum, Dtype local_rate) {
  CUDA_KERNEL_LOOP(i, N) {
    float hi = h[i];
    float hi_new = h[i] = momentum * hi + local_rate * g[i];
    g[i] = (1+momentum) * hi_new - momentum * hi;
  }
}
template <typename Dtype>
void nesterov_update_gpu(int N, Dtype* g, Dtype* h, Dtype momentum,
    Dtype local_rate) {
  NesterovUpdate<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
      <<<CAFFE_GET_BLOCKS(N), CAFFE_CUDA_NUM_THREADS>>>(
      N, g, h, momentum, local_rate);
  CUDA_POST_KERNEL_CHECK;
}
template void nesterov_update_gpu<float>(int, float*, float*, float, float);
template void nesterov_update_gpu<double>(int, double*, double*, double,
    double);

}  // namespace caffe
